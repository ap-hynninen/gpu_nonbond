#include "hip/hip_runtime.h"

#include <cassert>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "CudaDomdecBonded.h"

__global__ void build_tbl_kernel(const int nbond_tbl, int* __restrict__ bond_tbl,
				 const int nureyb_tbl, int* __restrict__ ureyb_tbl,
				 const int nangle_tbl, int* __restrict__ angle_tbl,
				 const int ndihe_tbl, int* __restrict__ dihe_tbl,
				 const int nimdihe_tbl, int* __restrict__ imdihe_tbl,
				 const int ncmap_tbl, int* __restrict__ cmap_tbl) {
  const int pos = threadIdx.x + blockIdx.x*blockDim.x;
  if (pos < nbond_tbl) {
    bond_tbl[pos] = pos;
  } else if (pos < nbond_tbl + nureyb_tbl) {
    ureyb_tbl[pos-nbond_tbl] = pos-nbond_tbl;
  } else if (pos < nbond_tbl + nureyb_tbl + nangle_tbl) {
    angle_tbl[pos-nbond_tbl-nureyb_tbl] = pos-nbond_tbl-nureyb_tbl;
  } else if (pos < nbond_tbl + nureyb_tbl + nangle_tbl + ndihe_tbl) {
    dihe_tbl[pos-nbond_tbl-nureyb_tbl-nangle_tbl] = pos-nbond_tbl-nureyb_tbl-nangle_tbl;
  } else if (pos < nbond_tbl + nureyb_tbl + nangle_tbl + ndihe_tbl + nimdihe_tbl) {
    imdihe_tbl[pos-nbond_tbl-nureyb_tbl-nangle_tbl-ndihe_tbl] = pos-nbond_tbl-nureyb_tbl-nangle_tbl-ndihe_tbl;
  } else if (pos < nbond_tbl + nureyb_tbl + nangle_tbl + ndihe_tbl + nimdihe_tbl + ncmap_tbl) {
    cmap_tbl[pos-nbond_tbl-nureyb_tbl-nangle_tbl-ndihe_tbl-nimdihe_tbl] = 
      pos-nbond_tbl-nureyb_tbl-nangle_tbl-ndihe_tbl-nimdihe_tbl;
  }
}

//############################################################################################
//############################################################################################
//############################################################################################

//
// Class creator
//
CudaDomdecBonded::CudaDomdecBonded(const int nbond, const bond_t* h_bond,
				   const int nureyb, const bond_t* h_ureyb,
				   const int nangle, const angle_t* h_angle,
				   const int ndihe, const dihe_t* h_dihe,
				   const int nimdihe, const dihe_t* h_imdihe,
				   const int ncmap, const cmap_t* h_cmap) {  
  assert((nureyb == 0) || (nureyb > 0 && nureyb == nangle));

  bond = NULL;
  ureyb = NULL;
  angle = NULL;
  dihe = NULL;
  imdihe = NULL;
  cmap = NULL;

  this->nbond = nbond;
  if (nbond > 0) {
    allocate<bond_t>(&bond, nbond);
    copy_HtoD<bond_t>(h_bond, bond, nbond);
  }

  this->nureyb = nureyb;
  if (nureyb > 0) {
    allocate<bond_t>(&ureyb, nureyb);
    copy_HtoD<bond_t>(h_ureyb, ureyb, nureyb);
  }

  this->nangle = nangle;
  if (nangle > 0) {
    allocate<angle_t>(&angle, nangle);
    copy_HtoD<angle_t>(h_angle, angle, nangle);
  }

  this->ndihe = ndihe;
  if (ndihe > 0) {
    allocate<dihe_t>(&dihe, ndihe);
    copy_HtoD<dihe_t>(h_dihe, dihe, ndihe);
  }

  this->nimdihe = nimdihe;
  if (nimdihe > 0) {
    allocate<dihe_t>(&imdihe, nimdihe);
    copy_HtoD<dihe_t>(h_imdihe, imdihe, nimdihe);
  }

  this->ncmap = ncmap;
  if (ncmap > 0) {
    allocate<cmap_t>(&cmap, ncmap);
    copy_HtoD<cmap_t>(h_cmap, cmap, ncmap);
  }

  nbond_tbl = 0;
  bond_tbl_len = 0;
  bond_tbl = NULL;

  nureyb_tbl = 0;
  ureyb_tbl_len = 0;
  ureyb_tbl = NULL;

  nangle_tbl = 0;
  angle_tbl_len = 0;
  angle_tbl = NULL;

  ndihe_tbl = 0;
  dihe_tbl_len = 0;
  dihe_tbl = NULL;

  nimdihe_tbl = 0;
  imdihe_tbl_len = 0;
  imdihe_tbl = NULL;

  ncmap_tbl = 0;
  cmap_tbl_len = 0;
  cmap_tbl = NULL;
}

//
// Class destructor
//
CudaDomdecBonded::~CudaDomdecBonded() {
  if (bond != NULL) deallocate<bond_t>(&bond);
  if (ureyb != NULL) deallocate<bond_t>(&ureyb);
  if (angle != NULL) deallocate<angle_t>(&angle);
  if (dihe != NULL) deallocate<dihe_t>(&dihe);
  if (imdihe != NULL) deallocate<dihe_t>(&imdihe);
  if (cmap != NULL) deallocate<cmap_t>(&cmap);

  if (bond_tbl != NULL) deallocate<int>(&bond_tbl);
  if (ureyb_tbl != NULL) deallocate<int>(&ureyb_tbl);
  if (angle_tbl != NULL) deallocate<int>(&angle_tbl);
  if (dihe_tbl != NULL) deallocate<int>(&dihe_tbl);
  if (imdihe_tbl != NULL) deallocate<int>(&imdihe_tbl);
  if (cmap_tbl != NULL) deallocate<int>(&cmap_tbl);
}

//
// Build tables
//
void CudaDomdecBonded::build_tbl(const CudaDomdec *domdec, const int *zone_patom,
				 hipStream_t stream) {

  if (domdec->numnode == 1) {
    nbond_tbl = nbond;
    nureyb_tbl = nureyb;
    nangle_tbl = nangle;
    ndihe_tbl = ndihe;
    nimdihe_tbl = nimdihe;
    ncmap_tbl = ncmap;

    if (nbond_tbl > 0) reallocate<int>(&bond_tbl, &bond_tbl_len, nbond_tbl, 1.2f);
    if (nureyb_tbl > 0) reallocate<int>(&ureyb_tbl, &ureyb_tbl_len, nureyb_tbl, 1.2f);
    if (nangle_tbl > 0) reallocate<int>(&angle_tbl, &angle_tbl_len, nangle_tbl, 1.2f);
    if (ndihe_tbl > 0) reallocate<int>(&dihe_tbl, &dihe_tbl_len, ndihe_tbl, 1.2f);
    if (nimdihe_tbl > 0) reallocate<int>(&imdihe_tbl, &imdihe_tbl_len, nimdihe_tbl, 1.2f);
    if (ncmap_tbl > 0) reallocate<int>(&cmap_tbl, &cmap_tbl_len, ncmap_tbl, 1.2f);

    int nthread = 512;
    int nblock = (nbond_tbl + nureyb_tbl + nangle_tbl + 
		  ndihe_tbl + nimdihe_tbl + ncmap_tbl -1)/nthread + 1;
    build_tbl_kernel<<< nblock, nthread, 0, stream >>>
      (nbond_tbl, bond_tbl,
       nureyb_tbl, ureyb_tbl,
       nangle_tbl, angle_tbl,
       ndihe_tbl, dihe_tbl,
       nimdihe_tbl, imdihe_tbl,
       ncmap_tbl, cmap_tbl);
    cudaCheck(hipGetLastError());

  } else {
    std::cerr << "CudaDomdecBonded::build_tbl, numnode > 1 not implemented yet" << std::endl;
    exit(1);
  }

}
