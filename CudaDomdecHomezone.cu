#include "hip/hip_runtime.h"
#include <cassert>
#include <algorithm>
#include "CudaDomdecHomezone.h"
#include "CudaMPI.h"
#include "mpi_utils.h"
#include "gpu_utils.h"

//
// Returns index from (dix, diy, diz)
//
//  int nxt = min(3, nx);
//  int nxyt = nxt*min(3, ny);
__host__ __device__ inline int dix2ind(int dix, int diy, int diz,
				       const int nx, const int ny, const int nz,
				       const int nxt, const int nxyt) {
  // (1 < dix) - (dix < -1) returns: 1 if dix > 1
  //                                -1 if dix < -1
  //                                 0 otherwise
  dix -= nx*((1 < dix) - (dix < -1));
  diy -= ny*((1 < diy) - (diy < -1));
  diz -= nz*((1 < diz) - (diz < -1));
  // After these, dix = {0, .., 2} or {0, .., nx-1}
  dix = (dix+1) % nx;
  diy = (diy+1) % ny;
  diz = (diz+1) % nz;
  // Get neighboring node index, ind = 0...nneigh-1
  return dix + diy*nxt + diz*nxyt;
}


//
// Update homezone atomlist. Simple version using atomicAdd (does this have to be faster?)
//
__global__ void fill_send_kernel(const int ncoord,
				 const double* __restrict__ xin,
				 const double* __restrict__ yin,
				 const double* __restrict__ zin,
				 const double inv_boxx, const double inv_boxy, const double inv_boxz,
				 const double* __restrict__ lohi_buf,
				 const int nx, const int ny, const int nz, const int nneigh,
				 int* __restrict__ num_send,
				 int* __restrict__ destind) {
  // Shared memory
  // Requires: 2*3*sizeof(double) + 2*9*sizeof(double) + 2*27*sizeof(double) + nneigh*sizeof(int)
  extern __shared__ double sh_buf[];
  double *sh_lox = &sh_buf[0];      // In total this is 2*3+2*9+2*27=78 doubles
  double *sh_hix = &sh_buf[3];
  double *sh_loy = &sh_buf[6];
  double *sh_hiy = &sh_buf[15];
  double *sh_loz = &sh_buf[24];
  double *sh_hiz = &sh_buf[51];
  int* sh_num_send = (int *)&sh_buf[78];

  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  int nxt = min(3, nx);
  int nxyt = nxt*min(3, ny);
  if (threadIdx.x < 78) {
    sh_buf[threadIdx.x] = lohi_buf[threadIdx.x];
  }
  const int warpstart = ((78-1)/warpsize+1)*warpsize;
  if (threadIdx.x >= warpstart && threadIdx.x-warpstart < nneigh) {
    sh_num_send[threadIdx.x-warpstart] = 0;
  }
  __syncthreads();
  bool error = false;
  if (i < ncoord) {
    double x = xin[i]*inv_boxx + 0.5;
    double y = yin[i]*inv_boxy + 0.5;
    double z = zin[i]*inv_boxz + 0.5;
    x -= floor(x);
    y -= floor(y);
    z -= floor(z);

    int dix=3;
    if (x >= sh_lox[0] && x < sh_hix[0]) dix = 0;
    if (x >= sh_lox[1] && x < sh_hix[1]) dix = 1;
    if (x >= sh_lox[2] && x < sh_hix[2]) dix = 2;
    if (dix == 3) error = true;
    dix = (dix % 3);   // simple error recovery

    double *sh_loyp = &sh_loy[dix*3];
    double *sh_hiyp = &sh_hiy[dix*3];
    int diy=3;
    if (y >= sh_loyp[0] && y < sh_hiyp[0]) diy = 0;
    if (y >= sh_loyp[1] && y < sh_hiyp[1]) diy = 1;
    if (y >= sh_loyp[2] && y < sh_hiyp[2]) diy = 2;
    if (diy == 3) error = true;
    diy = (diy % 3);

    double *sh_lozp = &sh_loz[dix*9 + diy*3];
    double *sh_hizp = &sh_hiz[dix*9 + diy*3];
    int diz=3;
    if (z >= sh_lozp[0] && z < sh_hizp[0]) diz = 0;
    if (z >= sh_lozp[1] && z < sh_hizp[1]) diz = 1;
    if (z >= sh_lozp[2] && z < sh_hizp[2]) diz = 2;
    if (diz == 3) error = true;
    diz = (diz % 3);

    // Transform into (-1,0,1)
    dix--;
    diy--;
    diz--;

    int ind = dix2ind(dix, diy, diz, nx, ny, nz, nxt, nxyt);

    atomicAdd(&sh_num_send[ind], 1);
    destind[i] = ind;
  }

  __syncthreads();
  if (threadIdx.x < nneigh) {
    atomicAdd(&num_send[threadIdx.x], sh_num_send[threadIdx.x]);
  }

  // Set error flag into num_send[nneigh]
  if (error) {
    num_send[nneigh] = 1;
  }
}

//
// Calculates pos_send[0...nneigh] using exclusive cumulative sum
// Launched with blockDim.x = 32 and nblock = 1
//
__global__ void calc_pos_send_kernel(const int nneigh,
				     const int* __restrict__ num_send,
				     int* __restrict__ pos_send) {

  // This kernel is so simple, we'll just loop. No fancy stuff here.
  if (threadIdx.x == 0) {
    pos_send[0] = 0;
    for (int i=0;i < nneigh;i++) pos_send[i+1] = pos_send[i] + num_send[i];
  }

  /*
  // Shared memory
  // Requires: 27*sizeof(int)
  __shared__ int sh_pos_send[27];
  // Calculate inclusive scan and then shift to make it exclusive scan
  // Calculate positions into sh_pos_send
  if (threadIdx.x < nneigh) sh_pos_send[threadIdx.x] = num_send[threadIdx.x];
  if (threadIdx.x == 0) printf("num_send = %d %d\n",num_send[0],num_send[1]);
  __syncthreads();
  for (int d=1;d < nneigh;d *= 2) {
    int t = threadIdx.x + d;
    int val = (t < nneigh) ? sh_pos_send[t] : 0;
    __syncthreads();
    if (threadIdx.x < nneigh) sh_pos_send[threadIdx.x] += val;
    __syncthreads();
  }
  if (threadIdx.x < nneigh) {
    // Shift & store result to get exclusive cumulative sum in global memory
    if (threadIdx.x == 0) pos_send[0] = 0;
    pos_send[threadIdx.x+1] = sh_pos_send[threadIdx.x];
  }
  */
}

//
// Packs send -buffer
//
__global__ void pack_send_kernel(const int ncoord,
				 const double* __restrict__ x1,
				 const double* __restrict__ y1,
				 const double* __restrict__ z1,
				 const double* __restrict__ x2,
				 const double* __restrict__ y2,
				 const double* __restrict__ z2,
				 const int* __restrict__ destind,
				 const int* __restrict__ loc2glo,
				 int* __restrict__ pos_send,
				 CudaDomdecHomezone::neighcomm_t* __restrict__ send) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < ncoord) {
    int ind = destind[i];
    int pos = atomicAdd(&pos_send[ind], 1);
    send[pos].gloind = loc2glo[i];
    send[pos].x1 = x1[i];
    send[pos].y1 = y1[i];
    send[pos].z1 = z1[i];
    send[pos].x2 = x2[i];
    send[pos].y2 = y2[i];
    send[pos].z2 = z2[i];
  }
}

//
// Unpacks received data
//
__global__ void unpack_recv_kernel(const int num_recv_tot,
				   const CudaDomdecHomezone::neighcomm_t* __restrict__ recv,
				   double* __restrict__ x1,
				   double* __restrict__ y1,
				   double* __restrict__ z1,
				   double* __restrict__ x2,
				   double* __restrict__ y2,
				   double* __restrict__ z2,
				   int* __restrict__ loc2glo) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < num_recv_tot) {
    loc2glo[i]= recv[i].gloind;
    x1[i]     = recv[i].x1;
    y1[i]     = recv[i].y1;
    z1[i]     = recv[i].z1;
    x2[i]     = recv[i].x2;
    y2[i]     = recv[i].y2;
    z2[i]     = recv[i].z2;
  }
}

//################################################################################
//################################################################################
//################################################################################

//
// Class creator
//
CudaDomdecHomezone::CudaDomdecHomezone(Domdec& domdec, CudaMPI& cudaMPI) : 
  domdec(domdec), cudaMPI(cudaMPI) {

  int nxt = min(3, domdec.get_nx());
  int nyt = min(3, domdec.get_ny());
  int nzt = min(3, domdec.get_nz());
  nneigh = nxt*nyt*nzt;

  allocate<int>(&num_send, nneigh+1);
  allocate<int>(&pos_send, nneigh+1);
  allocate_host<int>(&h_num_send, nneigh+1);

  allocate<double>(&lohi_buf, 78);
  allocate_host<double>(&h_lohi_buf, 78);

  destind_len = 0;
  destind = NULL;

  send_len = 0;
  send = NULL;
  h_send_len = 0;
  h_send = NULL;

  recv_len = 0;
  recv = NULL;

  h_recv_len = 0;
  h_recv = NULL;

  neighnode.resize(nneigh);
  request.resize((nneigh-1)*2);

  h_pos_send = new int[nneigh+1];
  num_recv.resize(nneigh);
  pos_recv.resize(nneigh+1);

  /*
  int ixl = -(nxt-1)/2;
  int ixh = ixl + nxt-1;

  int iyl = -(nyt-1)/2;
  int iyh = iyl + nyt-1;

  int izl = -(nzt-1)/2;
  int izh = izl + nzt-1;
  */

  std::fill(neighnode.begin(), neighnode.end(), -1);
  for (int diz=-1;diz <= 1;diz++) {
    for (int diy=-1;diy <= 1;diy++) {
      for (int dix=-1;dix <= 1;dix++) {
	int k = dix2ind(dix, diy, diz,
			domdec.get_nx(), domdec.get_ny(), domdec.get_nz(),
			nxt, nxt*nyt);
	neighnode.at(k) = domdec.get_nodeind_pbc(dix + domdec.get_homeix(),
						 diy + domdec.get_homeiy(),
						 diz + domdec.get_homeiz());
      }
    }
  }

  imynode = dix2ind(0, 0, 0, domdec.get_nx(), domdec.get_ny(), domdec.get_nz(),
		    nxt, nxt*nyt);

  if (neighnode.at(imynode) != domdec.get_mynode()) {
    std::cout << "CudaDomdecHomezone::CudaDomdecHomezone, error in setting neighnode(1)" << std::endl;
    exit(1);
  }

  for (int i=0;i < nneigh;i++) {
    if (neighnode.at(i) == -1) {
      std::cout << "CudaDomdecHomezone::CudaDomdecHomezone, error in setting neighnode(2)"
		<< std::endl;
      exit(1);
    }
  }

  //
  // Neighbor node communication order. This is setup such that the self node comes first and the
  // rest of the nodes are according to their MPI rank
  //
  // Get an ordered array of neighbor nodes without the self node
  std::vector<int> neighnode_sort(neighnode);
  neighnode_sort.erase(neighnode_sort.begin() + imynode);
  std::sort(neighnode_sort.begin(), neighnode_sort.end());
  // Place the self node at front, and the rest of the neighbors in sorted order after it
  comm_order.resize(nneigh);
  comm_order.at(0) = imynode;
  for (int i=1;i < nneigh;i++) {
    // MPI rank we're looking for
    int rank = neighnode_sort.at(i-1);
    comm_order.at(i) = (int)(std::find(neighnode.begin(), neighnode.end(), rank) - neighnode.begin());
  }

}

//
// Class destructor
//
CudaDomdecHomezone::~CudaDomdecHomezone() {
  deallocate<int>(&num_send);
  deallocate<int>(&pos_send);
  deallocate_host<int>(&h_num_send);
  deallocate<double>(&lohi_buf);
  deallocate_host<double>(&h_lohi_buf);
  delete [] h_pos_send;
  if (destind != NULL) deallocate<int>(&destind);
  if (send != NULL) deallocate<neighcomm_t>(&send);
  if (recv != NULL) deallocate<neighcomm_t>(&recv);
  if (h_send != NULL) deallocate_host<neighcomm_t>(&h_send);
  if (h_recv != NULL) deallocate_host<neighcomm_t>(&h_recv);
}

//
// Build Homezone, assigns coordinates into sub-boxes. Done on the CPU
// Creates new loc2glo, DOES NOT re-create h_coord according to the new loc2glo
// Returns the number of coordinates in the homezone
// NOTE: h_coord is the global array
//
int CudaDomdecHomezone::build(hostXYZ<double>& h_coord) {

  int nx = domdec.get_nx();
  int ny = domdec.get_ny();
  int nz = domdec.get_nz();
  int homeix = domdec.get_homeix();
  int homeiy = domdec.get_homeiy();
  int homeiz = domdec.get_homeiz();
  double inv_boxx = domdec.get_inv_boxx();
  double inv_boxy = domdec.get_inv_boxy();
  double inv_boxz = domdec.get_inv_boxz();

  int *h_loc2glo = new int[h_coord.size()];

  double lox = domdec.get_lo_bx();
  double hix = domdec.get_hi_bx();
  double loy = domdec.get_lo_by();
  double hiy = domdec.get_hi_by();
  double loz = domdec.get_lo_bz();
  double hiz = domdec.get_hi_bz();

  // Find coordinates that are in this sub-box
  int nloc = 0;
  for (int i=0;i < h_coord.size();i++) {
    double x = h_coord.x()[i]*inv_boxx + 0.5;
    double y = h_coord.y()[i]*inv_boxy + 0.5;
    double z = h_coord.z()[i]*inv_boxz + 0.5;    
    x -= floor(x);
    y -= floor(y);
    z -= floor(z);
    if (x >= lox && x < hix && y >= loy && y < hiy && z >= loz && z < hiz) {
      h_loc2glo[nloc++] = i;
    }
  }

  loc2glo.resize(nloc);

  copy_HtoD_sync<int>(h_loc2glo, get_loc2glo_ptr(), nloc);
  delete [] h_loc2glo;

  return nloc;
}

//
// Update Homezone
// creates new loc2glo, re-creates coord and coord2 accoring to the new loc2glo
// Returns: the number of coordinates in the homezone
//
int CudaDomdecHomezone::update(const int ncoord, cudaXYZ<double>& coord, cudaXYZ<double>& coord2,
			       hipStream_t stream) {
  assert(ncoord <= coord.size());
  assert(ncoord <= coord2.size());

  // Allocate to #coordinates to avoid busting the buffer limits
  reallocate<int>(&destind, &destind_len, ncoord, 1.2f);
  reallocate<neighcomm_t>(&send, &send_len, ncoord, 1.2f);

  clear_gpu_array<int>(num_send, nneigh+1, stream);

  // Get a copy of lohi_buf and copy it to GPU
  domdec.copy_lohi_buf(h_lohi_buf);
  copy_HtoD<double>(h_lohi_buf, lohi_buf, 78, stream);

  int nthread = 1024;
  int nblock = (ncoord - 1)/nthread + 1;
  int shmem = 2*3*sizeof(double) + 2*9*sizeof(double) + 2*27*sizeof(double) + nneigh*sizeof(int);

  // Assign coordinates into neighboring, or home, sub-boxes
  fill_send_kernel<<< nblock, nthread, shmem, stream >>>
    (ncoord, coord.x(), coord.y(), coord.z(),     
     domdec.get_inv_boxx(), domdec.get_inv_boxy(), domdec.get_inv_boxz(),
     lohi_buf,
     domdec.get_nx(), domdec.get_ny(), domdec.get_nz(), nneigh,
     num_send, destind);
  cudaCheck(hipGetLastError());

  // Copy num_send => h_num_send
  copy_DtoH<int>(num_send, h_num_send, nneigh+1, stream);

  // Calculate positions for send buffer
  calc_pos_send_kernel<<< 1, 32, 0, stream >>>(nneigh, num_send, pos_send);
  cudaCheck(hipGetLastError());

  // Pack coordinate data into send buffer
  pack_send_kernel<<< nblock, nthread, 0, stream >>>
    (ncoord, coord.x(), coord.y(), coord.z(), coord2.x(), coord2.y(), coord2.z(),
     destind, get_loc2glo_ptr(), pos_send, send);
  cudaCheck(hipGetLastError());

  // Wait here for the stream to finish
  cudaCheck(hipStreamSynchronize(stream));

  // Check for error flag
  if (h_num_send[nneigh] != 0) {
    std::cerr << "CudaDomdecHomezone::update, atom(s) moved more than a single box length"
	      << std::endl;
    exit(1);
  }

  // Compute positions h_pos_send from h_num_send
  // NOTE: h_pos_send[0] = 0 and h_pos_send[nneigh] = total number to send
  h_pos_send[0] = 0;
  for (int i=0;i < nneigh;i++) h_pos_send[i+1] = h_pos_send[i] + h_num_send[i];

  // Total number of coordinates to send is h_pos_send[nneigh]
  if (!cudaMPI.isCudaAware()) {
    reallocate_host<neighcomm_t>(&h_send, &h_send_len, h_pos_send[nneigh], 1.4f);
  }

  const int COUNT_TAG = 1;
  int nrequest = 0;
  // Send number of coordinates
  for (int i=0;i < nneigh;i++) {
    if (neighnode.at(i) != domdec.get_mynode()) {
      MPICheck(MPI_Isend(&h_num_send[i], 1, MPI_INT, neighnode.at(i), COUNT_TAG,
			 cudaMPI.get_comm(), &request.at(nrequest)));
      nrequest++;
    }
  }  

  // Receive number of coordinates
  for (int i=0;i < nneigh;i++) {
    if (neighnode.at(i) != domdec.get_mynode()) {
      MPICheck(MPI_Irecv(&num_recv.at(i), 1, MPI_INT, neighnode.at(i), COUNT_TAG,
			 cudaMPI.get_comm(), &request.at(nrequest)));
      nrequest++;
    } else {
      num_recv.at(i) = h_num_send[i];
    }
  }

  // Wait for communication to finish
  MPICheck(MPI_Waitall(nrequest, request.data(), MPI_STATUSES_IGNORE));

  pos_recv.at(0) = 0;
  for (int i=0;i < nneigh;i++) pos_recv.at(i+1) = pos_recv.at(i) + num_recv.at(i);
  int num_recv_tot = pos_recv.at(nneigh);

  // Re-allocate memory as needed
  reallocate<neighcomm_t>(&recv, &recv_len, num_recv_tot, 1.2f);
  loc2glo.resize(num_recv_tot);
  if (!cudaMPI.isCudaAware()) {
    reallocate_host<neighcomm_t>(&h_recv, &h_recv_len, num_recv_tot, 1.2f);
  }

  const int COORD_TAG = 1;

  // Send & Recv coordinate data
  for (int j=0;j < nneigh;j++) {
    int i = comm_order.at(j);
    if (neighnode.at(i) != domdec.get_mynode()) {
      if (h_num_send[i] > 0 && num_recv.at(i) > 0) {
	MPICheck(cudaMPI.Sendrecv(&send[h_pos_send[i]], h_num_send[i]*sizeof(neighcomm_t), 
				  neighnode.at(i), COORD_TAG,
				  &recv[pos_recv.at(i)], num_recv.at(i)*sizeof(neighcomm_t),
				  neighnode.at(i), COORD_TAG, MPI_STATUS_IGNORE,
				  &h_send[h_pos_send[i]], &h_recv[pos_recv.at(i)]));
      } else if (h_num_send[i] > 0) {
	MPICheck(cudaMPI.Send(&send[h_pos_send[i]], h_num_send[i]*sizeof(neighcomm_t), 
			      neighnode.at(i), COORD_TAG,
			      &h_send[h_pos_send[i]]));
      } else if (num_recv.at(i) > 0) {
	MPICheck(cudaMPI.Recv(&recv[pos_recv.at(i)], num_recv.at(i)*sizeof(neighcomm_t),
			      neighnode.at(i), COORD_TAG, MPI_STATUS_IGNORE,
			      &h_recv[pos_recv.at(i)]));
      }
    } else if (num_recv.at(i) > 0) {
      // Copy data from local (home) sub-box
      copy_DtoD<neighcomm_t>(&send[h_pos_send[i]], &recv[pos_recv.at(i)], num_recv.at(i), stream);
    }
  }
  // Wait until the copy_DtoD above finishes
  cudaCheck(hipStreamSynchronize(stream));

  // Re-allocate coord and coord2 if needed
  coord.realloc(num_recv_tot);
  coord2.realloc(num_recv_tot);

  // Unpack data on GPU
  unpack_recv_kernel<<< nblock, nthread, 0, stream >>>
    (num_recv_tot, recv, coord.x(), coord.y(), coord.z(), coord2.x(), coord2.y(), coord2.z(),
     get_loc2glo_ptr());
  cudaCheck(hipGetLastError());

  // Wait here for the stream to finish
  cudaCheck(hipStreamSynchronize(stream));

  return num_recv_tot;
}
