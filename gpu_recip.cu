#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "gpu_utils.h"
#include "XYZQ.h"
#include "Bspline.h"
#include "Grid.h"

void time_transpose();
void test();

int main(int argc, char *argv[]) {

  int gpu_ind = 3;
  cudaCheck(hipSetDevice(gpu_ind));

  cudaCheck(hipDeviceSynchronize());
  
  hipDeviceProp_t gpu_prop;
  cudaCheck(hipGetDeviceProperties(&gpu_prop, gpu_ind));

  printf("Using CUDA device (%d) %s\n",gpu_ind,gpu_prop.name);

  //  time_transpose();

  test();

  return 0;
}

//
//
//
void time_transpose() {

  const int NUM_REP = 100;
  const int nfftx = 64;
  const int nffty = 64;
  const int nfftz = 64;
  Matrix3d<float> A(nfftx, nffty, nfftz, "test_data/q_real_double.txt");
  //  Matrix3d<float> A(nfftx, nffty, nfftz);
  Matrix3d<float> B(nfftx, nffty, nfftz);
  Matrix3d<float> C(nfftx, nffty, nfftz);

  hipEvent_t start_event, stop_event;
  cudaCheck(hipEventCreate(&start_event));
  cudaCheck(hipEventCreate(&stop_event));
  float ms;
  double max_diff;

  // Copy
  A.copy(&B);
  cudaCheck(hipEventRecord(start_event,0));
  for (int i=0;i < NUM_REP;i++)
    A.copy(&B);
  cudaCheck(hipEventRecord(stop_event,0));
  cudaCheck(hipEventSynchronize(stop_event));
  cudaCheck(hipEventElapsedTime(&ms, start_event, stop_event));
  std::cout << "copy:" << std::endl;
  std::cout << "time (ms) = " << ms << std::endl;
  std::cout << "GB/s = " << 2*nfftx*nffty*nfftz*sizeof(float)*1e-6*NUM_REP/ms << std::endl;

  // Transpose (x,y,z) -> (y,z,x)
  A.transpose_xyz_yzx(&B);
  cudaCheck(hipEventRecord(start_event,0));
  for (int i=0;i < NUM_REP;i++)
    A.transpose_xyz_yzx(&B);
  cudaCheck(hipEventRecord(stop_event,0));
  cudaCheck(hipEventSynchronize(stop_event));
  cudaCheck(hipEventElapsedTime(&ms, start_event, stop_event));
  A.transpose_xyz_yzx_host(&C);
  if (!B.compare(&C, 0.0, max_diff)) {
    std::cout << "Error in transpose_xyz_yzx" << std::endl;
    return;
  }
  std::cout << "transpose_xyz_yzx:" << std::endl;
  std::cout << "time (ms) = " << ms << std::endl;
  std::cout << "GB/s = " << 2*nfftx*nffty*nfftz*sizeof(float)*1e-6*NUM_REP/ms << std::endl;

  // Transpose (x,y,z) -> (z,x,y)
  A.transpose_xyz_zxy(&B);
  cudaCheck(hipEventRecord(start_event,0));
  for (int i=0;i < NUM_REP;i++)
    A.transpose_xyz_zxy(&B);
  cudaCheck(hipEventRecord(stop_event,0));
  cudaCheck(hipEventSynchronize(stop_event));
  cudaCheck(hipEventElapsedTime(&ms, start_event, stop_event));
  A.transpose_xyz_zxy_host(&C);
  if (!B.compare(&C, 0.0, max_diff)) {
    std::cout << "Error in transpose_xyz_zxy" << std::endl;
    return;
  }
  std::cout << "transpose_xyz_zxy:" << std::endl;
  std::cout << "time (ms) = " << ms << std::endl;
  std::cout << "GB/s = " << 2*nfftx*nffty*nfftz*sizeof(float)*1e-6*NUM_REP/ms << std::endl;
}

//
// Test the code using data in test_data/ -directory
//
void test() {

  // Settings for the data:
  const double boxx = 62.23;
  const double boxy = 62.23;
  const double boxz = 62.23;
  const double kappa = 0.320;
  const int ncoord = 23558;
  const int nfftx = 64;
  const int nffty = 64;
  const int nfftz = 64;
  const int order = 4;

  // Number of MPI nodes & current node index
  int nnode = 1;
  int mynode = 0;

  // Setup reciprocal vectors
  double recip[9];
  for (int i=0;i < 9;i++) recip[i] = 0.0;
  recip[0] = 1.0/boxx;
  recip[4] = 1.0/boxy;
  recip[8] = 1.0/boxz;

  // Load comparison data
  Matrix3d<float> q(nfftx, nffty, nfftz, "test_data/q_real_double.txt");
  Matrix3d<float2> q_xfft(nfftx/2+1, nffty, nfftz, "test_data/q_comp1_double.txt");
  Matrix3d<float2> q_zfft(nfftz, nfftx/2+1, nffty, "test_data/q_comp5_double.txt");
  Matrix3d<float2> q_zfft_summed(nfftz, nfftx/2+1, nffty, "test_data/q_comp6_double.txt");

  // Load coordinates
  XYZQ xyzq("test_data/xyzq.txt");

  // Create Bspline and Grid objects
  Bspline<float> bspline(ncoord, order, nfftx, nffty, nfftz);
  Grid<long long int, float, float2> grid(nfftx, nffty, nfftz, order, nnode, mynode);

  bspline.set_recip<double>(recip);

  grid.make_fft_plans();
  grid.print_info();

  bspline.fill_bspline(xyzq.xyzq, xyzq.ncoord);
  bspline.calc_prefac();

  grid.spread_charge(xyzq.ncoord, bspline);

  double tol = 1.0e-5;
  double max_diff;

  if (!q.compare(grid.charge_grid, tol, max_diff)) {
    std::cout<< "q comparison FAILED" << std::endl;
    return;
  } else {
    std::cout<< "q comparison OK (tolerance " << tol << " max difference "<< max_diff << ")" << std::endl;
  }

  tol = 0.002;
  grid.r2c_fft();
  if (!q_zfft.compare(grid.zfft_grid, tol, max_diff)) {
    std::cout<< "q_zfft comparison FAILED" << std::endl;
    return;
  } else {
    std::cout<< "q_zfft comparison OK (tolerance " << tol << " max difference " << max_diff << ")" << std::endl;
  }

  tol = 1.0e-6;
  grid.scalar_sum(recip, kappa, bspline.prefac_x, bspline.prefac_y, bspline.prefac_z);
  if (!q_zfft_summed.compare(grid.zfft_grid, tol, max_diff)) {
    std::cout<< "q_zfft_summed comparison FAILED" << std::endl;
    q_zfft_summed.print(0,10,0,0,0,0);
    std::cout<<"====================================="<<std::endl;
    grid.zfft_grid->print(0,10,0,0,0,0);
    return;
  } else {
    std::cout<< "q_zfft_summed comparison OK (tolerance " << tol << " max difference " << max_diff << ")" << std::endl;
  }

  // Allocate GPU memory for forces
  long long int *force;
  allocate<long long int>(&force, 3*ncoord);

  grid.gather_force(ncoord, bspline, force);

  // Deallocate GPU memory
  deallocate<long long int>(&force);
}
