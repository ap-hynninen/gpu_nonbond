#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "CudaLeapfrogIntegrator.h"
#include "cuda_utils.h"
#include "gpu_utils.h"

//
// Storage
//
static __device__ CudaLeapfrogIntegrator_storage_t d_CudaLeapfrogIntegrator_storage;

//
// Calculates: a = b + c
//
__global__ void add_coord_kernel(const int n,
				 const double* __restrict__ bx,
				 const double* __restrict__ by,
				 const double* __restrict__ bz,
				 const double* __restrict__ cx,
				 const double* __restrict__ cy,
				 const double* __restrict__ cz,
				 double* __restrict__ ax,
				 double* __restrict__ ay,
				 double* __restrict__ az) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < n) {
    ax[tid] = bx[tid] + cx[tid];
    ay[tid] = by[tid] + cy[tid];
    az[tid] = bz[tid] + cz[tid];
  }
}

//
// Calculates: a = b - c
//
__global__ void sub_coord_kernel(const int n,
				 const double* __restrict__ bx,
				 const double* __restrict__ by,
				 const double* __restrict__ bz,
				 const double* __restrict__ cx,
				 const double* __restrict__ cy,
				 const double* __restrict__ cz,
				 double* __restrict__ ax,
				 double* __restrict__ ay,
				 double* __restrict__ az) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < n) {
    ax[tid] = bx[tid] - cx[tid];
    ay[tid] = by[tid] - cy[tid];
    az[tid] = bz[tid] - cz[tid];
  }
}

//
// Calculates the next step vector using forces:
// step = prev_step - force*dt^2/mass
// gamma_val = dt^2/mass
//
__global__ void calc_step_kernel(const int ncoord, const int stride, 
				 const double dtsq,
				 const double* __restrict__ force,
				 const double* __restrict__ prev_step_x,
				 const double* __restrict__ prev_step_y,
				 const double* __restrict__ prev_step_z,
				 const float* __restrict__ mass,
				 double* __restrict__ step_x,
				 double* __restrict__ step_y,
				 double* __restrict__ step_z) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    double gamma_val = dtsq/(double)mass[tid];
    step_x[tid] = prev_step_x[tid] - force[tid]*gamma_val;
    step_y[tid] = prev_step_y[tid] - force[tid+stride]*gamma_val;
    step_z[tid] = prev_step_z[tid] - force[tid+stride*2]*gamma_val;
  }
}

//
// Calculates kinetic energy
//
__global__ void calc_kine_kernel(const int ncoord,
				 const double fac,
				 const float* __restrict__ mass,
				 const double* __restrict__ prev_step_x,
				 const double* __restrict__ prev_step_y,
				 const double* __restrict__ prev_step_z,
				 const double* __restrict__ step_x,
				 const double* __restrict__ step_y,
				 const double* __restrict__ step_z) {

  // Required shared memory:
  // blockDim.x*sizeof(double)
  extern __shared__ double sh_kine[];

  const int tid = threadIdx.x + blockIdx.x*blockDim.x;

  double kine = 0.0;
  if (tid < ncoord) {
    double vx = (prev_step_x[tid] + step_x[tid])*fac;
    double vy = (prev_step_y[tid] + step_y[tid])*fac;
    double vz = (prev_step_z[tid] + step_z[tid])*fac;
    kine = ((double)mass[tid])*(vx*vx + vy*vy + vz*vz);
  }

  sh_kine[threadIdx.x] = kine;
  __syncthreads();
  for (int d=1;d < blockDim.x;d*=2) {
    int t = threadIdx.x + d;
    double kine_val = (t < blockDim.x) ? sh_kine[t] : 0.0;
    __syncthreads();
    sh_kine[threadIdx.x] += kine_val;
    __syncthreads();
  }
  
  if (threadIdx.x == 0) {
    atomicAdd(&d_CudaLeapfrogIntegrator_storage.kine, sh_kine[0]);
  }

}

//##################################################################################################

//
// Class creator
//
CudaLeapfrogIntegrator::CudaLeapfrogIntegrator(HoloConst *holoconst, hipStream_t stream) {
  this->holoconst = holoconst;
  this->stream = stream;
  cudaCheck(hipEventCreate(&copy_rms_work_done_event));
  cudaCheck(hipEventCreate(&copy_temp_ekin_done_event));
  cudaCheck(hipEventCreate(&done_integrate_event));
  global_mass = NULL;
  mass_len = 0;
  mass = NULL;
  allocate_host<CudaLeapfrogIntegrator_storage_t>(&h_CudaLeapfrogIntegrator_storage, 1);
}

//
// Class destructor
//
CudaLeapfrogIntegrator::~CudaLeapfrogIntegrator() {
  cudaCheck(hipEventDestroy(copy_rms_work_done_event));
  cudaCheck(hipEventDestroy(copy_temp_ekin_done_event));
  cudaCheck(hipEventDestroy(done_integrate_event));
  if (global_mass != NULL) deallocate<float>(&global_mass);
  if (mass != NULL) deallocate<float>(&mass);
  deallocate_host<CudaLeapfrogIntegrator_storage_t>(&h_CudaLeapfrogIntegrator_storage);
}

//
// Initialize integrator
//
void CudaLeapfrogIntegrator::spec_init(const double *x, const double *y, const double *z,
				       const double *dx, const double *dy, const double *dz,
				       const double *h_mass) {
  if (forcefield == NULL) {
    std::cerr << "CudaLeapfrogIntegrator::spec_init, no forcefield set!" << std::endl;
    exit(1);
  }

  // Create temporary host array for coordinates
  hostXYZ<double> h_prev_coord(ncoord_glo, NON_PINNED);
  h_prev_coord.set_data_fromhost(ncoord_glo, x, y, z);

  // Initialize force field coordinate arrays and divide atoms to nodes
  std::vector<int> h_loc2glo;
  CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
  p->assignCoordToNodes(h_prev_coord, h_loc2glo);

  // Realloc and set arrays
  step.realloc(h_loc2glo.size());
  step.clear();

  prev_step.realloc(h_loc2glo.size());
  prev_step.set_data_sync(h_loc2glo, dx, dy, dz);

  coord.realloc(h_loc2glo.size());
  coord.clear();

  prev_coord.realloc(h_loc2glo.size());
  prev_coord.set_data_sync(h_loc2glo, x, y, z);

  force.realloc(h_loc2glo.size());

  // Make global mass array
  float *h_mass_f = new float[ncoord_glo];
  for (int i=0;i < ncoord_glo;i++) {
    h_mass_f[i] = (float)h_mass[i];
  }
  allocate<float>(&global_mass, ncoord_glo);
  copy_HtoD<float>(h_mass_f, global_mass, ncoord_glo);
  delete [] h_mass_f;

  // Host versions of coordinate, step, and force arrays
  // NOTE: These are used for copying coordinates, so they must be global size
  h_coord.realloc(ncoord_glo);
  h_step.realloc(ncoord_glo);
  h_force.realloc(ncoord_glo);

}

//
// Swaps step <=> prev_step
//
void CudaLeapfrogIntegrator::swap_step() {
  assert(step.match(prev_step));

  // Wait here until work on stream has stopped
  cudaCheck(hipStreamSynchronize(stream));

  step.swap(prev_step);

}

//
// Swaps coord <=> prev_coord
//
void CudaLeapfrogIntegrator::swap_coord() {
  assert(coord.match(prev_coord));

  // Wait here until work on stream has stopped
  cudaCheck(hipStreamSynchronize(stream));

  coord.swap(prev_coord);

}

//
// Calculates new current coordinate positions (cur) using 
// the previous coordinates (prev) and the step vector (step)
// coord = prev_coord + prev_step
//
void CudaLeapfrogIntegrator::take_step() {

  add_coord(prev_coord, prev_step, coord);

  cudaCheck(hipEventRecord(done_integrate_event, stream));
}

//
// Calculate step
//
void CudaLeapfrogIntegrator::calc_step() {
  assert(prev_coord.match(step));

  int nthread = 512;
  int nblock = (step.size() - 1)/nthread + 1;

  double dtsq = timestep_akma*timestep_akma;

  calc_step_kernel<<< nblock, nthread, 0, stream >>>
    (step.size(), force.stride(), dtsq, (double *)force.xyz(), 
     prev_step.x(), prev_step.y(), prev_step.z(), mass,
     step.x(), step.y(), step.z());
  
  cudaCheck(hipGetLastError());
}

//
// Calculate forces
//

void CudaLeapfrogIntegrator::pre_calc_force() {
  if (forcefield != NULL) {
    CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
    //cudaCheck(hipStreamWaitEvent(stream, done_integrate_event, 0));
    cudaCheck(hipStreamSynchronize(stream));
    p->pre_calc(coord, prev_step);
  }
}

void CudaLeapfrogIntegrator::calc_force(const bool calc_energy, const bool calc_virial) {
  if (forcefield != NULL) {
    CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
    p->calc(calc_energy, calc_virial, force);
  }
}

void CudaLeapfrogIntegrator::post_calc_force() {
  if (forcefield != NULL) {
    CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
    reallocate<float>(&mass, &mass_len, coord.size());
    p->post_calc(global_mass, mass, holoconst);
    p->wait_calc(stream);
  }
}

void CudaLeapfrogIntegrator::stop_calc_force() {
  if (forcefield != NULL) {
    CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
    p->stop_calc();
  }
}

//
// Calculate temperature
//
void CudaLeapfrogIntegrator::calc_temperature() {
  // Clear kinetic energy accumulator
  h_CudaLeapfrogIntegrator_storage->kine = 0.0;
  cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_CudaLeapfrogIntegrator_storage),
				    h_CudaLeapfrogIntegrator_storage,
				    sizeof(CudaLeapfrogIntegrator_storage_t),
				    0, hipMemcpyHostToDevice, stream));
  // Calculate kinetic energy
  int nthread = 512;
  int nblock = (step.size() - 1)/nthread + 1;
  int shmem_size = nthread*sizeof(double);
  double fac = 0.5/timestep_akma;
  calc_kine_kernel<<< nblock, nthread, shmem_size, stream >>>
    (step.size(), fac, mass, 
     prev_step.x(), prev_step.y(), prev_step.z(),
     step.x(), step.y(), step.z());
  cudaCheck(hipGetLastError());
  // Retrieve result
  cudaCheck(hipMemcpyFromSymbol(h_CudaLeapfrogIntegrator_storage,
				 HIP_SYMBOL(d_CudaLeapfrogIntegrator_storage),
				 sizeof(CudaLeapfrogIntegrator_storage_t),
				 0, hipMemcpyDeviceToHost));
  //std::cout << "kinetic energy = " << 0.5*h_CudaLeapfrogIntegrator_storage->kine << std::endl;
}

//
// Do holonomic constraints
//
void CudaLeapfrogIntegrator::do_holoconst() {
  if (holoconst != NULL) {
    // prev_coord = coord + step
    add_coord(coord, step, prev_coord);
    // holonomic constraint, result in prev_coord
    holoconst->apply(coord, prev_coord, stream);
    // step = prev_coord - coord
    sub_coord(prev_coord, coord, step);
  }
}

//
// Calculates: a = b + c
//
void CudaLeapfrogIntegrator::add_coord(cudaXYZ<double> &b, cudaXYZ<double> &c,
				       cudaXYZ<double> &a) {
  assert(b.match(c));
  assert(b.match(a));

  int nthread = 512;
  int nblock = (a.size() - 1)/nthread + 1;

  add_coord_kernel<<< nblock, nthread, 0, stream >>>
    (a.size(), b.x(), b.y(), b.z(), c.x(), c.y(), c.z(), a.x(), a.y(), a.z() );

  cudaCheck(hipGetLastError());
}

//
// Calculates: a = b - c
//
void CudaLeapfrogIntegrator::sub_coord(cudaXYZ<double> &b, cudaXYZ<double> &c,
				       cudaXYZ<double> &a) {
  assert(b.match(c));
  assert(b.match(a));

  int nthread = 512;
  int nblock = (a.size() - 1)/nthread + 1;

  sub_coord_kernel<<< nblock, nthread, 0, stream >>>
    (a.size(), b.x(), b.y(), b.z(), c.x(), c.y(), c.z(), a.x(), a.y(), a.z());

  cudaCheck(hipGetLastError());
}

//
// Do constant pressure
//
void CudaLeapfrogIntegrator::do_pressure() {
}

//
// Returns true if constant pressure is ON
//
bool CudaLeapfrogIntegrator::const_pressure() {
  return false;
}

//
// Do constant temperature
//
void CudaLeapfrogIntegrator::do_temperature() {
}

//
// Print energy & other info on screen
//
void CudaLeapfrogIntegrator::do_print_energy(int step) {
  if (forcefield != NULL) {
    CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
    p->print_energy_virial(step);
  }
}

//
// Get coordinates to host memory
//
void CudaLeapfrogIntegrator::get_restart_data(double *x, double *y, double *z,
					      double *dx, double *dy, double *dz,
					      double *fx, double *fy, double *fz) {

  if (forcefield != NULL) {
    h_coord.set_data_sync(coord);
    h_step.set_data_sync(step);
    h_force.set_data_sync(force.size(),
			  (double *)(force.xyz()),
			  (double *)(force.xyz()+force.stride()),
			  (double *)(force.xyz()+2*force.stride()));
    CudaForcefield *p = static_cast<CudaForcefield*>(forcefield);
    p->get_restart_data(h_coord, h_step, h_force, x, y, z, dx, dy, dz, fx, fy, fz);
  }
  
}

