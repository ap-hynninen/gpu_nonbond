#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include "gpu_utils.h"
#include "reduce.h"
#include "cuda_utils.h"
#include "Force.h"


template <typename T>
int Force<T>::calc_stride() {
  const int sizeof_T = 4;
  //return ((ncoord*sizeof(T) - 1)/256 + 1)*256/sizeof(T);
  return ((ncoord*sizeof_T - 1)/256 + 1)*256/sizeof_T;
}

//
// Class creators
//
template <typename T>
Force<T>::Force() {
  ncoord = 0;
  stride = 0;
  data_len = 0;
  data = NULL;
}

template <typename T>
Force<T>::Force(const int ncoord) : ncoord(ncoord) {
  stride = calc_stride();
  data_len = 3*stride;
  allocate<T>(&data, data_len);
}

template <typename T>
Force<T>::Force(const char *filename) {
  std::ifstream file(filename);
  if (file.is_open()) {
    
    T fx, fy, fz;
    
    // Count number of coordinates
    ncoord = 0;
    while (file >> fx >> fy >> fz) ncoord++;

    stride = calc_stride();

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);
    
    // Allocate CPU memory
    T *data_cpu = new T[3*stride];
    
    // Read coordinates
    int i=0;
    while (file >> data_cpu[i] >> data_cpu[i+stride] >> data_cpu[i+stride*2]) i++;
    
    // Allocate GPU memory
    data_len = 3*stride;
    allocate<T>(&data, data_len);

    // Copy coordinates from CPU to GPU
    copy_HtoD<T>(data_cpu, data, 3*stride);

    // Deallocate CPU memory
    delete [] data_cpu;
    
  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Class destructor
//
template <typename T>
Force<T>::~Force() {
  if (data != NULL) deallocate<T>(&data);
}

//
// Sets force data to zero
//
template <typename T>
void Force<T>::clear() {
  clear_gpu_array<T>(data, 3*stride, get_direct_nonbond_stream());
}

//
// Compares two force arrays, returns true if the difference is within tolerance
// NOTE: Comparison is done in double precision
//
template <typename T>
bool Force<T>::compare(Force<T>* force, const double tol, double& max_diff) {

  assert(force->ncoord == ncoord);

  T *h_data1 = new T[3*stride];
  T *h_data2 = new T[3*force->stride];

  copy_DtoH<T>(data,        h_data1, 3*stride);
  copy_DtoH<T>(force->data, h_data2, 3*force->stride);

  bool ok = true;

  max_diff = 0.0;

  int i;
  double fx1, fy1, fz1;
  double fx2, fy2, fz2;
  double diff;
  try {
    for (i=0;i < ncoord;i++) {
      fx1 = (double)h_data1[i];
      fy1 = (double)h_data1[i + stride];
      fz1 = (double)h_data1[i + 2*stride];
      fx2 = (double)h_data2[i];
      fy2 = (double)h_data2[i + force->stride];
      fz2 = (double)h_data2[i + 2*force->stride];
      if (isnan(fx1) || isnan(fy1) || isnan(fz1) || isnan(fx2) || isnan(fy2) || isnan(fz2)) throw 1;
      diff = max(fabs(fx1-fx2), max(fabs(fy1-fy2), fabs(fz1-fz2)));
      max_diff = max(diff, max_diff);
      if (diff > tol) throw 2;
    }
  }
  catch (int a) {
    std::cout << "i = "<< i << std::endl;
    std::cout << "this: fx1 fy1 fz1 = " << fx1 << " "<< fy1 << " "<< fz1 << std::endl;
    std::cout << "force:fx2 fy2 fz2 = " << fx2 << " "<< fy2 << " "<< fz2 << std::endl;
    if (a == 2) std::cout << "difference: " << diff << std::endl;
    ok = false;
  }

  delete [] h_data1;
  delete [] h_data2;
  
  return ok;
}

//
// Sets the size of the force array
//
template <typename T>
void Force<T>::set_ncoord(int ncoord, float fac) {
  this->ncoord = ncoord;
  stride = calc_stride();
  reallocate<T>(&data, &data_len, 3*stride, fac);
}

//
// Returns stride
//
template <typename T>
int Force<T>::get_stride() {
  return stride;
}

//
// Copies force to host
//
template <typename T>
void Force<T>::get_force(T *h_data) {
  copy_DtoH<T>(data, h_data, 3*stride);
}

//
// Converts one type of force array to another. Result is in "force"
//
template <typename T>
template <typename T2>
void Force<T>::convert(Force<T2>* force) {

  assert(force->ncoord == ncoord);
  assert(force->stride == stride);

  int nthread = 512;
  int nblock = (3*stride - 1)/nthread + 1;

  reduce_data<T, T2>
    <<< nblock, nthread >>>(3*stride, this->data, force->data);
}

//
// Converts one type of force array to another. Result is in "this"
// NOTE: Only works when the size of the types T and T2 match
//
template <typename T>
template <typename T2>
void Force<T>::convert() {

  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (3*stride - 1)/nthread + 1;

  reduce_data<T, T2>
    <<< nblock, nthread >>>(3*stride, this->data);
}

//
// Converts one type of force array to another and adds force to the result.
// Result is in "this"
// NOTE: Only works when the size of the types T and T2 match
//
template <typename T>
template <typename T2, typename T3>
void Force<T>::convert_add(Force<T3> *force) {

  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (3*stride - 1)/nthread + 1;

  reduce_add_data<T, T2, T3>
    <<< nblock, nthread >>>(3*stride, force->data, this->data);
}

//
// Explicit instances of Force class
//
template class Force<long long int>;
template class Force<double>;
template class Force<float>;
template void Force<long long int>::convert<float>(Force<float>* force);
template void Force<long long int>::convert<double>();
template void Force<long long int>::convert_add<double>(Force<float> *force);
