#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include "gpu_utils.h"
#include "reduce.h"
#include "cuda_utils.h"
#include "Force.h"
#include "hostXYZ.h"

template <typename T>
Force<T>::Force(const char *filename) {
  _size = 0;
  _stride = 0;
  _capacity = 0;
  _xyz = NULL;

  std::ifstream file(filename);
  if (file.is_open()) {
    
    T fx, fy, fz;
    
    // Count number of coordinates
    int nforce = 0;
    while (file >> fx >> fy >> fz) nforce++;

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);
    
    // Allocate CPU memory
    hostXYZ<T> xyz_cpu(nforce, NON_PINNED);

    // Read coordinates
    int i=0;
    while (file >> xyz_cpu.x()[i] >> xyz_cpu.y()[i] >> xyz_cpu.z()[i]) i++;

    // Allocate GPU memory
    this->resize(nforce);

    // Copy coordinates from CPU to GPU
    copy_HtoD_sync<T>(xyz_cpu.x(), this->x(), nforce);
    copy_HtoD_sync<T>(xyz_cpu.y(), this->y(), nforce);
    copy_HtoD_sync<T>(xyz_cpu.z(), this->z(), nforce);

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Compares two force arrays, returns true if the difference is within tolerance
// NOTE: Comparison is done in double precision
//
template <typename T>
bool Force<T>::compare(Force<T>& force, const double tol, double& max_diff) {
  assert(force.size() == this->size());

  hostXYZ<T> xyz1(this->size(), NON_PINNED);
  hostXYZ<T> xyz2(force.size(), NON_PINNED);
  xyz1.set_data_sync(force.size(), force.x(), force.y(), force.z());
  xyz2.set_data_sync(this->size(), this->x(), this->y(), this->z());

  bool ok = true;

  max_diff = 0.0;

  int i;
  double fx1, fy1, fz1;
  double fx2, fy2, fz2;
  double diff;
  try {
    for (i=0;i < this->size();i++) {
      fx1 = (double)(xyz1.x()[i]);
      fy1 = (double)(xyz1.y()[i]);
      fz1 = (double)(xyz1.z()[i]);
      fx2 = (double)(xyz2.x()[i]);
      fy2 = (double)(xyz2.y()[i]);
      fz2 = (double)(xyz2.z()[i]);
      if (isnan(fx1) || isnan(fy1) || isnan(fz1) || isnan(fx2) || isnan(fy2) || isnan(fz2)) throw 1;
      diff = max(fabs(fx1-fx2), max(fabs(fy1-fy2), fabs(fz1-fz2)));
      max_diff = max(diff, max_diff);
      if (diff > tol) throw 2;
    }
  }
  catch (int a) {
    std::cout << "i = "<< i << std::endl;
    std::cout << "this: fx1 fy1 fz1 = " << fx1 << " "<< fy1 << " "<< fz1 << std::endl;
    std::cout << "force:fx2 fy2 fz2 = " << fx2 << " "<< fy2 << " "<< fz2 << std::endl;
    if (a == 2) std::cout << "difference: " << diff << std::endl;
    ok = false;
  }

  return ok;
}

//
// Converts one type of force array to another. Result is in "force"
//
template <typename T>
template <typename T2>
void Force<T>::convert(Force<T2>& force, hipStream_t stream) {

  assert(force.size() == this->size());

  if (force.stride() == this->stride()) {
    int nthread = 512;
    int nblock = (3*this->stride() - 1)/nthread + 1;
    reduce_force<T, T2>
      <<< nblock, nthread, 0, stream >>>(3*this->stride(), this->xyz(), force.xyz());
    cudaCheck(hipGetLastError());
  } else {
    int nthread = 512;
    int nblock = (this->size() - 1)/nthread + 1;
    reduce_force<T, T2>
      <<< nblock, nthread, 0, stream >>>(this->size(), this->stride(), this->xyz(),
					 force.stride(), force.xyz());
    cudaCheck(hipGetLastError());
  }
}

//
// Converts one type of force array to another. Result is in "force"
//
template <typename T>
template <typename T2, typename T3>
void Force<T>::convert_to(Force<T3>& force, hipStream_t stream) {

  assert(force.size() == this->size());
  assert(force.stride() == this->stride());
  assert(sizeof(T2) == sizeof(T3));

  int nthread = 512;
  int nblock = (3*this->stride() - 1)/nthread + 1;

  reduce_force<T, T2>
    <<< nblock, nthread, 0, stream >>>(3*this->stride(), this->xyz(), (T2 *)force.xyz());
  cudaCheck(hipGetLastError());
}

//
// Converts one type of force array to another. Result is in "this"
// NOTE: Only works when the size of the types T and T2 match
//
template <typename T>
template <typename T2>
void Force<T>::convert(hipStream_t stream) {

  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (3*this->stride() - 1)/nthread + 1;

  reduce_force<T, T2>
    <<< nblock, nthread, 0, stream >>>(3*this->stride(), this->xyz());
  cudaCheck(hipGetLastError());
}

//
// Converts one type of force array to another and adds force to the result.
// Result is in "this"
// NOTE: Only works when the size of the types T and T2 match
//
template <typename T>
template <typename T2, typename T3>
void Force<T>::convert_add(Force<T3>& force, hipStream_t stream) {
  assert(force.stride() == this->stride());
  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (3*this->stride() - 1)/nthread + 1;

  reduce_add_force<T, T2, T3>
    <<< nblock, nthread, 0, stream >>>(3*this->stride(), force.xyz(), this->xyz());
  cudaCheck(hipGetLastError());
}

//
// Adds non-strided force_data
//
template <typename T>
template <typename T2>
void Force<T>::add(float3 *force_data, int force_n, hipStream_t stream) {

  assert(force_n <= this->size());
  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (force_n - 1)/nthread + 1;

  add_nonstrided_force<<< nblock, nthread, 0, stream >>>
    (force_n, force_data, this->stride(), (double *)this->xyz());
  cudaCheck(hipGetLastError());
}

//
// Explicit instances of Force class
//
template class Force<long long int>;
template class Force<double>;
template class Force<float>;
template void Force<long long int>::convert<double>(hipStream_t stream);
template void Force<long long int>::convert_add<double>(Force<float>& force, hipStream_t stream);
template void Force<long long int>::convert<float>(Force<float>& force, hipStream_t stream);
template void Force<long long int>::convert<double>(Force<double>& force, hipStream_t stream);
template void Force<float>::convert_to<double>(Force<long long int>& force, hipStream_t stream);
template void Force<long long int>::add<double>(float3 *force_data, int force_n, hipStream_t stream);
