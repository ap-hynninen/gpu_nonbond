#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "XYZQ.h"

//
// XYZQ class method definitions
//
// (c) Antti-Pekka Hynninen, 2013, aphynninen@hotmail.com
//
//

//
// Copies x, y, z coordinates into xyzq -array
//
__global__ void set_xyz_kernel(const int ncoord, const int stride,
			       const double* __restrict__ xyz,
			       float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float x = xyz[tid];
    float y = xyz[tid + stride];
    float z = xyz[tid + stride*2];
    xyzq[tid].x = x;
    xyzq[tid].y = y;
    xyzq[tid].z = z;
  }
}

//##########################################################################################
//##########################################################################################
//##########################################################################################

int XYZQ::get_xyzq_len() {
  return ((ncoord-1)/align+1)*align;
}

//
// Class creator
//
XYZQ::XYZQ() {
  ncoord = 0;
  xyzq_len = 0;
  align = 32;
  xyzq = NULL;
}

//
// Class creator
//
XYZQ::XYZQ(int ncoord, int align) : ncoord(ncoord), align(align) {
  xyzq_len = get_xyzq_len();
  allocate<float4>(&xyzq, xyzq_len);
}

//
// Class creator
//
XYZQ::XYZQ(const char *filename, int align) : align(align) {
  
  std::ifstream file(filename);
  if (file.is_open()) {
    
    float x, y, z, q;
    
    // Count number of coordinates
    ncoord = 0;
    while (file >> x >> y >> z >> q) ncoord++;

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);
    
    // Allocate CPU memory
    float4 *xyzq_cpu = new float4[ncoord];
    
    // Read coordinates
    int i=0;
    while (file >> xyzq_cpu[i].x >> xyzq_cpu[i].y >> xyzq_cpu[i].z >> xyzq_cpu[i].w) i++;
    
    // Allocate GPU memory
    xyzq_len = get_xyzq_len();
    allocate<float4>(&xyzq, xyzq_len);

    // Copy coordinates from CPU to GPU
    copy_HtoD<float4>(xyzq_cpu, xyzq, ncoord);

    // Deallocate CPU memory
    delete [] xyzq_cpu;
    
  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }
  
}

//
// Class destructor
//
XYZQ::~XYZQ() {
  if (xyzq != NULL) deallocate<float4>(&xyzq);
}

//
// Set ncoord
//
void XYZQ::set_ncoord(int ncoord, float fac) {
  this->ncoord = ncoord;
  int req_xyzq_len = get_xyzq_len();
  
  reallocate<float4>(&xyzq, &xyzq_len, req_xyzq_len, fac);
}

//
// Copies xyzq from host
// NOTE: Does not reallocate xyzq
//
void XYZQ::set_xyzq(int ncopy, float4 *h_xyzq, size_t offset, hipStream_t stream) {
  copy_HtoD<float4>(&h_xyzq[offset], &xyzq[offset], ncopy, stream);
}

//
// Copies xyz (on device) into the coordinate slots
//
void XYZQ::set_xyz(const double *xyz, const int stride, hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyz_kernel<<< nblock, nthread, 0, stream >>>(ncoord, stride, xyz, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Compares two XYZQ arrays
//
bool XYZQ::compare(XYZQ& xyzq_in, const double tol, double& max_diff) {
  assert(xyzq_in.ncoord == ncoord);

  float4 *h_xyzq = new float4[ncoord];
  float4 *h_xyzq_in = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);
  copy_DtoH<float4>(xyzq_in.xyzq, h_xyzq_in, ncoord);

  bool ok = true;

  max_diff = 0.0;
  int i;
  double dx, dy, dz, dq;
  double diff;
  try {
    for (i=0;i < ncoord;i++) {
      dx = fabs(h_xyzq[i].x - h_xyzq_in[i].x);
      dy = fabs(h_xyzq[i].y - h_xyzq_in[i].y);
      dz = fabs(h_xyzq[i].z - h_xyzq_in[i].z);
      dq = fabs(h_xyzq[i].w - h_xyzq_in[i].w);
      diff = max(dx, max(dy, dz));
      max_diff = max(max_diff, diff);
      if (diff > tol || dq > 0.0) throw 1;
    }
  }
  catch (int a) {
    std::cout << "i = " << i << std::endl;
    std::cout << "this: x,y,z,q = " << h_xyzq[i].x << " " << h_xyzq[i].y
	      << " " << h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
    std::cout << "in  : x,y,z,q = " << h_xyzq_in[i].x << " " << h_xyzq_in[i].y
	      << " " << h_xyzq_in[i].z << " " << h_xyzq_in[i].w << std::endl;
    ok = false;
  }

  delete [] h_xyzq;
  delete [] h_xyzq_in;

  return ok;
}
