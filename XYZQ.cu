#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "XYZQ.h"

//
// XYZQ class method definitions
//
// (c) Antti-Pekka Hynninen, 2013, aphynninen@hotmail.com
//
//

//
// Copies x, y, z coordinates into xyzq -array
//
__global__ void set_xyz_kernel(const int ncoord, const int stride,
			       const double* __restrict__ xyz,
			       float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float x = xyz[tid];
    float y = xyz[tid + stride];
    float z = xyz[tid + stride*2];
    xyzq[tid].x = x;
    xyzq[tid].y = y;
    xyzq[tid].z = z;
  }
}

//
// Copies (x, y, z, q) into xyzq -array
//
__global__ void set_xyzq_kernel(const int ncoord, const int stride,
				const double* __restrict__ xyz,
				const float* __restrict__ q,
				float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    xyzq_val.x = xyz[tid];
    xyzq_val.y = xyz[tid + stride];
    xyzq_val.z = xyz[tid + stride*2];
    xyzq_val.w = q[tid];
    xyzq[tid] = xyzq_val;
  }
}

//
// Copies (x, y, z, q) into xyzq -array and also shifts (x, y, z)
//
__global__ void set_xyzq_shift_kernel(const int ncoord, const int stride,
				      const double* __restrict__ xyz,
				      const float* __restrict__ q,
				      const int* __restrict__ loc2glo,
				      const float3* __restrict__ xyz_shift,
				      const double boxx, const double boxy, const double boxz,
				      float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    float3 shift = xyz_shift[tid];
    xyzq_val.x = (float)(xyz[tid]            + ((double)shift.x)*boxx);
    xyzq_val.y = (float)(xyz[tid + stride]   + ((double)shift.y)*boxy);
    xyzq_val.z = (float)(xyz[tid + stride*2] + ((double)shift.z)*boxz);
    xyzq_val.w = q[loc2glo[tid]];
    xyzq[tid] = xyzq_val;
  }
}

//
// Copies (x, y, z) into xyzq -array and also shifts (x, y, z)
//
__global__ void set_xyz_shift_kernel(const int ncoord, const int stride,
				     const double* __restrict__ xyz,
				     const float3* __restrict__ xyz_shift,
				     const double boxx, const double boxy, const double boxz,
				     float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    float3 shift = xyz_shift[tid];
    xyzq_val.x = (float)(xyz[tid]            + ((double)shift.x)*boxx);
    xyzq_val.y = (float)(xyz[tid + stride]   + ((double)shift.y)*boxy);
    xyzq_val.z = (float)(xyz[tid + stride*2] + ((double)shift.z)*boxz);
    xyzq[tid].x = xyzq_val.x;
    xyzq[tid].y = xyzq_val.y;
    xyzq[tid].z = xyzq_val.z;
  }
}

//##########################################################################################
//##########################################################################################
//##########################################################################################

//
// Return xyzq length that has extra align:
// ncoord-1 = last possible index
//
int XYZQ::get_xyzq_len() {
  return (ncoord-1 + align);
}

//
// Class creator
//
XYZQ::XYZQ() {
  ncoord = 0;
  xyzq_len = 0;
  align = 32;
  xyzq = NULL;
}

//
// Class creator
//
XYZQ::XYZQ(int ncoord, int align) : ncoord(ncoord), align(align) {
  xyzq_len = get_xyzq_len();
  allocate<float4>(&xyzq, xyzq_len);
}

//
// Class creator
//
XYZQ::XYZQ(const char *filename, int align) : align(align) {
  
  std::ifstream file(filename);
  if (file.is_open()) {
    
    float x, y, z, q;
    
    // Count number of coordinates
    ncoord = 0;
    while (file >> x >> y >> z >> q) ncoord++;

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);
    
    // Allocate CPU memory
    float4 *xyzq_cpu = new float4[ncoord];
    
    // Read coordinates
    int i=0;
    while (file >> xyzq_cpu[i].x >> xyzq_cpu[i].y >> xyzq_cpu[i].z >> xyzq_cpu[i].w) i++;
    
    // Allocate GPU memory
    xyzq_len = get_xyzq_len();
    allocate<float4>(&xyzq, xyzq_len);

    // Copy coordinates from CPU to GPU
    copy_HtoD<float4>(xyzq_cpu, xyzq, ncoord);

    // Deallocate CPU memory
    delete [] xyzq_cpu;
    
  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }
  
}

//
// Class destructor
//
XYZQ::~XYZQ() {
  if (xyzq != NULL) deallocate<float4>(&xyzq);
}

//
// Set ncoord
//
void XYZQ::set_ncoord(int ncoord, float fac) {
  this->ncoord = ncoord;
  int req_xyzq_len = get_xyzq_len();
  
  reallocate<float4>(&xyzq, &xyzq_len, req_xyzq_len, fac);
}

//
// Copies xyzq from host
// NOTE: Does not reallocate xyzq
//
void XYZQ::set_xyzq(int ncopy, float4 *h_xyzq, size_t offset, hipStream_t stream) {
  copy_HtoD<float4>(&h_xyzq[offset], &xyzq[offset], ncopy, stream);
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyzq(const cudaXYZ<double> *coord, const float *q, hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyzq_kernel<<< nblock, nthread, 0, stream >>>(coord->n, coord->stride, coord->data, q, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyzq(const cudaXYZ<double> *coord, const float *q, const int *loc2glo,
		    const float3 *xyz_shift,
		    const double boxx, const double boxy, const double boxz, hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyzq_shift_kernel<<< nblock, nthread, 0, stream >>>(coord->n, coord->stride, coord->data, q,
							  loc2glo, xyz_shift, boxx, boxy, boxz, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z (on device) into the coordinate slots
//
void XYZQ::set_xyz(const cudaXYZ<double> *coord, hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyz_kernel<<< nblock, nthread, 0, stream >>>(coord->n, coord->stride, coord->data, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyz(const cudaXYZ<double> *coord, const float3 *xyz_shift,
		    const double boxx, const double boxy, const double boxz, hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyz_shift_kernel<<< nblock, nthread, 0, stream >>>(coord->n, coord->stride, coord->data,
							 xyz_shift, boxx, boxy, boxz, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Compares two XYZQ arrays
//
bool XYZQ::compare(XYZQ& xyzq_in, const double tol, double& max_diff) {
  assert(xyzq_in.ncoord == ncoord);

  float4 *h_xyzq = new float4[ncoord];
  float4 *h_xyzq_in = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);
  copy_DtoH<float4>(xyzq_in.xyzq, h_xyzq_in, ncoord);

  bool ok = true;

  max_diff = 0.0;
  int i;
  double dx, dy, dz, dq;
  double diff;
  try {
    for (i=0;i < ncoord;i++) {
      dx = fabs(h_xyzq[i].x - h_xyzq_in[i].x);
      dy = fabs(h_xyzq[i].y - h_xyzq_in[i].y);
      dz = fabs(h_xyzq[i].z - h_xyzq_in[i].z);
      dq = fabs(h_xyzq[i].w - h_xyzq_in[i].w);
      diff = max(dx, max(dy, dz));
      max_diff = max(max_diff, diff);
      if (diff > tol || dq > 0.0) throw 1;
    }
  }
  catch (int a) {
    std::cout << "i = " << i << std::endl;
    std::cout << "this: x,y,z,q = " << h_xyzq[i].x << " " << h_xyzq[i].y
	      << " " << h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
    std::cout << "in  : x,y,z,q = " << h_xyzq_in[i].x << " " << h_xyzq_in[i].y
	      << " " << h_xyzq_in[i].z << " " << h_xyzq_in[i].w << std::endl;
    ok = false;
  }

  delete [] h_xyzq;
  delete [] h_xyzq_in;

  return ok;
}

void XYZQ::print(const int start, const int end, std::ostream& out) {

  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);

  for (int i=start;i <= end;i++) {
    out << i << " " << h_xyzq[i].x << " " << h_xyzq[i].y << " "
	<< h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
  }

  delete [] h_xyzq;
}
