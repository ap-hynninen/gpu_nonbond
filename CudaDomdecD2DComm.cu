#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/transform.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/gather.h>
#include "CudaDomdecD2DComm.h"
#include "mpi_utils.h"
#include "hostXYZ.h"

//################################################################################
//################################################################################
//################################################################################

CudaDomdecD2DComm::CudaDomdecD2DComm(Domdec& domdec, CudaMPI& cudaMPI) : 
  DomdecD2DComm(domdec), cudaMPI(cudaMPI) {

  // Send
  sendbuf_len = 0;
  sendbuf = NULL;

  h_sendbuf_len = 0;
  h_sendbuf = NULL;

  z_send_loc.resize(nz_comm);
  
  // Recv
  recvbuf_len = 0;
  recvbuf = NULL;

  h_recvbuf_len = 0;
  h_recvbuf = NULL;

  // MPI requests
  //int max_n_comm = std::max(std::max(nx_comm,ny_comm), nz_comm);
  //request.reserve(2*max_n_comm);

}

CudaDomdecD2DComm::~CudaDomdecD2DComm() {
  if (sendbuf != NULL) deallocate<char>(&sendbuf);
  if (h_sendbuf != NULL) deallocate_host<char>(&h_sendbuf);
  if (recvbuf != NULL) deallocate<char>(&recvbuf);
  if (h_recvbuf != NULL) deallocate_host<char>(&h_recvbuf);
}

struct z_pick_functor {
  const double zb, inv_boxz;

  z_pick_functor(double zb, double inv_boxz) : zb(zb), inv_boxz(inv_boxz) {}

  //
  // Returns 1 if coordinate is within the z boundary (zb)
  //
  __host__ __device__ unsigned char operator()(const double& z) const {
    double zf = z*inv_boxz + 0.5;
    zf -= floor(zf);
    // Now zf = (0.0 ... 1.0)
    return (zf < zb);
  }

};

inline int alignInt(const int pos, const int align) {
  return ((pos-1)/align+1)*align;
}

//
// Communicate coordinates
//
void CudaDomdecD2DComm::comm_coord(cudaXYZ<double>& coord, thrust::device_vector<int>& loc2glo,
				   const bool update) {

  double rnl = domdec.get_rnl();
  double inv_boxx = domdec.get_inv_boxx();
  double inv_boxy = domdec.get_inv_boxy();
  double inv_boxz = domdec.get_inv_boxz();
  int homeix = domdec.get_homeix();
  int homeiy = domdec.get_homeiy();
  int homeiz = domdec.get_homeiz();

  const int COUNT_TAG = 1, DATA_TAG = 2;
  
  // Size of each buffer elements
  const int buf_elem_size = update ? (sizeof(int) + 3*sizeof(double)) : (3*sizeof(double));

  // Resize arrays
  if (nx_comm + ny_comm + nz_comm > 0 && update) {
    atom_pick.resize(coord.n+1);
    atom_pos.resize(coord.n+1);
  }

  /*
  if (domdec.get_mynode() == 0) {
    hostXYZ<double> h_coord(coord);
    for (int i=0;i < 20;i++) {
      double z = h_coord.data[i+h_coord.stride*2];
      double zf = z*inv_boxz + 0.5;
      zf -= floor(zf);
      fprintf(stderr,"%d %lf %lf\n",i,z,zf);
    }
  }
  */

  if (nz_comm > 0) {

    double rnl_grouped = rnl;
    int pos = 0;
    z_psend.at(0) = 0;
    for (int i=0;i < nz_comm;i++) {
      
      if (update) {
	// Neighborlist has been updated => update communicated atoms
	double zf;
	get_fz_boundary(homeix, homeiy, homeiz-(i+1), rnl, rnl_grouped, zf);
	//if (homeiz-(i+1) < 0) zf -= 1.0;

	fprintf(stderr,"%d: homeiz=%d zf=%lf\n",domdec.get_mynode(),homeiz,zf);

	// Get pointer to z coordinates
	thrust::device_ptr<double> z_ptr(&coord.data[coord.stride*2]);

	// Pick atoms that are in the communication region
	thrust::transform(z_ptr, z_ptr + coord.n, atom_pick.begin(),
			  z_pick_functor(zf + rnl*inv_boxz, inv_boxz));

	// atom_pick[] now contains atoms that are picked for z-communication
	// Exclusive cumulative sum to find picked atom positions
	thrust::exclusive_scan(atom_pick.begin(), atom_pick.end(), atom_pos.begin());
	
	// Count the number of atoms we are adding to the buffer
	z_nsend.at(i) = atom_pos[coord.n];
	z_psend.at(i+1) = z_psend.at(i) + z_nsend.at(i);

	fprintf(stderr,"%d: z_nsend=%d\n",domdec.get_mynode(),z_nsend.at(i));

	z_send_loc.at(i).resize(z_nsend.at(i));

	// atom_pos[] now contains position to store each atom
	// Scatter to produce packed atom index table
	thrust::scatter_if(thrust::make_counting_iterator(0),
			   thrust::make_counting_iterator(coord.n),
			   atom_pos.begin(), atom_pick.begin(),
			   z_send_loc.at(i).begin());
	
	// z_send_loc[i][] now contains the local indices of atoms

	// Re-allocate sendbuf if needed
	int req_sendbuf_len = pos + alignInt(z_nsend.at(i),2)*sizeof(int) + 
	  z_nsend.at(i)*3*sizeof(double);
	reallocate<char>(&sendbuf, &sendbuf_len, req_sendbuf_len, 1.5f);

	//	fprintf(stderr,"%d: pos=%d size=%d buflen=%d\n",domdec.get_mynode(),
	//	pos,z_send_loc.at(i).size()*sizeof(int),sendbuf_len);

	// Get int pointer to sendbuf
	thrust::device_ptr<int> sendbuf_ind_ptr((int *)&sendbuf[pos]);
	
	// Pack in atom global indices to sendbuf[]
	thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(),
		       loc2glo.begin(), sendbuf_ind_ptr);

	// Advance sendbuf position
	pos += alignInt(z_nsend.at(i),2)*sizeof(int);
      }

      // Get double pointer to send buffer
      thrust::device_ptr<double> sendbuf_xyz_ptr((double *)&sendbuf[pos]);

      // Get pointer to coordinates
      thrust::device_ptr<double> xyz_ptr(&coord.data[0]);

      fprintf(stderr,"%d: pos=%d size=%d buflen=%d\n",domdec.get_mynode(),
	      pos,z_send_loc.at(i).size()*3*sizeof(double),sendbuf_len);
      
      // Pack in coordinates to sendbuf[]
      thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(),
		     xyz_ptr, sendbuf_xyz_ptr);

      thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(), xyz_ptr + coord.stride,
		     sendbuf_xyz_ptr + z_nsend.at(i));

      thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(), xyz_ptr + coord.stride*2,
		     sendbuf_xyz_ptr + 2*z_nsend.at(i));

      pos += z_nsend[i]*3*sizeof(double);
    } // for (int i=1;i < nz_comm;i++)

    // Compute byte positions
    computeByteNumPos(nz_comm, z_nsend, nsend, psend, update);
    if (pos != psend.at(nz_comm)) {
      std::cout << "CudaDomdecD2DComm::comm_coord, invalid pos (z)" << std::endl;
      exit(1);
    }

    if (update) {
      // Re-allocate h_sendbuf if needed
      if (!cudaMPI.isCudaAware()) {
	reallocate_host<char>(&h_sendbuf, &h_sendbuf_len, psend.at(nz_comm), 1.2f);
      }
      // Send & receive data counts
      for (int i=0;i < nz_comm;i++) {
	fprintf(stderr,"%d: nsend=%d send_node=%d recv_node=%d\n",domdec.get_mynode(),z_nsend.at(i),
		z_send_node.at(i),z_recv_node.at(i));
	MPICheck(MPI_Sendrecv(&z_nsend.at(i), 1, MPI_INT, z_send_node.at(i), COUNT_TAG,
			      &z_nrecv.at(i), 1, MPI_INT, z_recv_node.at(i), COUNT_TAG,
			      cudaMPI.get_comm(), MPI_STATUS_IGNORE));
      }
      // Compute positions
      z_precv.at(0) = 0;
      for (int i=0;i < nz_comm;i++) z_precv.at(i+1) = z_precv.at(i) + z_nrecv.at(i);
    }

    // Compute byte positions
    computeByteNumPos(nz_comm, z_nrecv, nrecv, precv, update);

    if (update) {
      // Re-allocate receive buffers
      reallocate<char>(&recvbuf, &recvbuf_len, precv.at(nz_comm), 1.2f);
      if (!cudaMPI.isCudaAware()) {
	reallocate_host<char>(&h_recvbuf, &h_recvbuf_len, precv.at(nz_comm), 1.2f);
      }
    }

    // Send & Recv data
    for (int i=0;i < nz_comm;i++) {
      if (nsend.at(i) > 0 && nrecv.at(i) > 0) {
	MPICheck(cudaMPI.Sendrecv(&sendbuf[psend.at(i)], nsend.at(i),
				  z_send_node.at(i), DATA_TAG,
				  &recvbuf[precv.at(i)], nrecv.at(i),
				  z_recv_node.at(i), DATA_TAG, MPI_STATUS_IGNORE,
				  &h_sendbuf[psend.at(i)], &h_recvbuf[precv.at(i)]));

      } else if (nsend.at(i) > 0) {
	MPICheck(cudaMPI.Send(&sendbuf[psend.at(i)], nsend.at(i),
			      z_send_node.at(i), DATA_TAG, &h_sendbuf[psend.at(i)]));
      } else if (nrecv.at(i) > 0) {
	MPICheck(cudaMPI.Recv(&recvbuf[precv.at(i)], nrecv.at(i),
			      z_recv_node.at(i), DATA_TAG, MPI_STATUS_IGNORE,
			      &h_recvbuf[precv.at(i)]));
      }
    }    

    /*
    int nrequest = 0;
    // Receive data
    for (int i=0;i < nz_comm;i++) {
      if (z_nrecv[i] > 0) {
	MPICheck(cudaMPI.Irecv(&recvbuf[z_precv[i]], z_nrecv[i]*buf_elem_size, z_recv_node[i], DATA_TAG,
			       &request[nrequest++], &h_recvbuf[z_precv[i]]));
      }
    }

    // Send data
    for (int i=0;i < nz_comm;i++) {
      if (z_nsend[i] > 0) {
	MPICheck(cudaMPI.Isend(&sendbuf[z_psend[i]], z_nsend[i]*buf_elem_size, z_send_node[i], DATA_TAG,
			       &request[nrequest++], &h_sendbuf[z_psend[i]]));
      }
    }

    // For for send and receive to finish
    MPICheck(MPI_Waitall(nrequest, request.data(), MPI_STATUSES_IGNORE));
    */

    // Unpack data from +z-direction into correct arrays
    for (int i=0;i < nz_comm;i++) {
      if (update) {
      }
    }

  } // if (nz_comm > 0)
  
  if (ny_comm > 0) {
    std::cout << "CudaDomdecD2DComm::comm_coord, y-communication not yet implemented" << std::endl;
    exit(1);
  }

  if (nx_comm > 0) {
    std::cout << "CudaDomdecD2DComm::comm_coord, x-communication not yet implemented" << std::endl;
    exit(1);
  }

}

void CudaDomdecD2DComm::computeByteNumPos(const int nc_comm, std::vector<int>& c_nsend,
					  std::vector<int>& nsend, std::vector<int>& psend,
					  const bool update) {
  nsend.resize(nc_comm);
  psend.resize(nc_comm+1);
  psend.at(0) = 0;
  if (update) {
    for (int i=0;i < nc_comm;i++) {
      nsend.at(i) = alignInt(c_nsend.at(i),2)*sizeof(int) + c_nsend.at(i)*3*sizeof(double);
      psend.at(i+1) = psend.at(i) + nsend.at(i);
    }
  } else {
    for (int i=0;i < nc_comm;i++) {
      nsend.at(i) = c_nsend.at(i)*3*sizeof(double);
      psend.at(i+1) = psend.at(i) + nsend.at(i);
    }
  }
}
