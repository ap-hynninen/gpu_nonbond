#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/transform.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/gather.h>
#include "CudaDomdecD2DComm.h"
#include "mpi_utils.h"

//################################################################################
//################################################################################
//################################################################################

CudaDomdecD2DComm::CudaDomdecD2DComm(Domdec& domdec, CudaMPI& cudaMPI) : 
  DomdecD2DComm(domdec), cudaMPI(cudaMPI) {

  // Send
  sendbuf_len = 0;
  sendbuf = NULL;

  h_sendbuf_len = 0;
  h_sendbuf = NULL;

  z_send_loc.resize(nz_comm);
  
  // Recv
  recvbuf_len = 0;
  recvbuf = NULL;

  h_recvbuf_len = 0;
  h_recvbuf = NULL;

  // MPI requests
  //int max_n_comm = std::max(std::max(nx_comm,ny_comm), nz_comm);
  //request.reserve(2*max_n_comm);

}

CudaDomdecD2DComm::~CudaDomdecD2DComm() {
  if (sendbuf != NULL) deallocate<char>(&sendbuf);
  if (h_sendbuf != NULL) deallocate_host<char>(&h_sendbuf);
  if (recvbuf != NULL) deallocate<char>(&recvbuf);
  if (h_recvbuf != NULL) deallocate_host<char>(&h_recvbuf);
}

struct z_pick_functor {
  const double zb, inv_boxz;

  z_pick_functor(double zb, double inv_boxz) : zb(zb), inv_boxz(inv_boxz) {}

  //
  // Returns 1 if coordinate is within the z boundary (zb)
  //
  __host__ __device__ unsigned char operator()(const double& z) const {
    double zf = z*inv_boxz + 0.5;
    zf -= floor(zf);
    // Now zf = (0.0 ... 1.0)
    return (zf < zb);
  }

};

inline int alignInt(const int pos, const int align) {
  return ((pos-1)/align+1)*align;
}

//
// Communicate coordinates
//
void CudaDomdecD2DComm::comm_coord(cudaXYZ<double>& coord, thrust::device_vector<int>& loc2glo,
				   const bool update) {

  double rnl = domdec.get_rnl();
  double inv_boxx = domdec.get_inv_boxx();
  double inv_boxy = domdec.get_inv_boxy();
  double inv_boxz = domdec.get_inv_boxz();
  int homeix = domdec.get_homeix();
  int homeiy = domdec.get_homeiy();
  int homeiz = domdec.get_homeiz();

  const int COUNT_TAG = 1, DATA_TAG = 2;
  
  // Size of each buffer elements
  const int buf_elem_size = update ? (sizeof(int) + 3*sizeof(double)) : (3*sizeof(double));

  // Resize arrays
  if (nx_comm + ny_comm + nz_comm > 0 && update) {
    atom_pick.resize(coord.size()+1);
    atom_pos.resize(coord.size()+1);
  }

  if (nz_comm > 0) {

    double rnl_grouped = rnl;
    int pos = 0;
    z_psend.at(0) = 0;
    for (int i=0;i < nz_comm;i++) {
      
      if (update) {
	// Neighborlist has been updated => update communicated atoms
	double zf;
	get_fz_boundary(homeix, homeiy, homeiz-(i+1), rnl, rnl_grouped, zf);
	//if (homeiz-(i+1) < 0) zf -= 1.0;

	fprintf(stderr,"%d: homeiz=%d zf=%lf\n",domdec.get_mynode(),homeiz,zf);

	// Get pointer to z coordinates
	thrust::device_ptr<double> z_ptr(coord.z());

	// Pick atoms that are in the communication region
	thrust::transform(z_ptr, z_ptr + coord.size(), atom_pick.begin(),
			  z_pick_functor(zf + rnl*inv_boxz, inv_boxz));

	// atom_pick[] now contains atoms that are picked for z-communication
	// Exclusive cumulative sum to find picked atom positions
	thrust::exclusive_scan(atom_pick.begin(), atom_pick.end(), atom_pos.begin());
	
	// Count the number of atoms we are adding to the buffer
	z_nsend.at(i) = atom_pos[coord.size()];
	z_psend.at(i+1) = z_psend.at(i) + z_nsend.at(i);

	z_send_loc.at(i).resize(z_nsend.at(i));

	// atom_pos[] now contains position to store each atom
	// Scatter to produce packed atom index table
	thrust::scatter_if(thrust::make_counting_iterator(0),
			   thrust::make_counting_iterator(coord.size()),
			   atom_pos.begin(), atom_pick.begin(),
			   z_send_loc.at(i).begin());
	
	// z_send_loc[i][] now contains the local indices of atoms

	// Re-allocate sendbuf if needed
	int req_sendbuf_len = pos + alignInt(z_nsend.at(i),2)*sizeof(int) + 
	  z_nsend.at(i)*3*sizeof(double);
	reallocate<char>(&sendbuf, &sendbuf_len, req_sendbuf_len, 1.5f);

	// Get int pointer to sendbuf
	thrust::device_ptr<int> sendbuf_ind_ptr((int *)&sendbuf[pos]);
	
	// Pack in atom global indices to sendbuf[]
	thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(),
		       loc2glo.begin(), sendbuf_ind_ptr);

	// Advance sendbuf position
	pos += alignInt(z_nsend.at(i),2)*sizeof(int);
      }

      // Get double pointer to send buffer
      thrust::device_ptr<double> sendbuf_xyz_ptr((double *)&sendbuf[pos]);

      // Pack in coordinates to sendbuf[]
      thrust::device_ptr<double> x_ptr(coord.x());
      thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(),
		     x_ptr, sendbuf_xyz_ptr);

      thrust::device_ptr<double> y_ptr(coord.y());
      thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(),
		     y_ptr, sendbuf_xyz_ptr + z_nsend.at(i));

      thrust::device_ptr<double> z_ptr(coord.z());
      thrust::gather(z_send_loc.at(i).begin(), z_send_loc.at(i).end(),
		     z_ptr, sendbuf_xyz_ptr + 2*z_nsend.at(i));

      pos += z_nsend[i]*3*sizeof(double);
    } // for (int i=1;i < nz_comm;i++)

    // Compute byte positions
    computeByteNumPos(nz_comm, z_nsend, nsend, psend, update);
    if (pos != psend.at(nz_comm)) {
      std::cout << "CudaDomdecD2DComm::comm_coord, invalid pos (z)" << std::endl;
      exit(1);
    }

    if (update) {
      // Re-allocate h_sendbuf if needed
      if (!cudaMPI.isCudaAware()) {
	reallocate_host<char>(&h_sendbuf, &h_sendbuf_len, psend.at(nz_comm), 1.2f);
      }
      // Send & receive data counts
      for (int i=0;i < nz_comm;i++) {
	MPICheck(MPI_Sendrecv(&z_nsend.at(i), 1, MPI_INT, z_send_node.at(i), COUNT_TAG,
			      &z_nrecv.at(i), 1, MPI_INT, z_recv_node.at(i), COUNT_TAG,
			      cudaMPI.get_comm(), MPI_STATUS_IGNORE));
      }
      // Compute positions
      z_precv.at(0) = 0;
      for (int i=0;i < nz_comm;i++) z_precv.at(i+1) = z_precv.at(i) + z_nrecv.at(i);
    }

    // Compute byte positions
    computeByteNumPos(nz_comm, z_nrecv, nrecv, precv, update);

    if (update) {
      // Re-allocate receive buffers
      reallocate<char>(&recvbuf, &recvbuf_len, precv.at(nz_comm), 1.2f);
      if (!cudaMPI.isCudaAware()) {
	reallocate_host<char>(&h_recvbuf, &h_recvbuf_len, precv.at(nz_comm), 1.2f);
      }
      z_recv_ind.resize(precv.at(nz_comm));
      coord.resize(coord.size()+precv.at(nz_comm));
    }

    // Send & Recv data
    for (int i=0;i < nz_comm;i++) {
      if (nsend.at(i) > 0 && nrecv.at(i) > 0) {
	MPICheck(cudaMPI.Sendrecv(&sendbuf[psend.at(i)], nsend.at(i),
				  z_send_node.at(i), DATA_TAG,
				  &recvbuf[precv.at(i)], nrecv.at(i),
				  z_recv_node.at(i), DATA_TAG, MPI_STATUS_IGNORE,
				  &h_sendbuf[psend.at(i)], &h_recvbuf[precv.at(i)]));

      } else if (nsend.at(i) > 0) {
	MPICheck(cudaMPI.Send(&sendbuf[psend.at(i)], nsend.at(i),
			      z_send_node.at(i), DATA_TAG, &h_sendbuf[psend.at(i)]));
      } else if (nrecv.at(i) > 0) {
	MPICheck(cudaMPI.Recv(&recvbuf[precv.at(i)], nrecv.at(i),
			      z_recv_node.at(i), DATA_TAG, MPI_STATUS_IGNORE,
			      &h_recvbuf[precv.at(i)]));
      }
    }    

    //----------------------------------------------------
    // Unpack data from +z-direction into correct arrays
    //----------------------------------------------------
    // Position where we start adding coordinates
    int cpos = domdec.get_zone_pcoord()[0];
    for (int i=0;i < nz_comm;i++) {
      int pos = 0;
      int src_pos = precv.at(i);
      if (update) {
	// Copy coordinates indices to z_recv_ind
	// format = indices[alignInt(nrecv.at(i),2) x int]
	thrust::device_ptr<double> ind_ptr((double *)&recvbuf[src_pos]);
	thrust::copy(ind_ptr, ind_ptr+z_nrecv.at(i), z_recv_ind.begin()+pos);
	pos += z_nrecv.at(i);
	src_pos += alignInt(z_nrecv.at(i),2)*sizeof(int);
      }
      // Unpack coordinates
      // format = X[nrecv.at(i) x double] | Y[nrecv.at(i) x double] | Z[nrecv.at(i) x double]
      
      thrust::device_ptr<double> x_src_ptr((double *)&recvbuf[src_pos]);
      thrust::device_ptr<double> x_dst_ptr(coord.x() + cpos);
      thrust::copy(x_src_ptr, x_src_ptr+z_nrecv.at(i), x_dst_ptr);
      src_pos += z_nrecv.at(i)*sizeof(double);

      thrust::device_ptr<double> y_src_ptr((double *)&recvbuf[src_pos]);
      thrust::device_ptr<double> y_dst_ptr(coord.y() + cpos);
      thrust::copy(y_src_ptr, y_src_ptr+z_nrecv.at(i), y_dst_ptr);
      src_pos += z_nrecv.at(i)*sizeof(double);

      thrust::device_ptr<double> z_src_ptr((double *)&recvbuf[src_pos]);
      thrust::device_ptr<double> z_dst_ptr(coord.z() + cpos);
      thrust::copy(z_src_ptr, z_src_ptr+z_nrecv.at(i), z_dst_ptr);
      src_pos += z_nrecv.at(i)*sizeof(double);
      cpos += z_nrecv.at(i);
    }
  } // if (nz_comm > 0)
  
  if (ny_comm > 0) {
    std::cout << "CudaDomdecD2DComm::comm_coord, y-communication not yet implemented" << std::endl;
    exit(1);
  }

  if (nx_comm > 0) {
    std::cout << "CudaDomdecD2DComm::comm_coord, x-communication not yet implemented" << std::endl;
    exit(1);
  }

}

void CudaDomdecD2DComm::computeByteNumPos(const int nc_comm, std::vector<int>& c_nsend,
					  std::vector<int>& nsend, std::vector<int>& psend,
					  const bool update) {
  nsend.resize(nc_comm);
  psend.resize(nc_comm+1);
  psend.at(0) = 0;
  if (update) {
    for (int i=0;i < nc_comm;i++) {
      nsend.at(i) = alignInt(c_nsend.at(i),2)*sizeof(int) + c_nsend.at(i)*3*sizeof(double);
      psend.at(i+1) = psend.at(i) + nsend.at(i);
    }
  } else {
    for (int i=0;i < nc_comm;i++) {
      nsend.at(i) = c_nsend.at(i)*3*sizeof(double);
      psend.at(i+1) = psend.at(i) + nsend.at(i);
    }
  }
}
