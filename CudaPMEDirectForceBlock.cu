#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <math.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "CudaPMEDirectForceBlock.h"

extern __constant__ DirectSettings_t d_setup;
extern __device__ DirectEnergyVirial_t d_energy_virial;

#ifndef USE_TEXTURE_OBJECTS
// VdW parameter texture reference
static texture<float2, 1, hipReadModeElementType> vdwparam_block_texref;
static bool vdwparam_block_texref_bound = false;
static texture<float2, 1, hipReadModeElementType> vdwparam14_block_texref;
static bool vdwparam14_block_texref_bound = false;
static texture<float, 1, hipReadModeElementType> blockparam_texref;
#endif

#ifndef USE_TEXTURE_OBJECTS
#define VDWPARAM_TEXREF vdwparam_block_texref
#define VDWPARAM14_TEXREF vdwparam14_block_texref
#endif

#define USE_BLOCK
#include "CudaDirectForce_util.h"
#undef USE_BLOCK

//########################################################################################
//########################################################################################
//########################################################################################

//
// Class creator
//
template <typename AT, typename CT>
CudaPMEDirectForceBlock<AT, CT>::CudaPMEDirectForceBlock(int nblock) {
  assert(nblock >= 1);
  blocktype_len = 0;
  blocktype = NULL;
  allocate<float>(&blockparam, nblock*(nblock+1)/2);
#ifdef USE_TEXTURE_OBJECTS
  blockparam_tex = 0;

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = blockparam;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = sizeof(CT)*8;
  resDesc.res.linear.sizeInBytes = nblock*(nblock+1)/2*sizeof(CT);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  hipCreateTextureObject(&blockparam_tex, &resDesc, &texDesc, NULL);
#else
  // Bind blockparam texture
  memset(&blockparam_texref, 0, sizeof(blockparam_texref));
  blockparam_texref.normalized = 0;
  blockparam_texref.filterMode = hipFilterModePoint;
  blockparam_texref.addressMode[0] = hipAddressModeClamp;
  blockparam_texref.channelDesc.x = 32;
  blockparam_texref.channelDesc.y = 0;
  blockparam_texref.channelDesc.z = 0;
  blockparam_texref.channelDesc.w = 0;
  blockparam_texref.channelDesc.f = hipChannelFormatKindFloat;
  cudaCheck(hipBindTexture(NULL, blockparam_texref, blockparam, nblock*(nblock+1)/2*sizeof(float)));
#endif
}

//
// Class destructor
//
template <typename AT, typename CT>
CudaPMEDirectForceBlock<AT, CT>::~CudaPMEDirectForceBlock() {
  if (blocktype != NULL) deallocate<int>(&blocktype);
#ifdef USE_TEXTURE_OBJECTS
  if (blockparam_tex != 0) hipDestroyTextureObject(blockparam_tex);
#else
  cudaCheck(hipUnbindTexture(blockparam_texref));
#endif
  deallocate<float>(&blockparam);
}

//
// Sets blocktype array from host memory
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::set_blocktype(const int ncoord, const int *h_blocktype) {
  // Align ncoord to warpsize
  int ncoord_aligned = ((ncoord-1)/warpsize+1)*warpsize;
  reallocate<int>(&blocktype, &blocktype_len, ncoord_aligned, 1.2f);
  copy_HtoD<int>(h_blocktype, blocktype, ncoord);
}

//
// Sets block parameters by copying them from CPU
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::set_blockparam(const CT *h_blockparam) {
  copy_HtoD<CT>(h_blockparam, blockparam, nblock*(nblock+1)/2);
}

//
// Calculates 1-4 exclusions and interactions
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::calc_14_force(const float4 *xyzq,
						    const bool calc_energy, const bool calc_virial,
						    const int stride, AT *force, hipStream_t stream) {

#ifdef USE_TEXTURE_OBJECTS
  if (this->vdwparam14_tex == 0) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_14_force, vdwparam14_tex must be created" << std::endl;
    exit(1);
  }
  //if (blockparam_tex == 0) {
  //std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_14_force, blockparam_tex must be created" << std::endl;
  //exit(1);
  //}
#else
  if (!vdwparam14_block_texref_bound) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_14_force, vdwparam14_block_texref must be bound"
	      << std::endl;
    exit(1);
  }
#endif

  int nthread = 512;
  int nin14block = (this->nin14list - 1)/nthread + 1;
  int nex14block = (this->nex14list - 1)/nthread + 1;
  int nblock = nin14block + nex14block;
  int shmem_size = 0;
  if (calc_energy) {
    shmem_size = nthread*sizeof(double2);
  }

  int vdw_model_loc = this->calc_vdw ? this->vdw_model : NONE;
  int elec_model_loc = this->calc_elec ? this->elec_model : NONE;
  if (elec_model_loc == NONE && vdw_model_loc == NONE) return;

#ifdef USE_TEXTURE_OBJECTS
  CREATE_KERNELS(CREATE_KERNEL14, calc_14_force_kernel, this->vdwparam14_tex,
		 this->nin14list, this->nex14list, nin14block, this->in14list, this->ex14list,
		 this->vdwtype, this->vdwparam14, xyzq, stride, force);
#else
  CREATE_KERNELS(CREATE_KERNEL14, calc_14_force_kernel,
		 this->nin14list, this->nex14list, nin14block, this->in14list, this->ex14list,
		 this->vdwtype, this->vdwparam14, xyzq, stride, force);
#endif

  cudaCheck(hipGetLastError());
}

//
// Calculates direct force
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::calc_force(const float4 *xyzq,
						 const CudaNeighborListBuild<32> *nlist,
						 const bool calc_energy,
						 const bool calc_virial,
						 const int stride, AT *force, hipStream_t stream) {

  const int tilesize = 32;

#ifdef USE_TEXTURE_OBJECTS
  if (this->vdwparam_tex == 0) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_force, vdwparam_tex must be created" << std::endl;
    exit(1);
  }
  if (blockparam_tex == 0) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_force, blockparam_tex must be created" << std::endl;
    exit(1);
  }
#else
  if (!vdwparam_block_texref_bound) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_force, vdwparam_block_texref must be bound"
	      << std::endl;
    exit(1);
  }
#endif

  if (nlist->get_n_ientry() == 0) return;
  int vdw_model_loc = this->calc_vdw ? this->vdw_model : NONE;
  int elec_model_loc = this->calc_elec ? this->elec_model : NONE;
  if (elec_model_loc == NONE && vdw_model_loc == NONE) return;

  int nwarp = 2;
  if (get_cuda_arch() < 300) {
    nwarp = 2;
  } else {
    nwarp = 4;
  }
  int nthread = warpsize*nwarp;
  int nblock_tot = (nlist->get_n_ientry()-1)/(nthread/warpsize)+1;

  int shmem_size = 0;
  // (sh_xi, sh_yi, sh_zi, sh_qi, sh_vdwtypei, sh_blocktypei)
  if (get_cuda_arch() < 300)
    shmem_size += (nthread/warpsize)*tilesize*(sizeof(float)*4 + sizeof(int) + sizeof(float));
  // (sh_fix, sh_fiy, sh_fiz)
  shmem_size += (nthread/warpsize)*warpsize*sizeof(AT)*3;
  // If no texture fetch for vdwparam:
  //shmem_size += nvdwparam*sizeof(float);

  if (calc_energy) shmem_size = max(shmem_size, (int)(nthread*sizeof(double)*2));
  if (calc_virial) shmem_size = max(shmem_size, (int)(nthread*sizeof(double)*3));

  int3 max_nblock3 = get_max_nblock();
  unsigned int max_nblock = max_nblock3.x;
  unsigned int base = 0;

  while (nblock_tot != 0) {

    int nblock = (nblock_tot > max_nblock) ? max_nblock : nblock_tot;
    nblock_tot -= nblock;

#ifdef USE_TEXTURE_OBJECTS
    CREATE_KERNELS(CREATE_KERNEL, calc_force_kernel, this->vdwparam_tex,
		   base, nlist->get_n_ientry(), nlist->get_ientry(), nlist->get_tile_indj(),
		   nlist->get_tile_excl(), stride, this->vdwparam, this->nvdwparam, xyzq, this->vdwtype,
		   this->blocktype, blockparam_tex, force);
#else
    CREATE_KERNELS(CREATE_KERNEL, calc_force_kernel,
		   base, nlist->get_n_ientry(), nlist->get_ientry(), nlist->get_tile_indj(),
		   nlist->get_tile_excl(), stride, this->vdwparam, this->nvdwparam, xyzq, this->vdwtype,
		   this->blocktype, force);
#endif

    base += (nthread/warpsize)*nblock;

    cudaCheck(hipGetLastError());
  }

}

//
// Explicit instances of CudaPMEDirectForceBlock
//
template class CudaPMEDirectForceBlock<long long int, float>;

#ifndef USE_TEXTURE_OBJECTS
#undef VDWPARAM_TEXREF
#undef VDWPARAM14_TEXREF
#endif
