#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <math.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"
#include "CudaPMEDirectForceBlock.h"

extern __constant__ DirectSettings_t d_setup;
extern __device__ DirectEnergyVirial_t d_energy_virial;

#define USE_BLOCK
#include "CudaDirectForce_util.h"
#undef USE_BLOCK

//########################################################################################
//########################################################################################
//########################################################################################

//
// Class creator
//
template <typename AT, typename CT>
CudaPMEDirectForceBlock<AT, CT>::CudaPMEDirectForceBlock(int nblock) {
  assert(nblock >= 1);
  blocktype_len = 0;
  blocktype = NULL;
  blockparam_tex = 0;
  allocate<float>(&blockparam, nblock*(nblock+1)/2);

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = blockparam;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = sizeof(CT)*8;
  resDesc.res.linear.sizeInBytes = nblock*(nblock+1)/2*sizeof(CT);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  hipCreateTextureObject(&blockparam_tex, &resDesc, &texDesc, NULL);

  // Bind blockparam texture
  //blockparam_texref.normalized = 0;
  //blockparam_texref.filterMode = hipFilterModePoint;
  //blockparam_texref.addressMode[0] = hipAddressModeClamp;
  //blockparam_texref.channelDesc.x = 32;
  //blockparam_texref.channelDesc.y = 0;
  //blockparam_texref.channelDesc.z = 0;
  //blockparam_texref.channelDesc.w = 0;
  //blockparam_texref.channelDesc.f = hipChannelFormatKindFloat;
  //cudaCheck(hipBindTexture(NULL, blockparam_texref, blockparam, nblock*(nblock+1)/2*sizeof(float)));
}

//
// Class destructor
//
template <typename AT, typename CT>
CudaPMEDirectForceBlock<AT, CT>::~CudaPMEDirectForceBlock() {
  if (blocktype != NULL) deallocate<int>(&blocktype);
  if (blockparam_tex != 0) hipDestroyTextureObject(blockparam_tex);
  // Unbind texture
  //cudaCheck(hipUnbindTexture(blockparam_texref));
  deallocate<float>(&blockparam);
}

//
// Sets blocktype array from host memory
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::set_blocktype(const int ncoord, const int *h_blocktype) {
  // Align ncoord to warpsize
  int ncoord_aligned = ((ncoord-1)/warpsize+1)*warpsize;
  reallocate<int>(&blocktype, &blocktype_len, ncoord_aligned, 1.2f);
  copy_HtoD<int>(h_blocktype, blocktype, ncoord);
}

//
// Sets block parameters by copying them from CPU
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::set_blockparam(const CT *h_blockparam) {
  copy_HtoD<CT>(h_blockparam, blockparam, nblock*(nblock+1)/2);
}

//
// Calculates 1-4 exclusions and interactions
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::calc_14_force(const float4 *xyzq,
						    const bool calc_energy, const bool calc_virial,
						    const int stride, AT *force, hipStream_t stream) {

  if (this->vdwparam14_tex == 0) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_14_force, vdwparam14_tex must be created" << std::endl;
    exit(1);
  }

  int nthread = 512;
  int nin14block = (this->nin14list - 1)/nthread + 1;
  int nex14block = (this->nex14list - 1)/nthread + 1;
  int nblock = nin14block + nex14block;
  int shmem_size = 0;
  if (calc_energy) {
    shmem_size = nthread*sizeof(double2);
  }

  int vdw_model_loc = this->calc_vdw ? this->vdw_model : NONE;
  int elec_model_loc = this->calc_elec ? this->elec_model : NONE;
  if (elec_model_loc == NONE && vdw_model_loc == NONE) return;

  CREATE_KERNELS(CREATE_KERNEL14, calc_14_force_kernel, this->vdwparam14_tex,
		 this->nin14list, this->nex14list, nin14block, this->in14list, this->ex14list,
		 this->vdwtype, this->vdwparam14, xyzq, stride, force);

  cudaCheck(hipGetLastError());
}

//
// Calculates direct force
//
template <typename AT, typename CT>
void CudaPMEDirectForceBlock<AT, CT>::calc_force(const float4 *xyzq,
						 const NeighborList<32> *nlist,
						 const bool calc_energy,
						 const bool calc_virial,
						 const int stride, AT *force, hipStream_t stream) {

  const int tilesize = 32;

  if (this->vdwparam_tex == 0) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_force, vdwparam_tex must be created" << std::endl;
    exit(1);
  }

  if (blockparam_tex == 0) {
    std::cerr << "CudaPMEDirectForceBlock<AT, CT>::calc_force, blockparam_tex must be created" << std::endl;
    exit(1);
  }

  if (nlist->n_ientry == 0) return;
  int vdw_model_loc = this->calc_vdw ? this->vdw_model : NONE;
  int elec_model_loc = this->calc_elec ? this->elec_model : NONE;
  if (elec_model_loc == NONE && vdw_model_loc == NONE) return;

  int nwarp = 2;
  if (get_cuda_arch() < 300) {
    nwarp = 2;
  } else {
    nwarp = 4;
  }
  int nthread = warpsize*nwarp;
  int nblock_tot = (nlist->n_ientry-1)/(nthread/warpsize)+1;

  int shmem_size = 0;
  // (sh_xi, sh_yi, sh_zi, sh_qi, sh_vdwtypei, sh_blocktypei)
  if (get_cuda_arch() < 300)
    shmem_size += (nthread/warpsize)*tilesize*(sizeof(float)*4 + sizeof(int) + sizeof(float));
  // (sh_fix, sh_fiy, sh_fiz)
  shmem_size += (nthread/warpsize)*warpsize*sizeof(AT)*3;
  // If no texture fetch for vdwparam:
  //shmem_size += nvdwparam*sizeof(float);

  if (calc_energy) shmem_size = max(shmem_size, (int)(nthread*sizeof(double)*2));
  if (calc_virial) shmem_size = max(shmem_size, (int)(nthread*sizeof(double)*3));

  int3 max_nblock3 = get_max_nblock();
  unsigned int max_nblock = max_nblock3.x;
  unsigned int base = 0;

  while (nblock_tot != 0) {

    int nblock = (nblock_tot > max_nblock) ? max_nblock : nblock_tot;
    nblock_tot -= nblock;

    CREATE_KERNELS(CREATE_KERNEL, calc_force_kernel, this->vdwparam_tex,
		   base, nlist->n_ientry, nlist->ientry, nlist->tile_indj,
		   nlist->tile_excl, stride, this->vdwparam, this->nvdwparam, xyzq, this->vdwtype,
		   this->blocktype, blockparam_tex, force);

    base += (nthread/warpsize)*nblock;

    cudaCheck(hipGetLastError());
  }

}

//
// Explicit instances of CudaPMEDirectForceBlock
//
template class CudaPMEDirectForceBlock<long long int, float>;
