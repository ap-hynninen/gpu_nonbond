#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include "CudaPMEForcefield.h"
#include "cuda_utils.h"
#include "gpu_utils.h"

__global__ void heuristic_check_kernel(const int ncoord, const int stride,
				       const double* __restrict__ coord,
				       const double* __restrict__ ref_coord,
				       const float rsq_limit,
				       int* global_flag) {
  // Required shared memory:
  // blockDim.x/warpsize*sizeof(int)
  extern __shared__ int sh_flag[];
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int stride2 = stride*2;
  const int sh_flag_size = blockDim.x/warpsize;

  float dx = 0.0f;
  float dy = 0.0f;
  float dz = 0.0f;
  if (tid < ncoord) {
    dx = (float)(coord[tid]         - ref_coord[tid]);
    dy = (float)(coord[tid+stride]  - ref_coord[tid+stride]);
    dz = (float)(coord[tid+stride2] - ref_coord[tid+stride2]);
  }

  float rsq = dx*dx + dy*dy + dz*dz;
  // flag = 1 update is needed
  //      = 0 no update needed
  int flag = (rsq > rsq_limit);
  // Reduce flag, packed into bits.
  // NOTE: this assumes that warpsize <= 32
  sh_flag[threadIdx.x/warpsize] = (flag << (threadIdx.x % warpsize));
  __syncthreads();
  if (threadIdx.x < sh_flag_size) {
    for (int d=1;d < sh_flag_size;d *= 2) {
      int t = threadIdx.x + d;
      int flag_val = (t < sh_flag_size) ? sh_flag[t] : 0;
      __syncthreads();
      sh_flag[threadIdx.x] |= flag_val;
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      int flag_val = sh_flag[0];
      atomicOr(global_flag, flag_val);
    }
  }

}

//############################################################################################
//############################################################################################
//############################################################################################

//
// Class creator
//
CudaPMEForcefield::CudaPMEForcefield(CudaDomdec& domdec, CudaDomdecBonded& domdec_bonded,
				     NeighborList<32>& nlist,
				     const int nbondcoef, const float2 *h_bondcoef,
				     const int nureybcoef, const float2 *h_ureybcoef,
				     const int nanglecoef, const float2 *h_anglecoef,
				     const int ndihecoef, const float4 *h_dihecoef,
				     const int nimdihecoef, const float4 *h_imdihecoef,
				     const int ncmapcoef, const float2 *h_cmapcoef,
				     const double roff, const double ron,
				     const double kappa, const double e14fac,
				     const int vdw_model, const int elec_model,
				     const int nvdwparam, const float *h_vdwparam,
				     const float *h_vdwparam14,
				     const int *h_glo_vdwtype, const float *h_glo_q,
				     CudaDomdecRecip* recip, CudaDomdecRecipComm& recipComm) : 
  domdec(domdec), recip(recip), domdec_bonded(domdec_bonded), nlist(nlist), recipComm(recipComm),
  kappa(kappa), recip_force_len(0), recip_force(NULL) {

  // Create streams
  cudaCheck(hipStreamCreate(&direct_stream[0]));
  cudaCheck(hipStreamCreate(&direct_stream[1]));
  cudaCheck(hipStreamCreate(&recip_stream));
  cudaCheck(hipStreamCreate(&in14_stream));
  cudaCheck(hipStreamCreate(&bonded_stream));

  // Create events
  cudaCheck(hipEventCreate(&done_direct_event));
  cudaCheck(hipEventCreate(&done_recip_event));
  cudaCheck(hipEventCreate(&done_in14_event));
  cudaCheck(hipEventCreate(&done_bonded_event));
  cudaCheck(hipEventCreate(&done_calc_event));

  // Set energy term flags
  calc_bond = true;
  calc_ureyb = true;
  calc_angle = true;
  calc_dihe = true;
  calc_imdihe = true;
  calc_cmap = true;

  // Bonded coefficients
  bonded.setup_coef(nbondcoef, h_bondcoef, nureybcoef, h_ureybcoef,
		    nanglecoef, h_anglecoef, ndihecoef, h_dihecoef,
		    nimdihecoef, h_imdihecoef, ncmapcoef, h_cmapcoef);
  
  // Direct non-bonded interactions
  setup_direct_nonbonded(roff, ron, kappa, e14fac, vdw_model, elec_model,
			 nvdwparam, h_vdwparam, h_vdwparam14, h_glo_vdwtype);

  // Set stream for reciprocal calculation
  if (recip != NULL) recip->set_stream(recip_stream);

  // Copy charges
  allocate<float>(&glo_q, domdec.get_ncoord_glo());
  copy_HtoD<float>(h_glo_q, glo_q, domdec.get_ncoord_glo());

  allocate<int>(&d_heuristic_flag, 1);
  allocate_host<int>(&h_heuristic_flag, 1);

  h_loc2glo_len = 0;
  h_loc2glo = NULL;
}

//
// Class destructor
//
CudaPMEForcefield::~CudaPMEForcefield() {
  deallocate<int>(&d_heuristic_flag);
  deallocate_host<int>(&h_heuristic_flag);
  deallocate<float>(&glo_q);
  deallocate<int>(&glo_vdwtype);
  if (recip_force != NULL) deallocate<float3>(&recip_force);
  if (h_loc2glo != NULL) delete [] h_loc2glo;
  // Destroy streams
  cudaCheck(hipStreamDestroy(direct_stream[0]));
  cudaCheck(hipStreamDestroy(direct_stream[1]));
  cudaCheck(hipStreamDestroy(recip_stream));
  cudaCheck(hipStreamDestroy(in14_stream));
  cudaCheck(hipStreamDestroy(bonded_stream));
  // Destroy events
  cudaCheck(hipEventDestroy(done_direct_event));
  cudaCheck(hipEventDestroy(done_recip_event));
  cudaCheck(hipEventDestroy(done_in14_event));
  cudaCheck(hipEventDestroy(done_bonded_event));
  cudaCheck(hipEventDestroy(done_calc_event));
}

//
// Setup direct non-bonded interactions.
//
void CudaPMEForcefield::setup_direct_nonbonded(const double roff, const double ron,
					       const double kappa, const double e14fac,
					       const int vdw_model, const int elec_model,
					       const int nvdwparam, const float *h_vdwparam,
					       const float *h_vdwparam14, const int *h_glo_vdwtype) {

  this->roff = roff;
  this->ron = ron;

  dir.setup(domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), kappa, roff, ron,
	    e14fac, vdw_model, elec_model);

  dir.set_vdwparam(nvdwparam, h_vdwparam);
  dir.set_vdwparam14(nvdwparam, h_vdwparam14);

  allocate<int>(&glo_vdwtype, domdec.get_ncoord_glo());
  copy_HtoD<int>(h_glo_vdwtype, glo_vdwtype, domdec.get_ncoord_glo());
}


//
// Pre-process force calculation
//
void CudaPMEForcefield::pre_calc(cudaXYZ<double>& coord, cudaXYZ<double>& prev_step) {

  // Check for neighborlist heuristic update
  if (heuristic_check(coord, direct_stream[0])) {
    neighborlist_updated = true;

    std::cout << "  Building neighborlist" << std::endl;

    // Update homezone coordinates (coord) and step vector (prev_step)
    // NOTE: Builds domdec.loc2glo
    domdec.update_homezone(coord, prev_step);

    fprintf(stderr,"%d: domdec.get_ncoord()=%d\n",domdec.get_mynode(),domdec.get_ncoord());

    // Communicate coordinates
    // NOTE: Builds rest of domdec.loc2glo and domdec.xyz_shift
    domdec.comm_coord(coord, true);

    return;

    // Copy: coord => xyzq_copy
    // NOTE: coord and xyz_shift are already in the order determined by domdec.loc2glo,
    //       however, glo_q is in the original global order.
    xyzq_copy.set_xyzq(coord, glo_q, domdec.get_loc2glo_ptr(), domdec.get_xyz_shift(),
		       domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz());

    // Sort coordinates
    // NOTE: Builds domdec.loc2glo and nlist->glo2loc
    nlist.sort(domdec.get_zone_pcoord(), xyzq_copy.xyzq, xyzq.xyzq, domdec.get_loc2glo_ptr());

    // Build neighborlist
    nlist.build(domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), domdec.get_rnl(),
		xyzq.xyzq, domdec.get_loc2glo_ptr());

    //nlist.test_build(domdec.get_zone_pcoord(), domdec.get_boxx(), domdec.get_boxy(),
    //domdec.get_boxz(), domdec.get_rnl(), xyzq.xyzq, domdec.get_loc2glo());

    // Build bonded tables
    domdec_bonded.build_tbl(&domdec, domdec.get_zone_pcoord());

    // Setup bonded interaction lists
    bonded.setup_list(xyzq.xyzq, domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
		      nlist.get_glo2loc(),
		      domdec_bonded.get_nbond_tbl(), domdec_bonded.get_bond_tbl(),
		      domdec_bonded.get_bond(),
		      domdec_bonded.get_nureyb_tbl(), domdec_bonded.get_ureyb_tbl(),
		      domdec_bonded.get_ureyb(),
		      domdec_bonded.get_nangle_tbl(), domdec_bonded.get_angle_tbl(),
		      domdec_bonded.get_angle(),
		      domdec_bonded.get_ndihe_tbl(), domdec_bonded.get_dihe_tbl(),
		      domdec_bonded.get_dihe(),
		      domdec_bonded.get_nimdihe_tbl(), domdec_bonded.get_imdihe_tbl(),
		      domdec_bonded.get_imdihe(),
		      domdec_bonded.get_ncmap_tbl(), domdec_bonded.get_cmap_tbl(),
		      domdec_bonded.get_cmap());

    // Set vdwtype for Direct non-bonded interactions
    dir.set_vdwtype(domdec.get_ncoord_tot(), glo_vdwtype, domdec.get_loc2glo_ptr());

    // Setup 1-4 interaction lists
    dir.set_14_list(xyzq.xyzq, domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
		    nlist.get_glo2loc(),
		    domdec_bonded.get_nin14_tbl(), domdec_bonded.get_in14_tbl(),
		    domdec_bonded.get_in14(),
		    domdec_bonded.get_nex14_tbl(), domdec_bonded.get_ex14_tbl(),
		    domdec_bonded.get_ex14());

    // Re-order prev_step vector:
    domdec.reorder_coord(prev_step, ref_coord, nlist.get_ind_sorted());
    prev_step.set_data(ref_coord);

    // Re-order coordinates (coord) and copy to reference coordinates (ref_coord)
    domdec.reorder_coord(coord, ref_coord, nlist.get_ind_sorted());
    coord.set_data(ref_coord);

  } else {
    neighborlist_updated = false;
    // Copy local coordinates to xyzq -array
    xyzq.set_xyz(coord, 0, domdec.get_ncoord()-1, domdec.get_xyz_shift(),
		 domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), direct_stream[0]);
    // Communicate coordinates between direct nodes
    domdec.comm_coord(coord, false);
    // Copy import volume coordinates to xyzq -array
    xyzq.set_xyz(coord, domdec.get_ncoord(), domdec.get_ncoord_glo()-1, domdec.get_xyz_shift(),
		 domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), direct_stream[0]);
  }

}

//
// Calculate forces
//
void CudaPMEForcefield::calc(const bool calc_energy, const bool calc_virial, Force<long long int>& force) {

  bool do_recipcomm = recipComm.get_hasPureRecip() || 
    (recipComm.get_num_recip() > 0  && recipComm.get_num_direct() > 1);

  if (do_recipcomm) {
    if (recipComm.get_isRecip() && recip == NULL) {
      std::cout << "CudaPMEForcefield::calc, missing recip object" << std::endl;
      exit(1);
    }
    //-------------------------------------
    // Send coordinates to recip node(s)
    //-------------------------------------
    // Send header
    if (recipComm.get_hasPureRecip()) {
      recipComm.send_header(domdec.get_ncoord(), domdec.get_inv_boxx(), domdec.get_inv_boxy(),
			    domdec.get_inv_boxz(), calc_energy, calc_virial);
    } else if (neighborlist_updated) {
      if (recipComm.get_isRecip()) {
	recipComm.recv_ncoord(domdec.get_ncoord());
      } else {
	recipComm.send_ncoord(domdec.get_ncoord());
      }
    }
    // Resize recip_xyzq and recip_force if needed
    if (recipComm.get_isRecip() && recipComm.get_num_direct() > 1) {
      recip_xyzq.set_ncoord(recipComm.get_ncoord());
    }
    reallocate<float3>(&recip_force, &recip_force_len, recipComm.get_ncoord(), 1.0f);
    // Send coordinates
    recipComm.send_coord(xyzq.xyzq);
    // Receive coordinates
    if (recipComm.get_isRecip()) recipComm.recv_coord(recip_xyzq.xyzq);
    //-------------------------------------
  }

  force.clear(direct_stream[0]);

  // Clear energy and virial variables
  if (calc_energy || calc_virial) {
    dir.clear_energy_virial();
    bonded.clear_energy_virial();
    if (recipComm.get_isRecip()) recip->clear_energy_virial();
  }

  // Direct non-bonded force
  dir.calc_force(xyzq.xyzq, nlist, calc_energy, calc_virial, force.xyz.stride, force.xyz.data,
		 direct_stream[0]);
  cudaCheck(hipEventRecord(done_direct_event, direct_stream[0]));

  // 1-4 interactions
  dir.calc_14_force(xyzq.xyzq, calc_energy, calc_virial, force.xyz.stride, force.xyz.data,
		    in14_stream);
  cudaCheck(hipEventRecord(done_in14_event, in14_stream));

  // Bonded forces
  bonded.calc_force(xyzq.xyzq, domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
  		    calc_energy, calc_virial, force.xyz.stride, force.xyz.data,
		    calc_bond, calc_ureyb, calc_angle, calc_dihe, calc_imdihe, calc_cmap,
		    bonded_stream);
  cudaCheck(hipEventRecord(done_bonded_event, bonded_stream));

  // Reciprocal force (Only reciprocal nodes calculate this)
  if (recipComm.get_isRecip()) {
    if (recipComm.get_num_recip() == 1) {
      if (recipComm.get_num_direct() == 1) {
	// Single Direct+Recip node => add to total force and be done
	recip->calc(domdec.get_inv_boxx(), domdec.get_inv_boxy(), domdec.get_inv_boxz(),
		    xyzq.xyzq, xyzq.ncoord,
		    calc_energy, calc_virial, force);
      } else {
	recip->calc(domdec.get_inv_boxx(), domdec.get_inv_boxy(), domdec.get_inv_boxz(),
		    recipComm.get_coord_ptr(), recipComm.get_ncoord(),
		    calc_energy, calc_virial, recip_force);
      }
    } else if (recipComm.get_num_recip() > 1) {
      // For #recip > 1, we need another force buffer (force_recip) and then need to combine results
      // to the total force
      std::cout << "CudaPMEForcefield::calc, #recip > 1 not implemented yet" << std::endl;
      exit(1);
    } else {
      std::cout << "CudaPMEForcefield::calc, #nrecip = 0, but recip defined should not end up here"
		<< std::endl;
      exit(1);
    }
  }

  cudaCheck(hipEventRecord(done_recip_event, recip_stream));

  // Make GPU wait until all computation is done
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_in14_event, 0));
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_bonded_event, 0));
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_recip_event, 0));
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_direct_event, 0));

  // Convert forces from FP to DP
  force.convert<double>(direct_stream[0]);

  bonded.get_energy_virial(calc_energy, calc_virial,
			   &energy_bond, &energy_ureyb,
			   &energy_angle,
			   &energy_dihe, &energy_imdihe,
			   &energy_cmap,
			   sforcex, sforcey, sforcez);

  dir.get_energy_virial(calc_energy, calc_virial,
			&energy_vdw, &energy_elec,
			&energy_excl, vir);

  if (recipComm.get_isRecip()) {
    recip->get_energy_virial(calc_energy, calc_virial, energy_ewksum, energy_ewself, vir);
  }

  // Communicate Direct-Direct
  domdec.comm_force(force);

  if (do_recipcomm) {
    // Communicate Direct-Recip forces
    if (recipComm.get_isRecip()) recipComm.send_force(recip_force);
    recipComm.recv_force(recip_force);
    // Add Recip force to the total force
    force.add<double>(recipComm.get_force_ptr(), domdec.get_ncoord(), direct_stream[0]);
  }

}

//
// Post-process force calculation. Used for array re-ordering after neighborlist search
//
void CudaPMEForcefield::post_calc(const float *global_mass, float *mass) {

  if (neighborlist_updated) {

    // Re-order xyz_shift
    domdec.reorder_xyz_shift(nlist.get_ind_sorted());

    // Re-order mass
    //domdec.reorder_mass(mass, nlist.get_ind_sorted());
    map_to_local_array<float>(domdec.get_ncoord(), domdec.get_loc2glo_ptr(), global_mass, mass);
  }

  cudaCheck(hipEventRecord(done_calc_event, direct_stream[0]));
}

//
// Make stream "stream" wait until calc - routine is done
//
void CudaPMEForcefield::wait_calc(hipStream_t stream) {
  cudaCheck(hipStreamWaitEvent(stream, done_calc_event, 0));
}

//
// Assigns coordinates to nodes
// NOTE: All nodes receive all coordinates here. Domdec distributes them across the nodes
//
void CudaPMEForcefield::assignCoordToNodes(hostXYZ<double>& coord, std::vector<int>& h_loc2glo) {
  // Build loc2glo for the homezone, we now know the number of coordinates at the homezone
  domdec.build_homezone(coord);
  // Copy loc2glo to h_loc2glo
  h_loc2glo.resize(domdec.get_ncoord());
  copy_DtoH<int>(domdec.get_loc2glo_ptr(), h_loc2glo.data(), domdec.get_ncoord());
  // Resize coordinate arrays to the new homezone size
  ref_coord.resize(domdec.get_ncoord());
  ref_coord.clear();
  xyzq.set_ncoord(domdec.get_ncoord());
  xyzq_copy.set_ncoord(domdec.get_ncoord());
}

//
// Checks if non-bonded list needs to be updated
// Returns true if update is needed
//
bool CudaPMEForcefield::heuristic_check(const cudaXYZ<double>& coord, hipStream_t stream) {
  assert(ref_coord.match(&coord));
  assert(warpsize <= 32);

  double rsq_limit_dbl = fabs(domdec.get_rnl() - roff)/2.0;
  rsq_limit_dbl *= rsq_limit_dbl;
  float rsq_limit = (float)rsq_limit_dbl;

  int ncoord = ref_coord.n;
  int stride = ref_coord.stride;
  int nthread = 512;
  int nblock = (ncoord - 1)/nthread + 1;

  int shmem_size = (nthread/warpsize)*sizeof(int);

  *h_heuristic_flag = 0;
  copy_HtoD<int>(h_heuristic_flag, d_heuristic_flag, 1, stream);

  heuristic_check_kernel<<< nblock, nthread, shmem_size, stream >>>
    (ncoord, stride, coord.data, ref_coord.data, rsq_limit, d_heuristic_flag);

  cudaCheck(hipGetLastError());

  copy_DtoH_sync<int>(d_heuristic_flag, h_heuristic_flag, 1);
  
  return (*h_heuristic_flag != 0);
}

//
// Print energies and virials on screen
//
void CudaPMEForcefield::print_energy_virial(int step) {
  double tol = 0.0;

  double energy_kin = 0.0;
  double energy = energy_bond + energy_angle + energy_ureyb + energy_dihe + energy_imdihe +
    energy_vdw + energy_elec + energy_ewksum + energy_ewself + energy_excl;
  double energy_tot = energy + energy_kin;
  double temp = 0.0;

  printf("DYNA>     %d %lf %lf %lf %lf\n",step, energy_tot, energy_kin, energy, temp);

  if (fabs(energy_bond) >= tol || fabs(energy_angle) >= tol || fabs(energy_ureyb) >= tol ||
      fabs(energy_dihe) >= tol || fabs(energy_imdihe) >= tol) {
    printf("DYNA INTERN> %lf %lf %lf %lf %lf\n",
	   energy_bond, energy_angle, energy_ureyb, energy_dihe, energy_imdihe);
  }

  if (fabs(energy_vdw) >= tol || fabs(energy_elec) >= tol) {
    printf("DYNA EXTERN> %lf %lf\n",energy_vdw, energy_elec);
  }

  if (fabs(energy_ewksum) >= tol || fabs(energy_ewself) >= tol || fabs(energy_excl) >= tol) {
    printf("DYNA EWALD> %lf %lf %lf\n",energy_ewksum, energy_ewself, energy_excl);
  }

}

//
// Copies restart data into host buffers
//
void CudaPMEForcefield::get_restart_data(hostXYZ<double> *h_coord, hostXYZ<double> *h_step,
					 hostXYZ<double> *h_force,
					 double *x, double *y, double *z, double *dx, double *dy, double *dz,
					 double *fx, double *fy, double *fz) {

  int ncoord = domdec.get_ncoord();

  if (h_loc2glo != NULL && h_loc2glo_len < ncoord) {
    delete [] h_loc2glo;
    h_loc2glo = NULL;
    h_loc2glo_len = 0;
  }
  if (h_loc2glo == NULL) {
    h_loc2glo_len = min(domdec.get_ncoord_glo(), (int)(ncoord*1.2));
    h_loc2glo = new int[h_loc2glo_len];
  }
  copy_DtoH_sync<int>(domdec.get_loc2glo_ptr(), h_loc2glo, ncoord);

  int coord_stride  = h_coord->stride;
  int coord_stride2 = h_coord->stride*2;
  int step_stride  = h_step->stride;
  int step_stride2 = h_step->stride*2;
  int force_stride  = h_force->stride;
  int force_stride2 = h_force->stride*2;

  for (int i=0;i < ncoord;i++) {
    int j = h_loc2glo[i];
    x[j] = h_coord->data[i];
    y[j] = h_coord->data[i + coord_stride];
    z[j] = h_coord->data[i + coord_stride2];
    dx[j] = h_step->data[i];
    dy[j] = h_step->data[i + step_stride];
    dz[j] = h_step->data[i + step_stride2];
    fx[j] = h_force->data[i];
    fy[j] = h_force->data[i + force_stride];
    fz[j] = h_force->data[i + force_stride2];
  }

}
