#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include "CudaPMEForcefield.h"
#include "cuda_utils.h"
#include "gpu_utils.h"

__global__ void heuristic_check_kernel(const int ncoord, const int stride,
				       const double* __restrict__ coord,
				       const double* __restrict__ ref_coord,
				       const float rsq_limit,
				       int* global_flag) {
  // Required shared memory:
  // blockDim.x/warpsize*sizeof(int)
  extern __shared__ int sh_flag[];
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int stride2 = stride*2;
  const int sh_flag_size = blockDim.x/warpsize;

  float dx = 0.0f;
  float dy = 0.0f;
  float dz = 0.0f;
  if (tid < ncoord) {
    dx = (float)(coord[tid]         - ref_coord[tid]);
    dy = (float)(coord[tid+stride]  - ref_coord[tid+stride]);
    dz = (float)(coord[tid+stride2] - ref_coord[tid+stride2]);
  }

  float rsq = dx*dx + dy*dy + dz*dz;
  // flag = 1 update is needed
  //      = 0 no update needed
  int flag = (rsq > rsq_limit);
  // Reduce flag, packed into bits.
  // NOTE: this assumes that warpsize <= 32
  sh_flag[threadIdx.x/warpsize] = (flag << (threadIdx.x % warpsize));
  __syncthreads();
  if (threadIdx.x < sh_flag_size) {
    for (int d=1;d < sh_flag_size;d *= 2) {
      int t = threadIdx.x + d;
      int flag_val = (t < sh_flag_size) ? sh_flag[t] : 0;
      __syncthreads();
      sh_flag[threadIdx.x] |= flag_val;
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      int flag_val = sh_flag[0];
      atomicOr(global_flag, flag_val);
    }
  }

}

//############################################################################################
//############################################################################################
//############################################################################################

//
// Class creator
//
CudaPMEForcefield::CudaPMEForcefield(CudaDomdec *domdec, CudaDomdecBonded *domdec_bonded,
				     NeighborList<32> *nlist,
				     const int nbondcoef, const float2 *h_bondcoef,
				     const int nureybcoef, const float2 *h_ureybcoef,
				     const int nanglecoef, const float2 *h_anglecoef,
				     const int ndihecoef, const float4 *h_dihecoef,
				     const int nimdihecoef, const float4 *h_imdihecoef,
				     const int ncmapcoef, const float2 *h_cmapcoef,
				     const double roff, const double ron,
				     const double kappa, const double e14fac,
				     const int vdw_model, const int elec_model,
				     const int nvdwparam, const float *h_vdwparam,
				     const float *h_vdwparam14,
				     const int *h_glo_vdwtype, const float *h_q,
				     const int nfftx, const int nffty, const int nfftz,
				     const int order) {

  // Create streams
  cudaCheck(hipStreamCreate(&direct_stream[0]));
  cudaCheck(hipStreamCreate(&direct_stream[1]));
  cudaCheck(hipStreamCreate(&recip_stream));
  cudaCheck(hipStreamCreate(&in14_stream));
  cudaCheck(hipStreamCreate(&bonded_stream));

  // Create events
  cudaCheck(hipEventCreate(&done_direct_event));
  cudaCheck(hipEventCreate(&done_recip_event));
  cudaCheck(hipEventCreate(&done_in14_event));
  cudaCheck(hipEventCreate(&done_bonded_event));
  cudaCheck(hipEventCreate(&done_calc_event));

  // Set energy term flags
  calc_bond = true;
  calc_ureyb = true;
  calc_angle = true;
  calc_dihe = true;
  calc_imdihe = true;
  calc_cmap = true;

  // Domain decomposition
  this->domdec = domdec;
  this->domdec_bonded = domdec_bonded;

  // Neighborlist
  this->nlist = nlist;

  // Bonded coefficients
  bonded.setup_coef(nbondcoef, h_bondcoef, nureybcoef, h_ureybcoef,
		    nanglecoef, h_anglecoef, ndihecoef, h_dihecoef,
		    nimdihecoef, h_imdihecoef, ncmapcoef, h_cmapcoef);
  
  // Direct non-bonded interactions
  setup_direct_nonbonded(roff, ron, kappa, e14fac, vdw_model, elec_model,
			 nvdwparam, h_vdwparam, h_vdwparam14, h_glo_vdwtype);

  // Copy charges
  allocate<float>(&q, domdec->get_ncoord_glo());
  copy_HtoD<float>(h_q, q, domdec->get_ncoord_glo());

  // Recip non-bonded interactions
  setup_recip_nonbonded(kappa, nfftx, nffty, nfftz, order);

  allocate<int>(&d_heuristic_flag, 1);
  allocate_host<int>(&h_heuristic_flag, 1);

  h_loc2glo_len = 0;
  h_loc2glo = NULL;
}

//
// Class destructor
//
CudaPMEForcefield::~CudaPMEForcefield() {
  deallocate<int>(&d_heuristic_flag);
  deallocate_host<int>(&h_heuristic_flag);
  deallocate<float>(&q);
  if (grid != NULL) delete grid;
  if (h_loc2glo != NULL) delete [] h_loc2glo;
  // Destroy streams
  cudaCheck(hipStreamDestroy(direct_stream[0]));
  cudaCheck(hipStreamDestroy(direct_stream[1]));
  cudaCheck(hipStreamDestroy(recip_stream));
  cudaCheck(hipStreamDestroy(in14_stream));
  cudaCheck(hipStreamDestroy(bonded_stream));
  // Destroy events
  cudaCheck(hipEventDestroy(done_direct_event));
  cudaCheck(hipEventDestroy(done_recip_event));
  cudaCheck(hipEventDestroy(done_in14_event));
  cudaCheck(hipEventDestroy(done_bonded_event));
  cudaCheck(hipEventDestroy(done_calc_event));
}

//
// Setup direct non-bonded interactions.
//
void CudaPMEForcefield::setup_direct_nonbonded(const double roff, const double ron,
					       const double kappa, const double e14fac,
					       const int vdw_model, const int elec_model,
					       const int nvdwparam, const float *h_vdwparam,
					       const float *h_vdwparam14, const int *h_glo_vdwtype) {

  this->roff = roff;
  this->ron = ron;

  dir.setup(domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz(), kappa, roff, ron,
	    e14fac, vdw_model, elec_model);

  dir.set_vdwparam(nvdwparam, h_vdwparam);
  dir.set_vdwparam14(nvdwparam, h_vdwparam14);

  allocate<int>(&glo_vdwtype, domdec->get_ncoord_glo());
  copy_HtoD<int>(h_glo_vdwtype, glo_vdwtype, domdec->get_ncoord_glo());
}

//
// Setup recip non-bonded interactions.
//
void CudaPMEForcefield::setup_recip_nonbonded(const double kappa,
					      const int nfftx, const int nffty, const int nfftz,
					      const int order) {

  this->kappa = kappa;

  if (nfftx > 0 && nffty > 0 && nfftz > 0 && order > 0) {
    const FFTtype fft_type = BOX;
    grid = new Grid<int, float, float2>(nfftx, nffty, nfftz, order, fft_type, 1, 0, recip_stream);
  } else {
    grid = NULL;
  }

}

//
// Pre-process force calculation
//
void CudaPMEForcefield::pre_calc(cudaXYZ<double> *coord, cudaXYZ<double> *prev_step) {

  // Check for neighborlist heuristic update
  if (heuristic_check(coord, direct_stream[0])) {
    neighborlist_updated = true;

    std::cout << "  Building neighborlist" << std::endl;

    // Update homezone coordinates (coord) and step vector (prev_step)
    // NOTE: Builds domdec->loc2glo
    domdec->update_homezone(coord, prev_step);

    // Communicate coordinates
    // NOTE: Builds rest of domdec->loc2glo and domdec->xyz_shift
    domdec->comm_coord(coord, true);

    // Copy: coord => xyzq_copy
    // NOTE: coord and xyz_shift are already in the order determined by domdec->loc2glo,
    //       however, q is in the original global order.
    xyzq_copy.set_xyzq(coord, q, domdec->get_loc2glo(), domdec->get_xyz_shift(),
		       domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz());

    // Sort coordinates
    // NOTE: Builds domdec->loc2glo and nlist->glo2loc
    nlist->sort(domdec->get_zone_pcoord(), xyzq_copy.xyzq, xyzq.xyzq, domdec->get_loc2glo());

    // Build neighborlist
    nlist->build(domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz(), domdec->get_rnl(),
		 xyzq.xyzq, domdec->get_loc2glo());

    //nlist->test_build(domdec->get_zone_pcoord(), domdec->get_boxx(), domdec->get_boxy(),
    //domdec->get_boxz(), domdec->get_rnl(), xyzq.xyzq, domdec->get_loc2glo());

    // Build bonded tables
    domdec_bonded->build_tbl(domdec, domdec->get_zone_pcoord());

    // Setup bonded interaction lists
    bonded.setup_list(xyzq.xyzq, domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz(),
		      nlist->get_glo2loc(),
		      domdec_bonded->get_nbond_tbl(), domdec_bonded->get_bond_tbl(),
		      domdec_bonded->get_bond(),
		      domdec_bonded->get_nureyb_tbl(), domdec_bonded->get_ureyb_tbl(),
		      domdec_bonded->get_ureyb(),
		      domdec_bonded->get_nangle_tbl(), domdec_bonded->get_angle_tbl(),
		      domdec_bonded->get_angle(),
		      domdec_bonded->get_ndihe_tbl(), domdec_bonded->get_dihe_tbl(),
		      domdec_bonded->get_dihe(),
		      domdec_bonded->get_nimdihe_tbl(), domdec_bonded->get_imdihe_tbl(),
		      domdec_bonded->get_imdihe(),
		      domdec_bonded->get_ncmap_tbl(), domdec_bonded->get_cmap_tbl(),
		      domdec_bonded->get_cmap());

    // Set vdwtype for Direct non-bonded interactions
    dir.set_vdwtype(domdec->get_ncoord_tot(), glo_vdwtype, domdec->get_loc2glo());

    // Setup 1-4 interaction lists
    dir.set_14_list(xyzq.xyzq, domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz(),
		    nlist->get_glo2loc(),
		    domdec_bonded->get_nin14_tbl(), domdec_bonded->get_in14_tbl(),
		    domdec_bonded->get_in14(),
		    domdec_bonded->get_nex14_tbl(), domdec_bonded->get_ex14_tbl(),
		    domdec_bonded->get_ex14());

    // Re-order prev_step vector:
    domdec->reorder_coord(prev_step, &ref_coord, nlist->get_ind_sorted());
    prev_step->set_data(ref_coord);

    // Re-order coordinates (coord) and copy to reference coordinates (ref_coord)
    domdec->reorder_coord(coord, &ref_coord, nlist->get_ind_sorted());
    coord->set_data(ref_coord);

  } else {
    neighborlist_updated = false;
    // Communicate coordinates
    domdec->comm_coord(coord, false);
    // Copy coordinates to xyzq -array
    xyzq.set_xyz(coord, domdec->get_xyz_shift(),
		 domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz(), direct_stream[0]);
  }

}

//
// Calculate forces
//
void CudaPMEForcefield::calc(const bool calc_energy, const bool calc_virial, Force<long long int> *force) {

  force->clear(direct_stream[0]);

  // Clear energy and virial variables
  if (calc_energy || calc_virial) {
    dir.clear_energy_virial();
    bonded.clear_energy_virial();
    if (grid != NULL) grid->clear_energy_virial();
  }

  // Direct non-bonded force
  dir.calc_force(xyzq.xyzq, nlist, calc_energy, calc_virial, force->xyz.stride, force->xyz.data,
		 direct_stream[0]);
  cudaCheck(hipEventRecord(done_direct_event, direct_stream[0]));

  // 1-4 interactions
  dir.calc_14_force(xyzq.xyzq, calc_energy, calc_virial, force->xyz.stride, force->xyz.data,
		    in14_stream);
  cudaCheck(hipEventRecord(done_in14_event, in14_stream));

  // Bonded forces
  bonded.calc_force(xyzq.xyzq, domdec->get_boxx(), domdec->get_boxy(), domdec->get_boxz(),
  		    calc_energy, calc_virial, force->xyz.stride, force->xyz.data,
		    calc_bond, calc_ureyb, calc_angle, calc_dihe, calc_imdihe, calc_cmap,
		    bonded_stream);
  cudaCheck(hipEventRecord(done_bonded_event, bonded_stream));

  // Reciprocal forces (Only reciprocal nodes calculate these)
  if (grid != NULL) {
    double recip[9];
    for (int i=0;i < 9;i++) recip[i] = 0;
    recip[0] = 1.0/domdec->get_boxx();
    recip[4] = 1.0/domdec->get_boxy();
    recip[8] = 1.0/domdec->get_boxz();
    grid->spread_charge(xyzq.xyzq, xyzq.ncoord, recip);
    grid->r2c_fft();
    grid->scalar_sum(recip, kappa, calc_energy, calc_virial);
    grid->c2r_fft();
    if (domdec->get_numnode() == 1) {
      grid->gather_force(xyzq.xyzq, xyzq.ncoord, recip, force->xyz.stride, force->xyz.data);
    } else {
      //grid->gather_force(xyzq.xyzq, xyzq.ncoord, recip, recip_force.xyz.stride, recip_force.xyz.data);
    }
    if (calc_energy) grid->calc_self_energy(xyzq.xyzq, xyzq.ncoord);
  }
  cudaCheck(hipEventRecord(done_recip_event, recip_stream));

  // Make GPU wait until all computation is done
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_in14_event, 0));
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_bonded_event, 0));
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_recip_event, 0));
  cudaCheck(hipStreamWaitEvent(direct_stream[0], done_direct_event, 0));

  // Convert forces from FP to DP
  force->convert<double>(direct_stream[0]);

  bonded.get_energy_virial(calc_energy, calc_virial,
			   &energy_bond, &energy_ureyb,
			   &energy_angle,
			   &energy_dihe, &energy_imdihe,
			   &energy_cmap,
			   sforcex, sforcey, sforcez);

  dir.get_energy_virial(calc_energy, calc_virial,
			&energy_vdw, &energy_elec,
			&energy_excl, vir);

  grid->get_energy_virial(kappa, calc_energy, calc_virial, &energy_ewksum, &energy_ewself, vir);

  // Communicate forces (After this all nodes have their correct total force)
  domdec->comm_force(force);

}

//
// Post-process force calculation. Used for array re-ordering after neighborlist search
//
void CudaPMEForcefield::post_calc(const float *global_mass, float *mass) {

  if (neighborlist_updated) {

    // Re-order xyz_shift
    domdec->reorder_xyz_shift(nlist->get_ind_sorted());

    // Re-order mass
    //domdec->reorder_mass(mass, nlist->get_ind_sorted());
    map_to_local_array<float>(domdec->get_ncoord(), domdec->get_loc2glo(), global_mass, mass);
  }

  cudaCheck(hipEventRecord(done_calc_event, direct_stream[0]));
}

//
// Make stream "stream" wait until calc - routine is done
//
void CudaPMEForcefield::wait_calc(hipStream_t stream) {
  cudaCheck(hipStreamWaitEvent(stream, done_calc_event, 0));
}

//
// Initializes coordinates.
// NOTE: All nodes receive all coordinates here. Domdec distributes them across the nodes
//
void CudaPMEForcefield::init_coord(cudaXYZ<double> *coord) {
  domdec->build_homezone(coord);
  ref_coord.resize(coord->n);
  ref_coord.clear();
  xyzq.set_ncoord(coord->n);
  xyzq_copy.set_ncoord(coord->n);
}

//
// Checks if non-bonded list needs to be updated
// Returns true if update is needed
//
bool CudaPMEForcefield::heuristic_check(const cudaXYZ<double> *coord, hipStream_t stream) {
  assert(ref_coord.match(coord));
  assert(warpsize <= 32);

  double rsq_limit_dbl = fabs(domdec->get_rnl() - roff)/2.0;
  rsq_limit_dbl *= rsq_limit_dbl;
  float rsq_limit = (float)rsq_limit_dbl;

  int ncoord = ref_coord.n;
  int stride = ref_coord.stride;
  int nthread = 512;
  int nblock = (ncoord - 1)/nthread + 1;

  int shmem_size = (nthread/warpsize)*sizeof(int);

  *h_heuristic_flag = 0;
  copy_HtoD<int>(h_heuristic_flag, d_heuristic_flag, 1, stream);

  heuristic_check_kernel<<< nblock, nthread, shmem_size, stream >>>
    (ncoord, stride, coord->data, ref_coord.data, rsq_limit, d_heuristic_flag);

  cudaCheck(hipGetLastError());

  copy_DtoH_sync<int>(d_heuristic_flag, h_heuristic_flag, 1);
  
  return (*h_heuristic_flag != 0);
}

//
// Print energies and virials on screen
//
void CudaPMEForcefield::print_energy_virial(int step) {
  double tol = 0.0;

  double energy_kin = 0.0;
  double energy = energy_bond + energy_angle + energy_ureyb + energy_dihe + energy_imdihe +
    energy_vdw + energy_elec + energy_ewksum + energy_ewself + energy_excl;
  double energy_tot = energy + energy_kin;
  double temp = 0.0;

  printf("DYNA>     %d %lf %lf %lf %lf\n",step, energy_tot, energy_kin, energy, temp);

  if (fabs(energy_bond) >= tol || fabs(energy_angle) >= tol || fabs(energy_ureyb) >= tol ||
      fabs(energy_dihe) >= tol || fabs(energy_imdihe) >= tol) {
    printf("DYNA INTERN> %lf %lf %lf %lf %lf\n",
	   energy_bond, energy_angle, energy_ureyb, energy_dihe, energy_imdihe);
  }

  if (fabs(energy_vdw) >= tol || fabs(energy_elec) >= tol) {
    printf("DYNA EXTERN> %lf %lf\n",energy_vdw, energy_elec);
  }

  if (fabs(energy_ewksum) >= tol || fabs(energy_ewself) >= tol || fabs(energy_excl) >= tol) {
    printf("DYNA EWALD> %lf %lf %lf\n",energy_ewksum, energy_ewself, energy_excl);
  }

}

//
// Copies restart data into host buffers
//
void CudaPMEForcefield::get_restart_data(hostXYZ<double> *h_coord, hostXYZ<double> *h_step,
					 hostXYZ<double> *h_force,
					 double *x, double *y, double *z, double *dx, double *dy, double *dz,
					 double *fx, double *fy, double *fz) {

  int ncoord = domdec->get_ncoord();

  if (h_loc2glo != NULL && h_loc2glo_len < ncoord) {
    delete [] h_loc2glo;
    h_loc2glo = NULL;
    h_loc2glo_len = 0;
  }
  if (h_loc2glo == NULL) {
    h_loc2glo_len = min(domdec->get_ncoord_glo(), (int)(ncoord*1.2));
    h_loc2glo = new int[h_loc2glo_len];
  }
  copy_DtoH_sync<int>(domdec->get_loc2glo(), h_loc2glo, ncoord);

  int coord_stride  = h_coord->stride;
  int coord_stride2 = h_coord->stride*2;
  int step_stride  = h_step->stride;
  int step_stride2 = h_step->stride*2;
  int force_stride  = h_force->stride;
  int force_stride2 = h_force->stride*2;

  for (int i=0;i < ncoord;i++) {
    int j = h_loc2glo[i];
    x[j] = h_coord->data[i];
    y[j] = h_coord->data[i + coord_stride];
    z[j] = h_coord->data[i + coord_stride2];
    dx[j] = h_step->data[i];
    dy[j] = h_step->data[i + step_stride];
    dz[j] = h_step->data[i + step_stride2];
    fx[j] = h_force->data[i];
    fy[j] = h_force->data[i + force_stride];
    fz[j] = h_force->data[i + force_stride2];
  }

}
