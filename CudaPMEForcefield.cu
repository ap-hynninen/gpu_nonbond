#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include "CudaPMEForcefield.h"
#include "cuda_utils.h"
#include "gpu_utils.h"

__global__ void heuristic_check_kernel(const int ncoord,
				       const double* __restrict__ x,
				       const double* __restrict__ y,
				       const double* __restrict__ z,
				       const double* __restrict__ ref_x,
				       const double* __restrict__ ref_y,
				       const double* __restrict__ ref_z,
				       const float rsq_limit,
				       int* global_flag) {
  // Required shared memory:
  // blockDim.x/warpsize*sizeof(int)
  extern __shared__ int sh_flag[];
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int sh_flag_size = blockDim.x/warpsize;

  float dx = 0.0f;
  float dy = 0.0f;
  float dz = 0.0f;
  if (tid < ncoord) {
    dx = (float)(x[tid] - ref_x[tid]);
    dy = (float)(y[tid] - ref_y[tid]);
    dz = (float)(z[tid] - ref_z[tid]);
  }

  float rsq = dx*dx + dy*dy + dz*dz;
  // flag = 1 update is needed
  //      = 0 no update needed
  int flag = (rsq > rsq_limit);
  // Reduce flag, packed into bits.
  // NOTE: this assumes that warpsize <= 32
  sh_flag[threadIdx.x/warpsize] = (flag << (threadIdx.x % warpsize));
  __syncthreads();
  if (threadIdx.x < sh_flag_size) {
    for (int d=1;d < sh_flag_size;d *= 2) {
      int t = threadIdx.x + d;
      int flag_val = (t < sh_flag_size) ? sh_flag[t] : 0;
      __syncthreads();
      sh_flag[threadIdx.x] |= flag_val;
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      int flag_val = sh_flag[0];
      atomicOr(global_flag, flag_val);
    }
  }

}

//############################################################################################
//############################################################################################
//############################################################################################

//
// Class creator
//
CudaPMEForcefield::CudaPMEForcefield(CudaDomdec& domdec, CudaDomdecGroups& domdecGroups,
				     const CudaTopExcl& topExcl,
				     const int nbondcoef, const float2 *h_bondcoef,
				     const int nureybcoef, const float2 *h_ureybcoef,
				     const int nanglecoef, const float2 *h_anglecoef,
				     const int ndihecoef, const float4 *h_dihecoef,
				     const int nimdihecoef, const float4 *h_imdihecoef,
				     const int ncmapcoef, const float2 *h_cmapcoef,
				     const double roff, const double ron,
				     const double kappa, const double e14fac,
				     const int vdw_model, const int elec_model,
				     const int nvdwparam, const float *h_vdwparam,
				     const float *h_vdwparam14,
				     const int *h_glo_vdwtype, const float *h_glo_q,
				     CudaDomdecRecip* recip, CudaDomdecRecipComm& recipComm) : 
  domdec(domdec), recip(recip), domdecGroups(domdecGroups), 
  nlist(topExcl, domdec.get_nx(), domdec.get_ny(), domdec.get_nz()),
  recipComm(recipComm), kappa(kappa), recip_force_len(0), recip_force(NULL) {

  // Create streams
  cudaCheck(hipStreamCreateWithPriority(&direct_stream[0], hipStreamNonBlocking, low_stream_priority()));
  cudaCheck(hipStreamCreateWithPriority(&direct_stream[1], hipStreamNonBlocking, high_stream_priority()));
  cudaCheck(hipStreamCreateWithFlags(&recip_stream, hipStreamNonBlocking));
  cudaCheck(hipStreamCreateWithFlags(&in14_stream, hipStreamNonBlocking));
  cudaCheck(hipStreamCreateWithFlags(&bonded_stream, hipStreamNonBlocking));

  // Create events
  cudaCheck(hipEventCreate(&done_direct_event[0]));
  cudaCheck(hipEventCreate(&done_direct_event[1]));
  cudaCheck(hipEventCreate(&done_recip_event));
  cudaCheck(hipEventCreate(&done_in14_event));
  cudaCheck(hipEventCreate(&done_bonded_event));
  cudaCheck(hipEventCreate(&done_force_clear_event));
  cudaCheck(hipEventCreate(&xyzq_ready_event[0]));
  cudaCheck(hipEventCreate(&xyzq_ready_event[1]));
  cudaCheck(hipEventCreate(&recip_coord_ready_event));
  cudaCheck(hipEventCreate(&setup_bond_done_event));
  cudaCheck(hipEventCreate(&setup_nonbond_done_event));
  cudaCheck(hipEventCreate(&setup_14_done_event));

  // Set energy term flags
  calc_bond = true;
  calc_ureyb = true;
  calc_angle = true;
  calc_dihe = true;
  calc_imdihe = true;
  calc_cmap = true;

  // Neighborlists
  std::vector<int> numIntZone(8, 0);
  std::vector< std::vector<int> > intZones(8, std::vector<int>() );
  // Create I vs. I interaction
  numIntZone.at(0) = 1;
  intZones.at(0).push_back(Domdec::I);
  nlist.registerList(numIntZone, intZones);
  if (domdec.get_numnode() > 1) {
    // NOTE: getImportIntZones will clear contents of numIntZone & intZones
    domdec.getImportIntZones(numIntZone, intZones);
    nlist.registerList(numIntZone, intZones);
  }

  // Bonded coefficients
  bonded.setup_coef(nbondcoef, h_bondcoef, nureybcoef, h_ureybcoef,
		    nanglecoef, h_anglecoef, ndihecoef, h_dihecoef,
		    nimdihecoef, h_imdihecoef, ncmapcoef, h_cmapcoef);
  
  // Direct non-bonded interactions
  setup_direct_nonbonded(roff, ron, kappa, e14fac, vdw_model, elec_model,
			 nvdwparam, h_vdwparam, h_vdwparam14, h_glo_vdwtype);

  // Set stream for reciprocal calculation
  if (recip != NULL) recip->set_stream(recip_stream);

  // Copy charges
  allocate<float>(&glo_q, domdec.get_ncoord_glo());
  copy_HtoD<float>(h_glo_q, glo_q, domdec.get_ncoord_glo());

  allocate<int>(&d_heuristic_flag, 1);
  allocate_host<int>(&h_heuristic_flag, 1);

}

//
// Class destructor
//
CudaPMEForcefield::~CudaPMEForcefield() {
  deallocate<int>(&d_heuristic_flag);
  deallocate_host<int>(&h_heuristic_flag);
  deallocate<float>(&glo_q);
  deallocate<int>(&glo_vdwtype);
  if (recip_force != NULL) deallocate<float3>(&recip_force);
  // Destroy streams
  cudaCheck(hipStreamDestroy(direct_stream[0]));
  cudaCheck(hipStreamDestroy(direct_stream[1]));
  cudaCheck(hipStreamDestroy(recip_stream));
  cudaCheck(hipStreamDestroy(in14_stream));
  cudaCheck(hipStreamDestroy(bonded_stream));
  // Destroy events
  cudaCheck(hipEventDestroy(done_direct_event[0]));
  cudaCheck(hipEventDestroy(done_direct_event[1]));
  cudaCheck(hipEventDestroy(done_recip_event));
  cudaCheck(hipEventDestroy(done_in14_event));
  cudaCheck(hipEventDestroy(done_bonded_event));
  cudaCheck(hipEventDestroy(done_force_clear_event));
  cudaCheck(hipEventDestroy(xyzq_ready_event[0]));
  cudaCheck(hipEventDestroy(xyzq_ready_event[1]));
  cudaCheck(hipEventDestroy(recip_coord_ready_event));
  cudaCheck(hipEventDestroy(setup_bond_done_event));
  cudaCheck(hipEventDestroy(setup_nonbond_done_event));
  cudaCheck(hipEventDestroy(setup_14_done_event));
}

//
// Setup direct non-bonded interactions.
//
void CudaPMEForcefield::setup_direct_nonbonded(const double roff, const double ron,
					       const double kappa, const double e14fac,
					       const int vdw_model, const int elec_model,
					       const int nvdwparam, const float *h_vdwparam,
					       const float *h_vdwparam14, const int *h_glo_vdwtype) {

  this->roff = roff;
  this->ron = ron;

  dir.setup(domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), kappa, roff, ron,
	    e14fac, vdw_model, elec_model);

  dir.set_vdwparam(nvdwparam, h_vdwparam);
  dir.set_vdwparam14(nvdwparam, h_vdwparam14);

  allocate<int>(&glo_vdwtype, domdec.get_ncoord_glo());
  copy_HtoD<int>(h_glo_vdwtype, glo_vdwtype, domdec.get_ncoord_glo());
}

int nstep=0;

//
// Calculate forces
//
void CudaPMEForcefield::calc(const bool calc_energy, const bool calc_virial,
			     cudaXYZ<double>& coord, cudaXYZ<double>& prev_step, 
			     Force<long long int>& force, hipStream_t stream) {

  // Check for neighborlist heuristic update
  if (heuristic_check(coord, stream)) {
    neighborlist_updated = true;

    if (domdec.get_mynode() == 0) fprintf(stderr,"Building neighborlist %d\n",nstep);

    // Update homezone coordinates (coord) and step vector (prev_step)
    // NOTE: Builds domdec.loc2glo
    domdec.update_homezone(coord, prev_step, stream);

    // ... (here we can sort & build neighborlist for local coordinates) ...

    // Communicate coordinates
    // NOTE: Builds rest of domdec.loc2glo and domdec.xyz_shift
    domdec.comm_coord(coord, true, stream);

    fprintf(stderr,"%d: domdec.get_ncoord()=%d domdec.get_ncoord_tot()=%d\n",
	    domdec.get_mynode(),domdec.get_ncoord(),domdec.get_ncoord_tot());

    // Re-allocate ref_coord and force
    ref_coord.realloc(domdec.get_ncoord_tot());
    force.realloc(domdec.get_ncoord_tot());

    // Clear energy and virial variables
    // NOTE: done_force_clear_event also waits for the energy & virial to clear
    if (calc_energy || calc_virial) {
      dir.clear_energy_virial(direct_stream[0]);
    }

    // Clear forces
    force.clear(direct_stream[0]);
    cudaCheck(hipEventRecord(done_force_clear_event, direct_stream[0]));

    // Re-allocate (xyzq, xyzq_copy)
    xyzq.realloc(domdec.get_ncoord_tot());
    xyzq_copy.realloc(domdec.get_ncoord_tot());

    // Copy: coord => xyzq_copy
    // NOTE: coord and xyz_shift are already in the order determined by domdec.loc2glo,
    //       however, glo_q is in the original global order.
    xyzq_copy.set_xyzq(coord, glo_q, domdec.get_loc2glo_ptr(), domdec.get_xyz_shift(),
		       domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), stream);

    //nlist.set_test(true);
    // Sort coordinates
    // NOTE: Builds domdec.loc2glo and nlist->glo2loc
    nlist.sort(0, domdec.get_zone_pcoord(), xyzq_copy.xyzq, xyzq.xyzq, domdec.get_loc2glo_ptr(), stream);
    cudaCheck(hipEventRecord(xyzq_ready_event[0], stream));

    // Build neighborlist
    nlist.build(0, domdec.get_zone_pcoord(), domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
		domdec.get_rnl(), xyzq.xyzq, domdec.get_loc2glo_ptr(), stream);

    if (nlist.getNumList() > 1) {
      nlist.sort(1, domdec.get_zone_pcoord(), xyzq_copy.xyzq, xyzq.xyzq, domdec.get_loc2glo_ptr(), stream);
      cudaCheck(hipEventRecord(xyzq_ready_event[1], stream));
      nlist.build(1, domdec.get_zone_pcoord(), domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
		  domdec.get_rnl(), xyzq.xyzq, domdec.get_loc2glo_ptr(), stream);
    }

    // Build bonded tables
    domdecGroups.buildGroupTables(stream);
    domdecGroups.syncGroupTables(stream);

    // Check the total number of groups
    if (!domdec.checkNumGroups(domdecGroups.get_atomGroupVector())) exit(1);
    
    // Setup bonded interaction lists
    bonded.setup_list(xyzq.xyzq, domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
		      nlist.get_glo2loc(),
		      domdecGroups.getNumGroupTable(BOND), domdecGroups.getGroupTable(BOND),
		      domdecGroups.getGroupList<bond_t>(BOND),
		      domdecGroups.getNumGroupTable(UREYB), domdecGroups.getGroupTable(UREYB),
		      domdecGroups.getGroupList<bond_t>(UREYB),
		      domdecGroups.getNumGroupTable(ANGLE), domdecGroups.getGroupTable(ANGLE),
		      domdecGroups.getGroupList<angle_t>(ANGLE),
		      domdecGroups.getNumGroupTable(DIHE), domdecGroups.getGroupTable(DIHE),
		      domdecGroups.getGroupList<dihe_t>(DIHE),
		      domdecGroups.getNumGroupTable(IMDIHE), domdecGroups.getGroupTable(IMDIHE),
		      domdecGroups.getGroupList<dihe_t>(IMDIHE),
		      domdecGroups.getNumGroupTable(CMAP), domdecGroups.getGroupTable(CMAP),
		      domdecGroups.getGroupList<cmap_t>(CMAP), stream);
    cudaCheck(hipEventRecord(setup_bond_done_event, stream));

    // Set vdwtype for Direct non-bonded interactions
    dir.set_vdwtype(domdec.get_ncoord_tot(), glo_vdwtype, domdec.get_loc2glo_ptr(), stream);
    cudaCheck(hipEventRecord(setup_nonbond_done_event, stream));

    // Setup 1-4 interaction lists
    dir.set_14_list(xyzq.xyzq, domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
		    nlist.get_glo2loc(),
		    domdecGroups.getNumGroupTable(IN14), domdecGroups.getGroupTable(IN14),
		    domdecGroups.getGroupList<xx14_t>(IN14),
		    domdecGroups.getNumGroupTable(EX14), domdecGroups.getGroupTable(EX14),
		    domdecGroups.getGroupList<xx14_t>(EX14), stream);
    cudaCheck(hipEventRecord(setup_14_done_event, stream));

    // Re-order prev_step vector, using ref_coord as temporary storage
    // NOTE: We only re-order up to homezone
    domdec.reorder_coord(domdec.get_ncoord(), prev_step, ref_coord, nlist.get_ind_sorted(), stream);

    // Re-order coordinates (coord), using ref_coord as temporary storage
    // NOTE: We re-order all coordinates, homezone+import. This is done so that coord is
    //       correctly set up e.g. for constraint communication
    domdec.reorder_coord(domdec.get_ncoord_tot(), coord, ref_coord, nlist.get_ind_sorted(), stream);

    // NOTE: Now ref_coord contains the correct current coordinates that are then used in
    //       the heuristic check in the next step

    // Update and re-order communication buffers
    domdec.comm_update(nlist.get_glo2loc(), stream);
    //domdec.test_comm_coord(nlist.get_glo2loc(), coord);

    // Wait for non-bonded setup to finish
    cudaCheck(hipStreamWaitEvent(direct_stream[0], setup_nonbond_done_event, 0));
    cudaCheck(hipStreamWaitEvent(direct_stream[0], xyzq_ready_event[0], 0));
    // Direct non-bonded force (homebox)
    dir.calc_force(xyzq.xyzq, nlist.getBuilder(0), calc_energy, calc_virial, force.stride(), force.xyz(),
		   direct_stream[0]);
    cudaCheck(hipEventRecord(done_direct_event[0], direct_stream[0]));

    if (nlist.getNumList() > 1) {
      // Wait for force clear finish
      cudaCheck(hipStreamWaitEvent(direct_stream[1], done_force_clear_event, 0));
      // Wait for non-bonded setup to finish
      cudaCheck(hipStreamWaitEvent(direct_stream[1], setup_nonbond_done_event, 0));
      cudaCheck(hipStreamWaitEvent(direct_stream[1], xyzq_ready_event[1], 0));
      // Direct non-bonded force (import)
      dir.calc_force(xyzq.xyzq, nlist.getBuilder(1), calc_energy, calc_virial, force.stride(), force.xyz(),
		     direct_stream[1]);
      cudaCheck(hipEventRecord(done_direct_event[1], direct_stream[1]));
    }

  } else {
    neighborlist_updated = false;

    // Clear energy and virial variables
    // NOTE: done_force_clear_event also waits for the energy & virial to clear
    if (calc_energy || calc_virial) {
      dir.clear_energy_virial(direct_stream[0]);
    }

    // Clear forces
    force.clear(direct_stream[0]);
    cudaCheck(hipEventRecord(done_force_clear_event, direct_stream[0]));

    // Copy local coordinates to xyzq -array
    xyzq.set_xyz(coord, 0, domdec.get_ncoord()-1, domdec.get_xyz_shift(),
		 domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), direct_stream[0]);
    // Record event "xyzq local coordinates are ready"
    cudaCheck(hipEventRecord(xyzq_ready_event[0], direct_stream[0]));
    // Direct non-bonded force (homebox)
    dir.calc_force(xyzq.xyzq, nlist.getBuilder(0), calc_energy, calc_virial, force.stride(), force.xyz(),
		   direct_stream[0]);
    cudaCheck(hipEventRecord(done_direct_event[0], direct_stream[0]));

    // Communicate coordinates between direct nodes
    domdec.comm_coord(coord, false, direct_stream[1]);

    // Copy import volume coordinates to xyzq -array
    xyzq.set_xyz(coord, domdec.get_ncoord(), domdec.get_ncoord_tot()-1, domdec.get_xyz_shift(),
		 domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(), direct_stream[1]);
    // Record event "xyzq array is ready"
    cudaCheck(hipEventRecord(xyzq_ready_event[1], direct_stream[1]));
    if (nlist.getNumList() > 1) {
      // Wait for force clear finish
      cudaCheck(hipStreamWaitEvent(direct_stream[1], done_force_clear_event, 0));
      // Direct non-bonded force (import)
      dir.calc_force(xyzq.xyzq, nlist.getBuilder(1), calc_energy, calc_virial, force.stride(), force.xyz(),
		     direct_stream[1]);
      cudaCheck(hipEventRecord(done_direct_event[1], direct_stream[1]));
    }
  }

  nstep++;

  //-------------------------------------------------------------------------------

  bool do_recipcomm = recipComm.get_hasPureRecip() || 
    (recipComm.get_num_recip() > 0 && recipComm.get_num_direct() > 1);

  if (do_recipcomm) {
    if (recipComm.get_isRecip() && recip == NULL) {
      std::cout << "CudaPMEForcefield::calc, missing recip object" << std::endl;
      exit(1);
    }
    //-------------------------------------
    // Send coordinates to recip node(s)
    //-------------------------------------
    // Send header
    if (recipComm.get_hasPureRecip()) {
      recipComm.send_header(domdec.get_ncoord(), domdec.get_inv_boxx(), domdec.get_inv_boxy(),
			    domdec.get_inv_boxz(), calc_energy, calc_virial);
    } else if (neighborlist_updated) {
      if (recipComm.get_isRecip()) {
	recipComm.recv_ncoord(domdec.get_ncoord());
      } else {
	recipComm.send_ncoord(domdec.get_ncoord());
      }
    }
    if (recipComm.get_isRecip() && recipComm.get_num_recip() == 1) {
      assert(recipComm.get_ncoord() == domdec.get_ncoord_glo());
    }
    // Re-allocate recip_xyzq and recip_force if needed
    if (recipComm.get_isRecip() && recipComm.get_num_direct() > 1) {
      recip_xyzq.realloc(recipComm.get_ncoord());
    }
    reallocate<float3>(&recip_force, &recip_force_len, recipComm.get_ncoord(), 1.0f);
    // Send coordinates
    cudaCheck(hipStreamWaitEvent(stream, xyzq_ready_event[0], 0));
    recipComm.send_coord(xyzq.xyzq, stream);
    // Receive coordinates
    if (recipComm.get_isRecip()) recipComm.recv_coord(recip_xyzq.xyzq, stream);
    //-------------------------------------
    cudaCheck(hipEventRecord(recip_coord_ready_event, stream));
  }

  // --------------- Use of stream "stream" stops here --------------

  // Clear energy and virial variables
  if (calc_energy || calc_virial) {
    bonded.clear_energy_virial(bonded_stream);
    if (recipComm.get_isRecip()) recip->clear_energy_virial();
  }

  // Wait for xyzq coordinates to be ready
  //cudaCheck(hipStreamWaitEvent(direct_stream[0], xyzq_ready_event[0], 0));
  //cudaCheck(hipStreamWaitEvent(direct_stream[0], setup_nonbond_done_event, 0));
  // .... (here we'll calculate the local forces) ...
  //cudaCheck(hipStreamWaitEvent(direct_stream[1], xyzq_ready_event[1], 0));
  //cudaCheck(hipStreamWaitEvent(direct_stream[1], setup_nonbond_done_event, 0));

  /*
  // Direct non-bonded force
  dir.calc_force(xyzq.xyzq, nlist.getBuilder(0), calc_energy, calc_virial, force.stride(), force.xyz(),
  		 direct_stream[0]);
  cudaCheck(hipEventRecord(done_direct_event[0], direct_stream[0]));

  if (nlist.getNumList() > 1) {
    dir.calc_force(xyzq.xyzq, nlist.getBuilder(1), calc_energy, calc_virial, force.stride(), force.xyz(),
    		   direct_stream[1]);
    cudaCheck(hipEventRecord(done_direct_event[1], direct_stream[1]));
  }
  */

  // 1-4 interactions
  // NOTE: we make GPU wait until force.cleap() is done
  cudaCheck(hipStreamWaitEvent(in14_stream, xyzq_ready_event[1], 0));
  cudaCheck(hipStreamWaitEvent(in14_stream, done_force_clear_event, 0));
  cudaCheck(hipStreamWaitEvent(in14_stream, setup_14_done_event, 0));
  dir.calc_14_force(xyzq.xyzq, calc_energy, calc_virial, force.stride(), force.xyz(),
  		    in14_stream);
  cudaCheck(hipEventRecord(done_in14_event, in14_stream));

  // Bonded forces
  cudaCheck(hipStreamWaitEvent(bonded_stream, xyzq_ready_event[1], 0));
  cudaCheck(hipStreamWaitEvent(bonded_stream, done_force_clear_event, 0));
  cudaCheck(hipStreamWaitEvent(bonded_stream, setup_bond_done_event, 0));
  bonded.calc_force(xyzq.xyzq, domdec.get_boxx(), domdec.get_boxy(), domdec.get_boxz(),
    		    calc_energy, calc_virial, force.stride(), force.xyz(),
  		    calc_bond, calc_ureyb, calc_angle, calc_dihe, calc_imdihe, calc_cmap,
  		    bonded_stream);
  cudaCheck(hipEventRecord(done_bonded_event, bonded_stream));

  // Reciprocal force (Only reciprocal nodes calculate this)
  if (recipComm.get_isRecip()) {
    if (recipComm.get_num_recip() == 1) {
      if (recipComm.get_num_direct() == 1) {
	// Single node that is a Direct+Recip node => add to total force and be done
	cudaCheck(hipStreamWaitEvent(recip_stream, xyzq_ready_event[0], 0));
	cudaCheck(hipStreamWaitEvent(recip_stream, done_force_clear_event, 0));
	recip->calc(domdec.get_inv_boxx(), domdec.get_inv_boxy(), domdec.get_inv_boxz(),
		    xyzq.xyzq, xyzq.ncoord,
		    calc_energy, calc_virial, force);
      } else {
	cudaCheck(hipStreamWaitEvent(recip_stream, recip_coord_ready_event, 0));
	recip->calc(domdec.get_inv_boxx(), domdec.get_inv_boxy(), domdec.get_inv_boxz(),
		    recipComm.get_coord_ptr(), recipComm.get_ncoord(),
		    calc_energy, calc_virial, recip_force);
      }
    } else if (recipComm.get_num_recip() > 1) {
      // For #recip > 1, we need another force buffer (force_recip) and then need to combine results
      // to the total force
      std::cout << "CudaPMEForcefield::calc, #recip > 1 not implemented yet" << std::endl;
      exit(1);
    } else {
      std::cout << "CudaPMEForcefield::calc, #nrecip = 0, but recip defined should not end up here"
		<< std::endl;
      exit(1);
    }
  }
  cudaCheck(hipEventRecord(done_recip_event, recip_stream));

  // --------------- stream "stream" picks up here --------------

  // Make integrator stream "stream" wait until all computation is done
  // i.e. we are pinning all computation to this stream
  cudaCheck(hipStreamWaitEvent(stream, done_in14_event, 0));
  cudaCheck(hipStreamWaitEvent(stream, done_bonded_event, 0));
  cudaCheck(hipStreamWaitEvent(stream, done_recip_event, 0));
  cudaCheck(hipStreamWaitEvent(stream, done_direct_event[0], 0));
  cudaCheck(hipStreamWaitEvent(stream, done_direct_event[1], 0));

  // Convert forces from FP to DP
  force.convert<double>(stream);

  // If energy or virial is requested, make CPU wait here for computation to finish
  if (calc_energy || calc_virial) {
    cudaCheck(hipEventSynchronize(done_in14_event));
    cudaCheck(hipEventSynchronize(done_bonded_event));
    if (recipComm.get_isRecip()) cudaCheck(hipEventSynchronize(done_recip_event));
    cudaCheck(hipEventSynchronize(done_direct_event[0]));
    cudaCheck(hipEventSynchronize(done_direct_event[1]));
  }

  bonded.get_energy_virial(calc_energy, calc_virial,
			   &energy_bond, &energy_ureyb,
			   &energy_angle,
			   &energy_dihe, &energy_imdihe,
			   &energy_cmap,
			   sforce);

  dir.get_energy_virial(calc_energy, calc_virial,
			&energy_vdw, &energy_elec,
			&energy_excl, vir);

  if (recipComm.get_isRecip()) {
    recip->get_energy_virial(calc_energy, calc_virial, energy_ewksum, energy_ewself, vir);
  }

  // Communicate Direct-Direct
  // NOTE: Synchronization on stream is done in comm_force
  // NOTE2: Due to the GPU waits above, this implies that all the other streams have
  //        finished their computation as well
  domdec.comm_force(force, stream);

  if (do_recipcomm) {
    cudaCheck(hipStreamSynchronize(recip_stream));
    // Communicate Direct-Recip forces
    if (recipComm.get_isRecip()) recipComm.send_force(recip_force, stream);
    recipComm.recv_force(recip_force, stream);
    // Add Recip force to the total force
    force.add<double>(recipComm.get_force_ptr(), domdec.get_ncoord(), stream);
  }

}

//
// Post-process force calculation. Used for array re-ordering after neighborlist search
// Updates holonomic constraint tables if neccessary
//
void CudaPMEForcefield::post_calc(const float *global_mass, float *mass, HoloConst* holoconst,
				  hipStream_t stream) {

  if (neighborlist_updated) {
    // Re-order xyz_shift
    domdec.reorder_xyz_shift(nlist.get_ind_sorted(), stream);

    // Re-do mass array
    map_to_local_array<float>(domdec.get_ncoord(), domdec.get_loc2glo_ptr(), global_mass, mass, stream);

    if (holoconst != NULL) {
      holoconst->setup_list(nlist.get_glo2loc(),
			    domdecGroups.getNumGroupTable(PAIR), domdecGroups.getGroupTable(PAIR),
			    domdecGroups.getGroupList<bond_t>(PAIR),
			    domdecGroups.getNumGroupTable(TRIP), domdecGroups.getGroupTable(TRIP),
			    domdecGroups.getGroupList<angle_t>(TRIP),
			    domdecGroups.getNumGroupTable(QUAD), domdecGroups.getGroupTable(QUAD),
			    domdecGroups.getGroupList<dihe_t>(QUAD),
			    domdecGroups.getNumGroupTable(SOLVENT), domdecGroups.getGroupTable(SOLVENT),
			    domdecGroups.getGroupList<solvent_t>(SOLVENT), stream);
      domdec.constCommSetup(domdecGroups.getNeighPos(), domdecGroups.getCoordInd(),
			    nlist.get_glo2loc(), stream);
    }
  }

}

//
// Communicate constraint coordinates in direction "dir" =-1 or +1
//
void CudaPMEForcefield::constComm(const int dir, cudaXYZ<double>& coord, hipStream_t stream) {
  domdec.constCommDo(dir, coord, stream);
}

//
// Assigns coordinates to nodes
// NOTE: All nodes receive all coordinates here. Domdec distributes them across the nodes
//
void CudaPMEForcefield::assignCoordToNodes(hostXYZ<double>& coord, std::vector<int>& h_loc2glo) {
  // Build loc2glo for the homezone, we now know the number of coordinates at the homezone
  domdec.build_homezone(coord);
  // Copy loc2glo to h_loc2glo
  h_loc2glo.resize(domdec.get_ncoord());
  copy_DtoH<int>(domdec.get_loc2glo_ptr(), h_loc2glo.data(), domdec.get_ncoord());
  // Re-allocate coordinate arrays to the new homezone size
  ref_coord.realloc(domdec.get_ncoord());
  ref_coord.clear();
  xyzq.realloc(domdec.get_ncoord());
  xyzq_copy.realloc(domdec.get_ncoord());
}

//
// Checks if non-bonded list needs to be updated
// Returns true if update is needed
//
bool CudaPMEForcefield::heuristic_check(const cudaXYZ<double>& coord, hipStream_t stream) {
  assert(domdec.get_ncoord() <= ref_coord.size());
  assert(domdec.get_ncoord() <= coord.size());
  assert(warpsize <= 32);

  double rsq_limit_dbl = fabs(domdec.get_rnl() - roff)/2.0;
  rsq_limit_dbl *= rsq_limit_dbl;
  float rsq_limit = (float)rsq_limit_dbl;

  int nthread = 512;
  int nblock = (domdec.get_ncoord() - 1)/nthread + 1;
  int shmem_size = (nthread/warpsize)*sizeof(int);

  *h_heuristic_flag = 0;
  copy_HtoD<int>(h_heuristic_flag, d_heuristic_flag, 1, stream);

  heuristic_check_kernel<<< nblock, nthread, shmem_size, stream >>>
    (domdec.get_ncoord(), coord.x(), coord.y(), coord.z(),
     ref_coord.x(), ref_coord.y(), ref_coord.z(),
     rsq_limit, d_heuristic_flag);
  cudaCheck(hipGetLastError());

  copy_DtoH<int>(d_heuristic_flag, h_heuristic_flag, 1, stream);

  // Make sure above kernel calls finish
  cudaCheck(hipStreamSynchronize(stream));

  return domdec.checkHeuristic((*h_heuristic_flag != 0));
}

//
// Print energies and virials on screen
//
void CudaPMEForcefield::print_energy_virial(int step, const double energy_kin) {
  double tol = 0.0;

  double energy = energy_bond + energy_angle + energy_ureyb + energy_dihe + energy_imdihe +
    energy_vdw + energy_elec + energy_ewksum + energy_ewself + energy_excl;
  double energy_tot = energy + energy_kin;
  double temp = 0.0;

  printf("DYNA>     %d %lf %lf %lf %lf\n",step, energy_tot, energy_kin, energy, temp);

  if (fabs(energy_bond) >= tol || fabs(energy_angle) >= tol || fabs(energy_ureyb) >= tol ||
      fabs(energy_dihe) >= tol || fabs(energy_imdihe) >= tol) {
    printf("DYNA INTERN> %lf %lf %lf %lf %lf\n",
	   energy_bond, energy_angle, energy_ureyb, energy_dihe, energy_imdihe);
  }

  if (fabs(energy_vdw) >= tol || fabs(energy_elec) >= tol) {
    printf("DYNA EXTERN> %lf %lf\n",energy_vdw, energy_elec);
  }

  if (fabs(energy_ewksum) >= tol || fabs(energy_ewself) >= tol || fabs(energy_excl) >= tol) {
    printf("DYNA EWALD> %lf %lf %lf\n",energy_ewksum, energy_ewself, energy_excl);
  }

}

//
// Copies restart data into host buffers
//
void CudaPMEForcefield::get_restart_data(cudaXYZ<double>& coord, cudaXYZ<double>& step,
					 Force<long long int>& force,
					 double *x, double *y, double *z,
					 double *dx, double *dy, double *dz,
					 double *fx, double *fy, double *fz) {

  int* loc2glo = new int[domdec.get_ncoord()];
  int* loc2glo_glo = new int[domdec.get_ncoord_glo()];
  int* nrecv = new int[domdec.get_numnode()];
  int* precv = new int[domdec.get_numnode()];
  double* recvbuf = new double[domdec.get_ncoord_glo()];
  hostXYZ<double> hXYZ(domdec.get_ncoord(), NON_PINNED);

  copy_DtoH_sync<int>(domdec.get_loc2glo_ptr(), loc2glo, domdec.get_ncoord());
  domdec.buildGlobal_loc2glo(loc2glo, loc2glo_glo, nrecv, precv);

  hXYZ.set_data_sync(domdec.get_ncoord(), coord);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.x(), x);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.y(), y);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.z(), z);
  hXYZ.set_data_sync(domdec.get_ncoord(), step);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.x(), dx);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.y(), dy);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.z(), dz);
  hXYZ.set_data_sync(domdec.get_ncoord(), (double *)force.x(), (double *)force.y(), (double *)force.z());
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.x(), fx);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.y(), fy);
  domdec.combineData(loc2glo_glo, nrecv, precv, recvbuf, hXYZ.z(), fz);

  delete [] loc2glo;
  delete [] loc2glo_glo;
  delete [] nrecv;
  delete [] precv;
  delete [] recvbuf;

}
