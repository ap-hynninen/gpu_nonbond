#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <math.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"
#include "DirectForce.h"

template <typename AT, typename CT>
__forceinline__ __device__ void write_force(const CT fx, const CT fy, const CT fz,
					    const int ind, const int stride,
					    AT* force) {
  // The generic version can not be used
}

// Template specialization for 64bit integer = "long long int"
template <>
__forceinline__ __device__ void write_force <long long int, float> (const float fx, const float fy, const float fz,
								    const int ind, const int stride,
								    long long int* force) {

  atomicAdd((unsigned long long int *)&force[ind           ], llitoulli(fx));
  atomicAdd((unsigned long long int *)&force[ind + stride  ], llitoulli(fy));
  atomicAdd((unsigned long long int *)&force[ind + stride*2], llitoulli(fz));
}

/*
template <>
__forceinline__ __device__ void write_force <double, float>() {
    // Reduce forces and then do atomicAdd from a single thread
    // Write to shared memory
    fj_tmp[tid].x = fjx;
    fj_tmp[tid].y = fjy;
    fj_tmp[tid].z = fjz;
    if (threadIdx.x == 0) {
      FORCE_T f_red[3] = {0.0f, 0.0f, 0.0f};
      for (int i=sh_start;i < sh_start + threadIdx.x;i++) {
	f_red[0] += fj_tmp[i].x;
	f_red[1] += fj_tmp[i].y;
	f_red[2] += fj_tmp[i].z;
      }
      atomicAdd(&force[blockIdx.x*stride3 +           indj + threadIdx.x], f_red[0]);
      atomicAdd(&force[blockIdx.x*stride3 + stride  + indj + threadIdx.x], f_red[1]);
      atomicAdd(&force[blockIdx.x*stride3 + stride2 + indj + threadIdx.x], f_red[2]);
    }
    //    force[blockIdx.x*stride3 +           indj + threadIdx.x] += fjx;
    //    force[blockIdx.x*stride3 + stride +  indj + threadIdx.x] += fjy;
    //    force[blockIdx.x*stride3 + stride2 + indj + threadIdx.x] += fjz;
}
*/

template <typename AT, typename CT>
__forceinline__ __device__
void calc_component_force(CT fij,
			  const CT dx, const CT dy, const CT dz,
			  AT &fxij, AT &fyij, AT &fzij) {
  fxij = (AT)(fij*dx);
  fyij = (AT)(fij*dy);
  fzij = (AT)(fij*dz);
}

template <>
__forceinline__ __device__
void calc_component_force<long long int, float>(float fij,
						const float dx, const float dy, const float dz,
						long long int &fxij, long long int &fyij, long long int &fzij) {
  fij *= FORCE_SCALE;
  fxij = lliroundf(fij*dx);
  fyij = lliroundf(fij*dy);
  fzij = lliroundf(fij*dz);
}

class vdw_base {
public:
  virtual void setup(float ron2, float roff2) = 0;
  //virtual float pair_force() = 0;
};

class vdw_vsh : public vdw_base {
public:
  float roffinv6;
  float roffinv12;
  float roffinv18;

  void setup(float ron2, float roff2) {
    roffinv6 = 1.0f/(roff2*roff2*roff2);
    roffinv12 = roffinv6*roffinv6;
    roffinv18 = roffinv12*roffinv6;
  }
};


enum {NONE, EWALD, EWALD_LOOKUP, VDW_CUT, VDW_VSH, VDW_VSW, VDW_VFSW};

//#define WARPSIZE 32                             // Number of threads per warp
//#define TILESIZE 32                             // Number of atoms per tile direction
//#define NUM_EXCL ((32*32-1)/32 + 1) // Number of exclusion mask integers
//#define NUM_THREAD_PER_EXCL (32/NUM_EXCL)       // Number of threads per exclusion mask integer

struct DirectSettings_t {
  float kappa;
  float kappa2;

  float boxx;
  float boxy;
  float boxz;

  float roff2;
  float ron2;

  float roffinv6;
  float roffinv12;
  float roffinv18;

  float inv_roff2_ron2;
};

// Settings for direct computation in host memory
static DirectSettings_t h_setup;

// Settings for direct computation in device memory
static __constant__ DirectSettings_t d_setup;

static texture<float2, 1, hipReadModeElementType> vdwparam_texref;
static bool vdwparam_texref_bound = false;

//
// Calculates VdW pair force & energy
//
template <int vdw_model, bool calc_energy>
__forceinline__ __device__
float pair_vdw_force(float r2, float r, float rinv, float rinv2, float c6, float c12, double &vdwpotl) {

  float fij_vdw;

  if (vdw_model == VDW_VSH) {
    float r6 = r2*r2*r2;
    float rinv6 = rinv2*rinv2*rinv2;
    float rinv12 = rinv6*rinv6;
	    
    if (calc_energy) {
      const float one_twelve = 0.0833333333333333f;
      const float one_six = 0.166666666666667f;
      vdwpotl += (double)(c12*one_twelve*(rinv12 + 2.0f*r6*d_setup.roffinv18 - 3.0f*d_setup.roffinv12) - 
			  c6*one_six*(rinv6 + r6*d_setup.roffinv12 - 2.0f*d_setup.roffinv6));
    }
	  
    fij_vdw = c6*(rinv6 - r6*d_setup.roffinv12) - c12*(rinv12 + r6*d_setup.roffinv18);
  } else if (vdw_model == VDW_VSW) {
    float roff2_r2_sq = d_setup.roff2 - r2;
    roff2_r2_sq *= roff2_r2_sq;
    float sw = (r2 <= d_setup.ron2) ? 1.0f : roff2_r2_sq*(d_setup.roff2 + 2.0f*r2 - 
							  3.0f*d_setup.ron2)*d_setup.inv_roff2_ron2;
    float dsw = (r2 <= d_setup.ron2) ? 0.0f : 6.0f*(d_setup.roff2-r2)*(d_setup.ron2-r2)*d_setup.inv_roff2_ron2;
    float rinv6 = rinv2*rinv2*rinv2;
    if (calc_energy) {
      vdwpotl += (double)((c12*rinv6 - c6)*rinv6*sw);
    }
    sw *= 3.0f*rinv2;
    fij_vdw = 2.0f*r*rinv6*(c12*rinv6*(-2.0f*sw + dsw)
			    + c6*(sw - dsw));
  } else if (vdw_model == VDW_CUT) {
    float rinv6 = rinv2*rinv2*rinv2;
	  
    if (calc_energy) {
      const float one_twelve = 0.0833333333333333f;
      const float one_six = 0.166666666666667f;
      float rinv12 = rinv6*rinv6;
      vdwpotl += (double)(c12*one_twelve*rinv12 - c6*one_six*rinv6);
      fij_vdw = c6*rinv6 - c12*rinv12;
    } else {
      fij_vdw = c6*rinv6 - c12*rinv6*rinv6;
    }

  } else if (vdw_model == NONE) {
    fij_vdw = 0.0f;
  }

  return fij_vdw;
}

//
// Calculates electrostatic force & energy
//
template <int elec_model, bool calc_energy>
__forceinline__ __device__
float pair_elec_force(float r2, float r, float rinv, float qi, float qj, double &coulpotl) {

  float fij_elec;

  float qq = qi*qj;

  if (elec_model == EWALD_LOOKUP) {
    fij_elec = 0.0f; //qq*lookup_force(r, kappa);
  } else if (elec_model == EWALD) {
    float erfc_val = fasterfc(d_setup.kappa*r);
    float exp_val = expf(-d_setup.kappa2*r2);
    if (calc_energy) {
      coulpotl += (double)(qq*erfc_val*rinv);
    }
    const float two_sqrtpi = 1.12837916709551f;    // 2/sqrt(pi)
    fij_elec = qq*(two_sqrtpi*d_setup.kappa*exp_val + erfc_val*rinv);
  } else if (elec_model == NONE) {
    fij_elec = 0.0f;
  }

  return fij_elec;
}

//
// Nonbonded force kernel
//
template <typename AT, typename CT, int tilesize, int vdw_model, int elec_model,
	  bool calc_energy, bool tex_vdwparam>
__global__ void calc_force_kernel(const int ni, const ientry_t *ientry,
				  const int *tile_indj, const tile_excl_t<tilesize> *tile_excl,
				  const int stride,
				  const float *vdwparam, const int nvdwparam,
				  const float4 *xyzq, const int *vdwtype,
				  AT *force) {

  // Pre-computed constants
  const int warpsize = 32;
  const int num_excl = ((tilesize*tilesize-1)/32 + 1);
  const int num_thread_per_excl = (32/num_excl);

  //
  // Shared data, common for the entire block
  //
   extern __shared__ char shmem[];

   volatile float4 *xyzq_i = (float4 *)&shmem[0];                   // tilesize*blockDim.y
   volatile int *vdwtype_i = (int *)&xyzq_i[tilesize*blockDim.y];   // tilesize*blockDim.y
   volatile AT *fix = (AT *)&vdwtype_i[tilesize*blockDim.y];        // WARPSIZE*blockDim.y
   volatile AT *fiy = &fix[warpsize*blockDim.y];                    // WARPSIZE*blockDim.y
   volatile AT *fiz = &fiy[warpsize*blockDim.y];                    // WARPSIZE*blockDim.y
   volatile float *vdwparam_sh;

   if (tex_vdwparam) {
     vdwparam_sh = (float *)&fiz[warpsize*blockDim.y];
   }

   /*
  __shared__ float4 xyzq_i[TILESIZE*TILEX_NBLOCK];
  __shared__ int vdwtype_i[TILESIZE*TILEX_NBLOCK];
  __shared__ AT fix[WARPSIZE*TILEX_NBLOCK];
  __shared__ AT fiy[WARPSIZE*TILEX_NBLOCK];
  __shared__ AT fiz[WARPSIZE*TILEX_NBLOCK];
   */

  /*
#ifdef PREC_SPDP
  __shared__ FORCE3_T fj_tmp[WARPSIZE*TILEX_NBLOCK];
#endif
#ifndef TEX_FETCH_VDWPARAM
  __shared__ float vdwparam_sh[MAX_NVDWPARAM];
#endif
  */

  // Load ientry
  const unsigned int ientry_ind = threadIdx.y + blockDim.y*blockIdx.x;

  int indi, ish, startj, endj;
  if (ientry_ind < ni) {
    indi   = ientry[ientry_ind].indi;
    ish    = ientry[ientry_ind].ish;
    startj = ientry[ientry_ind].startj;
    endj   = ientry[ientry_ind].endj;
  } else {
    indi = 0;
    ish  = 0;
    startj = 1;
    endj = 0;
  }

  // Calculate shift for i-atom
  float shz = (ish/9 - 1)*d_setup.boxz;
  ish -= (ish/9)*9;
  float shy = (ish/3 - 1)*d_setup.boxy;
  ish -= (ish/3)*3;
  float shx = (ish - 1)*d_setup.boxx;

  const unsigned int sh_start = tilesize*threadIdx.y;
  const unsigned int tid = threadIdx.x + blockDim.x*threadIdx.y;

  unsigned int load_ij;
  if (tilesize == 16) {
    load_ij = threadIdx.x % tilesize;
  } else {
    load_ij = threadIdx.x;
  }

  // Load i-atom data to shared memory (and shift coordinates)
  float4 xyzq_tmp = xyzq[indi + load_ij];
  xyzq_i[sh_start + load_ij].x = xyzq_tmp.x + shx;
  xyzq_i[sh_start + load_ij].y = xyzq_tmp.y + shy;
  xyzq_i[sh_start + load_ij].z = xyzq_tmp.z + shz;
  xyzq_i[sh_start + load_ij].w = xyzq_tmp.w;

  vdwtype_i[sh_start + load_ij] = vdwtype[indi + load_ij];

  fix[tid] = (AT)0;
  fiy[tid] = (AT)0;
  fiz[tid] = (AT)0;

  if (!tex_vdwparam) {
    // Copy vdwparam to shared memory
    if (tid < nvdwparam)
      vdwparam_sh[tid] = vdwparam[tid];
  }

  __syncthreads();

  //  float roff2 = roff*roff;

  //  vdw.setup(ron2, roff2);

  /*
#if (VDWTYPE == VSH)
  // roffinv6  = 1/roff^6
  // roffinv12 = 1/roff^12
  // roffinv18 = 1/roff^18
  float roffinv6 = 1.0f/(roff2*roff2*roff2);
  float roffinv12 = roffinv6*roffinv6;
  float roffinv18 = roffinv12*roffinv6;
#elif (VDWTYPE == VSW)
  float ron2 = ron*ron;
  //  inv_roff2_ron2 = 1.0/(roff2 - ron2)^3
  float inv_roff2_ron2 = roff2 - ron2;
  inv_roff2_ron2 = 1.0f/(inv_roff2_ron2*inv_roff2_ron2*inv_roff2_ron2);
#endif

#if (ELECTYPE == EWALD)
  float kappa2 = kappa*kappa;
#endif

#ifdef CALC_ENERGY
  double vdwpotl = 0.0;
  double coulpotl = 0.0;
#endif
  */

  double vdwpotl;
  double coulpotl;
  if (calc_energy) {
    vdwpotl = 0.0;
    coulpotl = 0.0;
  }

  for (int jtile=startj;jtile <= endj;jtile++) {

    // Load j-atom starting index and exclusion mask
    unsigned int excl;
    if (tilesize == 16) {
      // For 16x16 tile, the exclusion mask per is 8 bits per thread:
      // NUM_THREAD_PER_EXCL = 4
      excl = tile_excl[jtile].excl[threadIdx.x/num_thread_per_excl] >> 
	((threadIdx.x % num_thread_per_excl)*num_excl);
    } else {
      excl = tile_excl[jtile].excl[load_ij];
    }
    int indj = tile_indj[jtile];

    // Skip empty tile
    if (__all(~excl == 0)) continue;

    float4 xyzq_j = xyzq[indj + load_ij];
    int ja = vdwtype[indj + load_ij];

    // Clear j forces
    AT fjx = (AT)0;
    AT fjy = (AT)0;
    AT fjz = (AT)0;

    for (int t=0;t < num_excl;t++) {
      
      unsigned int excl_bit = !(excl & 1);

      if (excl_bit) {
	
	int ii;
	if (tilesize == 16) {
	  ii = sh_start + (threadIdx.x + t*2 + (threadIdx.x/tilesize)*(tilesize-1)) % tilesize;
	} else {
	  ii = sh_start + ((threadIdx.x + t) % tilesize);
	}
	
	float dx = xyzq_i[ii].x - xyzq_j.x;
	float dy = xyzq_i[ii].y - xyzq_j.y;
	float dz = xyzq_i[ii].z - xyzq_j.z;
	
	float r2 = dx*dx + dy*dy + dz*dz;

	if (r2 < d_setup.roff2) {

	  int ia = vdwtype_i[ii];
	  int aa = (ja > ia) ? ja : ia;      // aa = max(ja,ia)
	  int ivdw = (aa*(aa-3) + 2*(ja + ia) - 2) >> 1;

	  float c6, c12;
	  if (tex_vdwparam) {
	    float2 c6c12 = tex1Dfetch(vdwparam_texref, ivdw);
	    c6  = c6c12.x;
	    c12 = c6c12.y;
	  } else {
	    c6 = vdwparam_sh[ivdw];
	    c12 = vdwparam_sh[ivdw+1];
	  }

	  float rinv = rsqrtf(r2);
	  float rinv2 = rinv*rinv;
	  float r = r2*rinv;

	  float fij_vdw = pair_vdw_force<vdw_model, calc_energy>(r2, r, rinv, rinv2, c6, c12, vdwpotl);

	  float fij_elec = pair_elec_force<elec_model, calc_energy>(r2, r, rinv, xyzq_i[ii].w, xyzq_j.w, coulpotl);

	  float fij = (fij_vdw - fij_elec)*rinv2;

	  AT fxij;
	  AT fyij;
	  AT fzij;
	  calc_component_force<AT, CT>(fij, dx, dy, dz, fxij, fyij, fzij);

	  fjx -= fxij;
	  fjy -= fyij;
	  fjz -= fzij;

	  if (tilesize == 16) {
	    // We need to re-calculate ii because ii must be warp sized in order to
	    // prevent race condition
	    int tmp = (threadIdx.x + t*2) % 16 + (threadIdx.x/16)*31;
	    ii = sh_start*2 + (tmp + (tmp/32)*16) % 32;
	  }

	  fix[ii] += fxij;
	  fiy[ii] += fyij;
	  fiz[ii] += fzij;
	}
      }

      // Advance exclusion mask
      excl >>= 1;
    }

    // Dump register forces (fjx, fjy, fjz)
    write_force<AT, CT>(fjx, fjy, fjz, indj+load_ij, stride, force);
  }

  // Dump shared memory force (fi)
  __syncthreads();
  write_force<AT, CT>(fix[tid], fiy[tid], fiz[tid], indi+load_ij, stride, force);

  if (calc_energy) {
    // Reduce energies to (pot)
    // Reduces within thread block, uses the "xyzq_i" shared memory buffer
    __syncthreads();          // NOTE: this makes sure we can write to xyzq_i 
    double2 *potbuf = (double2 *)(xyzq_i);
    potbuf[tid].x = vdwpotl;
    potbuf[tid].y = coulpotl;
    // sync to make sure all threads in block are finished writing share memory
    __syncthreads();
    const int nthreadblock = blockDim.x*blockDim.y;
    for (int i=1;i < nthreadblock;i *= 2) {
      int pos = tid + i;
      double vdwpot_val  = (pos < nthreadblock) ? potbuf[pos].x : 0.0;
      double coulpot_val = (pos < nthreadblock) ? potbuf[pos].y : 0.0;
      __syncthreads();
      potbuf[tid].x += vdwpot_val;
      potbuf[tid].y += coulpot_val;
      __syncthreads();
    }
    if (tid == 0) {
      atomicAdd((double *)&force[stride*3],   potbuf[0].x);
      atomicAdd((double *)&force[stride*3+1], potbuf[0].y);
    }

  }

}

//
// Class creator
//
template <typename AT, typename CT>
DirectForce<AT, CT>::DirectForce() {
  vdwparam = NULL;
  nvdwparam = 0;
  vdwparam_len = 0;
  use_tex_vdwparam = true;
  vdwparam_texref_bound = false;

  vdwtype = NULL;
  vdwtype_len = 0;

  set_calc_vdw(true);
  set_calc_elec(true);
}

//
// Class destructor
//
template <typename AT, typename CT>
DirectForce<AT, CT>::~DirectForce() {
  if (vdwparam != NULL) deallocate<CT>(&vdwparam);
  if (vdwtype != NULL) deallocate<int>(&vdwtype);
}

//
// Copies h_setup -> d_setup
//
void update_setup() {
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(&d_setup), &h_setup, sizeof(DirectSettings_t)));
}

//
// Sets parameters for the nonbonded computation
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::setup(CT boxx, CT boxy, CT boxz, 
				CT kappa,
				CT roff, CT ron,
				int vdw_model, int elec_model,
				bool calc_vdw, bool calc_elec) {
  h_setup.boxx = boxx;
  h_setup.boxy = boxy;
  h_setup.boxz = boxz;
  h_setup.kappa = kappa;
  h_setup.kappa2 = kappa*kappa;
  h_setup.roff2 = roff*roff;
  h_setup.ron2 = ron*ron;

  h_setup.roffinv6 = ((CT)1.0)/(h_setup.roff2*h_setup.roff2*h_setup.roff2);
  h_setup.roffinv12 = h_setup.roffinv6*h_setup.roffinv6;
  h_setup.roffinv18 = h_setup.roffinv12*h_setup.roffinv6;

  h_setup.inv_roff2_ron2 = h_setup.roff2 - h_setup.ron2;
  h_setup.inv_roff2_ron2 = ((CT)1.0)/(h_setup.inv_roff2_ron2*h_setup.inv_roff2_ron2*h_setup.inv_roff2_ron2);

  this->vdw_model = vdw_model;
  this->elec_model = elec_model;

  set_calc_vdw(calc_vdw);
  set_calc_elec(calc_elec);

  update_setup();
}

//
// Sets "calc_vdw" flag
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_calc_vdw(bool calc_vdw) {
  this->calc_vdw = calc_vdw;
}

//
// Sets "calc_elec" flag
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_calc_elec(bool calc_elec) {
  this->calc_elec = calc_elec;
}

//
// Sets box size
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_box_size(CT boxx, CT boxy, CT boxz) {
  h_setup.boxx = boxx;
  h_setup.boxy = boxy;
  h_setup.boxz = boxz;
  update_setup();
}

//
// Sets VdW parameters
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_vdwparam(int nvdwparam, CT *h_vdwparam) {

  this->nvdwparam = nvdwparam;

  // "Fix" vdwparam by multiplying c6 by 6.0 and c12 by 12.0
  // NOTE: this is done in order to avoid the multiplication in the inner loop
  CT *h_vdwparam_fixed = new CT[nvdwparam];
  for(int i=0;i < nvdwparam/2;i++) {
    h_vdwparam_fixed[i*2]   = ((CT)6.0)*h_vdwparam[i*2];
    h_vdwparam_fixed[i*2+1] = ((CT)12.0)*h_vdwparam[i*2+1];
  }

  bool vdwparam_reallocated = false;
  if (nvdwparam > vdwparam_len) {
    reallocate<CT>(&vdwparam, &vdwparam_len, nvdwparam, 1.0f);
    vdwparam_reallocated = true;
  }
  copy_HtoD<CT>(h_vdwparam_fixed, vdwparam, nvdwparam);
  delete [] h_vdwparam_fixed;

  if (use_tex_vdwparam && vdwparam_reallocated) {
    // Unbind texture
    if (vdwparam_texref_bound) {
      cudaCheck(hipUnbindTexture(vdwparam_texref));
      vdwparam_texref_bound = 0;
    }
    // Bind texture
    vdwparam_texref.normalized = 0;
    vdwparam_texref.filterMode = hipFilterModePoint;
    vdwparam_texref.addressMode[0] = hipAddressModeClamp;
    vdwparam_texref.channelDesc.x = 32;
    vdwparam_texref.channelDesc.y = 32;
    vdwparam_texref.channelDesc.z = 0;
    vdwparam_texref.channelDesc.w = 0;
    vdwparam_texref.channelDesc.f = hipChannelFormatKindFloat;
    cudaCheck(hipBindTexture(NULL, vdwparam_texref, vdwparam, 
			      nvdwparam*sizeof(float)));
    vdwparam_texref_bound = true;
  }
}

//
// Sets VdW parameters by loading them from a file
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_vdwparam(const char *filename) {
  
  int nvdwparam;
  CT *h_vdwparam;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> nvdwparam;

    h_vdwparam = new float[nvdwparam];

    for (int i=0;i < nvdwparam;i++) {
      file >> h_vdwparam[i];
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  set_vdwparam(nvdwparam, h_vdwparam);

  delete [] h_vdwparam;
}

//
// Sets vdwtype array
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_vdwtype(int ncoord, int *h_vdwtype) {
  reallocate<int>(&vdwtype, &vdwtype_len, ncoord, 1.2f);
  copy_HtoD<int>(h_vdwtype, vdwtype, ncoord);
}

//
// Sets vdwtype array by loading it from a file
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::set_vdwtype(const char *filename) {

  int ncoord;
  int *h_vdwtype;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> ncoord;

    h_vdwtype = new int[ncoord];

    for (int i=0;i < ncoord;i++) {
      file >> h_vdwtype[i];
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  set_vdwtype(ncoord, h_vdwtype);
  
  delete [] h_vdwtype;
}

//
// Calculates direct force
//
template <typename AT, typename CT>
void DirectForce<AT, CT>::calc_force(const int ncoord, const float4 *xyzq,				     
				     const NeighborList<32> *nlist, const bool calc_energy,
				     AT *force) {

  int stride = 0;

  dim3 nthread(32, 6, 1);
  dim3 nblock((nlist->ni-1)/nthread.y+1, 1, 1);

  int vdw_model_loc = calc_vdw ? vdw_model : NONE;
  int elec_model_loc = calc_elec ? elec_model : NONE;

  if (vdw_model_loc == VDW_VSH) {
    if (elec_model_loc == EWALD) {
      if (calc_energy) {
	calc_force_kernel <AT, CT, 32, VDW_VSH, EWALD, true, true>
	  <<< nblock, nthread >>>(nlist->ni, nlist->ientry, nlist->tile_indj, nlist->tile_excl,
				  stride, vdwparam, nvdwparam, xyzq, vdwtype,
				  force);
      } else {
	calc_force_kernel <AT, CT, 32, VDW_VSH, EWALD, false, true>
	  <<< nblock, nthread >>>(nlist->ni, nlist->ientry, nlist->tile_indj, nlist->tile_excl,
				  stride, vdwparam, nvdwparam, xyzq, vdwtype,
				  force);
      }
    } else {
      std::cout<<"DirectForce<AT, CT>::calc_force, Invalid EWALD model"<<std::endl;
      exit(1);
    }
  } else {
    std::cout<<"DirectForce<AT, CT>::calc_force, Invalid VDW model"<<std::endl;
    exit(1);
  }
}


//
// Explicit instances of DirectForce
//
template class DirectForce<long long int, float>;
