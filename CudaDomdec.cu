#include "hip/hip_runtime.h"
#include <cassert>
#include "gpu_utils.h"
#include "CudaDomdec.h"

//
// Calculates (x, y, z) shift
// (x0, y0, z0) = fractional origin
//
__global__ void calc_xyz_shift(const int ncoord, const int stride, const double* __restrict__ coord,
			       const double x0, const double y0, const double z0,
			       const double inv_boxx, const double inv_boxy, const double inv_boxz,
			       float3* __restrict__ xyz_shift) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < ncoord) {
    double x = coord[i]*inv_boxx;
    double y = coord[i+stride]*inv_boxy;
    double z = coord[i+stride*2]*inv_boxz;
    float3 shift;
    shift.x = ceilf(x0 - x);
    shift.y = ceilf(y0 - y);
    shift.z = ceilf(z0 - z);
    xyz_shift[i] = shift;
  }
}

//
// Re-order coordinates
//
__global__ void reorder_coord_kernel(const int ncoord, const int stride,
				     const int* __restrict__ ind_sorted,
				     const double* __restrict__ coord_src,
				     double* __restrict__ coord_dst) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  const int stride2 = stride*2;
  if (i < ncoord) {
    int j = ind_sorted[i];
    coord_dst[i]         = coord_src[j];
    coord_dst[i+stride]  = coord_src[j+stride];
    coord_dst[i+stride2] = coord_src[j+stride2];
  }
}

//
// Re-order xyz_shift
//
__global__ void reorder_xyz_shift_kernel(const int ncoord,
					 const int* __restrict__ ind_sorted,
					 const float3* __restrict__ xyz_shift_in,
					 float3* __restrict__ xyz_shift_out) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < ncoord) {
    int j = ind_sorted[i];
    xyz_shift_out[i] = xyz_shift_in[j];
  }
}

//
// Re-order mass
//
__global__ void reorder_mass_kernel(const int ncoord,
				    const int* __restrict__ ind_sorted,
				    const float* __restrict__ mass_in,
				    float* __restrict__ mass_out) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < ncoord) {
    int j = ind_sorted[i];
    mass_out[i] = mass_in[j];
  }
}

//
// Choose z coordinates
//
__global__ void choose_z_coord_kernel(const int ncoord, const float* __restrict__ zbound_f,
				      const float rcut_f, const float invz, const float* __restrict__ shz,
				      const double* __restrict__ coord,
				      unsigned char* __restrict__ coord_tags) {
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int izone = tid/ncoord;
  const int i = tid - izone*ncoord;

  float z = ((float)coord[i])*invz + 0.5f;
  z -= floor(z);
  z += shz[izone] - zbound_f[izone];
  z = max(0.0f, z);
  unsigned char tag;
  if (z < rcut_f) {
    // In the zone
    tag = 1;
  } else {
    tag = 0;
  }
  coord_tags[i + izone*ncoord] = tag;
}


//#############################################################################################
//#############################################################################################
//#############################################################################################

//
// Class creator
//
CudaDomdec::CudaDomdec(int ncoord_glo, double boxx, double boxy, double boxz, double rnl,
		       int nx, int ny, int nz, int mynode, CudaMPI& cudaMPI) : 
  Domdec(ncoord_glo, boxx, boxy, boxz, rnl, nx, ny, nz, mynode), homezone(*this, cudaMPI), 
  D2Dcomm(*this, cudaMPI) {

  xyz_shift0_len = 0;
  xyz_shift0 = NULL;

  xyz_shift1_len = 0;
  xyz_shift1 = NULL;

  mass_tmp_len = 0;
  mass_tmp = NULL;
}

//
// Class destructor
//
CudaDomdec::~CudaDomdec() {
  if (xyz_shift0 != NULL) deallocate<float3>(&xyz_shift0);
  if (xyz_shift1 != NULL) deallocate<float3>(&xyz_shift1);
  if (mass_tmp != NULL) deallocate<float>(&mass_tmp);
}

//
// Builds coordinate distribution across all nodes
// NOTE: Here all nodes have all coordinates.
// NOTE: Used only in the beginning of dynamics
//
void CudaDomdec::build_homezone(hostXYZ<double>& coord) {
  this->zone_ncoord[0] = homezone.build(coord);
  for (int i=1;i < 8;i++) zone_ncoord[i] = 0;
  this->update_zone_pcoord();
}

//
// Update coordinate distribution across all nodes
// Update is done according to coord, coord2 is a hangaround
// NOTE: Used during dynamics
//
void CudaDomdec::update_homezone(cudaXYZ<double>& coord, cudaXYZ<double>& coord2, hipStream_t stream) {
  if (numnode > 1) {
    this->zone_ncoord[0] = homezone.update(coord, coord2, stream);
    for (int i=1;i < 8;i++) zone_ncoord[i] = 0;
    this->update_zone_pcoord();
  }
}

//
// Communicate coordinates
//
void CudaDomdec::comm_coord(cudaXYZ<double>& coord, const bool update, hipStream_t stream) {

  D2Dcomm.comm_coord(coord, homezone.get_loc2glo(), update);

  // Calculate xyz_shift
  if (update) {
    int nthread, nblock;

    /*
    // ---------------- Z -------------------
    int nreq = 0;
    for (int i=0;i < nz_comm;i++) {
      if (z_recv_count[i] > 0) {
	cuda_irecv(z_recv_buf[i], z_recv_count[i], z_recv_node[i], &reqbuf[nreq]);
	nreq++;
      }
    }

    for (int i=0;i < nz_comm;i++)
      h_z_boundary[i] = get_fz_boundary(homeix, homeiy, homeiz-i, rnl, r_bonded);

    nthread = 512;
    nblock = (zone_ncoord[0] - 1)/nthread + 1;
    choose_z_coord_kernel<<< nblock, nthread, 0, stream >>>
      (zone_ncoord[0], z_boundary, rnl/boxz, coord->data, coord_tags);

    if (mpi_cuda_aware) {
      for (int i=0;i < nz_comm;i++) {
	if (z_send_count[i] > 0) {
	  cuda_isend(z_send_buf[i], z_send_count[i], z_send_node[i], &reqbuf[nreq]);
	  nreq++;
	}
      }
    } else {
      for (int i=0;i < nz_comm;i++) {
	if (z_send_count[i] > 0) {
	}
      }
    }

    cuda_waitall(nreq, reqbuf);

    // ---------------- Y -------------------

    for (int i=0;i < ny_comm;i++)
      h_y_boundary[i] = get_fy_boundary(homeix, homeiy-i, homeiz, rnl, r_bonded);

    for (int i=0;i < ny_comm;i++)
      get_ex_boundary(homeix, homeiy-i, homeiz, yf, zf, &
		      z_bonded, q_checkbonded, cut, rcut_bonded);

    // ---------------- X -------------------

    for (int i=0;i < nx_comm;i++)
      get_fx_boundary(homeix-i, xf);

    for (int i=0;i < nx_comm;i++)
      get_ez_boundary(homeix-i, homeiy, xf, yf, y_bonded, q_checkbonded);

    for (int i=0;i < nx_comm;i++)
      get_ey_boundary(homeix-i, homeiy, homeiz, xf, zf, z_bonded, q_checkbonded, cut);

    for (int i=0;i < nx_comm;i++) {
      get_c_boundary(homeix-i, homeiy, homeiz, xf, yf, zf, &
			  y_bonded, z_bonded, q_checkbonded);
      get_z0_for_c(homeix-i, homeiy, homeiz, z0);
    }
    */

    // Calculate xyz shift
    double x0 = 0.0;
    double y0 = 0.0;
    double z0 = 0.0;

    float fac = (numnode > 1) ? 1.2f : 1.0f;
    reallocate<float3>(&xyz_shift0, &xyz_shift0_len, zone_pcoord[7], fac);
    reallocate<float3>(&xyz_shift1, &xyz_shift1_len, zone_pcoord[7], fac);
    
    nthread = 512;
    nblock = (zone_pcoord[7] - 1)/nthread + 1;
    calc_xyz_shift<<< nblock, nthread, 0, stream >>>
      (zone_pcoord[7], coord.stride, coord.data,
       x0, y0, z0, this->get_inv_boxx(), this->get_inv_boxy(), this->get_inv_boxz(), xyz_shift0);
    cudaCheck(hipGetLastError());
  }

}

//
// Communicate forces
//
void CudaDomdec::comm_force(Force<long long int>& force, hipStream_t stream) {
}

//
// Re-order coordinates using ind_sorted: coord_src => coord_dst
//
void CudaDomdec::reorder_coord(cudaXYZ<double>& coord_src, cudaXYZ<double>& coord_dst,
			       const int* ind_sorted, hipStream_t stream) {
  assert(coord_src.match(&coord_dst));
  assert(zone_pcoord[7] == coord_src.n);

  if (numnode == 1) {
    int nthread = 512;
    int nblock = (zone_pcoord[7] - 1)/nthread + 1;
    reorder_coord_kernel<<< nblock, nthread, 0, stream >>>
      (zone_pcoord[7], coord_src.stride, ind_sorted, coord_src.data, coord_dst.data);
    cudaCheck(hipGetLastError());
  } else {
    std::cerr << "CudaDomdec::reorder_coord, not ready for numnode > 1" << std::endl;
    exit(1);
  }

}

//
// Re-order xyz_shift
//
void CudaDomdec::reorder_xyz_shift(const int* ind_sorted, hipStream_t stream) {

  int nthread = 512;
  int nblock = (zone_pcoord[7] - 1)/nthread + 1;
  reorder_xyz_shift_kernel<<< nblock, nthread, 0, stream >>>
    (zone_pcoord[7], ind_sorted, xyz_shift0, xyz_shift1);
  cudaCheck(hipGetLastError());

  float3 *p = xyz_shift0;
  xyz_shift0 = xyz_shift1;
  xyz_shift1 = p;

  int t = xyz_shift0_len;
  xyz_shift0_len = xyz_shift1_len;
  xyz_shift1_len = t;
}

//
// Re-order mass
//
void CudaDomdec::reorder_mass(float *mass, const int* ind_sorted, hipStream_t stream) {

  reallocate<float>(&mass_tmp, &mass_tmp_len, zone_pcoord[7], 1.2f);

  int nthread = 512;
  int nblock = (zone_pcoord[7] - 1)/nthread + 1;
  reorder_mass_kernel<<< nblock, nthread, 0, stream >>>
    (zone_pcoord[7], ind_sorted, mass, mass_tmp);
  cudaCheck(hipGetLastError());

  copy_DtoD<float>(mass_tmp, mass, zone_pcoord[7], stream);
}
