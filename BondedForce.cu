#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "BondedForce.h"

// Energy and virial in device memory
static __device__ BondedEnergyVirial_t d_energy_virial;

//
// Calculates box shift
// 
// On CPU this index is calculated as:
//
// ! shift index = 1...26*3+1
// calc_ishift_{P*} = (is(1)+1 + (is(2)+1)*3 + (is(3)+1)*9 + 1)*3 - 2
//
// where is(1:3) = {-1, 0, 1}
//
__forceinline__ __device__
float3 calc_box_shift(int ish,
		      const float boxx,
		      const float boxy,
		      const float boxz) {
  float3 sh;
  ish = (ish+2)/3 - 1;
  sh.z = (ish/9 - 1)*boxz;
  ish -= (ish/9)*9;
  sh.y = (ish/3 - 1)*boxy;
  ish -= (ish/3)*3;
  sh.x = (ish - 1)*boxx;
  return sh;
}

//
// Reduces energy values
//
__forceinline__ __device__
void reduce_energy(const double epot, volatile double *sh_epot,
		   double *global_epot) {
  sh_epot[threadIdx.x] = epot;
  __syncthreads();
  for (int i=1;i < blockDim.x;i *= 2) {
    int t = threadIdx.x + i;
    double epot_val  = (t < blockDim.x) ? sh_epot[t] : 0.0;
    __syncthreads();
    sh_epot[threadIdx.x] += epot_val;
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    double epot_val = sh_epot[0];
    atomicAdd(global_epot, epot_val);
  }
}

//
// Templated sqrt() -function
//
template <typename T>
__forceinline__ __device__
double sqrt_template(const T x) {
  if (sizeof(T) == 4) {
    return sqrtf(x);
  } else {
    return sqrt(x);
  }
}

template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__device__ void calc_bond_force_device(const int pos,
				       const bondlist_t* __restrict__ bondlist,
				       const float2* __restrict__ bondcoef,
				       const float4* __restrict__ xyzq,
				       const int stride,
				       const float boxx, const float boxy, const float boxz,
				       AT* __restrict__ force, double &epot) {

  int ii = bondlist[pos].i - 1;
  int jj = bondlist[pos].j - 1;
  int ic = bondlist[pos].itype - 1;
  int ish = bondlist[pos].ishift;

  // Calculate shift for i-atom
  float3 sh_xyz = calc_box_shift(ish, boxx, boxy, boxz);

  float4 xyzqi = xyzq[ii];
  float4 xyzqj = xyzq[jj];

  CT dx = xyzqi.x + sh_xyz.x - xyzqj.x;
  CT dy = xyzqi.y + sh_xyz.y - xyzqj.y;
  CT dz = xyzqi.z + sh_xyz.z - xyzqj.z;

  CT r = sqrt_template<CT>(dx*dx + dy*dy + dz*dz);

  float2 bondcoef_val = bondcoef[ic];
  CT db = r - (CT)bondcoef_val.x;
  CT fij = db*(CT)bondcoef_val.y;
  
  if (calc_energy) {
    epot += (double)(fij*db);
  }
  fij *= ((CT)2)/r;
  
  AT fxij, fyij, fzij;
  calc_component_force<AT, CT>(fij, dx, dy, dz, fxij, fyij, fzij);
  
  // Store forces
  write_force<AT>(fxij, fyij, fzij, ii, stride, force);
  write_force<AT>(-fxij, -fyij, -fzij, jj, stride, force);
  
  // Store shifted forces
  if (calc_virial) {
    //       sforce(is)   = sforce(is)   + fx
    //       sforce(is+1) = sforce(is+1) + fy
    //       sforce(is+2) = sforce(is+2) + fz
  }
  
}

//
// bondcoef.x = cbb
// bondcoef.y = cbc
//
template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_bond_force_kernel(
				       const int nbondlist,
				       const bondlist_t* __restrict__ bondlist,
				       const float2* __restrict__ bondcoef,
				       const float4* __restrict__ xyzq,
				       const int stride,
				       const float boxx, const float boxy, const float boxz,
				       AT* __restrict__ force
) {
  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) {
    epot = 0.0;
  }

  while (pos < nbondlist) {
    calc_bond_force_device<AT, CT, calc_energy, calc_virial>
      (pos, bondlist, bondcoef, xyzq,
       stride, boxx, boxy, boxz,
       force, epot);
    pos += blockDim.x*gridDim.x;
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_bond);
  }

}

//
// bondcoef.x = cbb
// bondcoef.y = cbc
//
template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_ureyb_force_kernel(
				       const int nureyblist,
				       const bondlist_t* __restrict__ ureyblist,
				       const float2* __restrict__ ureybcoef,
				       const float4* __restrict__ xyzq,
				       const int stride,
				       const float boxx, const float boxy, const float boxz,
				       AT* __restrict__ force
) {
  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) {
    epot = 0.0;
  }

  while (pos < nureyblist) {
    calc_bond_force_device<AT, CT, calc_energy, calc_virial>
      (pos, ureyblist, ureybcoef, xyzq,
       stride, boxx, boxy, boxz,
       force, epot);
    pos += blockDim.x*gridDim.x;
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_ureyb);
  }

}

template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__device__ void calc_angle_force_device(const int pos,
					const anglelist_t* __restrict__ anglelist,
					const float2* __restrict__ anglecoef,
					const float4* __restrict__ xyzq,
					const int stride,
					const float boxx, const float boxy, const float boxz,
					AT* __restrict__ force, double &epot) {

    int ii = anglelist[pos].i - 1;
    int jj = anglelist[pos].j - 1;
    int kk = anglelist[pos].k - 1;
    int ic = anglelist[pos].itype - 1;
    int ish = anglelist[pos].ishift1;
    int ksh = anglelist[pos].ishift2;

    // Calculate shift for i-atom
    float3 ish_xyz = calc_box_shift(ish, boxx, boxy, boxz);

    // Calculate shift for k-atom
    float3 ksh_xyz = calc_box_shift(ksh, boxx, boxy, boxz);

    CT dxij = xyzq[ii].x + ish_xyz.x - xyzq[jj].x;
    CT dyij = xyzq[ii].y + ish_xyz.y - xyzq[jj].y;
    CT dzij = xyzq[ii].z + ish_xyz.z - xyzq[jj].z;

    CT dxkj = xyzq[kk].x + ksh_xyz.x - xyzq[jj].x;
    CT dykj = xyzq[kk].y + ksh_xyz.y - xyzq[jj].y;
    CT dzkj = xyzq[kk].z + ksh_xyz.z - xyzq[jj].z;

    CT rij = sqrtf(dxij*dxij + dyij*dyij + dzij*dzij);
    CT rkj = sqrtf(dxkj*dxkj + dykj*dykj + dzkj*dzkj);

    CT rij_inv = ((CT)1)/rij;
    CT rkj_inv = ((CT)1)/rkj;

    CT dxijr = dxij*rij_inv;
    CT dyijr = dyij*rij_inv;
    CT dzijr = dzij*rij_inv;
    CT dxkjr = dxkj*rkj_inv;
    CT dykjr = dykj*rkj_inv;
    CT dzkjr = dzkj*rkj_inv;
    CT cst = dxijr*dxkjr + dyijr*dykjr + dzijr*dzkjr;

    // anglecoef.x = ctb
    // anglecoef.y = ctc
    float2 anglecoef_val = anglecoef[ic];

    // Restrict values of cst to the interval [-0.999 ... 0.999]
    // NOTE: we are ignoring the fancy stuff that is done on the CPU version
    cst = min((CT)0.999, max(-(CT)0.999, cst));

    CT at = acosf(cst);
    CT da = at - (CT)anglecoef_val.x;
    CT df = ((CT)anglecoef_val.y)*da;
    if (calc_energy) {
      epot += epot + (double)(df*da);
    }
    CT st2r = ((CT)1.0)/(((CT)1.0) - cst*cst);
    CT str = sqrtf(st2r);
    df = -((CT)2.0)*df*str;

    CT dtxi = rij_inv*(dxkjr - cst*dxijr);
    CT dtxj = rkj_inv*(dxijr - cst*dxkjr);
    CT dtyi = rij_inv*(dykjr - cst*dyijr);
    CT dtyj = rkj_inv*(dyijr - cst*dykjr);
    CT dtzi = rij_inv*(dzkjr - cst*dzijr);
    CT dtzj = rkj_inv*(dzijr - cst*dzkjr);

    AT AT_dtxi, AT_dtyi, AT_dtzi;
    AT AT_dtxj, AT_dtyj, AT_dtzj;
    calc_component_force<AT, CT>(df, dtxi, dtyi, dtzi, AT_dtxi, AT_dtyi, AT_dtzi);
    calc_component_force<AT, CT>(df, dtxj, dtyj, dtzj, AT_dtxj, AT_dtyj, AT_dtzj);

    write_force<AT>(AT_dtxi, AT_dtyi, AT_dtzi, ii, stride, force);
    write_force<AT>(AT_dtxj, AT_dtyj, AT_dtzj, kk, stride, force);
    write_force<AT>(-AT_dtxi-AT_dtxj, -AT_dtyi-AT_dtyj, -AT_dtzi-AT_dtzj, jj, stride, force);
    
    if (calc_virial) {
      //       sforce(is) = sforce(is) + dtxi
      //       sforce(ks) = sforce(ks) + dtxj
    }
    
    if (calc_virial) {
      //       sforce(is+1) = sforce(is+1) + dtxi
      //       sforce(ks+1) = sforce(ks+1) + dtxj
    }

    if (calc_virial) {
      //       sforce(is+2) = sforce(is+2) + dtxi
      //       sforce(ks+2) = sforce(ks+2) + dtxj
    }

}

//
// anglecoef.x = ctb
// anglecoef.y = ctc
//
template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_angle_force_kernel(const int nanglelist,
					const anglelist_t* __restrict__ anglelist,
					const float2* __restrict__ anglecoef,
					const float4* __restrict__ xyzq,
					const int stride,
					const float boxx, const float boxy, const float boxz,
					AT* __restrict__ force
) {
  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) epot = 0.0;

  while (pos < nanglelist) {
    calc_angle_force_device<AT, CT, calc_energy, calc_virial>
      (pos, anglelist, anglecoef, xyzq, stride,
       boxx, boxy, boxz, (AT *)force, epot);
    pos += blockDim.x*gridDim.x;
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_angle);
  }
}


//
// Dihedral potential
//
// dihecoef.x = cpd (integer)
// dihecoef.y = cpc
// dihecoef.z = cpsin
// dihecoef.w = cpcos
//
// Out: df, e
//
template <typename T, bool calc_energy>
__forceinline__ __device__
void dihe_pot(const float4* dihecoef, const int ic_in,
	      const T st, const T ct, T& df, double& e) {

  df = (T)0;
  if (calc_energy) e = 0.0;
  int ic = ic_in;

  bool lrep = true;
  while (lrep) {
    float4 dihecoef_val = dihecoef[ic];

    int iper = (int)dihecoef_val.x;
    lrep = (iper > 0) ? false : true;
    iper = abs(iper);

    T e1 = (T)1;
    T df1 = (T)0;
    T ddf1 = (T)0;

    // Calculation of cos(n*phi-phi0) and sin(n*phi-phi0).
    for (int nper=1;nper <= iper;nper++) {
      ddf1 = e1*ct - df1*st;
      df1 = e1*st + df1*ct;
      e1 = ddf1;
    }
    if (calc_energy) e1 = e1*dihecoef_val.w + df1*dihecoef_val.z;
    df1 = df1*dihecoef_val.w - ddf1*dihecoef_val.z;
    df1 = -iper*df1;
    if (calc_energy) {
      e1 += (T)1;
      if (iper == 0) e1 = (T)1;
    }

    float arg = dihecoef_val.y;
    if (calc_energy) e += arg*e1;
    df += arg*df1;

    ic++;
  }

}

//
// Improper dihedral potential
//
// imdihecoef.x = cid (integer)
// imdihecoef.y = cic
// imdihecoef.z = cisin
// imdihecoef.w = cicos
//
// Out: df, e
//
template<typename T, bool calc_energy>
__forceinline__ __device__
void imdihe_pot(const float4 *dihecoef, const int ic_in,
		const T st, const T ct, T& df, double& e) {
  df = (T)0;
  if (calc_energy) e = 0.0;

  float4 dihecoef_val = dihecoef[ic_in];

  if ((int)dihecoef_val.x != 0) {
    int ic = ic_in;
    bool lrep = true;
    while (lrep) {
      
      int iper = (int)dihecoef_val.x;
      lrep = (iper > 0) ? false : true;
      iper = abs(iper);
      
      T e1 = (T)1;
      T df1 = (T)0;
      T ddf1 = (T)0;
      
      // Calculation of cos(n*phi-phi0) and sin(n*phi-phi0).
      for (int nper=1;nper <= iper;nper++) {
	ddf1 = e1*ct - df1*st;
	df1 = e1*st + df1*ct;
	e1 = ddf1;
      }
      if (calc_energy) e1 = e1*dihecoef_val.w + df1*dihecoef_val.z;
      df1 = df1*dihecoef_val.w - ddf1*dihecoef_val.z;
      df1 = -iper*df1;
      if (calc_energy) {
	e1 += (T)1;
	if (iper == 0) e1 = (T)1;
      }
      
      float arg = dihecoef_val.y;
      if (calc_energy) e += arg*e1;
      df += arg*df1;
      
      ic++;
      if (lrep) dihecoef_val = dihecoef[ic];
    }       
    // use harmonic potential
  } else {
    // calcul of cos(phi-phi0),sin(phi-phi0) and (phi-phi0).

    T ca = ct*dihecoef_val.w + st*dihecoef_val.z;
    T sa = st*dihecoef_val.w - ct*dihecoef_val.z;
    T ap;
    if (ca > (T)0.1) {
      ap = asinf(sa);
    } else {
      //ap = sign(acos(max(ca,-(T)1)),sa);
      ap = acosf(max(ca,-(T)1));
      ap = (sa > (T)0) ? ap : -ap;
      // warning is now triggered at deltaphi=84.26...deg (used to be 90).
      //nbent = nbent + 1;
    }

    df = dihecoef_val.y*ap;
    if (calc_energy) e = df*ap;
    df *= (T)2;
  }
  
}

template <typename AT, typename CT, bool q_dihe, bool calc_energy, bool calc_virial>
__device__ void calc_dihe_force_device(const int pos,
				       const dihelist_t* __restrict__ dihelist,
				       const float4* __restrict__ dihecoef,
				       const float4* __restrict__ xyzq,
				       const int stride,
				       const float boxx, const float boxy, const float boxz,
				       AT* __restrict__ force, double &epot) {
  int ii = dihelist[pos].i - 1;
  int jj = dihelist[pos].j - 1;
  int kk = dihelist[pos].k - 1;
  int ll = dihelist[pos].l - 1;
  int ic = dihelist[pos].itype - 1;
  int ish = dihelist[pos].ishift1;
  int jsh = dihelist[pos].ishift2;
  int lsh = dihelist[pos].ishift3;

  // Calculate shift for i-atom
  float3 si = calc_box_shift(ish, boxx, boxy, boxz);

  // Calculate shift for j-atom
  float3 sj = calc_box_shift(jsh, boxx, boxy, boxz);

  // Calculate shift for l-atom
  float3 sl = calc_box_shift(lsh, boxx, boxy, boxz);

  CT fx = (xyzq[ii].x + si.x) - (xyzq[jj].x + sj.x);
  CT fy = (xyzq[ii].y + si.y) - (xyzq[jj].y + sj.y);
  CT fz = (xyzq[ii].z + si.z) - (xyzq[jj].z + sj.z);

  CT gx = xyzq[jj].x + sj.x - xyzq[kk].x;
  CT gy = xyzq[jj].y + sj.y - xyzq[kk].y;
  CT gz = xyzq[jj].z + sj.z - xyzq[kk].z;

  CT hx = xyzq[ll].x + sl.x - xyzq[kk].x;
  CT hy = xyzq[ll].y + sl.y - xyzq[kk].y;
  CT hz = xyzq[ll].z + sl.z - xyzq[kk].z;

  // A=F^G, B=H^G.
  CT ax = fy*gz - fz*gy;
  CT ay = fz*gx - fx*gz;
  CT az = fx*gy - fy*gx;
  CT bx = hy*gz - hz*gy;
  CT by = hz*gx - hx*gz;
  CT bz = hx*gy - hy*gx;

  CT ra2 = ax*ax + ay*ay + az*az;
  CT rb2 = bx*bx + by*by + bz*bz;
  CT rg = sqrtf(gx*gx + gy*gy + gz*gz);

  //    if((ra2 <= rxmin2) .or. (rb2 <= rxmin2) .or. (rg <= rxmin)) then
  //          nlinear = nlinear + 1
  //       endif

  CT rgr = 1.0f / rg;
  CT ra2r = 1.0f / ra2;
  CT rb2r = 1.0f / rb2;
  CT rabr = sqrtf(ra2r*rb2r);

  // ct=cos(phi)
  CT ct = (ax*bx + ay*by + az*bz)*rabr;
  //
  // ST=sin(phi), Note that sin(phi).G/|G|=B^A/(|A|.|B|)
  // which can be simplify to sin(phi)=|G|H.A/(|A|.|B|)
  CT st = rg*rabr*(ax*hx + ay*hy + az*hz);
  //
  //     Energy and derivative contributions.

  CT df;
  double e;
  if (q_dihe) {
    dihe_pot<CT, calc_energy>(dihecoef, ic, st, ct, df, e);
  } else {
    imdihe_pot<CT, calc_energy>(dihecoef, ic, st, ct, df, e);
  }

  if (calc_energy) epot += e;

  //
  //     Compute derivatives wrt catesian coordinates.
  //
  // GAA=dE/dphi.|G|/A^2, GBB=dE/dphi.|G|/B^2, FG=F.G, HG=H.G
  //  FGA=dE/dphi*F.G/(|G|A^2), HGB=dE/dphi*H.G/(|G|B^2)

  CT fg = fx*gx + fy*gy + fz*gz;
  CT hg = hx*gx + hy*gy + hz*gz;
  ra2r *= df;
  rb2r *= df;
  CT fga = fg*ra2r*rgr;
  CT hgb = hg*rb2r*rgr;
  CT gaa = ra2r*rg;
  CT gbb = rb2r*rg;
  // DFi=dE/dFi, DGi=dE/dGi, DHi=dE/dHi.

  // Store forces
  AT dfx, dfy, dfz;
  calc_component_force<AT, CT>(-gaa, ax, ay, az, dfx, dfy, dfz);
  write_force<AT>(dfx, dfy, dfz, ii, stride, force);

  AT dgx, dgy, dgz;
  calc_component_force<AT, CT>(fga, ax, ay, az, -hgb, bx, by, bz,
			       dgx, dgy, dgz);
  write_force<AT>(dgx-dfx, dgy-dfy, dgz-dfz, jj, stride, force);

  AT dhx, dhy, dhz;
  calc_component_force<AT, CT>(gbb, bx, by, bz, dhx, dhy, dhz);
  write_force<AT>(-dhx-dgx, -dhy-dgy, -dhz-dgz, kk, stride, force);
  write_force<AT>(dhx, dhy, dhz, ll, stride, force);

  if (calc_virial) {
    //       sforce(is)   = sforce(is)   + dfx
    //       sforce(is+1) = sforce(is+1) + dfy
    //       sforce(is+2) = sforce(is+2) + dfz
  }

  if (calc_virial) {
    //       sforce(js)   = sforce(js)   - dfx + dgx
    //       sforce(js+1) = sforce(js+1) - dfy + dgy
    //       sforce(js+2) = sforce(js+2) - dfz + dgz
  }
  if (calc_virial) {
    //       sforce(ls)   = sforce(ls)   + dhx
    //       sforce(ls+1) = sforce(ls+1) + dhy
    //       sforce(ls+2) = sforce(ls+2) + dhz
  }

}

//
// dihecoef.x = cpd (integer)
// dihecoef.y = cpc
// dihecoef.z = cpsin
// dihecoef.w = cpcos
//
//
template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_dihe_force_kernel(const int ndihelist,
				       const dihelist_t* __restrict__ dihelist,
				       const float4* __restrict__ dihecoef,
				       const float4* __restrict__ xyzq,
				       const int stride,
				       const float boxx, const float boxy, const float boxz,
				       AT* __restrict__ force
) {
  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) epot = 0.0;

  while (pos < ndihelist) {
    calc_dihe_force_device<AT, CT, true, calc_energy, calc_virial>
      (pos, dihelist, dihecoef, xyzq, stride,
       boxx, boxy, boxz, (AT *)force, epot);
    pos += blockDim.x*gridDim.x;
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_dihe);
  }
}

//
// dihecoef.x = cpd (integer)
// dihecoef.y = cpc
// dihecoef.z = cpsin
// dihecoef.w = cpcos
//
//
template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_imdihe_force_kernel(const int nimdihelist,
					 const dihelist_t* __restrict__ imdihelist,
					 const float4* __restrict__ imdihecoef,
					 const float4* __restrict__ xyzq,
					 const int stride,
					 const float boxx, const float boxy, const float boxz,
					 AT* __restrict__ force
) {
  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) epot = 0.0;

  while (pos < nimdihelist) {
    calc_dihe_force_device<AT, CT, false, calc_energy, calc_virial>
      (pos, imdihelist, imdihecoef, xyzq, stride,
       boxx, boxy, boxz, (AT *)force, epot);
    pos += blockDim.x*gridDim.x;
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_imdihe);
  }
}

/*
template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_all_forces_kernel() {

  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) {
    epot = 0.0;
  }

  if (pos < d_setup.nbondlist) {
    calc_bond_force_device<AT, CT, calc_energy, calc_virial>
      (pos, d_setup.bondlist, d_setup.bondcoef, d_setup.xyzq,
       d_setup.stride, d_setup.boxx, d_setup.boxy, d_setup.boxz,
       (AT *)d_setup.force, epot);
  } else if (pos < d_setup.nureyblist + d_setup.nbondlist) {
    calc_bond_force_device<AT, CT, calc_energy, calc_virial>
      (pos - d_setup.nbondlist, d_setup.ureyblist, d_setup.ureybcoef, d_setup.xyzq,
       d_setup.stride, d_setup.boxx, d_setup.boxy, d_setup.boxz,
       (AT *)d_setup.force, epot);
  } else if (pos < d_setup.nanglelist + d_setup.nureyblist + d_setup.nbondlist) {
    calc_angle_force_device<AT, CT, calc_energy, calc_virial>
      (pos - d_setup.nureyblist - d_setup.nbondlist,
       d_setup.anglelist, d_setup.anglecoef, d_setup.xyzq, d_setup.stride,
       d_setup.boxx, d_setup.boxy, d_setup.boxz, (AT *)d_setup.force, epot);
  } else if (pos < d_setup.ndihelist + d_setup.nanglelist + 
	     d_setup.nureyblist + d_setup.nbondlist) {
    calc_dihe_force_device<AT, CT, true, calc_energy, calc_virial>
      (pos - d_setup.nanglelist - d_setup.nureyblist - d_setup.nbondlist,
       d_setup.dihelist, d_setup.dihecoef, d_setup.xyzq, d_setup.stride,
       d_setup.boxx, d_setup.boxy, d_setup.boxz, (AT *)d_setup.force, epot);
  } else if (pos < d_setup.nimdihelist + d_setup.ndihelist + d_setup.nanglelist + 
	     d_setup.nureyblist + d_setup.nbondlist) {
    calc_dihe_force_device<AT, CT, false, calc_energy, calc_virial>
      (pos - d_setup.ndihelist - d_setup.nanglelist - d_setup.nureyblist - d_setup.nbondlist,
       d_setup.imdihelist, d_setup.imdihecoef, d_setup.xyzq, d_setup.stride,
       d_setup.boxx, d_setup.boxy, d_setup.boxz, (AT *)d_setup.force, epot);
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_bond);
  }

}
*/

template <typename AT, typename CT, bool calc_energy, bool calc_virial>
__global__ void calc_all_forces_kernel(
				       const int nbondlist,
				       const bondlist_t* __restrict__ bondlist,
				       const float2* __restrict__ bondcoef,

				       const int nureyblist,
				       const bondlist_t* __restrict__ ureyblist,
				       const float2* __restrict__ ureybcoef,

				       const int nanglelist,
				       const anglelist_t* __restrict__ anglelist,
				       const float2* __restrict__ anglecoef,

				       const int ndihelist,
				       const dihelist_t* __restrict__ dihelist,
				       const float4* __restrict__ dihecoef,

				       const int nimdihelist,
				       const dihelist_t* __restrict__ imdihelist,
				       const float4* __restrict__ imdihecoef,

				       const float4* __restrict__ xyzq,
				       const int stride,
				       const float boxx, const float boxy, const float boxz,
				       AT* __restrict__ force) {

  // Amount of shared memory required:
  // sh_epot: blockDim.x*sizeof(double)
  extern __shared__ double sh_epot[];

  int pos = threadIdx.x + blockIdx.x*blockDim.x;

  double epot;
  if (calc_energy) {
    epot = 0.0;
  }

  if (pos < nbondlist) {
    calc_bond_force_device<AT, CT, calc_energy, calc_virial>
      (pos, bondlist, bondcoef, xyzq,
       stride, boxx, boxy, boxz,
       force, epot);
  } else if (pos < nureyblist + nbondlist) {
    calc_bond_force_device<AT, CT, calc_energy, calc_virial>
      (pos - nbondlist, ureyblist, ureybcoef, xyzq,
       stride, boxx, boxy, boxz,
       force, epot);
  } else if (pos < nanglelist + nureyblist + nbondlist) {
    calc_angle_force_device<AT, CT, calc_energy, calc_virial>
      (pos - nureyblist - nbondlist,
       anglelist, anglecoef, xyzq, stride,
       boxx, boxy, boxz, force, epot);
  } else if (pos < ndihelist + nanglelist + 
	     nureyblist + nbondlist) {
    calc_dihe_force_device<AT, CT, true, calc_energy, calc_virial>
      (pos - nanglelist - nureyblist - nbondlist,
       dihelist, dihecoef, xyzq, stride,
       boxx, boxy, boxz, force, epot);
  } else if (pos < nimdihelist + ndihelist + nanglelist + 
	     nureyblist + nbondlist) {
    calc_dihe_force_device<AT, CT, false, calc_energy, calc_virial>
      (pos - ndihelist - nanglelist - nureyblist - nbondlist,
       imdihelist, imdihecoef, xyzq, stride,
       boxx, boxy, boxz, force, epot);
  }

  // Reduce energy
  if (calc_energy) {
    reduce_energy(epot, sh_epot, &d_energy_virial.energy_bond);
  }

}


//#############################################################################################

//
// Class creator
//
template <typename AT, typename CT>
BondedForce<AT, CT>::BondedForce() {
  nbondlist = 0;
  nbondcoef = 0;
  bondlist_len = 0;
  bondlist = NULL;
  bondcoef_len = 0;
  bondcoef = NULL;

  nureyblist = 0;
  nureybcoef = 0;
  ureyblist_len = 0;
  ureyblist = NULL;
  ureybcoef_len = 0;
  ureybcoef = NULL;

  nanglelist = 0;
  nanglecoef = 0;
  anglelist_len = 0;
  anglelist = NULL;
  anglecoef_len = 0;
  anglecoef = NULL;

  ndihelist = 0;
  ndihecoef = 0;
  dihelist_len = 0;
  dihelist = NULL;
  dihecoef_len = 0;
  dihecoef = NULL;

  nimdihelist = 0;
  nimdihecoef = 0;
  imdihelist_len = 0;
  imdihelist = NULL;
  imdihecoef_len = 0;
  imdihecoef = NULL;

  ncmaplist = 0;
  ncmapcoef = 0;
  cmaplist_len = 0;
  cmaplist = NULL;
  cmapcoef_len = 0;
  cmapcoef = NULL;

  allocate_host<BondedEnergyVirial_t>(&h_energy_virial, 1);
}

//
// Class destructor
//
template <typename AT, typename CT>
BondedForce<AT, CT>::~BondedForce() {
  if (bondlist != NULL) deallocate<bondlist_t>(&bondlist);
  if (bondcoef != NULL) deallocate<float2>(&bondcoef);

  if (ureyblist != NULL) deallocate<bondlist_t>(&ureyblist);
  if (ureybcoef != NULL) deallocate<float2>(&ureybcoef);

  if (anglelist != NULL) deallocate<anglelist_t>(&anglelist);
  if (anglecoef != NULL) deallocate<float2>(&anglecoef);

  if (dihelist != NULL) deallocate<dihelist_t>(&dihelist);
  if (dihecoef != NULL) deallocate<float4>(&dihecoef);

  if (imdihelist != NULL) deallocate<dihelist_t>(&imdihelist);
  if (imdihecoef != NULL) deallocate<float4>(&imdihecoef);

  if (cmaplist != NULL) deallocate<cmaplist_t>(&cmaplist);
  if (cmapcoef != NULL) deallocate<float2>(&cmapcoef);

  if (h_energy_virial != NULL) deallocate_host<BondedEnergyVirial_t>(&h_energy_virial);

}

//
// Setup coefficients (copies them from CPU to GPU)
// NOTE: This only has to be once in the beginning of the simulation
//
template <typename AT, typename CT>
void BondedForce<AT, CT>::setup_coef(int nbondcoef, float2 *h_bondcoef,
				     int nureybcoef, float2 *h_ureybcoef,
				     int nanglecoef, float2 *h_anglecoef,
				     int ndihecoef, float4 *h_dihecoef,
				     int nimdihecoef, float4 *h_imdihecoef,
				     int ncmapcoef, float2 *h_cmapcoef) {

  assert(nureybcoef == nanglecoef);

  this->nbondcoef = nbondcoef;
  if (nbondcoef > 0) {
    reallocate<float2>(&bondcoef, &bondcoef_len, nbondcoef, 1.2f);
    copy_HtoD<float2>(h_bondcoef, bondcoef, nbondcoef);
  }

  this->nureybcoef = nureybcoef;
  if (nureybcoef > 0) {
    reallocate<float2>(&ureybcoef, &ureybcoef_len, nureybcoef, 1.2f);
    copy_HtoD<float2>(h_ureybcoef, ureybcoef, nureybcoef);
  }

  this->nanglecoef = nanglecoef;
  if (nanglecoef > 0) {
    reallocate<float2>(&anglecoef, &anglecoef_len, nanglecoef, 1.2f);
    copy_HtoD<float2>(h_anglecoef, anglecoef, nanglecoef);
  }

  this->ndihecoef = ndihecoef;
  if (ndihecoef > 0) {
    reallocate<float4>(&dihecoef, &dihecoef_len, ndihecoef, 1.2f);
    copy_HtoD<float4>(h_dihecoef, dihecoef, ndihecoef);
  }

  this->nimdihecoef = nimdihecoef;
  if (nimdihecoef > 0) {
    reallocate<float4>(&imdihecoef, &imdihecoef_len, nimdihecoef, 1.2f);
    copy_HtoD<float4>(h_imdihecoef, imdihecoef, nimdihecoef);
  }

  this->ncmapcoef = ncmapcoef;
  if (ncmapcoef > 0) {
    reallocate<float2>(&cmapcoef, &cmapcoef_len, ncmapcoef, 1.2f);
    copy_HtoD<float2>(h_cmapcoef, cmapcoef, ncmapcoef);
  }

}

//
// Setup bondlists (copies them from CPU to GPU)
// NOTE: This has to be done after neighborlist update
//
template <typename AT, typename CT>
void BondedForce<AT, CT>::setup_list(int nbondlist, bondlist_t *h_bondlist, 
				     int nureyblist, bondlist_t *h_ureyblist,
				     int nanglelist, anglelist_t *h_anglelist,
				     int ndihelist, dihelist_t *h_dihelist,
				     int nimdihelist, dihelist_t *h_imdihelist,
				     int ncmaplist, cmaplist_t *h_cmaplist) {
  assert(nureyblist == nanglelist);

  this->nbondlist = nbondlist;
  if (nbondlist > 0) {
    reallocate<bondlist_t>(&bondlist, &bondlist_len, nbondlist, 1.2f);
    copy_HtoD<bondlist_t>(h_bondlist, bondlist, nbondlist);
  }

  this->nureyblist = nureyblist;
  if (nureyblist > 0) {
    reallocate<bondlist_t>(&ureyblist, &ureyblist_len, nureyblist, 1.2f);
    copy_HtoD<bondlist_t>(h_ureyblist, ureyblist, nureyblist);
  }

  this->nanglelist = nanglelist;
  if (nanglelist > 0) {
    reallocate<anglelist_t>(&anglelist, &anglelist_len, nanglelist, 1.2f);
    copy_HtoD<anglelist_t>(h_anglelist, anglelist, nanglelist);
  }

  this->ndihelist = ndihelist;
  if (ndihelist > 0) {
    reallocate<dihelist_t>(&dihelist, &dihelist_len, ndihelist, 1.2f);
    copy_HtoD<dihelist_t>(h_dihelist, dihelist, ndihelist);
  }

  this->nimdihelist = nimdihelist;
  if (nimdihelist > 0) {
    reallocate<dihelist_t>(&imdihelist, &imdihelist_len, nimdihelist, 1.2f);
    copy_HtoD<dihelist_t>(h_imdihelist, imdihelist, nimdihelist);
  }

  this->ncmaplist = ncmaplist;
  if (ncmaplist > 0) {
    reallocate<cmaplist_t>(&cmaplist, &cmaplist_len, ncmaplist, 1.2f);
    copy_HtoD<cmaplist_t>(h_cmaplist, cmaplist, ncmaplist);
  }

}

//
// Calculates forces
//
template <typename AT, typename CT>
void BondedForce<AT, CT>::calc_force(const float4 *xyzq,
				     const float boxx, const float boxy, const float boxz,
				     const bool calc_energy,
				     const bool calc_virial,
				     const int stride, AT *force,
				     const bool calc_bond, const bool calc_ureyb,
				     const bool calc_angle, const bool calc_dihe,
				     const bool calc_imdihe,
				     hipStream_t stream) {

  int nthread, nblock, shmem_size;

  nthread = 512;
  nblock = (nbondlist -1)/nthread + 1;
  shmem_size = 0;
  if (calc_energy) {
    shmem_size += nthread*sizeof(double);
  }

  if (calc_energy) {
    if (calc_virial) {
      std::cerr << "BondedForce<AT, CT>::calc_force, calc_virial not implemented yet" << std::endl;
    } else {

      if (calc_bond) {
	nthread = 512;
	nblock = (nbondlist -1)/nthread + 1;
	shmem_size = 0;
	if (calc_energy) shmem_size += nthread*sizeof(double);
	calc_bond_force_kernel<AT, CT, true, false >
	  <<< nblock, nthread, shmem_size, stream >>>
	  (nbondlist, bondlist, bondcoef, xyzq, stride, boxx, boxy, boxz, force);
	cudaCheck(hipGetLastError());
      }

      if (calc_ureyb) {
	nthread = 512;
	nblock = (nureyblist -1)/nthread + 1;
	shmem_size = 0;
	if (calc_energy) shmem_size += nthread*sizeof(double);
	calc_ureyb_force_kernel<AT, CT, true, false >
	  <<< nblock, nthread, shmem_size, stream >>>
	  (nureyblist, ureyblist, ureybcoef, xyzq, stride, boxx, boxy, boxz, force);
	cudaCheck(hipGetLastError());
      }

      if (calc_angle) {
	nthread = 512;
	nblock = (nanglelist -1)/nthread + 1;
	shmem_size = 0;
	if (calc_energy) shmem_size += nthread*sizeof(double);
	calc_angle_force_kernel<AT, CT, true, false >
	  <<< nblock, nthread, shmem_size, stream >>>
	  (nanglelist, anglelist, anglecoef, xyzq, stride, boxx, boxy, boxz, force);
	cudaCheck(hipGetLastError());
      }

      if (calc_dihe) {
	nthread = 512;
	nblock = (ndihelist -1)/nthread + 1;
	shmem_size = 0;
	if (calc_energy) shmem_size += nthread*sizeof(double);
	calc_dihe_force_kernel<AT, CT, true, false >
	  <<< nblock, nthread, shmem_size, stream >>>
	  (ndihelist, dihelist, dihecoef, xyzq, stride, boxx, boxy, boxz, force);
	cudaCheck(hipGetLastError());
      }

      if (calc_imdihe) {
	nthread = 512;
	nblock = (nimdihelist -1)/nthread + 1;
	shmem_size = 0;
	if (calc_energy) shmem_size += nthread*sizeof(double);
	calc_imdihe_force_kernel<AT, CT, true, false >
	  <<< nblock, nthread, shmem_size, stream >>>
	  (nimdihelist, imdihelist, imdihecoef, xyzq, stride, boxx, boxy, boxz, force);
	cudaCheck(hipGetLastError());
      }

    }
  } else {
    if (calc_virial) {
      std::cerr << "BondedForce<AT, CT>::calc_force, calc_virial not implemented yet" << std::endl;
    } else {

      int nbondlist_loc   = (calc_bond)   ? nbondlist   : 0;
      int nureyblist_loc  = (calc_ureyb)  ? nureyblist  : 0;
      int nanglelist_loc  = (calc_angle)  ? nanglelist  : 0;
      int ndihelist_loc   = (calc_dihe)   ? ndihelist   : 0;
      int nimdihelist_loc = (calc_imdihe) ? nimdihelist : 0;

      nthread = 512;
      nblock = (nbondlist_loc + nureyblist_loc + nanglelist_loc + 
		ndihelist_loc + nimdihelist_loc -1)/nthread + 1;      
      calc_all_forces_kernel<AT, CT, false, false>
	<<< nblock, nthread, shmem_size, stream>>>
	(nbondlist_loc, bondlist, bondcoef,
	 nureyblist_loc, ureyblist, ureybcoef,
	 nanglelist_loc, anglelist, anglecoef,
	 ndihelist_loc, dihelist, dihecoef,
	 nimdihelist_loc, imdihelist, imdihecoef,
	 xyzq, stride, boxx, boxy, boxz, force);

    }
  }

}

//
// Sets Energies and virials to zero
//
template <typename AT, typename CT>
void BondedForce<AT, CT>::clear_energy_virial() {
  h_energy_virial->energy_bond = 0.0;
  h_energy_virial->energy_ureyb = 0.0;
  h_energy_virial->energy_angle = 0.0;
  h_energy_virial->energy_dihe = 0.0;
  h_energy_virial->energy_imdihe = 0.0;
  h_energy_virial->energy_cmap = 0.0;
  for (int i=0;i < 27;i++) {
    h_energy_virial->sforcex[i] = 0.0;
    h_energy_virial->sforcey[i] = 0.0;
    h_energy_virial->sforcez[i] = 0.0;
  }
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_energy_virial), h_energy_virial, sizeof(BondedEnergyVirial_t)));
}

//
// Read Energies and virials
// prev_calc_energy = true, if energy was calculated when the force kernel was last called
// prev_calc_virial = true, if virial was calculated when the force kernel was last called
//
template <typename AT, typename CT>
void BondedForce<AT, CT>::get_energy_virial(bool prev_calc_energy, bool prev_calc_virial,
					    double *energy_bond, double *energy_ureyb,
					    double *energy_angle,
					    double *energy_dihe, double *energy_imdihe,
					    double *energy_cmap,
					    double *sforcex, double *sforcey, double *sforcez) {
  if (prev_calc_energy && prev_calc_virial) {
    cudaCheck(hipMemcpyFromSymbol(h_energy_virial, HIP_SYMBOL(d_energy_virial), sizeof(BondedEnergyVirial_t)));
  } else if (prev_calc_energy) {
    cudaCheck(hipMemcpyFromSymbol(h_energy_virial, HIP_SYMBOL(d_energy_virial), 6*sizeof(double)));
  } else if (prev_calc_virial) {
    cudaCheck(hipMemcpyFromSymbol(h_energy_virial, HIP_SYMBOL(d_energy_virial), 27*3*sizeof(double),
				   6*sizeof(double)));
  }
  *energy_bond = h_energy_virial->energy_bond;
  *energy_ureyb = h_energy_virial->energy_ureyb;
  *energy_angle = h_energy_virial->energy_angle;
  *energy_dihe = h_energy_virial->energy_dihe;
  *energy_imdihe = h_energy_virial->energy_imdihe;
  *energy_cmap = h_energy_virial->energy_cmap;
  for (int i=0;i < 27;i++) {
    sforcex[i] = h_energy_virial->sforcex[i];
    sforcey[i] = h_energy_virial->sforcey[i];
    sforcez[i] = h_energy_virial->sforcez[i];
  }
}

//
// Explicit instances of BondedForce
//
template class BondedForce<long long int, float>;
template class BondedForce<long long int, double>;

