#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <cstring>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "mpi_utils.h"
#include "CudaLeapfrogIntegrator.h"
#include "CudaDomdec.h"
#include "CudaDomdecGroups.h"
#include "CudaPMEForcefield.h"
#include "CudaDomdecRecipLooper.h"

int numnode=1, mynode=0;

bool test_cudaAware(const int mynode, const int numnode);
void test(const int nstep, const bool use_holoconst, const bool cudaAware, const bool use_pure_recip);

int main(int argc, char *argv[]) {

  int nstep = 1;
  bool use_holoconst = false;
  bool cudaAware = false;
  bool use_pure_recip = false;
  std::vector<int> devices;

  int iarg = 1;
  bool arg_ok = true;
  while (iarg < argc) {
    if (strcmp(argv[iarg],"-nstep")==0) {
      iarg++;
      if (iarg == argc) {
	arg_ok = false;
	break;
      }
      sscanf(argv[iarg],"%d",&nstep);
      iarg++;
    } else if (strcmp(argv[iarg],"-holoconst")==0) {
      iarg++;
      if (iarg == argc) {
	arg_ok = false;
	break;
      }
      if (strcmp(argv[iarg],"yes")==0) {
	use_holoconst = true;
      } else if (strcmp(argv[iarg],"no")==0) {
	use_holoconst = false;
      } else {
	arg_ok = false;
	break;
      }
      iarg++;
    } else if (strcmp(argv[iarg],"-cuda-aware")==0) {
      iarg++;
      if (iarg == argc) {
	arg_ok = false;
	break;
      }
      if (strcmp(argv[iarg],"yes")==0) {
	cudaAware = true;
      } else if (strcmp(argv[iarg],"no")==0) {
	cudaAware = false;
      } else {
	arg_ok = false;
	break;
      }
      iarg++;
    } else if (strcmp(argv[iarg],"-use-pure-recip")==0) {
      iarg++;
      if (iarg == argc) {
	arg_ok = false;
	break;
      }
      if (strcmp(argv[iarg],"yes")==0) {
	use_pure_recip = true;
      } else if (strcmp(argv[iarg],"no")==0) {
	use_pure_recip = false;
      } else {
	arg_ok = false;
	break;
      }
      iarg++;
    } else if (strcmp(argv[iarg],"-devices")==0) {
      iarg++;
      if (iarg == argc) {
	arg_ok = false;
	break;
      }
      while (iarg < argc && isdigit(argv[iarg][0])) {
	//fprintf(stderr,"%d\n",atoi(argv[iarg]));
	devices.push_back(atoi(argv[iarg]));
	iarg++;
      }
    } else {
      std::cout << "Invalid option " << argv[iarg] << std::endl;
      arg_ok = false;
      break;
    }
  }

  if (arg_ok) {
    // Get the local rank within this node from environmental variables
    int local_rank = get_env_local_rank();
    int local_size = get_env_local_size();

    if (local_rank < 0 && cudaAware) {
      std::cout << "Requesting CUDA aware MPI but local rank not defined by environment" << std::endl;
      std::cout << "=> defaulting back to non CUDA aware MPI" << std::endl;
      cudaAware = false;
    }

    std::cout << "local_rank=" << local_rank << " local_size=" << local_size << std::endl;
    if (local_rank >= 0 && local_size > 0) {
      start_gpu(local_size, local_rank, devices);
      start_mpi(argc, argv, numnode, mynode);
    } else {
      start_mpi(argc, argv, numnode, mynode);
      start_gpu(numnode, mynode, devices);
    }

    if (cudaAware) {
      if (!test_cudaAware(mynode, numnode)) {
	std::cout << "CUDA aware test FAILED => defaulting back to non CUDA aware MPI" << std::endl;
	cudaAware = false;
      } else {
	if (mynode == 0) {
	  std::cout << "CUDA aware tested OK" << std::endl;
	}
      }
    }

  }

  int ret_val = 0;

  if (arg_ok) {
    test(nstep, use_holoconst, cudaAware, use_pure_recip);
  } else {
    std::cout << "Usage: mpirun -n X gpu_dyna OPTIONS" << std::endl;
    std::cout << "OPTIONS:" << std::endl;
    std::cout << "-nstep N" << std::endl;
    std::cout << "-holoconst <yes|no> " << std::endl;
    std::cout << "-cuda-aware <yes|no> " << std::endl;
    std::cout << "-use-pure-recip <yes|no>" << std::endl;
    std::cout << "-devices 0 1 2 3" << std::endl;
    ret_val = 1;
  }

  if (arg_ok) {
    stop_mpi();
    stop_gpu();
  }

  return ret_val;
}

//
// Loads vector from file
//
template <typename T>
void load_vec(const int nind, const char *filename, const int n, T *ind) {
  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nind;k++) {
	if (!(file >> ind[i*nind+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Loads constraints and masses from file
//
void load_constr_mass(const int nconstr, const int nmass, const char *filename, const int n,
		      double *constr, double *mass) {

  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nconstr;k++) {
	if (!(file >> constr[i*nconstr+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
      for (int k=0;k < nmass;k++) {
	if (!(file >> mass[i*nmass+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Writes (x, y, z) into a file
//
void write_xyz(const int n, const double *x, const double *y, const double *z, const char *filename) {
  std::ofstream file(filename);
  if (file.is_open()) {
    file.precision(16);
    file.setf(std::ios::fixed, std::ios::floatfield);
    for (int i=0;i < n;i++) {
      file << x[i] << " " << y[i] << " " << z[i] << std::endl;
    }
  } else {
    std::cout << "write_xyz: Error opening file " << filename << std::endl;
    exit(1);
  }
}

//
// Reads (x, y, z) from a file
//
void read_xyz(const int n, double *x, double *y, double *z, const char *filename) {
  std::ifstream file(filename);
  if (file.is_open()) {
    for (int i=0;i < n;i++) {
      file >> x[i] >> y[i] >> z[i];
    }
  } else {
    std::cout << "write_xyz: Error opening file " << filename << std::endl;
    exit(1);
  }
}

//
// Checks holonomic constraints
//
void check_holoconst(const double* x, const double* y, const double* z,
		     const int npair, const bond_t* h_pair_indtype, const double* h_pair_constr, 
		     const int ntrip, const angle_t* h_trip_indtype, const double* h_trip_constr,
		     const int nquad, const dihe_t* h_quad_indtype, const double* h_quad_constr,
		     const int nsolvent, const solvent_t* h_solvent_ind,
		     const double rOHsq, const double rHHsq) {
  double tol = 1.0e-8;
  double max_err = 0.0;

  for (int i=0;i < npair;i++) {
    bond_t bond = h_pair_indtype[i];
    double dx = x[bond.i] - x[bond.j];
    double dy = y[bond.i] - y[bond.j];
    double dz = z[bond.i] - z[bond.j];
    double rsq = dx*dx + dy*dy + dz*dz;
    double err = fabs(rsq - h_pair_constr[bond.itype]);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in PAIR: err = " << err << std::endl;
      return;
    }
  }

  for (int i=0;i < ntrip;i++) {
    angle_t angle = h_trip_indtype[i];
    double dx = x[angle.i] - x[angle.j];
    double dy = y[angle.i] - y[angle.j];
    double dz = z[angle.i] - z[angle.j];
    double rsq = dx*dx + dy*dy + dz*dz;
    double err = fabs(rsq - h_trip_constr[angle.itype*2]);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in TRIP (i-j): err = " << err << std::endl;
      return;
    }
    dx = x[angle.i] - x[angle.k];
    dy = y[angle.i] - y[angle.k];
    dz = z[angle.i] - z[angle.k];
    rsq = dx*dx + dy*dy + dz*dz;
    err = fabs(rsq - h_trip_constr[angle.itype*2+1]);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in TRIP (i-k): err = " << err << std::endl;
      return;
    }
  }

  for (int i=0;i < nquad;i++) {
    dihe_t dihe = h_quad_indtype[i];
    double dx = x[dihe.i] - x[dihe.j];
    double dy = y[dihe.i] - y[dihe.j];
    double dz = z[dihe.i] - z[dihe.j];
    double rsq = dx*dx + dy*dy + dz*dz;
    double err = fabs(rsq - h_quad_constr[dihe.itype*3]);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in QUAD (i-j): err = " << err << std::endl;
      return;
    }
    dx = x[dihe.i] - x[dihe.k];
    dy = y[dihe.i] - y[dihe.k];
    dz = z[dihe.i] - z[dihe.k];
    rsq = dx*dx + dy*dy + dz*dz;
    err = fabs(rsq - h_quad_constr[dihe.itype*3+1]);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in QUAD (i-k): err = " << err << std::endl;
      return;
    }
    dx = x[dihe.i] - x[dihe.l];
    dy = y[dihe.i] - y[dihe.l];
    dz = z[dihe.i] - z[dihe.l];
    rsq = dx*dx + dy*dy + dz*dz;
    err = fabs(rsq - h_quad_constr[dihe.itype*3+2]);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in QUAD (i-l): err = " << err << std::endl;
      return;
    }
  }

  for (int i=0;i < nsolvent;i++) {
    solvent_t solvent = h_solvent_ind[i];
    double dx = x[solvent.i] - x[solvent.j];
    double dy = y[solvent.i] - y[solvent.j];
    double dz = z[solvent.i] - z[solvent.j];
    double rsq = dx*dx + dy*dy + dz*dz;
    double err = fabs(rsq - rOHsq);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in SOLVENT (O-H1): err = " << err << std::endl;
      return;
    }
    dx = x[solvent.i] - x[solvent.k];
    dy = y[solvent.i] - y[solvent.k];
    dz = z[solvent.i] - z[solvent.k];
    rsq = dx*dx + dy*dy + dz*dz;
    err = fabs(rsq - rOHsq);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in SOLVENT (O-H2): err = " << err << std::endl;
      return;
    }
    dx = x[solvent.j] - x[solvent.k];
    dy = y[solvent.j] - y[solvent.k];
    dz = z[solvent.j] - z[solvent.k];
    rsq = dx*dx + dy*dy + dz*dz;
    err = fabs(rsq - rHHsq);
    max_err = max(max_err, err);
    if (err > tol) {
      std::cout << "Error in SOLVENT (H1-H2): err = " << err << std::endl;
      return;
    }
  }

  std::cout << "check_holoconst OK (max_err = " << max_err << ")" << std::endl;
}

//
// Test the code using data in test_data/ -directory
//
void test(const int nstep, const bool use_holoconst, const bool cudaAware, const bool use_pure_recip) {

  // Settings for the data:
  const double boxx = 62.23;
  const double boxy = 62.23;
  const double boxz = 62.23;
  const double kappa = 0.320;
  const int nfftx = 64;
  const int nffty = 64;
  const int nfftz = 64;
  const int forder = 4;
  const double rnl = 11.0;
  const double roff = 9.0;
  const double ron = 7.5;
  const double e14fac = 1.0;
  const int ncoord = 23558;

  // Very simple node setup
  int nx;
  int ny;
  int nz;
  bool isDirect;
  bool isRecip;
  std::vector<int> direct_nodes;
  std::vector<int> recip_nodes;
  if (use_pure_recip && numnode > 1) {
    // Separate Recip node, last node
    direct_nodes.resize(numnode-1);
    recip_nodes.resize(1);
    nx = 1;
    ny = 1;
    nz = numnode-1;
    for (int i=0;i < numnode-1;i++) direct_nodes.at(i) = i;
    recip_nodes.at(0) = numnode-1;
    isDirect = false;
    isRecip = false;
    if (mynode == recip_nodes.at(0)) {
      isRecip = true;
    } else {
      isDirect = true;
    }
  } else {
    // Mixed Direct+Recip node, first node
    direct_nodes.resize(numnode);
    recip_nodes.resize(1);
    nx = 1;
    ny = 1;
    nz = numnode;
    isDirect = true;
    isRecip = (mynode == 0) ? true : false;
    for (int i=0;i < numnode;i++) direct_nodes.at(i) = i;
    recip_nodes.at(0) = 0;
  }

  if (isDirect && isRecip) {
    std::cout << "Node " << mynode << " is Direct+Recip" << std::endl;
  } else if (isDirect) {
    std::cout << "Node " << mynode << " is Direct" << std::endl;
  } else if (isRecip) {
    std::cout << "Node " << mynode << " is Recip" << std::endl;
  }

  // MPI communicators
  MPI_Comm comm_direct;
  MPI_Comm comm_recip;
  MPI_Comm comm_direct_recip = MPI_COMM_WORLD;
  
  MPI_Group group_world;
  MPI_Group group_direct;
  MPI_Group group_recip;
  
  // Get handle to the entire domain
  MPICheck(MPI_Comm_group(MPI_COMM_WORLD, &group_world));
  
  //if (isDirect) {
  MPICheck(MPI_Group_incl(group_world, direct_nodes.size(), direct_nodes.data(), &group_direct));
  MPICheck(MPI_Comm_create(MPI_COMM_WORLD, group_direct, &comm_direct));
  //}
  
  //if (isRecip) {
  MPICheck(MPI_Group_incl(group_world, recip_nodes.size(), recip_nodes.data(), &group_recip));
  MPICheck(MPI_Comm_create(MPI_COMM_WORLD, group_recip, &comm_recip));
  //}

  CudaDomdecRecip *recip = NULL;
  CudaDomdecRecipComm recipComm(comm_recip, comm_direct_recip,
				mynode, direct_nodes, recip_nodes, cudaAware);
  
  // Create reciprocal calculator
  if (isRecip) {
    recip = new CudaDomdecRecip(nfftx, nffty, nfftz, forder, kappa);
  }

  if (isDirect) {
    // --------------------------
    // Direct node
    // --------------------------

    const int nbond = 23592;
    const int nbondcoef = 129;

    const int nureyb = 11584;
    const int nureybcoef = 327;

    const int nangle = 11584;
    const int nanglecoef = 327;

    const int ndihe = 6701;
    const int ndihecoef = 438;

    const int nimdihe = 418;
    const int nimdihecoef = 40;

    bond_t *h_bond = new bond_t[nbond];
    load_vec<int>(3, "test_data/bond.txt", nbond, (int *)h_bond);
    float2 *h_bondcoef = new float2[nbondcoef];
    load_vec<float>(2, "test_data/bondcoef.txt", nbondcoef, (float *)h_bondcoef);

    bond_t *h_ureyb = new bond_t[nureyb];
    load_vec<int>(3, "test_data/ureyb.txt", nureyb, (int *)h_ureyb);
    float2 *h_ureybcoef = new float2[nureybcoef];
    load_vec<float>(2, "test_data/ureybcoef.txt", nureybcoef, (float *)h_ureybcoef);

    angle_t *h_angle = new angle_t[nangle];
    load_vec<int>(4, "test_data/angle.txt", nangle, (int *)h_angle);
    float2 *h_anglecoef = new float2[nanglecoef];
    load_vec<float>(2, "test_data/anglecoef.txt", nanglecoef, (float *)h_anglecoef);

    dihe_t *h_dihe = new dihe_t[ndihe];
    load_vec<int>(5, "test_data/dihe.txt", ndihe, (int *)h_dihe);
    float4 *h_dihecoef = new float4[ndihecoef];
    load_vec<float>(4, "test_data/dihecoef.txt", ndihecoef, (float *)h_dihecoef);

    dihe_t *h_imdihe = new dihe_t[nimdihe];
    load_vec<int>(5, "test_data/imdihe.txt", nimdihe, (int *)h_imdihe);
    float4 *h_imdihecoef = new float4[nimdihecoef];
    load_vec<float>(4, "test_data/imdihecoef.txt", nimdihecoef, (float *)h_imdihecoef);

    //-------------------------------------------------------------------------------------

    const int nvdwparam = 1260;
    float* h_vdwparam = new float[nvdwparam];
    float* h_vdwparam14 = new float[nvdwparam];
    load_vec<float>(1, "test_data/vdwparam.txt", nvdwparam, h_vdwparam);
    load_vec<float>(1, "test_data/vdwparam14.txt", nvdwparam, h_vdwparam14);

    int *h_vdwtype = new int[ncoord];
    load_vec<int>(1, "test_data/glo_vdwtype.txt", ncoord, h_vdwtype);

    //-------------------------------------------------------------------------------------

    const int niblo14 = 23558;
    const int ninb14 = 34709;
    int *h_iblo14 = new int[niblo14];
    int *h_inb14 = new int[ninb14];
    load_vec<int>(1, "test_data/iblo14.txt", niblo14, h_iblo14);
    load_vec<int>(1, "test_data/inb14.txt", ninb14, h_inb14);

    //-------------------------------------------------------------------------------------
  
    const int nin14 = 6556;
    const int nex14 = 28153;
    xx14_t *h_in14 = new xx14_t[nin14];
    xx14_t *h_ex14 = new xx14_t[nex14];
    load_vec<int>(2, "test_data/in14.txt", nin14, (int *)h_in14);
    load_vec<int>(2, "test_data/ex14.txt", nex14, (int *)h_ex14);

    //-------------------------------------------------------------------------------------

    const double mO = 15.9994;
    const double mH = 1.008;
    const double rOHsq = 0.91623184;
    const double rHHsq = 2.29189321;
    const int nsolvent = 7023;
    const int npair = 458;
    const int ntrip = 233;
    const int nquad = 99;
    const int npair_type = 9;
    const int ntrip_type = 3;
    const int nquad_type = 2;

    double *h_pair_constr = new double[npair_type];
    double *h_pair_mass = new double[npair_type*2];
    load_constr_mass(1, 2, "test_data/pair_types.txt", npair_type, h_pair_constr, h_pair_mass);
    bond_t* h_pair_indtype = new bond_t[npair];
    load_vec<int>(3, "test_data/pair_indtype.txt", npair, (int *)h_pair_indtype);

    double *h_trip_constr = new double[ntrip_type*2];
    double *h_trip_mass = new double[ntrip_type*5];
    load_constr_mass(2, 5, "test_data/trip_types.txt", ntrip_type, h_trip_constr, h_trip_mass);
    angle_t* h_trip_indtype = new angle_t[ntrip];
    load_vec<int>(4, "test_data/trip_indtype.txt", ntrip, (int *)h_trip_indtype);

    double *h_quad_constr = new double[nquad_type*3];
    double *h_quad_mass = new double[nquad_type*7];
    load_constr_mass(3, 7, "test_data/quad_types.txt", nquad_type, h_quad_constr, h_quad_mass);
    dihe_t* h_quad_indtype = new dihe_t[nquad];
    load_vec<int>(5, "test_data/quad_indtype.txt", nquad, (int *)h_quad_indtype);

    // Load constraint indices
    solvent_t *h_solvent_ind = new solvent_t[nsolvent];
    load_vec<int>(3, "test_data/solvent_ind.txt", nsolvent, (int *)h_solvent_ind);

    HoloConst* holoconst = NULL;
    if (use_holoconst) {
      holoconst = new HoloConst;;
      holoconst->setup_solvent_parameters(mO, mH, rOHsq, rHHsq);
      holoconst->setup_types(npair_type, h_pair_constr, h_pair_mass,
			     ntrip_type, h_trip_constr, h_trip_mass,
			     nquad_type, h_quad_constr, h_quad_mass);
    }
    //-------------------------------------------------------------------------------------

    // Topological exclusions
    CudaTopExcl topExcl(ncoord, h_iblo14, h_inb14);

    // Setup domain decomposition
    CudaMPI cudaMPI(cudaAware, comm_direct);
    CudaDomdec domdec(ncoord, boxx, boxy, boxz, rnl, nx, ny, nz, mynode, cudaMPI);

    // Setup "bonded" groups
    CudaDomdecGroups domdecGroups(domdec);

    CudaAtomGroup<bond_t> bondGroup(nbond, h_bond, "BOND");
    CudaAtomGroup<bond_t> ureybGroup(nureyb, h_ureyb, "UREYB");
    CudaAtomGroup<angle_t> angleGroup(nangle, h_angle, "ANGLE");
    CudaAtomGroup<dihe_t> diheGroup(ndihe, h_dihe, "DIHE");
    CudaAtomGroup<dihe_t> imdiheGroup(nimdihe, h_imdihe, "IMDIHE");
    CudaAtomGroup<xx14_t> in14Group(nin14, h_in14, "IN14");
    CudaAtomGroup<xx14_t> ex14Group(nex14, h_ex14, "EX14");
    CudaAtomGroup<bond_t>    pairGroup(npair, h_pair_indtype, "PAIR");
    CudaAtomGroup<angle_t>   tripGroup(ntrip, h_trip_indtype, "TRIP");
    CudaAtomGroup<dihe_t>    quadGroup(nquad, h_quad_indtype, "QUAD");
    CudaAtomGroup<solvent_t> solventGroup(nsolvent, h_solvent_ind, "SOLVENT");
    // Register groups
    // NOTE: the register IDs (BOND, UREYB, ...) must be unique
    domdecGroups.beginGroups();
    domdecGroups.insertGroup(BOND, bondGroup, h_bond);
    domdecGroups.insertGroup(UREYB, ureybGroup, h_ureyb);
    domdecGroups.insertGroup(ANGLE, angleGroup, h_angle);
    domdecGroups.insertGroup(DIHE, diheGroup, h_dihe);
    domdecGroups.insertGroup(IMDIHE, imdiheGroup, h_imdihe);
    domdecGroups.insertGroup(IN14, in14Group, h_in14);
    domdecGroups.insertGroup(EX14, ex14Group, h_ex14);
    if (use_holoconst) {
      domdecGroups.insertGroup(PAIR,    pairGroup, h_pair_indtype);
      domdecGroups.insertGroup(TRIP,    tripGroup, h_trip_indtype);
      domdecGroups.insertGroup(QUAD,    quadGroup, h_quad_indtype);
      domdecGroups.insertGroup(SOLVENT, solventGroup, h_solvent_ind);
    }
    domdecGroups.finishGroups();

    CudaLeapfrogIntegrator leapfrog(holoconst);

    // Charges
    float *h_q = new float[ncoord];
    load_vec<float>(1, "test_data/q.txt", ncoord, h_q);

    // Setup PME force field
    CudaPMEForcefield forcefield(// Domain decomposition
				 domdec, domdecGroups,
				 // Non-bonded topological exclusions
				 topExcl,
				 // Bonded
				 nbondcoef, h_bondcoef, nureybcoef, h_ureybcoef, nanglecoef, h_anglecoef,
				 ndihecoef, h_dihecoef, nimdihecoef, h_imdihecoef, 0, NULL,
				 // Direct non-bonded
				 roff, ron, kappa, e14fac, VDW_VSH, EWALD,
				 nvdwparam, h_vdwparam, h_vdwparam14, h_vdwtype, h_q,
				 // Recip non-bonded
				 recip, recipComm);

    delete [] h_q;

    leapfrog.set_forcefield(&forcefield);

    // Masses
    double *mass = new double[ncoord];
    load_vec<double>(1, "test_data/mass.txt", ncoord, mass);

    // Coordinates
    double *x = new double[ncoord];
    double *y = new double[ncoord];
    double *z = new double[ncoord];
    load_vec<double>(1, "test_data/x.txt", ncoord, x);
    load_vec<double>(1, "test_data/y.txt", ncoord, y);
    load_vec<double>(1, "test_data/z.txt", ncoord, z);

    // Step vector
    double *dx = new double[ncoord];
    double *dy = new double[ncoord];
    double *dz = new double[ncoord];
    load_vec<double>(1, "test_data/dx.txt", ncoord, dx);
    load_vec<double>(1, "test_data/dy.txt", ncoord, dy);
    load_vec<double>(1, "test_data/dz.txt", ncoord, dz);

    double *fx = new double[ncoord];
    double *fy = new double[ncoord];
    double *fz = new double[ncoord];

    leapfrog.init(ncoord, x, y, z, dx, dy, dz, mass);
    leapfrog.set_coord_buffers(x, y, z);
    leapfrog.set_step_buffers(dx, dy, dz);
    leapfrog.set_force_buffers(fx, fy, fz);
    double timestep = use_holoconst ? 2.0 : 1.0;
    leapfrog.set_timestep(timestep);
    MPICheck(MPI_Barrier( MPI_COMM_WORLD));
    double begin = MPI_Wtime();
    leapfrog.run(nstep, 0);
    double end = MPI_Wtime();
    double time_spent = end - begin;
    if (mynode == 0) {
      std::cout << "time_spent (sec) = " << time_spent << " ns/day = "
		<< (double)nstep*timestep*0.000001/(time_spent/(double)(60*60*24)) << std::endl;
    }

    cudaCheck(hipDeviceSynchronize());

    if (mynode == 0) {
      if (nstep == 100 || nstep == 20 || nstep == 10 || nstep == 2 
	  || (nstep == 1 && !use_holoconst)) {
	double* fxref = new double[ncoord];
	double* fyref = new double[ncoord];
	double* fzref = new double[ncoord];
	char filename[256];
	if (nstep == 100 && use_holoconst) {
	  sprintf(filename,"test_data/force_dyn%d_holoconst.txt",nstep);
	} else {
	  sprintf(filename,"test_data/force_dyn%d.txt",nstep);
	}
	read_xyz(ncoord, fxref, fyref, fzref, filename);
	double max_err = 0.0;
	double err_tol = 5.0e-6;
	for (int i=0;i < ncoord;i++) {
	  double dfx = fx[i] - fxref[i];
	  double dfy = fy[i] - fyref[i];
	  double dfz = fz[i] - fzref[i];
	  double err = dfx*dfx + dfy*dfy + dfz*dfz;
	  max_err = max(max_err, err);
	  if (err > err_tol) {
	    std::cout << "i = " << i << " err = " << err << std::endl;
	    break;
	  }
	}
	if (max_err < err_tol) {
	  std::cout << "Test OK, maximum error = " << max_err << std::endl;
	} else {
	  std::cout << "Test FAILED" << std::endl;
	}
	delete [] fxref;
	delete [] fyref;
	delete [] fzref;
      } else {
	std::cout << "Test NOT performed (nstep != 100)" << std::endl;
      }

      write_xyz(ncoord, x, y, z, "coord.txt");
      write_xyz(ncoord, dx, dy, dz, "step.txt");
      write_xyz(ncoord, fx, fy, fz, "force.txt");

      if (nstep != 1 && use_holoconst) {
	check_holoconst(x, y, z,
			npair, h_pair_indtype, h_pair_constr, 
			ntrip, h_trip_indtype, h_trip_constr,
			nquad, h_quad_indtype, h_quad_constr,
			nsolvent, h_solvent_ind, rOHsq, rHHsq);
      }
    }

    delete [] mass;

    delete [] x;
    delete [] y;
    delete [] z;

    delete [] dx;
    delete [] dy;
    delete [] dz;

    delete [] fx;
    delete [] fy;
    delete [] fz;

    //-------------------------------------------------------------------------------------

    if (h_bond != NULL) delete [] h_bond;
    delete [] h_bondcoef;
  
    if (h_ureyb != NULL) delete [] h_ureyb;
    delete [] h_ureybcoef;
  
    if (h_angle != NULL) delete [] h_angle;
    delete [] h_anglecoef;

    if (h_dihe != NULL) delete [] h_dihe;
    delete [] h_dihecoef;
  
    if (h_imdihe != NULL) delete [] h_imdihe;
    delete [] h_imdihecoef;

    //-------------------------------------------------------------------------------------

    delete [] h_vdwparam;
    delete [] h_vdwparam14;
    delete [] h_vdwtype;

    //-------------------------------------------------------------------------------------

    delete [] h_iblo14;
    delete [] h_inb14;

    //-------------------------------------------------------------------------------------

    delete [] h_in14;
    delete [] h_ex14;

    //-------------------------------------------------------------------------------------

    delete [] h_solvent_ind;

    delete [] h_pair_indtype;
    delete [] h_trip_indtype;
    delete [] h_quad_indtype;

    delete [] h_pair_constr;
    delete [] h_pair_mass;
    delete [] h_trip_constr;
    delete [] h_trip_mass;
    delete [] h_quad_constr;
    delete [] h_quad_mass;

    //-------------------------------------------------------------------------------------
    if (holoconst != NULL) delete holoconst;

  } else {
    // ------------------------------------------------------------
    // Pure recip node, loop here until Direct nodes say were done
    // ------------------------------------------------------------
    CudaDomdecRecipLooper looper(*recip, recipComm);
    looper.run();
  }

  if (recip != NULL) delete recip;

  if (isDirect) {
    MPICheck(MPI_Group_free(&group_direct));
  }

  if (isRecip) {
    MPICheck(MPI_Group_free(&group_recip));
  }

  return;
}

//
// Test if MPI is truly cuda aware by sending and receiving device buffer
//
bool test_cudaAware(const int mynode, const int numnode) {
  const int ncomm = 10;
  int *sendbuf, *recvbuf;
  allocate<int>(&sendbuf, ncomm);
  allocate<int>(&recvbuf, ncomm);
  int *h_sendbuf = new int[ncomm];
  int *h_recvbuf = new int[ncomm];
  for (int i=0;i < ncomm;i++) {
    h_sendbuf[i] = i;
    h_recvbuf[i] = 0;
  }
  copy_HtoD_sync<int>(h_sendbuf, sendbuf, ncomm);
  copy_HtoD_sync<int>(h_recvbuf, recvbuf, ncomm);

  const int TAG=1;
  MPICheck(MPI_Sendrecv(sendbuf, ncomm, MPI_INT, (mynode+1) % numnode, TAG,
			recvbuf, ncomm, MPI_INT, (mynode-1+numnode) % numnode, TAG,
			MPI_COMM_WORLD, MPI_STATUS_IGNORE));

  copy_DtoH_sync<int>(recvbuf, h_recvbuf, ncomm);
  bool ok=true;
  for (int i=0;i < ncomm;i++) {
    if (h_recvbuf[i] != i) ok = false;
  }

  deallocate<int>(&sendbuf);
  deallocate<int>(&recvbuf);
  delete [] h_sendbuf;
  delete [] h_recvbuf;

  return ok;
}
