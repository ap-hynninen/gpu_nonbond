#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include "HoloConst.h"

void test();

//
// Main
//
int main(int argc, char *argv[]) {

  int numnode = 1;
  int mynode = 0;

  start_gpu(numnode, mynode);
  
  test();

  return 0;
}

//
// Loads (x, y, z) coordinates from file
//
void load_coord(const char *filename, const int stride, double *xyz) {

  std::ifstream file(filename);
  if (file.is_open()) {

    int i = 0;
    while (file >> xyz[i] >> xyz[i+stride] >> xyz[i+stride*2]) i++;

    if (i > stride) {
      std::cerr<<"Too many lines in file "<<filename<<std::endl;
      exit(1);
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Loads indices from file
//
void load_ind(const int nind, const char *filename, const int n, int *ind) {

  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nind;k++) {
	if (!(file >> ind[i*nind+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Loads constraints and masses from file
//
void load_constr_mass(const int nconstr, const int nmass, const char *filename, const int n,
		      double *constr, double *mass) {

  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nconstr;k++) {
	if (!(file >> constr[i*nconstr+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
      for (int k=0;k < nmass;k++) {
	if (!(file >> mass[i*nmass+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Checks SETTLE and SHAKE results
//
bool check_result(const int nind, const int n, const int *ind,
		  const double *xyz, const double *xyz_ref, const int stride,
		  const double tol, double &max_diff) {

  double x1, y1, z1;
  double x2, y2, z2;
  double diff;
  int imol, j, i;

  try {
    for (imol=0;imol < n;imol++) {

      for (j=0;j < nind;j++) {
	i = ind[imol*nind+j];
	x1 = xyz[i];
	y1 = xyz[i + stride];
	z1 = xyz[i + 2*stride];
	x2 = xyz_ref[i];
	y2 = xyz_ref[i + stride];
	z2 = xyz_ref[i + 2*stride];
	
	if (isnan(x1) || isnan(y1) || isnan(z1) || isnan(x2) || isnan(y2) || isnan(z2)) throw 1;
	diff = max(fabs(x1-x2), max(fabs(y1-y2), fabs(z1-z2)));
	max_diff = max(diff, max_diff);
	if (diff > tol) throw 1;
      }
    }
  }
  catch (int a) {
    std::cout << "comparison FAILED, imol=" << imol << " diff=" << diff << std::endl;
    std::cout << "ind =";
    for (j=0;j < nind;j++) {
      std::cout << " " << ind[imol*nind+j];
    }
    std::cout << std::endl;
    std::cout << x1 << " "<< y1 << " "<< z1 << std::endl;
    std::cout << x2 << " "<< y2 << " "<< z2 << std::endl;
    return false;
  }

  return true;
}

//
// Test the code using data in test_data/ -directory
//
void test() {

  // Settings for the data:
  const double mO = 15.9994;
  const double mH = 1.008;
  const double rOHsq = 0.91623184;
  const double rHHsq = 2.29189321;
  const int ncoord = 23558;
  const int stride = ((ncoord-1)/32+1)*32;
  const int nsolvent = 7023;
  const int npair = 458;
  const int ntrip = 233;
  const int nquad = 99;

  // Load coordinates
  double *h_xyz0 = (double *)malloc(stride*3*sizeof(double));
  double *h_xyz1 = (double *)malloc(stride*3*sizeof(double));
  double *h_xyz_ref = (double *)malloc(stride*3*sizeof(double));
  load_coord("test_data/xyz0.txt", stride, h_xyz0);
  load_coord("test_data/xyz1.txt", stride, h_xyz1);
  load_coord("test_data/xyz_ref.txt", stride, h_xyz_ref);

  double *xyz0;
  double *xyz1;
  allocate<double>(&xyz0, stride*3);
  allocate<double>(&xyz1, stride*3);

  copy_HtoD<double>(h_xyz0, xyz0, stride*3);
  copy_HtoD<double>(h_xyz1, xyz1, stride*3);

  // Load constraint indices
  int *h_solvent_ind = (int *)malloc(nsolvent*3*sizeof(int));
  load_ind(3, "test_data/solvent_ind.txt", nsolvent, h_solvent_ind);

  int *h_pair_ind = (int *)malloc(npair*2*sizeof(int));
  load_ind(2, "test_data/pair_ind.txt", npair, h_pair_ind);

  int *h_trip_ind = (int *)malloc(ntrip*3*sizeof(int));
  load_ind(3, "test_data/trip_ind.txt", ntrip, h_trip_ind);

  int *h_quad_ind = (int *)malloc(nquad*4*sizeof(int));
  load_ind(4, "test_data/quad_ind.txt", nquad, h_quad_ind);

  // Load constraint distances and masses
  double *h_pair_constr = (double *)malloc(npair*sizeof(double));
  double *h_pair_mass = (double *)malloc(npair*2*sizeof(double));
  load_constr_mass(1, 2, "test_data/pair_constr_mass.txt", npair, h_pair_constr, h_pair_mass);

  double *h_trip_constr = (double *)malloc(ntrip*2*sizeof(double));
  double *h_trip_mass = (double *)malloc(ntrip*5*sizeof(double));
  load_constr_mass(2, 5, "test_data/trip_constr_mass.txt", ntrip, h_trip_constr, h_trip_mass);

  double *h_quad_constr = (double *)malloc(nquad*3*sizeof(double));
  double *h_quad_mass = (double *)malloc(nquad*7*sizeof(double));
  load_constr_mass(3, 7, "test_data/quad_constr_mass.txt", nquad, h_quad_constr, h_quad_mass);

  //--------------------------------------------------------------------------
  // Setup & Apply holonomic constraints
  //--------------------------------------------------------------------------

  HoloConst holoconst;

  // Setup
  holoconst.setup(mO, mH, rOHsq, rHHsq);
  holoconst.set_solvent_ind(nsolvent, (int3 *)h_solvent_ind);
  holoconst.set_pair_ind(npair, (int2 *)h_pair_ind, h_pair_constr, h_pair_mass);
  holoconst.set_trip_ind(ntrip, (int3 *)h_trip_ind, h_trip_constr, h_trip_mass);
  holoconst.set_quad_ind(nquad, (int4 *)h_quad_ind, h_quad_constr, h_quad_mass);
  
  // Apply holonomic constraints
  holoconst.apply(xyz0, xyz1, stride);

  copy_HtoD<double>(h_xyz1, xyz1, stride*3);

  holoconst.apply2(xyz0, xyz1, stride);

  copy_HtoD<double>(h_xyz1, xyz1, stride*3);

  holoconst.apply2(xyz0, xyz1, stride);


  //--------------------------------------------------------------------------
  // Check result
  //--------------------------------------------------------------------------
  copy_DtoH<double>(xyz1, h_xyz1, stride*3);

  double max_diff;
  double tol = 5.0e-14;

  max_diff = 0.0;
  if (check_result(3, nsolvent, h_solvent_ind, h_xyz1, h_xyz_ref, stride, tol, max_diff)) {
    std::cout<<"solvent SETTLE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  max_diff = 0.0;
  if (check_result(2, npair, h_pair_ind, h_xyz1, h_xyz_ref, stride, tol, max_diff)) {
    std::cout<<"pair SHAKE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  max_diff = 0.0;
  if (check_result(3, ntrip, h_trip_ind, h_xyz1, h_xyz_ref, stride, tol, max_diff)) {
    std::cout<<"trip SHAKE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  max_diff = 0.0;
  if (check_result(4, nquad, h_quad_ind, h_xyz1, h_xyz_ref, stride, tol, max_diff)) {
    std::cout<<"quad SHAKE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  free(h_xyz0);
  free(h_xyz1);
  free(h_xyz_ref);

  free(h_solvent_ind);

  free(h_pair_ind);
  free(h_trip_ind);
  free(h_quad_ind);

  free(h_pair_constr);
  free(h_pair_mass);
  free(h_trip_constr);
  free(h_trip_mass);
  free(h_quad_constr);
  free(h_quad_mass);

  deallocate<double>(&xyz0);
  deallocate<double>(&xyz1);
  
}
