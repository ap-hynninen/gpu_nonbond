#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include "gpu_utils.h"
//#include "const_reduce_lists.h"
#include "HoloConst.h"
#include "hostXYZ.h"

void test();

//
// Main
//
int main(int argc, char *argv[]) {

  int numnode = 1;
  int mynode = 0;

  start_gpu(numnode, mynode);
  
  test();

  return 0;
}

//
// Loads (x, y, z) coordinates from file
//
void load_coord(const char *filename, const int n, double *x, double *y, double *z) {

  std::ifstream file(filename);
  if (file.is_open()) {

    int i = 0;
    while (file >> x[i] >> y[i] >> z[i]) i++;

    if (i > n) {
      std::cerr<<"Too many lines in file "<<filename<<std::endl;
      exit(1);
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Loads indices from file
//
void load_ind(const int nind, const char *filename, const int n, int *ind) {

  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nind;k++) {
	if (!(file >> ind[i*nind+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Loads constraints and masses from file
//
void load_constr_mass(const int nconstr, const int nmass, const char *filename, const int n,
		      double *constr, double *mass) {

  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nconstr;k++) {
	if (!(file >> constr[i*nconstr+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
      for (int k=0;k < nmass;k++) {
	if (!(file >> mass[i*nmass+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Checks SETTLE and SHAKE results
//
bool check_result(const int nind, const int n, const int *ind,
		  const double *x, const double *y, const double *z,
		  const double *x_ref, const double *y_ref, const double *z_ref,
		  const double tol, double &max_diff) {

  double x1, y1, z1;
  double x2, y2, z2;
  double diff;
  int imol, j, i;

  try {
    for (imol=0;imol < n;imol++) {
      for (j=0;j < nind;j++) {
	i = ind[imol*nind+j];
	x1 = x[i];
	y1 = y[i];
	z1 = z[i];
	x2 = x_ref[i];
	y2 = y_ref[i];
	z2 = z_ref[i];
	if (isnan(x1) || isnan(y1) || isnan(z1) || isnan(x2) || isnan(y2) || isnan(z2)) throw 1;
	diff = max(fabs(x1-x2), max(fabs(y1-y2), fabs(z1-z2)));
	max_diff = max(diff, max_diff);
	if (diff > tol) throw 1;
      }
    }
  }
  catch (int a) {
    std::cout << "comparison FAILED, imol=" << imol << " diff=" << diff << std::endl;
    std::cout << "ind =";
    for (j=0;j < nind;j++) {
      std::cout << " " << ind[imol*nind+j];
    }
    std::cout << std::endl;
    std::cout << x1 << " "<< y1 << " "<< z1 << std::endl;
    std::cout << x2 << " "<< y2 << " "<< z2 << std::endl;
    return false;
  }

  return true;
}

//
// Test the code using data in test_data/ -directory
//
void test() {

  // Settings for the data:
  const double mO = 15.9994;
  const double mH = 1.008;
  const double rOHsq = 0.91623184;
  const double rHHsq = 2.29189321;
  const int ncoord = 23558;
  const int nsolvent = 7023;
  const int npair = 458;
  const int ntrip = 233;
  const int nquad = 99;

  cudaXYZ<double> xyz0(ncoord);
  cudaXYZ<double> xyz1(ncoord);

  // Load coordinates
  hostXYZ<double> h_xyz0(ncoord, NON_PINNED);
  hostXYZ<double> h_xyz1(ncoord, NON_PINNED);
  hostXYZ<double> h_xyz_ref(ncoord, NON_PINNED);
  load_coord("test_data/xyz0.txt", h_xyz0.size(), h_xyz0.x(), h_xyz0.y(), h_xyz0.z());
  load_coord("test_data/xyz1.txt", h_xyz1.size(), h_xyz1.x(), h_xyz1.y(), h_xyz1.z());
  load_coord("test_data/xyz_ref.txt", h_xyz_ref.size(), h_xyz_ref.x(), h_xyz_ref.y(), h_xyz_ref.z());

  xyz0.set_data_sync(h_xyz0);
  xyz1.set_data_sync(h_xyz1);

  // Load constraint indices
  int *h_solvent_ind = (int *)malloc(nsolvent*3*sizeof(int));
  load_ind(3, "test_data/solvent_ind.txt", nsolvent, h_solvent_ind);

  int *h_pair_ind = (int *)malloc(npair*2*sizeof(int));
  load_ind(2, "test_data/pair_ind.txt", npair, h_pair_ind);

  int *h_trip_ind = (int *)malloc(ntrip*3*sizeof(int));
  load_ind(3, "test_data/trip_ind.txt", ntrip, h_trip_ind);

  int *h_quad_ind = (int *)malloc(nquad*4*sizeof(int));
  load_ind(4, "test_data/quad_ind.txt", nquad, h_quad_ind);

  // Load constraint distances and masses
  double *h_pair_constr = (double *)malloc(npair*sizeof(double));
  double *h_pair_mass = (double *)malloc(npair*2*sizeof(double));
  load_constr_mass(1, 2, "test_data/pair_constr_mass.txt", npair, h_pair_constr, h_pair_mass);

  double *h_trip_constr = (double *)malloc(ntrip*2*sizeof(double));
  double *h_trip_mass = (double *)malloc(ntrip*5*sizeof(double));
  load_constr_mass(2, 5, "test_data/trip_constr_mass.txt", ntrip, h_trip_constr, h_trip_mass);

  double *h_quad_constr = (double *)malloc(nquad*3*sizeof(double));
  double *h_quad_mass = (double *)malloc(nquad*7*sizeof(double));
  load_constr_mass(3, 7, "test_data/quad_constr_mass.txt", nquad, h_quad_constr, h_quad_mass);

  //--------------------------------------------------------------------------
  // Reduce constr and mass lists to indexed lists
  //--------------------------------------------------------------------------

  /*
  int npair_constr;
  int npair_mass;
  double *h_pair_constr_red = NULL;
  double *h_pair_mass_red = NULL;
  int *h_pair_constr_indlist = NULL;
  int *h_pair_mass_indlist = NULL;

  int ntrip_constr;
  int ntrip_mass;
  double *h_trip_constr_red = NULL;
  double *h_trip_mass_red = NULL;
  int *h_trip_constr_indlist = NULL;
  int *h_trip_mass_indlist = NULL;

  int nquad_constr;
  int nquad_mass;
  double *h_quad_constr_red = NULL;
  double *h_quad_mass_red = NULL;
  int *h_quad_constr_indlist = NULL;
  int *h_quad_mass_indlist = NULL;

  reduce_lists(npair, h_pair_constr, h_pair_mass, ntrip, h_trip_constr, h_trip_mass,
	       nquad, h_quad_constr, h_quad_mass,
	       &npair_constr, &h_pair_constr_red, &h_pair_constr_indlist,
	       &npair_mass, &h_pair_mass_red, &h_pair_mass_indlist,
	       &ntrip_constr, &h_trip_constr_red, &h_trip_constr_indlist,
	       &ntrip_mass, &h_trip_mass_red, &h_trip_mass_indlist,
	       &nquad_constr, &h_quad_constr_red, &h_quad_constr_indlist,
	       &nquad_mass, &h_quad_mass_red, &h_quad_mass_indlist);
  */

  //--------------------------------------------------------------------------
  // Setup & Apply holonomic constraints
  //--------------------------------------------------------------------------

  HoloConst holoconst;

  // Setup
  holoconst.setup_solvent_parameters(mO, mH, rOHsq, rHHsq);
  holoconst.setup_ind_mass_constr(npair, (int2 *)h_pair_ind, h_pair_constr, h_pair_mass,
				  ntrip, (int3 *)h_trip_ind, h_trip_constr, h_trip_mass,
				  nquad, (int4 *)h_quad_ind, h_quad_constr, h_quad_mass,
				  nsolvent, (int3 *)h_solvent_ind);

  /*
  holoconst.set_pair_constr(npair_constr, h_pair_constr_red, h_pair_constr_indlist);
  holoconst.set_pair_mass(npair_mass, h_pair_mass_red, h_pair_mass_indlist);
  holoconst.set_trip_constr(ntrip_constr, h_trip_constr_red, h_trip_constr_indlist);
  holoconst.set_trip_mass(ntrip_mass, h_trip_mass_red, h_trip_mass_indlist);
  holoconst.set_quad_constr(nquad_constr, h_quad_constr_red, h_quad_constr_indlist);
  holoconst.set_quad_mass(nquad_mass, h_quad_mass_red, h_quad_mass_indlist);
  */

  // Apply holonomic constraints
  holoconst.apply(xyz0, xyz1);
  cudaCheck(hipDeviceSynchronize());

  //copy_HtoD<double>(h_xyz1, xyz1, stride*3);
  xyz1.set_data_sync(h_xyz1);
  holoconst.apply(xyz0, xyz1);
  cudaCheck(hipDeviceSynchronize());

  //--------------------------------------------------------------------------
  // Check result
  //--------------------------------------------------------------------------
  copy_DtoH<double>(xyz1.x(), h_xyz1.x(), xyz1.size());
  copy_DtoH<double>(xyz1.y(), h_xyz1.y(), xyz1.size());
  copy_DtoH<double>(xyz1.z(), h_xyz1.z(), xyz1.size());

  double max_diff;
  double tol = 5.0e-14;

  max_diff = 0.0;
  if (check_result(3, nsolvent, h_solvent_ind, h_xyz1.x(), h_xyz1.y(), h_xyz1.z(),
		   h_xyz_ref.x(), h_xyz_ref.y(), h_xyz_ref.z(), tol, max_diff)) {
    std::cout<<"solvent SETTLE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  max_diff = 0.0;
  if (check_result(2, npair, h_pair_ind, h_xyz1.x(), h_xyz1.y(), h_xyz1.z(),
		   h_xyz_ref.x(), h_xyz_ref.y(), h_xyz_ref.z(), tol, max_diff)) {
    std::cout<<"pair SHAKE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  max_diff = 0.0;
  if (check_result(3, ntrip, h_trip_ind, h_xyz1.x(), h_xyz1.y(), h_xyz1.z(),
		   h_xyz_ref.x(), h_xyz_ref.y(), h_xyz_ref.z(), tol, max_diff)) {
    std::cout<<"trip SHAKE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  max_diff = 0.0;
  if (check_result(4, nquad, h_quad_ind, h_xyz1.x(), h_xyz1.y(), h_xyz1.z(),
		   h_xyz_ref.x(), h_xyz_ref.y(), h_xyz_ref.z(), tol, max_diff)) {
    std::cout<<"quad SHAKE OK (tolerance " << tol << " max difference " << 
      max_diff << ")" << std::endl;
  }

  free(h_solvent_ind);

  free(h_pair_ind);
  free(h_trip_ind);
  free(h_quad_ind);

  free(h_pair_constr);
  free(h_pair_mass);
  free(h_trip_constr);
  free(h_trip_mass);
  free(h_quad_constr);
  free(h_quad_mass);

  /*
  free(h_pair_constr_red);
  free(h_pair_mass_red);
  free(h_pair_constr_indlist);
  free(h_pair_mass_indlist);

  free(h_trip_constr_red);
  free(h_trip_mass_red);
  free(h_trip_constr_indlist);
  free(h_trip_mass_indlist);

  free(h_quad_constr_red);
  free(h_quad_mass_red);
  free(h_quad_constr_indlist);
  free(h_quad_mass_indlist);
  */

}
