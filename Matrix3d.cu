#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include "gpu_utils.h"
#include "Matrix3d.h"

const int TILEDIM = 32;
const int TILEROWS = 8;

template <typename T>
__global__ void transpose_xyz_yzx_kernel() {
}

//
// Copies a 3d matrixL data_in(x, y, z) -> data_out(x, y, z)
//
template <typename T>
__global__ void copy_kernel(const int nx, const int ny, const int nz,
			    const int xsize, const int ysize, const int zsize,
			    const T* data_in, T* data_out) {

  const int x = blockIdx.x * TILEDIM + threadIdx.x;
  const int y = blockIdx.y * TILEDIM + threadIdx.y;
  const int z = blockIdx.z + threadIdx.z;

  for (int j=0;j < TILEDIM;j += TILEROWS)
    data_out[x + (y + j + z*ysize)*xsize] = data_in[x + (y + j + z*ysize)*xsize];

}


//
// Transposes a 3d matrix out-of-place: data_in(x, y, z) -> data_out(y, z, x)
//
template <typename T>
__global__ void transpose_xyz_yzx_kernel(const int nx, const int ny, const int nz,
					 const int xsize, const int ysize, const int zsize,
					 const T* data_in, T* data_out) {

  // Shared memory
  __shared__ T tile[TILEDIM][TILEDIM+1];

  int x = blockIdx.x * TILEDIM + threadIdx.x;
  int y = blockIdx.y * TILEDIM + threadIdx.y;
  int z = blockIdx.z           + threadIdx.z;

  // Read (x,y) data_in into tile (shared memory)
  for (int j=0;j < TILEDIM;j += TILEROWS)
    tile[threadIdx.y + j][threadIdx.x] = data_in[x + (y + j + z*ysize)*xsize];

  __syncthreads();

  // Write (y,x) tile into data_out
  y = blockIdx.y * TILEDIM + threadIdx.x;
  x = blockIdx.x * TILEDIM + threadIdx.y;
  for (int j=0;j < TILEDIM;j += TILEROWS)
    data_out[y + (z + (x+j)*zsize)*ysize] = tile[threadIdx.x][threadIdx.y + j];

}

template <typename T>
Matrix3d<T>::Matrix3d() : nx(0), ny(0), nz(0), xsize(0), ysize(0), zsize(0) {
  data = NULL;
  external_storage = false;
}

template <typename T>
Matrix3d<T>::Matrix3d(const int nx, const int ny, const int nz, T* ext_data) : 
  nx(nx), ny(ny), nz(nz), xsize(nx), ysize(ny), zsize(nz) {
  init(xsize*ysize*zsize, ext_data);
}

template <typename T>
Matrix3d<T>::Matrix3d(const int nx, const int ny, const int nz,
		      const int xsize, const int ysize, const int zsize, T* ext_data) : 
  nx(nx), ny(ny), nz(nz), xsize(xsize), ysize(ysize), zsize(zsize) {
  init(xsize*ysize*zsize, ext_data);
}

template <typename T>
Matrix3d<T>::Matrix3d(const int nx, const int ny, const int nz,
		      const char *filename, T* ext_data) : 
  nx(nx), ny(ny), nz(nz), xsize(nx), ysize(ny), zsize(nz) {
  init(xsize*ysize*zsize, ext_data);
  load(nx, ny, nz, filename);
}

template <typename T>
Matrix3d<T>::~Matrix3d() {
  if (!external_storage)
    deallocate<T>(&data);
}

template <typename T>
void Matrix3d<T>::init(const int size, T* ext_data) {
  if (ext_data == NULL) {
    allocate<T>(&data, size);
    external_storage = false;
  } else {
    data = ext_data;
    external_storage = true;
  }
}

//
// Prints matrix size on screen
//
template <typename T>
void Matrix3d<T>::print_info() {
  std::cout << "nx ny nz          = " << nx << " "<< ny << " "<< nz << std::endl;
  std::cout << "xsize ysize zsize = " << xsize << " "<< ysize << " "<< zsize << std::endl;
}

template <>
inline double Matrix3d<long long int>::norm(long long int a, long long int b) {
  return (double)llabs(a-b);
}

template <>
inline double Matrix3d<float>::norm(float a, float b) {
  return (double)fabsf(a-b);
}

template <>
inline double Matrix3d<float2>::norm(float2 a, float2 b) {
  return (double)max(fabsf(a.x-b.x), fabsf(a.y-b.y) );
}

std::ostream& operator<<(std::ostream& os, float2& a) {
  os << a.x << " " << a.y;
  return os;
}

std::istream& operator>>(std::istream& is, float2& a) {
  is >> a.x >> a.y;
  return is;
}

//
// Compares two matrices, returns true if the difference is within tolerance
// NOTE: Comparison is done in double precision
//
template <typename T>
bool Matrix3d<T>::compare(Matrix3d<T>* mat, const double tol, double& max_diff) {

  assert(mat->nx == nx);
  assert(mat->ny == ny);
  assert(mat->nz == nz);

  T *h_data1 = new T[xsize*ysize*zsize];
  T *h_data2 = new T[xsize*ysize*zsize];

  copy_DtoH<T>(data,      h_data1, xsize*ysize*zsize);
  copy_DtoH<T>(mat->data, h_data2, xsize*ysize*zsize);

  bool ok = true;

  max_diff = 0.0;

  int x, y, z;
  double diff;
  try {
    for (z=0;z < nz;z++)
      for (y=0;y < ny;y++)
	for (x=0;x < nx;x++) {
	  diff = norm(h_data1[x + (y + z*ysize)*xsize], h_data2[x + (y + z*ysize)*xsize]);
	  max_diff = (diff > max_diff) ? diff : max_diff;
	  if (diff > tol) throw 1;
	}
  }
  catch (int a) {
    std::cout << "x y z = " << x << " "<< y << " "<< z << std::endl;
    std::cout << "this: " << h_data1[x + (y + z*ysize)*xsize] << std::endl;
    std::cout << "mat:  " << h_data2[x + (y + z*ysize)*xsize] << std::endl;
    std::cout << "difference: " << diff << std::endl;
    ok = false;
  }

  delete [] h_data1;
  delete [] h_data2;
  
  return ok;
}

//
// Transposes a 3d matrix out-of-place: data(x, y, z) -> data(y, z, x)
// NOTE: this is a slow reference calculation performed on the host
//
template <typename T>
void Matrix3d<T>::transpose_xyz_yzx_host(Matrix3d<T>* mat) {

  assert(mat->nx == ny);
  assert(mat->ny == nz);
  assert(mat->nz == nx);

  T *h_data1 = new T[xsize*ysize*zsize];
  T *h_data2 = new T[xsize*ysize*zsize];

  copy_DtoH<T>(data,      h_data1, xsize*ysize*zsize);
  copy_DtoH<T>(mat->data, h_data2, xsize*ysize*zsize);

  for (int z=0;z < nz;z++)
    for (int y=0;y < ny;y++)
      for (int x=0;x < nx;x++)
	h_data2[y + (z + x*zsize)*ysize] = h_data1[x + (y + z*ysize)*xsize];

  copy_HtoD<T>(h_data1, data,      xsize*ysize*zsize);
  copy_HtoD<T>(h_data2, mat->data, xsize*ysize*zsize);

  delete [] h_data1;
  delete [] h_data2;

}

//
// Transposes a 3d matrix out-of-place: data(x, y, z) -> data(y, z, x)
//
template <typename T>
void Matrix3d<T>::transpose_xyz_yzx(Matrix3d<T>* mat) {

  assert(mat->nx == ny);
  assert(mat->ny == nz);
  assert(mat->nz == nx);

  dim3 nthread(TILEDIM, TILEROWS, 1);
  dim3 nblock((nx-1)/TILEDIM+1, (ny-1)/TILEDIM+1, nz);

  transpose_xyz_yzx_kernel<<< nblock, nthread >>>(nx, ny, nz, xsize, ysize, zsize,
						  data, mat->data);

  cudaCheck(hipGetLastError());

}

//
// Copies a 3d matrix data(x, y, z) -> data(x, y, z)
//
template <typename T>
void Matrix3d<T>::copy(Matrix3d<T>* mat) {

  assert(mat->nx == nx);
  assert(mat->ny == ny);
  assert(mat->nz == nz);

  dim3 nthread(TILEDIM, TILEROWS, 1);
  dim3 nblock((nx-1)/TILEDIM+1, (ny-1)/TILEDIM+1, nz);

  copy_kernel<<< nblock, nthread >>>(nx, ny, nz, xsize, ysize, zsize,
				     data, mat->data);

  cudaCheck(hipGetLastError());

}

//
// Prints part of matrix (x0:x1, y0:y1, z0:z1) on screen
//
template <typename T>
void Matrix3d<T>::print(const int x0, const int x1, 
			const int y0, const int y1,
			const int z0, const int z1) {
  T *h_data = new T[xsize*ysize*zsize];

  copy_DtoH<T>(data, h_data, xsize*ysize*zsize);

  for (int z=z0;z <= z1;z++)
    for (int y=y0;y <= y1;y++)
      for (int x=x0;x <= x1;x++)
	std::cout << h_data[x + (y + z*ysize)*xsize] << std::endl;

  delete [] h_data;
}

//
// Loads Matrix of size nx,ny,nz from file "filename"
//
template <typename T>
void Matrix3d<T>::load(const int nx, const int ny, const int nz,
		       const char *filename) {

  assert(this->nx == nx);
  assert(this->ny == ny);
  assert(this->nz == nz);

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    // Allocate CPU memory
    T *h_data = new T[xsize*ysize*zsize];
    
    // Read data
    for (int z=0;z < nz;z++)
      for (int y=0;y < ny;y++)
	for (int x=0;x < nx;x++)
	  file >> h_data[x + (y + z*ysize)*xsize];

    // Copy data from CPU to GPU
    copy_HtoD<T>(h_data, data, xsize*ysize*zsize);

    // Deallocate CPU memory
    delete [] h_data;

    // Close file
    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

}

//
// Explicit instances of Matrix3d
//
template class Matrix3d<float>;
template class Matrix3d<float2>;
template class Matrix3d<long long int>;
