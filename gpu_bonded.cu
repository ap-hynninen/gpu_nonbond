#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "XYZQ.h"
#include "Force.h"
#include "BondedForce.h"
#include "VirialPressure.h"

void test();

int numnode = 1;
int mynode = 0;

int main(int argc, char *argv[]) {

  start_gpu(numnode, mynode);
  
  test();

  return 0;
}

//
// Loads indices from file
//
template <typename T>
void load_ind(const int nind, const char *filename, const int n, T *ind) {
  std::ifstream file(filename);
  if (file.is_open()) {

    for (int i=0;i < n;i++) {
      for (int k=0;k < nind;k++) {
	if (!(file >> ind[i*nind+k])) {
	  std::cerr<<"Error reading file "<<filename<<std::endl;
	  exit(1);
	}
      }
    }

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Test the code using data in test_data/ -directory
//
void test() {

  // Settings for the data:
  const double boxx = 62.23;
  const double boxy = 62.23;
  const double boxz = 62.23;
  const int ncoord = 23558;

  const double energy_bond_ref = 715.08289;
  const double energy_ureyb_ref = 167.39536;
  const double energy_angle_ref = 1228.72913;
  const double energy_dihe_ref = 921.88694;
  const double energy_imdihe_ref = 102.07776;

  double sforce_ref[81];
  load_ind<double>(81, "test_data/sforce_bonded.txt", 1, sforce_ref);

  double sforcex[27], sforcey[27], sforcez[27];
  double energy_bond, energy_ureyb, energy_angle, energy_dihe, energy_imdihe, energy_cmap;

  Force<double> force_bonded("test_data/force_bonded.txt");
  Force<long long int> force_fp(ncoord);
  Force<double> force(ncoord);

  // Load coordinates
  XYZQ xyzq("test_data/xyzq.txt", 32);

  const int nbondlist = 23592;
  const int nbondcoef = 129;

  const int nureyblist = 11584;
  const int nureybcoef = 327;

  const int nanglelist = 11584;
  const int nanglecoef = 327;

  const int ndihelist = 6701;
  const int ndihecoef = 438;

  const int nimdihelist = 418;
  const int nimdihecoef = 40;

  const int ncmaplist = 0;
  const int ncmapcoef = 0;

  bondlist_t *h_bondlist = new bondlist_t[nbondlist];
  float2 *h_bondcoef = new float2[nbondcoef];
  load_ind<int>(4, "test_data/bondlist.txt", nbondlist, (int *)h_bondlist);
  load_ind<float>(2, "test_data/bondcoef.txt", nbondcoef, (float *)h_bondcoef);

  bondlist_t *h_ureyblist = new bondlist_t[nureyblist];
  float2 *h_ureybcoef = new float2[nureybcoef];
  load_ind<int>(4, "test_data/ureyblist.txt", nureyblist, (int *)h_ureyblist);
  load_ind<float>(2, "test_data/ureybcoef.txt", nureybcoef, (float *)h_ureybcoef);

  anglelist_t *h_anglelist = new anglelist_t[nanglelist];
  float2 *h_anglecoef = new float2[nanglecoef];
  load_ind<int>(6, "test_data/anglelist.txt", nanglelist, (int *)h_anglelist);
  load_ind<float>(2, "test_data/anglecoef.txt", nanglecoef, (float *)h_anglecoef);

  dihelist_t *h_dihelist = new dihelist_t[ndihelist];
  float4 *h_dihecoef = new float4[ndihecoef];
  load_ind<int>(8, "test_data/dihelist.txt", ndihelist, (int *)h_dihelist);
  load_ind<float>(4, "test_data/dihecoef.txt", ndihecoef, (float *)h_dihecoef);

  dihelist_t *h_imdihelist = new dihelist_t[nimdihelist];
  float4 *h_imdihecoef = new float4[nimdihecoef];
  load_ind<int>(8, "test_data/imdihelist.txt", nimdihelist, (int *)h_imdihelist);
  load_ind<float>(4, "test_data/imdihecoef.txt", nimdihecoef, (float *)h_imdihecoef);

  cmaplist_t *h_cmaplist = NULL; //new cmaplist_t[ncmaplist];
  float2 *h_cmapcoef = NULL;//new float2[ncmaplist];
  //load_ind<int>(8, "test_data/cmaplist_176k.txt", ncmaplist, (int *)h_cmaplist);
  //load_ind<float>(2, "test_data/cmapcoef_176k.txt", ncmaplist, (float *)h_cmapcoef);

  force_fp.clear();

  // Single precision
  {
    force_fp.clear();
    BondedForce<long long int, float> bondedforce;
    bondedforce.clear_energy_virial();
    bondedforce.setup_coef(nbondcoef, h_bondcoef,
			   nureybcoef, h_ureybcoef,
			   nanglecoef, h_anglecoef,
			   ndihecoef, h_dihecoef,
			   nimdihecoef, h_imdihecoef,
			   ncmapcoef, h_cmapcoef);
    bondedforce.setup_list(nbondlist, h_bondlist, 
			   nureyblist, h_ureyblist, 
			   nanglelist, h_anglelist, 
			   ndihelist, h_dihelist, 
			   nimdihelist, h_imdihelist, 
			   ncmaplist, h_cmaplist);

    bondedforce.calc_force(xyzq.xyzq, boxx, boxy, boxz, true, false,
			   force_fp.xyz.stride, force_fp.xyz.data,
			   true, true, true, true, true, true);
    bondedforce.get_energy_virial(true, false,
				  &energy_bond, &energy_ureyb,
				  &energy_angle,
				  &energy_dihe, &energy_imdihe,
				  &energy_cmap,
				  sforcex, sforcey, sforcez);
    force_fp.convert(&force);

    double max_diff;
    double tol = 0.0057;
    if (!force_bonded.compare(&force, tol, max_diff)) {
      std::cout << "(SP) Bonded force comparison FAILED " << std::endl;
    } else {
      std::cout<<"(SP) Bonded force comparison OK (tolerance " << tol << " max difference " 
	       << max_diff << ")" << std::endl;
    }

    max_diff = fabs(energy_bond_ref - energy_bond);
    if (max_diff > tol) {
      std::cout << "(SP) energy_bond comparison FAILED: ref = " << energy_bond_ref 
		<< " energy = " << energy_bond << std::endl;
    } else {
      std::cout << "(SP) energy_bond comparison OK (tolerance " << tol << " difference " 
		<< max_diff << ")" << std::endl;
    }

    max_diff = fabs(energy_ureyb_ref - energy_ureyb);
    if (max_diff > tol) {
      std::cout << "(SP) energy_ureyb comparison FAILED: ref = " << energy_ureyb_ref 
		<< " energy = " << energy_ureyb << std::endl;
    } else {
      std::cout << "(SP) energy_ureyb comparison OK (tolerance " << tol << " difference " 
		<< max_diff << ")" << std::endl;
    }

    max_diff = fabs(energy_angle_ref - energy_angle);
    if (max_diff > tol) {
      std::cout << "(SP) energy_angle comparison FAILED: ref = " << energy_angle_ref 
		<< " energy = " << energy_angle << std::endl;
    } else {
      std::cout << "(SP) energy_angle comparison OK (tolerance " << tol << " difference " 
		<< max_diff << ")" << std::endl;
    }

    max_diff = fabs(energy_dihe_ref - energy_dihe);
    if (max_diff > tol) {
      std::cout << "(SP) energy_dihe comparison FAILED: ref = " << energy_dihe_ref 
		<< " energy = " << energy_dihe << std::endl;
    } else {
      std::cout << "(SP) energy_dihe comparison OK (tolerance " << tol << " difference " 
		<< max_diff << ")" << std::endl;
    }

    max_diff = fabs(energy_imdihe_ref - energy_imdihe);
    if (max_diff > tol) {
      std::cout << "(SP) energy_imdihe comparison FAILED: ref = " << energy_imdihe_ref 
		<< " energy = " << energy_imdihe << std::endl;
    } else {
      std::cout << "(SP) energy_imdihe comparison OK (tolerance " << tol << " difference " 
		<< max_diff << ")" << std::endl;
    }

    force_fp.clear();
    bondedforce.calc_force(xyzq.xyzq, boxx, boxy, boxz, false, false,
			   force_fp.xyz.stride, force_fp.xyz.data,
			   true, true, true, true, true, true);
    force_fp.convert(&force);

    tol = 0.0057;
    if (!force_bonded.compare(&force, tol, max_diff)) {
      std::cout << "(SP) Bonded force comparison FAILED " << std::endl;
    } else {
      std::cout<<"(SP) Bonded force comparison OK (tolerance " << tol << " max difference " 
	       << max_diff << ")" << std::endl;
    }

  }

  // Double precision
  {
    force_fp.clear();
    BondedForce<long long int, double> bondedforce;
    bondedforce.clear_energy_virial();
    bondedforce.setup_coef(nbondcoef, h_bondcoef,
			   nureybcoef, h_ureybcoef,
			   nanglecoef, h_anglecoef,
			   ndihecoef, h_dihecoef,
			   nimdihecoef, h_imdihecoef,
			   ncmapcoef, h_cmapcoef);
    bondedforce.setup_list(nbondlist, h_bondlist, 
			   nureyblist, h_ureyblist, 
			   nanglelist, h_anglelist, 
			   ndihelist, h_dihelist, 
			   nimdihelist, h_imdihelist, 
			   ncmaplist, h_cmaplist);
    bondedforce.calc_force(xyzq.xyzq, boxx, boxy, boxz, true, false,
			   force_fp.xyz.stride, force_fp.xyz.data,
			   true, true, true, true, true, true);
    force_fp.convert(&force);

    double max_diff;
    double tol = 0.0058;
    if (!force_bonded.compare(&force, tol, max_diff)) {
      std::cout<<"(DP) Bonded force comparison FAILED"<<std::endl;
    } else {
      std::cout<<"(DP) Bonded force comparison OK (tolerance " << tol << " max difference " 
	       << max_diff << ")" << std::endl;
    }
  }

  delete [] h_bondlist;
  delete [] h_bondcoef;
  
  delete [] h_ureyblist;
  delete [] h_ureybcoef;
  
  delete [] h_anglelist;
  delete [] h_anglecoef;

  delete [] h_dihelist;
  delete [] h_dihecoef;
  
  delete [] h_imdihelist;
  delete [] h_imdihecoef;

  delete [] h_cmaplist;
  delete [] h_cmapcoef;
  
}
