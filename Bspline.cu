#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gpu_utils.h"
#include "Bspline.h"

template <typename T>
__global__ void fill_bspline_4(const float4 *xyzq, const int ncoord, const float *recip,
			       const int nfftx, const int nffty, const int nfftz,
			       gridp_t *gridp, float3 *theta, float3 *dtheta) {

  // Position to xyzq and atomgrid
  unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;

  while (pos < ncoord) {
    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    float w;
    // NOTE: I don't think we need the +2.0f here..
    w = x*recip[0] + y*recip[1] + z*recip[2] + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));

    w = x*recip[3] + y*recip[4] + z*recip[5] + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));

    w = x*recip[6] + y*recip[7] + z*recip[8] + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)(frx);
    int fryi = (int)(fry);
    int frzi = (int)(frz);

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    gridp[pos].x = frxi;
    gridp[pos].y = fryi;
    gridp[pos].z = frzi;
    gridp[pos].q = q;

    float3 theta_tmp[4];
    float3 dtheta_tmp[4];

    theta_tmp[3].x = 0.0f;
    theta_tmp[3].y = 0.0f;
    theta_tmp[3].z = 0.0f;
    theta_tmp[1].x = wx;
    theta_tmp[1].y = wy;
    theta_tmp[1].z = wz;
    theta_tmp[0].x = 1.0f - wx;
    theta_tmp[0].y = 1.0f - wy;
    theta_tmp[0].z = 1.0f - wz;

    // compute standard b-spline recursion
    theta_tmp[2].x = 0.5f*wx*theta_tmp[1].x;
    theta_tmp[2].y = 0.5f*wy*theta_tmp[1].y;
    theta_tmp[2].z = 0.5f*wz*theta_tmp[1].z;
       
    theta_tmp[1].x = 0.5f*((wx+1.0f)*theta_tmp[0].x + (2.0f-wx)*theta_tmp[1].x);
    theta_tmp[1].y = 0.5f*((wy+1.0f)*theta_tmp[0].y + (2.0f-wy)*theta_tmp[1].y);
    theta_tmp[1].z = 0.5f*((wz+1.0f)*theta_tmp[0].z + (2.0f-wz)*theta_tmp[1].z);
       
    theta_tmp[0].x = 0.5f*(1.0f-wx)*theta_tmp[0].x;
    theta_tmp[0].y = 0.5f*(1.0f-wy)*theta_tmp[0].y;
    theta_tmp[0].z = 0.5f*(1.0f-wz)*theta_tmp[0].z;
       
    // perform standard b-spline differentiationa
    dtheta_tmp[0].x = -theta_tmp[0].x;
    dtheta_tmp[0].y = -theta_tmp[0].y;
    dtheta_tmp[0].z = -theta_tmp[0].z;

    dtheta_tmp[1].x = theta_tmp[0].x - theta_tmp[1].x;
    dtheta_tmp[1].y = theta_tmp[0].y - theta_tmp[1].y;
    dtheta_tmp[1].z = theta_tmp[0].z - theta_tmp[1].z;

    dtheta_tmp[2].x = theta_tmp[1].x - theta_tmp[2].x;
    dtheta_tmp[2].y = theta_tmp[1].y - theta_tmp[2].y;
    dtheta_tmp[2].z = theta_tmp[1].z - theta_tmp[2].z;

    dtheta_tmp[3].x = theta_tmp[2].x - theta_tmp[3].x;
    dtheta_tmp[3].y = theta_tmp[2].y - theta_tmp[3].y;
    dtheta_tmp[3].z = theta_tmp[2].z - theta_tmp[3].z;
          
    // one more recursion
    theta_tmp[3].x = (1.0f/3.0f)*wx*theta_tmp[2].x;
    theta_tmp[3].y = (1.0f/3.0f)*wy*theta_tmp[2].y;
    theta_tmp[3].z = (1.0f/3.0f)*wz*theta_tmp[2].z;

    theta_tmp[2].x = (1.0f/3.0f)*((wx+1.0f)*theta_tmp[1].x + (3.0f-wx)*theta_tmp[2].x);
    theta_tmp[2].y = (1.0f/3.0f)*((wy+1.0f)*theta_tmp[1].y + (3.0f-wy)*theta_tmp[2].y);
    theta_tmp[2].z = (1.0f/3.0f)*((wz+1.0f)*theta_tmp[1].z + (3.0f-wz)*theta_tmp[2].z);

    theta_tmp[1].x = (1.0f/3.0f)*((wx+2.0f)*theta_tmp[0].x + (2.0f-wx)*theta_tmp[1].x);
    theta_tmp[1].y = (1.0f/3.0f)*((wy+2.0f)*theta_tmp[0].y + (2.0f-wy)*theta_tmp[1].y);
    theta_tmp[1].z = (1.0f/3.0f)*((wz+2.0f)*theta_tmp[0].z + (2.0f-wz)*theta_tmp[1].z);
       
    theta_tmp[0].x = (1.0f/3.0f)*(1.0f-wx)*theta_tmp[0].x;
    theta_tmp[0].y = (1.0f/3.0f)*(1.0f-wy)*theta_tmp[0].y;
    theta_tmp[0].z = (1.0f/3.0f)*(1.0f-wz)*theta_tmp[0].z;

    // Store theta_tmp and dtheta_tmp into global memory
    int pos4 = pos*4;
    theta[pos4]   = theta_tmp[0];
    theta[pos4+1] = theta_tmp[1];
    theta[pos4+2] = theta_tmp[2];
    theta[pos4+3] = theta_tmp[3];

    dtheta[pos4]   = dtheta_tmp[0];
    dtheta[pos4+1] = dtheta_tmp[1];
    dtheta[pos4+2] = dtheta_tmp[2];
    dtheta[pos4+3] = dtheta_tmp[3];

    pos += blockDim.x*gridDim.x;
  }

}

//
// Bspline class method definitions
//
// (c) Antti-Pekka Hynninen, 2013, aphynninen@hotmail.com
//

template <typename T>
void Bspline<T>::init(const int ncoord) {
  reallocate<T>(&theta, &theta_len, 3*ncoord*order, 1.2f);
  reallocate<T>(&dtheta, &dtheta_len, 3*ncoord*order, 1.2f);
  reallocate<gridp_t>(&gridp, &gridp_len, ncoord, 1.2f);  
}

template <typename T>
Bspline<T>::Bspline(const int ncoord, const int order, const double *h_recip) :
  theta(NULL), dtheta(NULL), gridp(NULL), order(order) {
  init(ncoord);
  allocate<T>(&recip, 9);
  set_recip(h_recip);
}
  
template <typename T>
Bspline<T>::~Bspline() {
  deallocate<T>(&theta);
  deallocate<T>(&dtheta);
  deallocate<gridp_t>(&gridp);
  deallocate<T>(&recip);
}

template <typename T>
template <typename B>
void Bspline<T>::set_recip(const B *h_recip) {
  T h_recip_T[9];
  for (int i=0;i < 9;i++) h_recip_T[i] = (T)h_recip[i];
  copy_HtoD<T>(h_recip_T, recip, 9);
}

template <typename T>
void Bspline<T>::fill_bspline(const float4 *xyzq, const int ncoord,
			      const int nfftx, const int nffty, const int nfftz) {
  int nthread = 64;
  int nblock = (ncoord-1)/nthread + 1;

  std::cout << "nblock=" << nblock << std::endl;

  //bool ortho = (recip[1] == recip[2] == recip[3] == recip[5] == recip[6] == recip[7] == 0.0f);
  
  switch(order) {
  case 4:
    fill_bspline_4<T> <<< nblock, nthread >>>(xyzq, ncoord, recip, 
					      nfftx, nffty, nfftz, gridp, 
					      (float3 *)theta, (float3 *)dtheta);
    break;
  default:
    exit(1);
  }
  
  cudaCheck(hipGetLastError());
}

void dftmod(double *bsp_mod, const double *bsp_arr, const int nfft) {

  const double rsmall = 1.0e-10;
  double nfftr = (2.0*3.14159265358979323846)/(double)nfft;

  for (int k=1;k <= nfft;k++) {
    double sum1 = 0.0;
    double sum2 = 0.0;
    double arg1 = (k-1)*nfftr;
    for (int j=1;j < nfft;j++) {
      double arg = arg1*(j-1);
      sum1 += bsp_arr[j-1]*cos(arg);
      sum2 += bsp_arr[j-1]*sin(arg);
    }
    bsp_mod[k-1] = sum1*sum1 + sum2*sum2;
  }

  for (int k=1;k <= nfft;k++)
    if (bsp_mod[k-1] < rsmall)
      bsp_mod[k-1] = 0.5*(bsp_mod[k-1-1] + bsp_mod[k+1-1]);

  for (int k=1;k <= nfft;k++)
    bsp_mod[k-1] = 1.0/bsp_mod[k-1];

}

//
// Calculates (prefac_x, prefac_y, prefac_z)
// NOTE: This calculation is done on the CPU since it is only done very infrequently
//
template <typename T>
void Bspline<T>::calc_prefac() {
  
  int max_nfft = max(nfftx, nffty, nfftz);
  double *bsp_arr = new double[max_nfft];
  double *bsp_mod = new double[max_nfft];

  for (int i=0;i < max_nfft;i++) bsp_arr[i] = 0.0;

  fill_bspline_host(w, order, array, darray);

  dftmod(bsp_mod, bsp_arr, nfftx);
  for (int i=0;i < nfftx;i++) h_prefac_x[i] = (T)bsp_mod[i];

  delete [] bsp_arr;
  delete [] bsp_mod;
}

//
// Explicit instances of Bspline
//
template class Bspline<float>;
