#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"

static __device__ NeighborListParam_t d_nlist_param;

//
// Sort atoms into z-columns
//
// col_natom[0..ncellx*ncelly-1] = number of atoms in each column
// atom_icol[istart..iend]     = column index for atoms 
//
__global__ void calc_z_column_index_kernel(const float4* __restrict__ xyzq,
					   int* __restrict__ col_natom,
					   int* __restrict__ atom_icol,
					   int3* __restrict__ col_xy_zone) {

  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  int ind0 = 0;
  for (int izone=0;izone < 8;izone++) {
    if (i < d_nlist_param.zone_patom[izone]) {
      float4 xyzq_val = xyzq[i];
      float x = xyzq_val.x;
      float y = xyzq_val.y;
      float3 minxyz = d_nlist_param.minxyz[izone];
      int ix = (int)((x - minxyz.x)*d_nlist_param.inv_celldx[izone]);
      int iy = (int)((y - minxyz.y)*d_nlist_param.inv_celldy[izone]);
      int ind = ind0 + ix + iy*d_nlist_param.ncellx[izone];
      atomicAdd(&col_natom[ind], 1);
      atom_icol[i] = ind;
      int3 col_xy_zone_val;
      col_xy_zone_val.x = ix;
      col_xy_zone_val.y = iy;
      col_xy_zone_val.z = izone;
      col_xy_zone[ind] = col_xy_zone_val;
      break;
    }
    ind0 += d_nlist_param.ncellx[izone]*d_nlist_param.ncelly[izone];
  }

}

/*
//
// Computes z column position using parallel exclusive prefix sum
//
// NOTE: Must have nblock = 1, we loop over buckets to avoid multiple kernel calls
//
__global__ void calc_z_column_pos_kernel(const int ncol_tot,
					 int* __restrict__ col_natom,
					 int* __restrict__ col_patom) {
  // Shared memory
  // Requires: blockDim.x*sizeof(int)
  extern __shared__ int shpos[];

  if (threadIdx.x == 0) col_patom[0] = 0;

  int offset = 0;
  for (int base=0;base < ncol_tot;base += blockDim.x) {
    int i = base + threadIdx.x;
    shpos[threadIdx.x] = (i < ncol_tot) ? col_natom[i] : 0;
    if (i < ncol_tot) col_natom[i] = 0;
    __syncthreads();

    for (int d=1;d < blockDim.x; d *= 2) {
      int tmp = (threadIdx.x >= d) ? shpos[threadIdx.x-d] : 0;
      __syncthreads();
      shpos[threadIdx.x] += tmp;
      __syncthreads();
    }

    // Write result into global memory
    if (i < ncol_tot) col_patom[i+1] = shpos[threadIdx.x] + offset;

    offset += shpos[blockDim.x-1];
  }

}
*/

//
// Computes z column position using parallel exclusive prefix sum
// Also computes the cell_patom, col_ncellz, and ncell
//
// NOTE: Must have nblock = 1, we loop over buckets to avoid multiple kernel calls
//
template <int tilesize>
__global__ void calc_z_column_pos_kernel(const int ncol_tot,
					 const int3* __restrict__ col_xy_zone,
					 int* __restrict__ col_natom,
					 int* __restrict__ col_patom,
					 int* __restrict__ cell_patom,
					 int* __restrict__ col_ncellz,
					 int4* __restrict__ cell_xyz_zone,
					 int* __restrict__ col_cell) {
  // Shared memory
  // Requires: blockDim.x*sizeof(int2)
  extern __shared__ int2 shpos2[];

  if (threadIdx.x == 0) {
    col_patom[0] = 0;
  }

  int2 offset = make_int2(0, 0);
  for (int base=0;base < ncol_tot;base += blockDim.x) {
    int i = base + threadIdx.x;
    int2 tmpval;
    tmpval.x = (i < ncol_tot) ? col_natom[i] : 0;  // Number of atoms in this column
    tmpval.y = (i < ncol_tot) ? (tmpval.x - 1)/tilesize + 1 : 0; // Number of z-cells in this column
    col_ncellz[i] = tmpval.y;                      // Set col_ncellz[icol]
    shpos2[threadIdx.x] = tmpval;
    if (i < ncol_tot) col_natom[i] = 0;
    __syncthreads();

    for (int d=1;d < blockDim.x; d *= 2) {
      int2 tmp = (threadIdx.x >= d) ? shpos2[threadIdx.x-d] : make_int2(0, 0);
      __syncthreads();
      shpos2[threadIdx.x].x += tmp.x;
      shpos2[threadIdx.x].y += tmp.y;
      __syncthreads();
    }

    if (i < ncol_tot) {
      // Write col_patom in global memory
      int2 val1 = shpos2[threadIdx.x];
      val1.x += offset.x;
      val1.y += offset.y;
      col_patom[i+1] = val1.x;
      // Write cell_patom in global memory
      // OPTIMIZATION NOTE: Is this looping too slow? Should we move this into a separate kernel?
      int2 val0 = (threadIdx.x > 0) ? shpos2[threadIdx.x - 1] : make_int2(0, 0);
      val0.x += offset.x;
      val0.y += offset.y;
      int icell0 = val0.y;
      int icell1 = val1.y;
      int iatom  = val0.x;
      // Write col_cell
      col_cell[i] = icell0;
      // col_xy_zone[icol].x = x coordinate for each column
      // col_xy_zone[icol].y = y coordinate for each column
      // col_xy_zone[icol].z = zone for each column
      int4 cell_xyz_zone_val;
      int3 col_xy_zone_val = col_xy_zone[i];
      cell_xyz_zone_val.x = col_xy_zone_val.x;
      cell_xyz_zone_val.y = col_xy_zone_val.y;
      cell_xyz_zone_val.z = 0;
      cell_xyz_zone_val.w = col_xy_zone_val.z;
      for (int icell=icell0;icell < icell1;icell++,iatom+=tilesize,cell_xyz_zone_val.z++) {
	cell_patom[icell] = iatom;
	cell_xyz_zone[icell] = cell_xyz_zone_val;
      }
    }
    
    // Add the last value to the offset for the next block
    int2 lastval = shpos2[blockDim.x-1];
    offset.x += lastval.x;
    offset.y += lastval.y;

    // Sync threads so that the next iteration can start writing in shared memory
    __syncthreads();
  }

  // Write ncell into global GPU buffer
  if (threadIdx.x == 0) {
    d_nlist_param.ncell = offset.y;
  }

}

//
// Calculates ncellz_max[izone].
//
// blockDim.x = max number of columns over all zones
// Each thread block calculates one zone (blockIdx.x = izone)
//
__global__ void calc_ncellz_max(const int* __restrict__ col_ncellz) {

  // Shared memory
  // Requires: blockDim.x*sizeof(int)
  extern __shared__ int sh_col_ncellz[];

  // ncol[izone] gives the cumulative sum of ncellx[izone]*ncelly[izone]
  int start = d_nlist_param.ncol[blockIdx.x];
  int end   = d_nlist_param.ncol[blockIdx.x+1];
  int n = end - start - 1;

  if (n > 0) {
    // Load col_ncellz into shared memory
    int col_ncellz_val = 0;
    if (threadIdx.x < end) col_ncellz_val = col_ncellz[start + threadIdx.x];
    sh_col_ncellz[threadIdx.x] = col_ncellz_val;
    __syncthreads();
    
    // Reduce
    for (int d=1;d < n;d *= 2) {
      int t = threadIdx.x + d;
      int val = (t < n) ? sh_col_ncellz[t] : 0;
      __syncthreads();
      sh_col_ncellz[threadIdx.x] = max(sh_col_ncellz[threadIdx.x], val);
      __syncthreads();
    }
    
    // Write into global memory
    if (threadIdx.x == 0) {
      d_nlist_param.ncellz_max[blockIdx.x] = sh_col_ncellz[0];
    }
  } else {
    if (threadIdx.x == 0) {
      d_nlist_param.ncellz_max[blockIdx.x] = 0;
    }
  }

}

//
// Re-order atoms according to pos
//
__global__ void reorder_atoms_z_column_kernel(const int ncoord,
					      const int* atom_icol,
					      int* col_natom,
					      const int* col_patom,
					      const float4* __restrict__ xyzq_in,
					      float4* __restrict__ xyzq_out,
					      int* __restrict__ loc2glo_ind) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < ncoord) {
    int ind = atom_icol[i];
    int pos = col_patom[ind];
    int n = atomicAdd(&col_natom[ind], 1);
    // new position = pos + n
    int newpos = pos + n;
    loc2glo_ind[newpos] = i;
    float4 xyzq_val = xyzq_in[i];
    xyzq_out[newpos] = xyzq_val;
  }

  /*
  // Setup startcell_zone[izone]
  if (i == 0) {
    int p = 0;
    for (int izone=0;izone <= 8;izone++) {
      d_nlist_param.startcell_zone[izone] = p;
      p += d_nlist_param.ncellx[izone]*d_nlist_param.ncelly[izone]*d_nlist_param.ncellz_max[izone];
    }
  }
  */

}

//
// Sorts atoms according to z coordinate
//
// Uses bitonic sort, see:
// http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
//
// Each thread block sorts a single z column
//
struct keyval_t {
  float key;
  int val;
};
__global__ void sort_z_column_kernel(const int* __restrict__ col_patom,
				     float4* __restrict__ xyzq,
				     int* __restrict__ loc2glo_ind) {

  // Shared memory
  // Requires: blockDim.x*sizeof(keyval_t)
  extern __shared__ keyval_t sh_keyval[];

  int col_patom0 = col_patom[blockIdx.x];
  int n = col_patom[blockIdx.x+1] - col_patom0;

  // Read keys and values into shared memory
  keyval_t keyval;
  keyval.key = (threadIdx.x < n) ? xyzq[threadIdx.x + col_patom0].z : 1.0e38;
  keyval.val = (threadIdx.x < n) ? (threadIdx.x + col_patom0) : (n-1);
  sh_keyval[threadIdx.x] = keyval;
  __syncthreads();

  for (int k = 2;k <= blockDim.x;k *= 2) {
    for (int j = k/2; j > 0;j /= 2) {
      int ixj = threadIdx.x ^ j;
      if (ixj > threadIdx.x && ixj < blockDim.x) {
	// asc = true for ascending order
	bool asc = ((threadIdx.x & k) == 0);
	
	// Read data
	keyval_t keyval1 = sh_keyval[threadIdx.x];
	keyval_t keyval2 = sh_keyval[ixj];
	
	float lo_key = asc ? keyval1.key : keyval2.key;
	float hi_key = asc ? keyval2.key : keyval1.key;
	
	if (lo_key > hi_key) {
	  // keys are in wrong order => exchange
	  sh_keyval[threadIdx.x] = keyval2;
	  sh_keyval[ixj]         = keyval1;
	}
	
	//if ((i&k)==0 && get(i)>get(ixj)) exchange(i,ixj);
	//if ((i&k)!=0 && get(i)<get(ixj)) exchange(i,ixj);
      }
      __syncthreads();
    }
  }

  // sh_keyval[threadIdx.x].val gives the mapping:
  //
  // xyzq_new[threadIdx.x + col_patom0]        = xyzq[sh_keyval[threadIdx.x].val]
  // loc2glo_ind_new[threadIdx.x + col_patom0] = loc2glo_ind[sh_keyval[threadIdx.x].val]
  //

  float4 xyzq_val;
  int ind_val;
  if (threadIdx.x < n) {
    int i = sh_keyval[threadIdx.x].val;
    ind_val = loc2glo_ind[i];
    xyzq_val = xyzq[i];
  }
  __syncthreads();
  if (threadIdx.x < n) {
    int newpos = threadIdx.x + col_patom0;
    xyzq[newpos] = xyzq_val;
    loc2glo_ind[newpos] = ind_val;
  }

}

//
// Setup n_int_zone[0:7] and int_zone[0:7][0:7]
// zone ordering is: I,FZ,FY,EX,FX,EZ,EY,C = 0,...7
//
template <int tilesize>
void NeighborList<tilesize>::set_int_zone(const int *zone_patom, int *n_int_zone, int int_zone[][8]) {
  const int I=0,FZ=1,FY=2,EX=3/*,FX=4,EZ=5,EY=6,C=7*/;
  // Setup interaction order that maximizes communication-computation overlap
  const int zones[8][5] = { {I, -1, -1, -1, -1},  // I-I
			    {I, -1, -1, -1, -1},  // I-FZ
			    {I, FZ, -1, -1, -1},  // I-FY, FZ-FY
			    {I, -1, -1, -1, -1},  // I-EX
			    {I, FZ, FY, EX, -1},  // I-FX, FZ-FX, FY-FX, EX-FX
			    {I, FZ, -1, -1, -1},  // I-EZ, FZ-EZ
			    {I, FY, -1, -1, -1},  // I-EY, FY-EY
			    {I, -1, -1, -1, -1}}; // I-C
  int ncoord_zone[8];

  // ncoord_zone[izone] = number of atoms in zone "izone"
  for (int izone=0;izone < 8;izone++) {
    int nstart;
    if (izone > 0) {
      nstart = zone_patom[izone-1];
    } else {
      nstart = 0;
    }
    ncoord_zone[izone] = zone_patom[izone] - nstart;
  }

  n_int_zone_max = 0;
  for (int izone=0;izone < 8;izone++) {
    n_int_zone[izone] = 0;
    int j = 0;
    while (zones[izone][j] > -1) {
      if (ncoord_zone[zones[izone][j]] > 0) {
	int_zone[izone][n_int_zone[izone]] = zones[izone][j];
	n_int_zone[izone]++;
      }
      j++;
    }
    n_int_zone_max = max(n_int_zone_max, n_int_zone[izone]);
  }

}

//
// Setup xy-cell sizes
//
template <int tilesize>
void NeighborList<tilesize>::set_cell_sizes(const int *zone_patom,
					    const float3 *max_xyz, const float3 *min_xyz,
					    int *ncellx, int *ncelly, int *ncellz_max,
					    float *celldx, float *celldy) {

  for (int izone=0;izone < 8;izone++) {
    int nstart;
    if (izone > 0) {
      nstart = zone_patom[izone-1];
    } else {
      nstart = 0;
    }
    // ncoord_zone = number of atoms in this zone
    int ncoord_zone = zone_patom[izone] - nstart;
    if (ncoord_zone > 0) {
      // NOTE: we increase the cell sizes here by 0.001 to make sure no atoms drop outside cells
      float xsize = max_xyz[izone].x - min_xyz[izone].x + 0.001f;
      float ysize = max_xyz[izone].y - min_xyz[izone].y + 0.001f;
      float zsize = max_xyz[izone].z - min_xyz[izone].z + 0.001f;
      float delta = powf(xsize*ysize*zsize*tilesize/(float)ncoord_zone, 1.0f/3.0f);
      ncellx[izone] = max(1, (int)(xsize/delta));
      ncelly[izone] = max(1, (int)(ysize/delta));
      // Approximation for ncellz = 2 x "uniform distribution of atoms"
      ncellz_max[izone] = max(1, 2*ncoord_zone/(ncellx[izone]*ncelly[izone]*tilesize));
      celldx[izone] = xsize/(float)(ncellx[izone]);
      celldy[izone] = ysize/(float)(ncelly[izone]);
    } else {
      ncellx[izone] = 0;
      ncelly[izone] = 0;
      celldx[izone] = 1.0f;
      celldy[izone] = 1.0f;
    }
  }

  std::cout << "celldx = " << celldx[0] << " ncellx[0] = " << ncellx[0] 
	    << " xsize = " << max_xyz[0].x - min_xyz[0].x + 0.001f << std::endl;

  std::cout << "celldy = " << celldy[0] << " ncelly[0] = " << ncelly[0] 
	    << " ysize = " << max_xyz[0].y - min_xyz[0].y + 0.001f << std::endl;

}

//
// Tests for z columns
//
template <int tilesize>
bool NeighborList<tilesize>::test_z_columns(const int* zone_patom,
					    const int* ncellx, const int* ncelly,
					    const int ncol_tot,
					    const float3* min_xyz,
					    const float* celldx, const float* celldy,
					    float4* xyzq, float4* xyzq_sorted,
					    int* col_patom, int* loc2glo_ind) {

  int ncoord = zone_patom[7];
  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);
  float4 *h_xyzq_sorted = new float4[ncoord];
  copy_DtoH<float4>(xyzq_sorted, h_xyzq_sorted, ncoord);

  int *h_col_patom = new int[ncol_tot+1];
  copy_DtoH<int>(col_patom, h_col_patom, ncol_tot+1);
  int *h_loc2glo_ind = new int[ncoord];
  copy_DtoH<int>(loc2glo_ind, h_loc2glo_ind, ncoord);

  bool ok = true;

  int izone, i, j;
  float x, y, xj, yj;
  int ix, iy, ind, lo_ind, hi_ind;
  try {
    int ind0 = 0;
    for (izone=0;izone < 8;izone++) {
      int istart, iend;
      if (izone > 0) {
	istart = zone_patom[izone-1];
      } else {
	istart = 0;
      }
      iend = zone_patom[izone] - 1;
      if (iend >= istart) {
	float x0 = min_xyz[izone].x;
	float y0 = min_xyz[izone].y;
	for (i=istart;i <= iend;i++) {
	  x = h_xyzq_sorted[i].x;
	  y = h_xyzq_sorted[i].y;
	  ix = (int)((x - x0)/celldx[izone]);
	  iy = (int)((y - y0)/celldy[izone]);
	  ind = ind0 + ix + iy*ncellx[izone];
	  lo_ind = h_col_patom[ind];
	  hi_ind = h_col_patom[ind+1] - 1;
	  if (i < lo_ind || i > hi_ind) throw 1;
	}
	for (i=istart;i <= iend;i++) {
	  x = h_xyzq_sorted[i].x;
	  y = h_xyzq_sorted[i].y;
	  j = h_loc2glo_ind[i];
	  xj = h_xyzq[j].x;
	  yj = h_xyzq[j].y;
	  if (x != xj || y != yj) throw 2;
	}	
	ind0 += ncellx[izone]*ncelly[izone];
      }
    }
  }
  catch (int a) {
    std::cout << "test_z_columns FAILED at i=" << i << std::endl;
    if (a == 1) {
      std::cout << "ind, lo_ind, hi_ind = " << ind << " " << lo_ind << " " << hi_ind << std::endl;
    } else if (a == 2) {
      std::cout << "x,y   =" << x << " " << y << std::endl;
      std::cout << "xj,yj =" << xj << " " << yj << std::endl;
    }
    ok = false;
  }

  if (ok) std::cout << "test_z_columns OK" << std::endl;

  delete [] h_xyzq;
  delete [] h_xyzq_sorted;
  delete [] h_col_patom;
  delete [] h_loc2glo_ind;

  return ok;
}

//
// Tests sort
//
template <int tilesize>
bool NeighborList<tilesize>::test_sort(const int* zone_patom,
				       const int* ncellx, const int* ncelly,
				       const int ncol_tot, const int ncell_max,
				       const float3* min_xyz,
				       const float* celldx, const float* celldy,
				       float4* xyzq, float4* xyzq_sorted,
				       int* col_patom, int* cell_patom,
				       int* loc2glo_ind) {

  int ncoord = zone_patom[7];
  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);
  float4 *h_xyzq_sorted = new float4[ncoord];
  copy_DtoH<float4>(xyzq_sorted, h_xyzq_sorted, ncoord);
  int *h_col_patom = new int[ncol_tot+1];
  copy_DtoH<int>(col_patom, h_col_patom, ncol_tot+1);
  int *h_loc2glo_ind = new int[ncoord];
  copy_DtoH<int>(loc2glo_ind, h_loc2glo_ind, ncoord);
  int *h_cell_patom = new int[ncell_max];
  copy_DtoH<int>(cell_patom, h_cell_patom, ncell_max);

  bool ok = true;

  int izone, i, j, k, prev_ind;
  float x, y, z, prev_z;
  float xj, yj, zj;
  int ix, iy, ind, lo_ind, hi_ind;
  try {

    k = 0;
    for (i=1;i < ncol_tot+1;i++) {
      for (j=h_col_patom[i-1];j < h_col_patom[i];j+=32) {
	if (j != h_cell_patom[k]) throw 4;
	k++;
      }
    }

    int ind0 = 0;
    for (izone=0;izone < 8;izone++) {
      int istart, iend;
      if (izone > 0) {
	istart = zone_patom[izone-1];
      } else {
	istart = 0;
      }
      iend = zone_patom[izone] - 1;
      if (iend >= istart) {
	float x0 = min_xyz[izone].x;
	float y0 = min_xyz[izone].y;
	prev_z = min_xyz[izone].z;
	prev_ind = ind0;
	for (i=istart;i <= iend;i++) {
	  x = h_xyzq_sorted[i].x;
	  y = h_xyzq_sorted[i].y;
	  z = h_xyzq_sorted[i].z;
	  
	  ix = (int)((x - x0)/celldx[izone]);
	  iy = (int)((y - y0)/celldy[izone]);
	  ind = ind0 + ix + iy*ncellx[izone];

	  if (prev_ind != ind) {
	    prev_z = min_xyz[izone].z;
	  }

	  lo_ind = h_col_patom[ind];
	  hi_ind = h_col_patom[ind+1] - 1;
	  if (i < lo_ind || i > hi_ind) throw 1;
	  if (z < prev_z) throw 2;
	  prev_z = z;
	  prev_ind = ind;
	}

	for (i=istart;i <= iend;i++) {
	  x = h_xyzq_sorted[i].x;
	  y = h_xyzq_sorted[i].y;
	  z = h_xyzq_sorted[i].z;	  
	  j = h_loc2glo_ind[i];
	  xj = h_xyzq[j].x;
	  yj = h_xyzq[j].y;
	  zj = h_xyzq[j].z;
	  if (x != xj || y != yj || z != zj) throw 3;
	}	

	ind0 += ncellx[izone]*ncelly[izone];
      }
    }
  }
  catch (int a) {
    std::cout << "test_sort FAILED at i=" << i << std::endl;
    if (a == 1) {
      std::cout << "ind, lo_ind, hi_ind = " << ind << " " << lo_ind << " " << hi_ind << std::endl;
    } else if (a == 2) {
      std::cout << "prev_z, z = " << prev_z << " " << z << std::endl;
    } else if (a == 3) {
      std::cout << "x,y,z   =" << x << " " << y << " " << z << std::endl;
      std::cout << "xj,yj,zj=" << xj << " " << yj << " " << zj << std::endl;
    } else if (a == 4) {
      std::cout << "j,k=" << j << " " << k << "cell_patom[k]=" << h_cell_patom[k] << std::endl;
    }
    ok = false;
  }
  catch(...) {
    std::cout << "default catch" << std::endl;
  }

  if (ok) std::cout << "test_sort OK" << std::endl;

  delete [] h_xyzq;
  delete [] h_xyzq_sorted;
  delete [] h_col_patom;
  delete [] h_cell_patom;
  delete [] h_loc2glo_ind;

  return ok;
}

//
// Copies h_nlist_param (CPU) -> d_nlist_param (GPU)
//
template <int tilesize>
void NeighborList<tilesize>::set_nlist_param(hipStream_t stream) {
  cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_nlist_param), h_nlist_param, sizeof(NeighborListParam_t),
  				    0, hipMemcpyHostToDevice, stream));
}

//
// Copies d_nlist_param (GPU) -> h_nlist_param (CPU)
//
template <int tilesize>
void NeighborList<tilesize>::get_nlist_param() {
  cudaCheck(hipMemcpyFromSymbol(h_nlist_param, HIP_SYMBOL(d_nlist_param), sizeof(NeighborListParam_t),
				 0, hipMemcpyDeviceToHost));
}


//
// Sorts atoms into tiles
//
template <int tilesize>
void NeighborList<tilesize>::sort(const int *zone_patom,
				  const float3 *max_xyz, const float3 *min_xyz,
				  float4 *xyzq,
				  float4 *xyzq_sorted,
				  hipStream_t stream) {

  int n_int_zone[8], int_zone[8][8];
  int ncellx[8], ncelly[8], ncellz_max[8];
  float celldx[8], celldy[8];

  int ncoord = zone_patom[7];

  set_int_zone(zone_patom, n_int_zone, int_zone);
  set_cell_sizes(zone_patom, max_xyz, min_xyz, ncellx, ncelly, ncellz_max, celldx, celldy);

  // Setup nlist_param and copy it to GPU
  int ncol = 0;
  int max_ncellxy = 0;
  for (int izone=0;izone < 8;izone++) {
    h_nlist_param->zone_patom[izone] = zone_patom[izone];
    h_nlist_param->n_int_zone[izone] = n_int_zone[izone];
    for (int jzone=0;jzone < 8;jzone++) {
      h_nlist_param->int_zone[izone][jzone] = int_zone[izone][jzone];
    }
    h_nlist_param->ncol[izone] = ncol;
    h_nlist_param->ncellx[izone] = ncellx[izone];
    h_nlist_param->ncelly[izone] = ncelly[izone];
    max_ncellxy = max(max_ncellxy, ncellx[izone]*ncelly[izone]);
    ncol += ncellx[izone]*ncelly[izone];
    h_nlist_param->celldx[izone] = celldx[izone];
    h_nlist_param->celldy[izone] = celldy[izone];
    h_nlist_param->inv_celldx[izone] = 1.0f/celldx[izone];
    h_nlist_param->inv_celldy[izone] = 1.0f/celldy[izone];
    h_nlist_param->minxyz[izone].x = min_xyz[izone].x;
    h_nlist_param->minxyz[izone].y = min_xyz[izone].y;
    h_nlist_param->minxyz[izone].z = min_xyz[izone].z;
  }
  h_nlist_param->ncol[8] = ncol;

  set_nlist_param(stream);

  int ncol_tot = 0;
  ncell_max = 0;
  for (int izone=0;izone < 8;izone++) {
    ncol_tot += ncellx[izone]*ncelly[izone];
    ncell_max += ncellx[izone]*ncelly[izone]*ncellz_max[izone];
  }
  // NOTE: ncell_max is an approximate upper bound for the number of cells,
  //       it is possible to blow this bound, so we should check for it

  reallocate<int>(&col_natom, &col_natom_len, ncol_tot, 1.2f);
  reallocate<int>(&col_patom, &col_patom_len, ncol_tot+1, 1.2f);
  reallocate<int>(&atom_icol, &atom_icol_len, ncoord, 1.2f);
  reallocate<int>(&loc2glo_ind, &loc2glo_ind_len, ncoord, 1.2f);
  reallocate<int>(&cell_patom, &cell_patom_len, ncell_max, 1.2f);
  reallocate<int4>(&cell_xyz_zone, &cell_xyz_zone_len, ncell_max, 1.2f);
  reallocate<float>(&cell_bz, &cell_bz_len, ncell_max, 1.2f);
  reallocate<int>(&col_ncellz, &col_ncellz_len, ncol_tot, 1.2f);
  reallocate<int3>(&col_xy_zone, &col_xy_zone_len, ncol_tot, 1.2f);
  reallocate<int>(&col_cell, &col_cell_len, ncol_tot, 1.2f);

  clear_gpu_array<int>(col_natom, ncol_tot, stream);

  int nthread, nblock;

  //
  // Calculate number of atoms in each z-column (col_natom) and the column index for each atom (atom_icol)
  //
  nthread = 512;
  nblock = (ncoord-1)/nthread+1;
  calc_z_column_index_kernel<<< nblock, nthread, 0, stream >>>
    (xyzq, col_natom, atom_icol, col_xy_zone);
  cudaCheck(hipGetLastError());

  /*
  int ind0 = 0;
  for (int izone=0;izone < 8;izone++) {
    int istart, iend;
    if (izone > 0) {
      istart = zone_patom[izone-1];
    } else {
      istart = 0;
    }
    iend = zone_patom[izone] - 1;
    if (iend >= istart) {

      nthread = 512;
      nblock = (ncoord-1)/nthread+1;

      calc_z_column_index_kernel<<< nblock, nthread, 0, stream >>>
	(istart, iend, xyzq, ind0, izone, col_natom, atom_icol);
      cudaCheck(hipGetLastError());

      ind0 += ncellx[izone]*ncelly[izone];
    }
  }
  */

  /*
  thrust::device_ptr<int> col_natom_ptr(col_natom);
  thrust::device_ptr<int> col_patom_ptr(col_patom);
  thrust::exclusive_scan(col_natom_ptr, col_natom_ptr + ncol_tot, col_patom_ptr);
  clear_gpu_array<int>(col_natom, ncol_tot, stream);
  */

  //
  // Calculate positions in z columns
  // NOTE: Clears col_natom and sets (col_patom, cell_patom, col_ncellz, d_nlist_param.ncell)
  //
  std::cout << "ncol_tot = " << ncol_tot << std::endl;
  nthread = min(((ncol_tot-1)/tilesize+1)*tilesize, get_max_nthread());
  std::cout << "nthread = " << nthread << std::endl;
  //int shmem_size = nthread*sizeof(int);
  //  calc_z_column_pos_kernel<<< 1, nthread, shmem_size, stream >>>
  //    (ncol_tot, col_natom, col_patom);
  int shmem_size = nthread*sizeof(int2);
  calc_z_column_pos_kernel<tilesize> <<< 1, nthread, shmem_size, stream >>>
    (ncol_tot, col_xy_zone, col_natom, col_patom, cell_patom, col_ncellz, cell_xyz_zone,
     col_cell);

  //
  // Calculate ncellz_max[izone]
  //
  nthread = ((max_ncellxy-1)/warpsize+1)*warpsize;
  nblock = 8;
  shmem_size = nthread*sizeof(int);
  calc_ncellz_max<<< nblock, nthread, shmem_size, stream >>>(col_ncellz);

  //
  // Reorder atoms into z-columns
  // NOTE: also sets up startcell_zone[izone]
  //
  nthread = 512;
  nblock = (ncoord-1)/nthread+1;
  reorder_atoms_z_column_kernel<<< nblock, nthread, 0, stream >>>
    (ncoord, atom_icol, col_natom, col_patom, xyzq, xyzq_sorted, loc2glo_ind);
  cudaCheck(hipGetLastError());

  // Test z columns
  cudaCheck(hipDeviceSynchronize());
  test_z_columns(zone_patom, ncellx, ncelly, ncol_tot, min_xyz, celldx, celldy, xyzq, xyzq_sorted,
		 col_patom, loc2glo_ind);

  // Now sort according to z coordinate
  nthread = 512; //11*tilesize;
  nblock = ncellx[0]*ncelly[0];
  if (nthread < get_max_nthread()) {
    shmem_size = nthread*sizeof(keyval_t);
    sort_z_column_kernel<<< nblock, nthread, shmem_size, stream >>>
      (col_patom, xyzq_sorted, loc2glo_ind);
    cudaCheck(hipGetLastError());
  } else {
    std::cerr << "Neighborlist::sort, this version of sort_z_column_kernel not implemented yet"
	      << std::endl;
  }

  // Test sort
  cudaCheck(hipDeviceSynchronize());
  test_sort(zone_patom, ncellx, ncelly, ncol_tot, ncell_max,
	    min_xyz, celldx, celldy, xyzq, xyzq_sorted,
	    col_patom, cell_patom, loc2glo_ind);

  //  reorder_atoms_kernel<<< nblock, nthread, 0, stream >>>
  //    (ncoord, tilex_val, xyzq, xyzq_sorted);
  //cudaCheck(hipGetLastError());
}

//
// Calculates bounding box (bb) and cell z-boundaries (cell_bz)
// NOTE: Each thread calculates one bounding box
//
template <int tilesize>
__global__ void calc_bounding_box_kernel(const int* __restrict__ cell_patom,
					 const float4* __restrict__ xyzq,
					 bb_t* __restrict__ bb,
					 float* __restrict__ cell_bz) {

  const int icell = threadIdx.x + blockIdx.x*blockDim.x;

  if (icell < d_nlist_param.ncell) {
    int istart = cell_patom[icell];
    int iend   = cell_patom[icell+1] - 1;
    float4 xyzq_val = xyzq[istart];
    float minx = xyzq_val.x;
    float miny = xyzq_val.y;
    float minz = xyzq_val.z;
    float maxx = xyzq_val.x;
    float maxy = xyzq_val.y;
    float maxz = xyzq_val.z;

    //int ix = (int)((minx - x0)*inv_dx);
    //int iy = (int)((miny - y0)*inv_dy);

    for (int i=istart+1;i < iend;i++) {
      /*
      if (i < 0 || i >= 23558) {
	printf("ERROR i = %d\n",i);
	return;
      }
      */
      xyzq_val = xyzq[i];
      minx = min(minx, xyzq_val.x);
      miny = min(miny, xyzq_val.y);
      minz = min(minz, xyzq_val.z);
      maxx = max(maxx, xyzq_val.x);
      maxy = max(maxy, xyzq_val.y);
      maxz = max(maxz, xyzq_val.z);
    }
    // Set the cell z-boundary equal to the z-coordinate of the last atom
    cell_bz[icell] = xyzq_val.z;
    bb_t bb_val;
    bb_val.x = 0.5f*(minx + maxx);
    bb_val.y = 0.5f*(miny + maxy);
    bb_val.z = 0.5f*(minz + maxz);
    bb_val.wx = 0.5f*(maxx - minx);
    bb_val.wy = 0.5f*(maxy - miny);
    bb_val.wz = 0.5f*(maxz - minz);
    bb[icell] = bb_val;
  }

}

//#######################################################################

//
// Class creator
//
template <int tilesize>
NeighborList<tilesize>::NeighborList(int nx, int ny, int nz) {
  ni = 0;
  ntot = 0;

  tile_excl = NULL;
  tile_excl_len = 0;

  ientry = NULL;
  ientry_len = 0;

  tile_indj = NULL;
  tile_indj_len = 0;

  // Sparse
  ni_sparse = 0;
  ntot_sparse = 0;

  pairs_len = 0;
  pairs = NULL;
  
  ientry_sparse_len = 0;
  ientry_sparse = NULL;

  tile_indj_sparse_len = NULL;
  tile_indj_sparse = NULL;

  // Neighbor list building
  col_natom_len = 0;
  col_natom = NULL;

  col_patom_len = 0;
  col_patom = NULL;

  atom_icol_len = 0;
  atom_icol = NULL;

  col_ncellz_len = 0;
  col_ncellz = NULL;

  col_xy_zone_len = 0;
  col_xy_zone = NULL;

  col_cell_len = 0;
  col_cell = NULL;

  loc2glo_ind_len = 0;
  loc2glo_ind = NULL;
  
  cell_patom_len = 0;
  cell_patom = NULL;

  cell_xyz_zone_len = 0;
  cell_xyz_zone = NULL;

  cell_bz_len = 0;
  cell_bz = NULL;

  bb_len = 0;
  bb = NULL;

  allocate_host<NeighborListParam_t>(&h_nlist_param, 1);

  h_nlist_param->imx_lo = 0;
  h_nlist_param->imx_hi = 0;
  h_nlist_param->imy_lo = 0;
  h_nlist_param->imy_hi = 0;
  h_nlist_param->imz_lo = 0;
  h_nlist_param->imz_hi = 0;
  if (nx == 1) {
    h_nlist_param->imx_lo = -1;
    h_nlist_param->imx_hi = 1;
  }
  if (ny == 1) {
    h_nlist_param->imy_lo = -1;
    h_nlist_param->imy_hi = 1;
  }
  if (nz == 1) {
    h_nlist_param->imz_lo = -1;
    h_nlist_param->imz_hi = 1;
  }

}

//
// Class destructor
//
template <int tilesize>
NeighborList<tilesize>::~NeighborList() {
  if (tile_excl != NULL) deallocate< tile_excl_t<tilesize> > (&tile_excl);
  if (ientry != NULL) deallocate<ientry_t>(&ientry);
  if (tile_indj != NULL) deallocate<int>(&tile_indj);
  // Sparse
  if (pairs != NULL) deallocate< pairs_t<tilesize> > (&pairs);
  if (ientry_sparse != NULL) deallocate<ientry_t>(&ientry_sparse);
  if (tile_indj_sparse != NULL) deallocate<int>(&tile_indj_sparse);
  // Neighbor list building
  if (col_natom != NULL) deallocate<int>(&col_natom);
  if (col_patom != NULL) deallocate<int>(&col_patom);
  if (atom_icol != NULL) deallocate<int>(&atom_icol);
  if (loc2glo_ind != NULL) deallocate<int>(&loc2glo_ind);
  if (cell_patom != NULL) deallocate<int>(&cell_patom);
  if (col_ncellz != NULL) deallocate<int>(&col_ncellz);
  if (col_xy_zone != NULL) deallocate<int3>(&col_xy_zone);
  if (col_cell != NULL) deallocate<int>(&col_cell);
  if (cell_xyz_zone != NULL) deallocate<int4>(&cell_xyz_zone);
  if (cell_bz != NULL) deallocate<float>(&cell_bz);
  if (bb != NULL) deallocate<bb_t>(&bb);
  deallocate_host<NeighborListParam_t>(&h_nlist_param);
}

//
// Sets ientry from host memory array
//
template <int tilesize>
void NeighborList<tilesize>::set_ientry(int ni, ientry_t *h_ientry, hipStream_t stream) {

  this->ni = ni;

  // Allocate & reallocate d_ientry
  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.4f);

  // Copy to device
  copy_HtoD<ientry_t>(h_ientry, ientry, ni, stream);
}

//----------------------------------------------------------------------------------------
//
// Builds neighborlist
//


//
// The entire warp enters here
// If IvsI = true, search within I zone
//
template <bool IvsI>
__device__
void get_cell_bounds_z(const int izone, const int jzone, const int icell, const int ncell,
		       const float x0, const float x1, const float* __restrict__ bx,
		       const float roff, int& jcell0, int& jcell1) {

  int jcell_start_left, jcell_start_right;

  if (IvsI) {
    // Search within a single zone (I)
    if (icell < 0) {
      // This is one of the image cells on the left =>
      // set the left cell boundary (jcell0) to 1 and start looking for the right
      // boundary from 1
      jcell_start_left = -1;         // with this value, we don't look for cells on the left
      jcell_start_right = 0;         // start looking for cells at right from 1
      jcell0 = 0;                    // left boundary set to minimum value
      jcell1 = -1;                   // set to "no cells" value
      //      dist[1] = 0.0f;
    } else if (icell >= ncell) {
      // This is one of the image cells on the right =>
      // set the right cell boundary (icell1) to ncell and start looking for the left
      // boundary from ncell
      jcell_start_left = ncell-1;    // start looking for cells at left from ncell
      jcell_start_right = ncell;     // with this value, we don't look for cells on the right
      jcell0 = ncell;                // set to "no cells" value
      jcell1 = ncell-1;              // right boundary set to maximum value
      //      dist[ncell] = 0.0f;
    } else {
      jcell_start_left = icell - 1;
      jcell_start_right = icell + 1;
      jcell0 = icell;
      jcell1 = icell;
      //      dist[icell] = 0.0f;
    }
  } else {
    /*
    // Search between two different zones
    if (bx(0) >= x1 || (bx(0) < x1 && bx(0) > x0)) {
      // j-zone is to the right of i-zone
      // => no left search, start right search from 1
      jcell_start_left = 0;
      jcell_start_right = 1;
      jcell0 = 1;
      jcell1 = 0;
    } else if (bx[ncell] <= x0 || (bx[ncell] > x0 && bx[ncell] < x1)) {
      // j-zone is to the left of i-zone
      // => no right search, start left search from ncell
      jcell_start_left = ncell;
      jcell_start_right = ncell + 1;
      jcell0 = ncell + 1;
      jcell1 = ncell;
    } else {
      // i-zone is between j-zones
      // => safe choice is to search the entire range
      jcell_start_left = ncell;
      jcell_start_right = 1;
      jcell0 = ncell;
      jcell1 = 1;
    }
    */
  }

  //
  // Check cells at left, stop once the distance to the cell right boundary
  // is greater than the cutoff.
  //
  // Cell right boundary is at bx[i]
  //
  for (int j=jcell_start_left;j >= 0;j--) {
    float d = x0 - bx[j];
    if (d > roff) break;
    //dist[j] = max(0.0f, d);
    jcell0 = j;
  }

  //
  // Check cells at right, stop once the distance to the cell left boundary
  // is greater than the cutoff.
  //
  // Cell left boundary is at bx[i-1]
  //
  for (int j=jcell_start_right;j < ncell;j++) {
    float bx_j = (j > 0) ? bx[j-1] : d_nlist_param.minxyz[jzone].z;
    float d = bx_j - x1;
    if (d > roff) break;
    //dist[j] = max(0.0f, d);
    jcell1 = j;
  }

  // Cell bounds are jcell0:jcell1
      
}

//
// The entire warp enters here
// If IvsI = true, search within I zone
//
template <bool IvsI>
__device__
void get_cell_bounds_xy(const int izone, const int jzone, const int icell,
			const int ncell,
			const float x0, const float x1,
			const float dx, const float roff,
			int& jcell0, int& jcell1) {

  if (IvsI) {
    // Search within a single zone (I)

    //
    // Check cells at left, stop once the distance to the cell right boundary 
    // is greater than the cutoff.
    //
    // Cell right boundary is at bx
    // portion inside i-cell is (x0-bx)
    // => what is left of roff on the left of i-cell is roff-(x0-bx)
    //
    float bx = d_nlist_param.minxyz[0].x + icell*dx;
    jcell0 = max(0, icell - (int)ceilf((roff - (x0 - bx))/dx));

    //
    // Check cells at right, stop once the distance to the cell left boundary
    // is greater than the cutoff.
    //
    // Cell left boundary is at bx
    // portion inside i-cell is (bx-x1)
    // => what is left of roff on the right of i-cell is roff-(bx-x1)
    //
    bx = d_nlist_param.minxyz[0].x + (icell+1)*dx;
    jcell1 = min(ncell-1, icell + (int)ceilf((roff - (bx - x1))/dx));

    //
    // Take care of the boundaries:
    //
    if (icell < 0) jcell0 = 0;
    if (icell >= ncell) jcell1 = ncell - 1;

    /*
    if (icell < 0) {
      // This is one of the image cells on the left =>
      // set the left cell boundary (jcell0) to 1 and start looking for the right
      // boundary from 1
      jcell_start_left = 0;       // with this value, we don't look for cells on the left
      jcell_start_right = 1;      // start looking for cells at right from 1
      jcell0 = 1;                  // left boundary set to minimum value
      jcell1 = 0;                    // set to "no cells" value
      dist[1] = 0.0f;
    } else if (icell >= ncell) {
      // This is one of the image cells on the right =>
      // set the right cell boundary (icell1) to ncell and start looking for the left
      // boundary from ncell
      jcell_start_left = ncell;      // start looking for cells at left from ncell
      jcell_start_right = ncell + 1; // with this value, we don't look for cells on the right
      jcell0 = ncell + 1;            // set to "no cells" value
      jcell1 = ncell;                // right boundary set to maximum value
      dist[ncell] = 0.0f;
    } else {
      jcell_start_left = icell - 1;
      jcell_start_right = icell + 1;
      jcell0 = icell;
      jcell1 = icell;
      dist[icell] = 0.0f;
    }
    */

  } else {
    /*
    if (bx(0) >= x1 || (bx(0) < x1 && bx(0) > x0)) {
      // j-zone is to the right of i-zone
      // => no left search, start right search from 1
      jcell_start_left = 0;
      jcell_start_right = 1;
      jcell0 = 1;
      jcell1 = 0;
    } else if (bx[ncell] <= x0 || (bx[ncell] > x0 && bx[ncell] < x1)) {
      // j-zone is to the left of i-zone
      // => no right search, start left search from ncell
      jcell_start_left = ncell;
      jcell_start_right = ncell + 1;
      jcell0 = ncell + 1;
      jcell1 = ncell;
    } else {
      // i-zone is between j-zones
      // => safe choice is to search the entire range
      jcell_start_left = ncell;
      jcell_start_right = 1;
      jcell0 = ncell;
      jcell1 = 1;
    }
    */
  }


  // Cell bounds are jcell0:jcell1
      
}

//
// Finds minimum of z0 and maximum of z1 across warp using __shfl -command
//
__forceinline__ __device__ void minmax_shfl(int z0, int z1, int &z0_min, int &z1_max) {
#if __CUDA_ARCH__ >= 300
  for (int i=16;i >= 1;i/=2) {
    z0_min = min(z0_min, __shfl_xor(z0, i));
    z1_max = max(z1_max, __shfl_xor(z1, i));
  }
#endif
}

//
// Calculates exclusive plus-scan across warp for binary (0 or 1) values
//
// wid = warp ID = threadIdx.x % warpsize
//
__forceinline__ __device__ int binary_scan(int val, int wid) {
  return __popc( __ballot(val) & ((1 << wid) - 1) );
}

//
// Calculates reduction across warp for binary (0 or 1) values
//
__forceinline__ __device__ int binary_reduce(int val) {
  return __popc(__ballot(val));
}

//
// Calculates distance exclusion mask using a single warp
//
// exclusion bits:
// 0 = no exclusion
// 1 = exclusion
//
// wid = warp thread index (0...warpSize-1)
//
template <int tilesize>
__device__ int get_dist_excl_mask(const int wid, const int icell, const int jcell,
				  const int ish,
				  const float boxx, const float boxy, const float boxz,
				  const float roff2,
				  const float4* __restrict__ xyzq,
				  const int* __restrict__ cell_patom,
				  volatile float3* __restrict__ sh_xyzi
				  ) {

  int istart = cell_patom[icell] - 1;
  int iend   = cell_patom[icell+1] - 2;

  int jstart = cell_patom[jcell] - 1;
  int jend   = cell_patom[jcell+1] - 2;

  // Load atom i coordinates to shared memory
  // NOTE: volatile -keyword 
  float4 xyzq_i;

  const unsigned int load_ij = threadIdx.x % tilesize;

  if (tilesize == 32 || wid < 16) {
    if (istart + load_ij <= iend) {
      xyzq_i = xyzq[istart + load_ij];
    } else {
      xyzq_i.x = -100000000.0f;
      xyzq_i.y = -100000000.0f;
      xyzq_i.z = -100000000.0f;
    }
    sh_xyzi[load_ij].x = xyzq_i.x;
    sh_xyzi[load_ij].y = xyzq_i.y;
    sh_xyzi[load_ij].z = xyzq_i.z;
  }

  // Load atom j coordinates
  float xj, yj, zj;
  //  const unsigned int loadj = (wid + (wid/TILESIZE)*(TILESIZE-1)) % TILESIZE;
  //  const unsigned int loadj = threadIdx.x % TILESIZE;
  if (jstart + load_ij <= jend) {
    float4 xyzq_j = xyzq[jstart + load_ij];
    xj = xyzq_j.x;
    yj = xyzq_j.y;
    zj = xyzq_j.z;
  } else {
    xj = 100000000.0f;
    yj = 100000000.0f;
    zj = 100000000.0f;
  }

  // Calculate shift
  int ish_t = ish;
  float zsh = (ish_t/9 - 1)*boxz;
  ish_t -= (ish_t/9)*9;
  float ysh = (ish_t/3 - 1)*boxy;
  ish_t -= (ish_t/3)*3;
  float xsh = (ish_t - 1)*boxx;

  xj -= xsh;
  yj -= ysh;
  zj -= zsh;
  
  int q_samecell = (icell == jcell);

  unsigned int excl = 0;
  int t;
  if (tilesize == 32) {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int i = ((threadIdx.x + t) % tilesize);
      float dx = sh_xyzi[i].x - xj;
      float dy = sh_xyzi[i].y - yj;
      float dz = sh_xyzi[i].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (wid <= i)) ) << t;
    }

  } else {

    /*
    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int load_i = (wid + t*2 + (wid/tilesize)*(tilesize-1)) % tilesize;
      int ii = sh_start + load_i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (load_ij <= load_i)) ) << t;
    }
    // excl is a 8 bit exclusion mask.
    // The full 32 bit exclusion mask is contained in 4 threads:
    // thread 0 contains the lowest 8 bits
    // thread 1 contains the next 8 bits, etc..
    
    excl <<= (threadIdx.x % num_thread_per_excl)*(num_excl<tilesize>::val);
    
    // Combine excl using shared memory
    const unsigned int sh_excl_ind = (threadIdx.x/warpsize)*(num_excl<tilesize>::val) + 
      (threadIdx.x % warpsize)/num_thread_per_excl;
    
    sh_excl[sh_excl_ind] = 0;
    __syncthreads();
    
    atomicOr(&sh_excl[sh_excl_ind], excl);
    
    // Make sure shared memory is written
    __syncthreads();
    
    // index to tile_excl.excl[] (0...7)
    const unsigned int excl_ind = (threadIdx.x % warpsize)/num_thread_per_excl;
    
    tile_indj[wid] = jstart;
    
    if ((threadIdx.x % num_thread_per_excl) == 0) {
      tile_excl[wid].excl[excl_ind] = sh_excl[sh_excl_ind];
    }
    */
  }

  return excl;
}

//
// Builds top_excl_pos[] and top_excl[] for local coordinate indexing
//
__global__ void build_local_top_excl(const int ncoord,
				     const int* __restrict__ loc2glo_ind,
				     const int* __restrict__ glo_top_excl_pos,
				     const int* __restrict__ glo_top_excl,
				     int* __restrict__ top_excl_pos,
				     int* __restrict__ top_excl) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < ncoord) {
    int j = loc2glo_ind[i];
    int jstart = glo_top_excl_pos[j];
    int jend   = glo_top_excl_pos[j+1] - 1;
    for (int j=jstart;j <= jend;j++) {
      int k = glo_top_excl[j];
      
    }
  }
}

//
// Returns topological exclusion mask
//
// NOTE: top_excl_pos[] and top_excl[] are in local coordinates
//
template <int tilesize>
__device__ int get_top_excl_mask(const int wid, const int icell, const int jcell,
				 const int* __restrict__ cell_patom,
				 const int* __restrict__ loc2glo_ind,
				 const int* __restrict__ top_excl_pos) {

  int istart = cell_patom[icell] - 1;
  int iend   = cell_patom[icell+1] - 2;

  int jstart = cell_patom[jcell] - 1;
  int jend   = cell_patom[jcell+1] - 2;

  int i = loc2glo_ind[istart + wid];
  int excl_start = top_excl_pos[i];
  int excl_end   = top_excl_pos[i+1] - 1;

  for (int excl_i = excl_start;excl_i <= excl_end;excl_i++) {
    int j = top_excl[excl_i];
  }

  return excl;
}

const int n_jlist_max = 100;

//
// Build neighborlist for one zone at the time
// One warp takes care of one cell
//
template < int tilesize, bool IvsI >
__global__
void build_kernel(const int4* __restrict__ cell_xyz_zone,
		  const int* __restrict__ col_ncellz,
		  const int* __restrict__ col_cell,
		  const float* __restrict__ cell_bz,
		  const int* __restrict__ cell_patom,
		  const float4* __restrict__ xyzq,
		  const float boxx, const float boxy, const float boxz,
		  const float roff, const float roff2,
		  const bb_t* __restrict__ bb) {

  // Shared memory
  extern __shared__ char shbuf[];

  // Index of the i-cell
  const int icell = (threadIdx.x + blockIdx.x*blockDim.x)/warpsize;

  if (icell >= d_nlist_param.ncell) return;

  // Warp index
  const int wid = threadIdx.x % warpsize;

  // Get (icellx, icelly, icellz, izone):
  int4 icell_xyz_zone = cell_xyz_zone[icell];
  int icellx = icell_xyz_zone.x;
  int icelly = icell_xyz_zone.y;
  int icellz = icell_xyz_zone.z;
  int izone  = IvsI ? 0 : icell_xyz_zone.w;

  int n_jzone = IvsI ? 1 : d_nlist_param.n_int_zone[izone];
  
  if (n_jzone == 0) return;

  // Load bounding box
  bb_t ibb = bb[icell];

  // ----------------------------------------------------------------
  // Calculate shared memory pointers:
  //
  // Total memory requirement:
  // (blockDim.x/warpsize)*( (~IvsI)*n_jzone*sizeof(int4) + n_jlist_max*sizeof(int) 
  //                         + tilesize*sizeof(float3))
  //
  // Required space:
  // jcellxy: (blockDim.x/warpsize)*n_jzone*sizeof(int4)
  // NOTE: Each warp has its own jcellxy[]
  volatile int4 *sh_jcellxy = (int4 *)&shbuf[(threadIdx.x/warpsize)*n_jzone*sizeof(int4)];
  int shbuf_pos;
  if (IvsI) {
    shbuf_pos = 0;
  } else {
    shbuf_pos = (blockDim.x/warpsize)*n_jzone*sizeof(int4);
  }

  // Temporary j-cell list. Each warp has its own jlist
  // sh_jlist: (blockDim.x/warpsize)*n_jlist_max*sizeof(int)
  volatile int *sh_jlist = (int *)&shbuf[shbuf_pos + (threadIdx.x/warpsize)*n_jlist_max*sizeof(int)];
  shbuf_pos += (blockDim.x/warpsize)*n_jlist_max*sizeof(int);

  // i-cell coordinates (x, y, z)
  // sh_xyzi: (blockDim.x/warpsize)*tilesize*sizeof(float3)
  volatile float3* sh_xyzi = (float3 *)&shbuf[shbuf_pos + 
					      (threadIdx.x/warpsize)*tilesize*sizeof(float3)];
  // ----------------------------------------------------------------

  for (int imx=d_nlist_param.imx_lo;imx <= d_nlist_param.imx_hi;imx++) {
    float imbbx0 = ibb.x + imx*boxx;
    int n_jcellx = 0;
    int jcellx_min, jcellx_max;
    if (IvsI) {
      get_cell_bounds_xy<IvsI>(0, 0, icellx + imx*d_nlist_param.ncellx[0],
			       d_nlist_param.ncellx[0], imbbx0-ibb.wx, imbbx0+ibb.wx,
			       d_nlist_param.celldx[0], roff, jcellx_min, jcellx_max);
      n_jcellx = max(0, jcellx_max - jcellx_min + 1);
      if (n_jcellx == 0) continue;
    } else {
      if (wid < n_jzone) {
	int jzone = d_nlist_param.int_zone[izone][wid];
	int jcellx0_t, jcellx1_t;
	get_cell_bounds_xy<IvsI>(izone, jzone, icellx + imx*d_nlist_param.ncellx[izone],
				 d_nlist_param.ncellx[jzone], imbbx0-ibb.wx, imbbx0+ibb.wx,
				 d_nlist_param.celldx[jzone], roff, jcellx0_t, jcellx1_t);
	n_jcellx = max(0, jcellx1_t-jcellx0_t+1);
	sh_jcellxy[wid].x = jcellx0_t;
	sh_jcellxy[wid].y = jcellx1_t;
      }
      if (__all(n_jcellx == 0)) continue;
    }
    
    for (int imy=d_nlist_param.imy_lo;imy <= d_nlist_param.imy_hi;imy++) {
      float imbby0 = ibb.y + imy*boxy;
      int n_jcelly = 0;
      int jcelly_min, jcelly_max;
      if (IvsI) {
	get_cell_bounds_xy<IvsI>(0, 0, icelly + imy*d_nlist_param.ncelly[0],
				 d_nlist_param.ncelly[0], imbby0-ibb.wy, imbby0+ibb.wy,
				 d_nlist_param.celldy[0], roff, jcelly_min, jcelly_max);
	n_jcelly = max(0, jcelly_max - jcelly_min + 1);
	if (n_jcelly == 0) continue;
      } else {
	if (wid < n_jzone) {
	  int jzone = IvsI ? 0 : d_nlist_param.int_zone[izone][wid];
	  int jcelly0_t, jcelly1_t;
	  get_cell_bounds_xy<IvsI>(izone, jzone, icelly + imy*d_nlist_param.ncelly[izone],
				   d_nlist_param.ncelly[jzone], imbby0-ibb.wy, imbby0+ibb.wy,
				   d_nlist_param.celldy[jzone], roff, jcelly0_t, jcelly1_t);
	  n_jcelly = max(0, jcelly1_t-jcelly0_t+1);
	  sh_jcellxy[wid].z = jcelly0_t;
	  sh_jcellxy[wid].w = jcelly1_t;
	}
	if (__all(n_jcelly == 0)) continue;
      }

      for (int imz=d_nlist_param.imz_lo;imz <= d_nlist_param.imz_hi;imz++) {
	float imbbz0 = ibb.z + imz*boxz;
	int ish = imx+1 + 3*(imy+1 + 3*(imz+1));

	int n_jlist = 0;

	if (IvsI) {
	  int total_xy = n_jcellx*n_jcelly;
	  if (threadIdx.x == 0 && icell == 0 && imz == 0) {
	    printf("%d %d %d icell = %d total_xy = %d\n",imx,imy,imz,icell,total_xy);
	  }
	  int jcellz_min=1000000, jcellz_max=0;
	  for (int ibase=0;ibase < total_xy;ibase+=warpsize) {
	    int i = ibase + wid;
	    int jcellz0_t=1000000, jcellz1_t=0;
	    if (i < total_xy) {
	      int jcelly = i/n_jcellx;
	      int jcellx = i - jcelly*n_jcellx;
	      jcellx += jcellx_min;
	      jcelly += jcelly_min;
	      int jcol = jcellx + jcelly*d_nlist_param.ncellx[0];
	      int cell0 = col_cell[jcol];
	      get_cell_bounds_z<IvsI>(0, 0, icellz + imz*col_ncellz[jcol],
				      col_ncellz[jcol], imbbz0-ibb.wz, imbbz0+ibb.wz,
				      &cell_bz[cell0], roff, jcellz0_t, jcellz1_t);
	      if (icell == 0 && imx == 0 && imy == 0 && imz == 0) {
		printf("jcell %d %d %d %d, %f %f cell_bz = %f %f %f\n",
		       jcellx,jcelly,jcellz0_t,jcellz1_t,imbbz0-ibb.wz, imbbz0+ibb.wz,
		       d_nlist_param.minxyz[0].z,
		       cell_bz[cell0],cell_bz[cell0+1]);
	      }
	    }
#if __CUDA_ARCH__ < 300
	    printf("build_kernel: minmax_shfl not implemented for __CUDA_ARCH__ < 300\n");
#else
	    minmax_shfl(jcellz0_t, jcellz1_t, jcellz_min, jcellz_max);
#endif
	  }

	  int n_jcellz = jcellz_max - jcellz_min + 1;
	  int total_xyz = n_jcellx*n_jcelly*n_jcellz;
	  if (threadIdx.x == 0 && icell == 0 && imz == 0) {
	    printf("%d %d %d icell = %d total_xyz = %d jcellz = %d %d\n",imx,imy,imz,icell,
		   total_xyz,jcellz_min,jcellz_max);
	  }
	  //
	  // Final loop that goes through the cells
	  //
	  // Cells are ordered in (y, x, z). (i.e. z first, x second, y third)
	  //
	  for (int ibase=0;ibase < total_xyz;ibase+=warpsize) {
	    int i = ibase + wid;
	    int ok = 0;
	    int jcell;
	    if (i < total_xyz) {
	      // Calculate (jcellx, jcelly, jcellz)
	      int it = i;	    
	      int jcelly = it/(n_jcellx*n_jcellz);
	      it -= jcelly*(n_jcellx*n_jcellz);
	      int jcellx = it/n_jcellz;
	      int jcellz = it - jcellx*n_jcellz;
	      jcellx += jcellx_min;
	      jcelly += jcelly_min;
	      jcellz += jcellz_min;
	      // Calculate column index "jcol" and final cell index "jcell"
	      int jcol = jcellx + jcelly*d_nlist_param.ncellx[0];
	      jcell = col_cell[jcol] + jcellz;
	      if (icell <= jcell) {
		// Read bounding box for j-cell
		bb_t jbb = bb[jcell];
		// Calculate distance between i-cell and j-cell bounding boxes
		float dx = max(0.0f, fabsf(imbbx0 - jbb.x) - ibb.wx - jbb.wx);
		float dy = max(0.0f, fabsf(imbby0 - jbb.y) - ibb.wy - jbb.wy);
		float dz = max(0.0f, fabsf(imbbz0 - jbb.z) - ibb.wz - jbb.wz);
		float r2 = dx*dx + dy*dy + dz*dz;
		if (r2 < roff2) {
		  if (icell == 0 && imx == 0 && imy == 0 && imz == 0) {
		    printf("jcell = %d, jcellxyz = %d %d %d r2 = %f %f %f %f\n",
			   jcell,jcellx,jcelly,jcellz,r2,dx,dy,dz);
		  }
		  ok = 1;
		}
	      }
	    } // if (i < total_xyz)
	    //
	    // Add j-cells into temporary list (in shared memory)
	    //
	    // First reduce to calculate position for each thread in warp
	    int pos = binary_scan(ok, wid);
	    if (ok) sh_jlist[n_jlist + pos] = jcell;
	    n_jlist += binary_reduce(ok);
	  }

	  // Calculate exclusion mask (=check for distance and topological exclusions)
	  for (int i=0;i < n_jlist;i++) {
	    int jcell = sh_jlist[i];
	    int excl = get_dist_excl_mask<tilesize>(wid, icell, jcell, ish, 
						    boxx, boxy, boxz, roff2, xyzq,
						    cell_patom, sh_xyzi);

	    if (icell == 0 && imx == 0 && imy == 0 && imz == 0) {
	      if (__all(excl == -1)) {
		if (wid == 0) printf("jcell = %d excl = %x\n",jcell,excl);
	      }
	    }

	    int top_excl = get_top_excl_mask<tilesize>();

	  }

	} else {
	  int n_jcellx_tot = n_jcellx;
	  int n_jcelly_tot = n_jcelly;
#if __CUDA_ARCH__ < 300
	  printf("build_kernel: this part not implemented (2)\n");
#else
	  for (int i=16;i >= 1;i /= 2) {
	    n_jcellx_tot += __shfl_xor(n_jcellx_tot, i);
	    n_jcelly_tot += __shfl_xor(n_jcelly_tot, i);
	  }
#endif
	  // Total amount of work
	  int total = n_jcellx_tot*n_jcelly_tot*n_jzone;
	}

	// Flush j-cells into global memory
	// j-cell list is in sh_jlist[0...n_jlist-1].
	

	//        for (int i = wid;i < total;i += warpsize) {
	//	}

      } // for (int imz=imz_lo;imz <= imz_hi;imz++)
    } // for (int imy=imy_lo;imy <= imy_hi;imy++)
  } // for (int imx=imx_lo;imx <= imx_hi;imx++)

  /*
  // Go through xy-images
  // NOTE: maximum nimgxy = 9
  int img = wid % d_nlist_param.nimgxy; // img = 0 ... d_nlist_param.nimgxy-1

  // Per thread shared memory array
  int4 *jcellxy_t = (int4 *)&shbuf[((threadIdx.x/warpsize)*d_nlist_param.nimgxy + img)
				   *n_jzone*sizeof(int4)];

  int *n_jcellxy_t = (int *)&shbuf[blockDim.x*d_nlist_param.nimgxy*n_jzone*sizeof(int4) +
				   ((threadIdx.x/warpsize)*d_nlist_param.nimgxy + img)
				   *n_jzone*sizeof(int)];

  int2 im = d_nlist_param.imgxy[img];
  float imbbx0 = ibb.x + im.x*boxx;
  float imbby0 = ibb.y + im.y*boxy;
  int n_jcellxy = 0;
  for (int jjzone=wid/d_nlist_param.nimgxy;jjzone < n_jzone;jjzone+=d_nlist_param.nimgxy) {
    int jzone = IvsI ? 0 : d_nlist_param.int_zone[izone][jjzone];
    
    int jcellx0, jcellx1;
    get_cell_bounds_xy<IvsI>(izone, jzone, icellx + im.x*d_nlist_param.ncellx[izone],
			     d_nlist_param.ncellx[jzone], imbbx0-ibb.wx, imbbx0+ibb.wx,
			     d_nlist_param.celldx[jzone], roff, jcellx0, jcellx1);
    int n_jcellxy_add = max(0, jcellx1-jcellx0+1);
    int4 jcellxy_tmp;
    jcellxy_tmp.x = jcellx0;
    jcellxy_tmp.y = jcellx1;

    int jcelly0, jcelly1;
    get_cell_bounds_xy<IvsI>(izone, jzone, icelly + im.y*d_nlist_param.ncelly[izone],
			     d_nlist_param.ncelly[jzone], imbby0-ibb.wy, imbby0+ibb.wy,
			     d_nlist_param.celldy[jzone], roff, jcelly0, jcelly1);
    n_jcellxy_add *= max(0, jcelly1-jcelly0+1);
    jcellxy_tmp.z = jcelly0;
    jcellxy_tmp.w = jcelly1;
    
    n_jcellxy += n_jcellxy_add;

    jcellxy_t[jzone] = jcell_val;
    n_jcellxy_t[jzone] = (n_jcellxy_add == 0) ? 0 : 1;

    // Found neighboring xy-cells:
    // (jcellx0 ... jcellx1) x (jcelly0 ... jcelly1)
  }
  // Neighboring xy-cells are in jcellxy_t[0...n_jzone-1]
  // Total list of xy-cells is in jcellxy[]

  volatile int4 *jcellxy = (int *)&shbuf[(threadIdx.x/warpsize)*d_nlist_param.nimgxy
					 *n_jzone*sizeof(int4)];

  // Found n_jcellxy xy-cells. Put them into a combined list in shared memory.
  // Position in the list is calculated using prefix sum.

  int pos_jcellxy;
#if __CUDA_ARCH__ < 300
  // NOT IMPLEMENTED YET
  //sh_n_jcellxy[wid] = n_jcellxy;
#else
  pos_jcellxy = n_jcellxy;
  for (int i=0;i < warpsize;i *= 2) {
    int val = __shfl_up(pos_jcellxy, 1);
    if (wid >= i) pos_jcellxy += val;
  }
  pos_jcellxy -= n_jcellxy;
#endif
  // Now pos_jcellxy gives the position where this thread is to write
  for (int i=0;i < n_jcellxy;i++) {
    jcellxy[pos_jcellxy + i] = jcellx0[cbase];
  }

  // Total number of cells to check = n_jcellxy_tot*nimgz*n_jzone
  */

  /*
  for (int jjzone = wid/nimgz;jjzone < n_jzone;jjzone += nimgz) {
    int jzone = IvsI ? 0 : d_nlist_param.int_zone[izone][jjzone];

    if (jcelly1[jzone] >= jcelly0[jzone] && jcellx1[jzone] >= jcellx0[jzone]) {

      // Loop over j-cells
      // NOTE: we do this in order y, x, z so that the resulting tile list
      //       is ordered
      
      for (int jcelly=jcelly0[jzone]; jcelly <= jcelly1(jzone);jcelly++) {
	float celldist1 = ydist[ydist_pos + jcelly];
	celldist1 *= celldist1;
	jcellx0_t = jcellx0[jzone];
	for (int jcellx=jcellx0_t; jcellx <= jcellx1[jzone]; jcellx++) {
	  
	}
      }
    }
  }



  for (int imx=imx_lo;imx <= imx_hi;imx++) {
    float imbbx0 = ibb.x + imx*boxx;
    int n_jcellx = 0;
    for (int jjzone=0;jjzone < n_jzone;jjzone++) {
      int jzone = IvsI ? 0 : d_nlist_param.int_zone[izone][jjzone];
      int jcellx0_t, jcellx1_t;
      get_cell_bounds<IvsI>(izone, jzone, icellx + imx*ncellx[izone], ncellx[jzone],
			    imbbx0-ibb.wx, imbbx0+ibb.wx, cellbx[jzone], roff,
			    jcellx0_t, jcellx1_t);
      n_jcellx += max(0, jcellx1_t-jcellx0_t+1);
      jcellx0[jzone] = jcellx0_t;
      jcellx1[jzone] = jcellx1_t;
    }

    for (int imy=imy_lo;imy <= imy_hi;imy++) {
      float imbby0 = ibb.y + imy*boxy;
      int n_jcelly = 0;
      for (int jjzone=0;jjzone < n_jzone;jjzone++) {
	int jzone = IvsI ? 0 : d_nlist_param.int_zone[izone][jjzone];
	int jcelly0_t, jcelly1_t;
	get_cell_bounds<IvsI>(izone, jzone, icelly + imy*ncelly[izone], ncelly[jzone],
			      imbby0-ibb.wy, imbby0+ibb.wy, cellby[jzone], roff,
			      jcelly0_t, jcelly1_t);
	n_jcelly += max(0, jcelly1_t-jcelly0_t+1);
	jcelly0[jzone] = jcelly0_t;
	jcelly1[jzone] = jcelly1_t;
      }
    } // for (int imy=imy_lo;imy <= imy_hi;imy++)

    for (int imz=imz_lo;imz <= imz_hi;imz++) {
	float imbbz0 = ibb.z + imz*boxz;
	
	int ish = imx+1 + 3*(imy+1 + 3*(imz+1));
	
	for (int jjzone=0;jjzone < n_jzone;jjzone++) {
	  int jzone = IvsI ? 0 : d_nlist_param.int_zone[izone][jjzone];

	  if (jcelly1[jzone] >= jcelly0[jzone] && jcellx1[jzone] >= jcellx0[jzone]) {
	    // Loop over j-cells
	    // NOTE: we do this in order y, x, z so that the resulting tile list
	    //       is ordered
	    for (int jcelly=jcelly0[jzone]; jcelly <= jcelly1(jzone);jcelly++) {
	      float celldist1 = ydist[ydist_pos + jcelly];
	      celldist1 *= celldist1;
	      jcellx0_t = jcellx0[jzone];
	      for (int jcellx=jcellx0_t; jcellx <= jcellx1[jzone]; jcellx++) {
		float celldist2 = celldist1 + xdist[xdist_pos + jcellx];
		celldist2 *= celldist2;
		if (celldist2 > cutsq) continue;
		// Get jcellz limits (jcellz0, jcellz1)
		pos_xy = jcellx + (jcelly-1)*ncellx[jzone];
		pos_cellbz = (max_ncellz(jzone)+1)*(pos_xy - 1);
		pos_ncellz = pos_xy + startcol_zone[jzone];
		get_cell_bounds<IvsI>(izone, jzone, icellz_im,
				      ncellz[pos_ncellz], imbbz0-ibb.wz, imbbz0+ibb.wz,
				      cellbz[jzone]%array(pos_cellbz:), roff,
				      jcellz0, jcellz1, zdist);
		for (int jcellz=jcellz0; jcellz <= jcellz1; jcellz++) {
		  if (celldist2 + zdist(jcellz)**2 > cutsq) continue;
		  // j-cell index is calculated as jcellz + start of the column cells
		  jcell = jcellz + startcell_col[pos_ncellz];

		  // Read bounding box for j-cell
		  bb_t jbb = bb[jcell];
                               
		  // Calculate distance between i- and j-cell bounding boxes
		  float bbxdist = max(0.0f, fabs(imbbx0 - jbb.x) - ibb.wx - jbb.wx);
		  float bbydist = max(0.0f, fabs(imbby0 - jbb.y) - ibb.wy - jbb.wy);
		  float bbzdist = max(0.0f, fabs(imbbz0 - jbb.z) - ibb.wz - jbb.wz);

		  if (bbxdist**2 + bbydist**2 + bbzdist**2 < cutsq) {
		  }


	}
	
    } // for (int imz=imz_lo;imz <= imz_hi;imz++)


  } // for (int imx=imx_lo;imx <= imx_hi;imx++)
  */

}

//
// Builds neighborlist
//
template <int tilesize>
void NeighborList<tilesize>::build(const float boxx, const float boxy, const float boxz,
				   const float roff,
				   const float4 *xyzq,
				   hipStream_t stream) {

  get_nlist_param();
  std::cout << "ncell = " << h_nlist_param->ncell << " ncell_max = " << ncell_max << std::endl;

  reallocate<bb_t>(&bb, &bb_len, ncell_max, 1.2f);

  int nthread = 512;
  int nblock = (ncell_max-1)/nthread + 1;

  calc_bounding_box_kernel<tilesize> <<< nblock, nthread, 0, stream >>>
    (cell_patom, xyzq, bb, cell_bz);
  cudaCheck(hipGetLastError());

  // Shared memory requirements:
  // (blockDim.x/warpsize)*( (~IvsI)*n_jzone*sizeof(int4) + n_jlist_max*sizeof(int) 
  //                         + tilesize*sizeof(float3))
  int shmem_size = (nthread/warpsize)*( n_jlist_max*sizeof(int) + tilesize*sizeof(float3));
  std::cout << "NeighborList::build, shmem_size = " << shmem_size << std::endl;

  build_kernel<tilesize, true>
    <<< nblock, nthread, shmem_size, stream >>>
    (cell_xyz_zone, col_ncellz, col_cell, cell_bz, cell_patom, xyzq,
     boxx, boxy, boxz, roff, roff*roff, bb);
  cudaCheck(hipGetLastError());

  //int shmem_size = (nthread/warpsize)*n_int_zone_max*sizeof(int4);

}

//----------------------------------------------------------------------------------------
//
// Builds tilex exclusion mask from ijlist[] based on distance and index
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
// Uses 32 threads to calculate the distances for a single ijlist -entry.
//
const int nwarp_build_excl_dist = 8;

template < int tilesize >
__global__ void build_excl_kernel(const unsigned int base_tid, const int n_ijlist,
				  const int3 *ijlist,
				  const int *cell_patom, const float4 *xyzq,
				  int *tile_indj,
				  tile_excl_t<tilesize> *tile_excl,
				  const float boxx, const float boxy, const float boxz,
				  const float roff2) {
  const int num_thread_per_excl = (32/(num_excl<tilesize>::val));

  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x + base_tid;
  // Global warp index
  const unsigned int wid = gtid / warpsize;
  // Local thread index (0...warpsize-1)
  const unsigned int tid = gtid % warpsize;
  // local thread index (0...tilesize-1)
  const unsigned int stid = gtid % tilesize;

  // Shared memory
  extern __shared__ char shmem[];
  volatile float3 *sh_xyzi = (float3 *)&shmem[0];    // nwarp_build_excl_dist*tilesize
  unsigned int *sh_excl = (unsigned int *)&sh_xyzi[nwarp_build_excl_dist*tilesize];

  //  __shared__ float3 sh_xyzi[nwarp_build_excl_dist*tilesize];
  //#if (tilesize == 16)
  //  __shared__ unsigned int sh_excl[nwarp_build_excl_dist*num_excl];
  //#endif

  if (wid >= n_ijlist) return;

  // Each warp computes one ijlist entry
  int3 ijlist_val = ijlist[wid];
  int icell = ijlist_val.x - 1;
  int ish   = ijlist_val.y;
  int jcell = ijlist_val.z - 1;

  int istart = cell_patom[icell] - 1;
  int iend   = cell_patom[icell+1] - 2;

  int jstart = cell_patom[jcell] - 1;
  int jend   = cell_patom[jcell+1] - 2;

  const unsigned int load_ij = threadIdx.x % tilesize;
  const int sh_start = (threadIdx.x/warpsize)*tilesize;

  // Load atom i coordinates to shared memory
  // NOTE: volatile qualifier in "sh_xyzi" guarantees that values are actually read/written from
  //       shared memory. Therefore, no __syncthreads() is needed.
  float4 xyzq_i;

  if (tilesize == 32 || tid < 16) {
    if (istart + load_ij <= iend) {
      xyzq_i = xyzq[istart + load_ij];
    } else {
      xyzq_i.x = -100000000.0f;
      xyzq_i.y = -100000000.0f;
      xyzq_i.z = -100000000.0f;
    }
    sh_xyzi[sh_start + load_ij].x = xyzq_i.x;
    sh_xyzi[sh_start + load_ij].y = xyzq_i.y;
    sh_xyzi[sh_start + load_ij].z = xyzq_i.z;
  }

  // Load atom j coordinates
  float xj, yj, zj;
  //  const unsigned int loadj = (tid + (tid/TILESIZE)*(TILESIZE-1)) % TILESIZE;
  //  const unsigned int loadj = threadIdx.x % TILESIZE;
  if (jstart + load_ij <= jend) {
    float4 xyzq_j = xyzq[jstart + load_ij];
    xj = xyzq_j.x;
    yj = xyzq_j.y;
    zj = xyzq_j.z;
  } else {
    xj = 100000000.0f;
    yj = 100000000.0f;
    zj = 100000000.0f;
  }

  // Calculate shift
  float zsh = (ish/9 - 1)*boxz;
  ish -= (ish/9)*9;
  float ysh = (ish/3 - 1)*boxy;
  ish -= (ish/3)*3;
  float xsh = (ish - 1)*boxx;

  xj -= xsh;
  yj -= ysh;
  zj -= zsh;
  
  // Make sure shared memory has been written
  // NOTE: since we're only operating within the warp, this __syncthreads() is just to make sure
  //       all values are actually written in shared memory and not kept in registers etc.
  //__syncthreads();

  int q_samecell = (icell == jcell);

  unsigned int excl = 0;
  int t;

  if (tilesize == 32) {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int i = ((threadIdx.x + t) % tilesize);
      int ii = sh_start + i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (tid <= i)) ) << t;
    }
    tile_indj[wid] = jstart;
    tile_excl[wid].excl[stid] = excl;

  } else {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int load_i = (tid + t*2 + (tid/tilesize)*(tilesize-1)) % tilesize;
      int ii = sh_start + load_i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (load_ij <= load_i)) ) << t;
    }
    // excl is a 8 bit exclusion mask.
    // The full 32 bit exclusion mask is contained in 4 threads:
    // thread 0 contains the lowest 8 bits
    // thread 1 contains the next 8 bits, etc..
    
    excl <<= (threadIdx.x % num_thread_per_excl)*(num_excl<tilesize>::val);
    
    // Combine excl using shared memory
    const unsigned int sh_excl_ind = (threadIdx.x/warpsize)*(num_excl<tilesize>::val) + 
      (threadIdx.x % warpsize)/num_thread_per_excl;
    
    sh_excl[sh_excl_ind] = 0;
    __syncthreads();
    
    atomicOr(&sh_excl[sh_excl_ind], excl);
    
    // Make sure shared memory is written
    __syncthreads();
    
    // index to tile_excl.excl[] (0...7)
    const unsigned int excl_ind = (threadIdx.x % warpsize)/num_thread_per_excl;
    
    tile_indj[wid] = jstart;
    
    if ((threadIdx.x % num_thread_per_excl) == 0) {
      tile_excl[wid].excl[excl_ind] = sh_excl[sh_excl_ind];
    }
  }

}

/*
void test_excl_dist_index(const int n_ijlist, const int3 *d_ijlist,
			  const int *d_cell_patom, const float4 *d_xyzq,
			  int *d_tile_indj,
			  tile_excl_t *d_tile_excl,
			  const float boxx, const float boxy, const float boxz,
			  const float roff2) {

  int3 *h_ijlist;
  int *h_cell_patom;
  float4 *h_xyzq;
  int *h_tile_indj;
  tile_excl_t *h_tile_excl;

  h_ijlist = (int3 *)malloc(n_ijlist*sizeof(int3));
  h_cell_patom = (int *)malloc(mdsim.ncell*sizeof(int));
  h_xyzq = (float4 *)malloc(mdsim.ncoord*sizeof(float4));
  h_tile_indj = (int *)malloc(n_ijlist*sizeof(int));
  h_tile_excl = (tile_excl_t *)malloc(n_ijlist*sizeof(tile_excl_t));

  cudaCheck(hipMemcpy(h_ijlist, d_ijlist, sizeof(int3)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_cell_patom, d_cell_patom, sizeof(int)*mdsim.ncell,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_xyzq, d_xyzq, sizeof(float4)*mdsim.ncoord,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_indj, d_tile_indj, sizeof(int)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_excl, d_tile_excl, sizeof(tile_excl_t)*n_ijlist,
		       hipMemcpyDeviceToHost));

  for (int wid=0;wid < n_ijlist;wid++) {

    int3 ijlist_val = h_ijlist[wid];
    int icell = ijlist_val.x - 1;
    int ish   = ijlist_val.y;
    int jcell = ijlist_val.z - 1;

    int istart = h_cell_patom[icell] - 1;
    int iend   = h_cell_patom[icell+1] - 2;

    int jstart = h_cell_patom[jcell] - 1;
    int jend   = h_cell_patom[jcell+1] - 2;

    int q_samecell = (icell == jcell);

    // Calculate shift
    float zsh = (ish/9 - 1)*boxz;
    ish -= (ish/9)*9;
    float ysh = (ish/3 - 1)*boxy;
    ish -= (ish/3)*3;
    float xsh = (ish - 1)*boxx;
    
    int i,j,ii,jj;

    for (ii=istart,i=0;ii <= iend;ii++,i++) {
      float4 xyzq_i = h_xyzq[ii];
      float xi = xyzq_i.x;
      float yi = xyzq_i.y;
      float zi = xyzq_i.z;
      for (jj=jstart,j=0;jj <= jend;jj++,j++) {
	float4 xyzq_j = h_xyzq[jj];
	float xj = xyzq_j.x - xsh;
	float yj = xyzq_j.y - ysh;
	float zj = xyzq_j.z - zsh;
	float dx = xi - xj;
	float dy = yi - yj;
	float dz = zi - zj;
	float r2 = dx*dx + dy*dy + dz*dz;
#if (TILESIZE == 16)
	int ttid = ((i+j) % 2)*16 + j;
	int iexcl = ttid/4;
	int tmp = i + 1 + j*15;
	int shbit = ((tmp/2) % 8) + (j % 4)*8;
#else
	int ij = i + j*TILESIZE - j;
	int iexcl = j;
	int shbit = (ij % TILESIZE);
#endif
	unsigned int ibit = 1 << shbit;
	unsigned int excl = ((r2 >= roff2) | (q_samecell && (j <= i)) ) << shbit;
	unsigned int excl_gpu = h_tile_excl[wid].excl[iexcl];
	if ( ((excl_gpu & ibit) ^ excl) != 0 && fabsf(r2-roff2) > 7.0e-5) {
	  printf("Error found in test_excl_dist_index:\n");
	  printf("wid = %d i = %d j = %d iexcl = %d shbit = %d\n",wid,i,j,iexcl,shbit);
	  printf("ii = %d jj = %d %d %d %d %d\n",ii,jj,r2 >= roff2,
		 (q_samecell && (j <= i)),icell,jcell);
	  printf("%x ^ %x = %x \n",excl_gpu & ibit, excl, (excl_gpu & ibit) ^ excl);
	  printf("i:  %f %f %f\n",xi,yi,zi);
	  printf("j:  %f %f %f\n",xj,yj,zj);
	  printf("jo: %f %f %f\n",xyzq_j.x,xyzq_j.y,xyzq_j.z);
	  printf("sh: %f %f %f\n",xsh,ysh,zsh);
	  printf("dx: %1.8f %1.8f %1.8f\n",dx,dy,dz);
	  printf("r2: %f %e\n",r2,fabsf(r2-roff2));
	  exit(1);
	}
      }
    }

  }

  free(h_ijlist);
  free(h_cell_patom);
  free(h_xyzq);
  free(h_tile_indj);
  free(h_tile_excl);

  printf("test_excl_dist_index OK\n");
}
*/

//
// Host wrapper for build_tilex_kernel
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
template <int tilesize>
void NeighborList<tilesize>::build_excl(const float boxx, const float boxy, const float boxz,
					const float roff,
					const int n_ijlist, const int3 *ijlist,
					const int *cell_patom,
					const float4 *xyzq,
					hipStream_t stream) {

  if (n_ijlist == 0) return;

  // Allocate & re-allocate (d_tile_indj, d_tile_excl)
  reallocate<int>(&tile_indj, &tile_indj_len, n_ijlist, 1.2f);
  reallocate<tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_ijlist, 1.2f);

  float roff2 = roff*roff;

  int nthread = nwarp_build_excl_dist*warpsize;
  int nblock_tot = (n_ijlist-1)/(nthread/warpsize) + 1;
  size_t shmem_size = nwarp_build_excl_dist*tilesize*sizeof(float3); 

  if (tilesize == 16) {
    shmem_size += nwarp_build_excl_dist*(num_excl<tilesize>::val)*sizeof(unsigned int);
  }

  int3 max_nblock3 = get_max_nblock();
  unsigned int max_nblock = max_nblock3.x;
  unsigned int base_tid = 0;

  while (nblock_tot != 0) {

    int nblock = (nblock_tot > max_nblock) ? max_nblock : nblock_tot;
    nblock_tot -= nblock;

    build_excl_kernel<tilesize>
      <<< nblock, nthread, shmem_size, stream >>>
      (base_tid, n_ijlist, ijlist, cell_patom,
       xyzq, tile_indj, tile_excl,
       boxx, boxy, boxz,
       roff2);

    base_tid += nblock*nthread;

    cudaCheck(hipGetLastError());
  }

  /*
  if (mdsim.q_test != 0) {
    test_excl_dist_index(mdsim.n_ijlist, mdsim.ijlist, mdsim.cell_patom,
			 mdsim.xyzq.xyzq, mdsim.tile_indj, mdsim.tile_excl,
			 boxx, boxy, boxz,
			 roff2);
  }
  */

}

//----------------------------------------------------------------------------------------
//
// Combines tile_excl_top on GPU
// One thread takes care of one integer in the exclusion mask, therefore:
//
// 32x32 tile, 32 integers per tile
// 16x16 tile, 8 integers per tile
//
template <int tilesize>
__global__ void add_tile_top_kernel(const int ntile_top,
				    const int *tile_ind_top,
				    const tile_excl_t<tilesize> *tile_excl_top,
				    tile_excl_t<tilesize> *tile_excl) {
  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x;
  // Index to tile_ind_top[]
  const unsigned int i = gtid / (num_excl<tilesize>::val);
  // Index to exclusion mask
  const unsigned int ix = gtid % (num_excl<tilesize>::val);

  if (i < ntile_top) {
    int ind = tile_ind_top[i];
    tile_excl[ind].excl[ix] |= tile_excl_top[i].excl[ix];
  }

}

//
// Host wrapper for add_tile_top_kernel
//
template <int tilesize>
void NeighborList<tilesize>::add_tile_top(const int ntile_top, const int *tile_ind_top,
					  const tile_excl_t<tilesize> *tile_excl_top,
					  hipStream_t stream) {
  int nthread = 256;
  int nblock = (ntile_top*(num_excl<tilesize>::val) - 1)/nthread + 1;
  
  add_tile_top_kernel<tilesize>
    <<< nblock, nthread, 0, stream >>>
    (ntile_top, tile_ind_top, tile_excl_top, tile_excl);
  
  cudaCheck(hipGetLastError());
}

static int BitCount(unsigned int u)
 {
         unsigned int uCount;

         uCount = u
                  - ((u >> 1) & 033333333333)
                  - ((u >> 2) & 011111111111);
         return
           ((uCount + (uCount >> 3))
            & 030707070707) % 63;
 }

/*
static int BitCount_ref(unsigned int u) {
  unsigned int x = u;
  int res = 0;
  while (x != 0) {
    res += (x & 1);
    x >>= 1;
  }
  return res;
}
*/

//
// Splits neighbor list into dense and sparse parts
//
template <int tilesize>
void NeighborList<tilesize>::split_dense_sparse(int npair_cutoff) {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_dense = new ientry_t[ni];
  int *h_tile_indj_dense = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl_dense = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_sparse = new ientry_t[ni];
  int *h_tile_indj_sparse = new int[ntot];
  pairs_t<tilesize> *h_pairs = new pairs_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  int ni_dense = 0;
  int ntot_dense = 0;
  ni_sparse = 0;
  ntot_sparse = 0;
  for (int i=0;i < ni;i++) {
    bool sparse_i_tiles = true;
    int startj_dense = ntot_dense;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	npair += 32 - n1bit;
      }

      if (npair <= npair_cutoff) {
	// Sparse
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  
	}
	h_tile_indj_sparse[ntot_sparse] = h_tile_indj[j];
	ntot_sparse++;
      } else {
	// Dense
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_dense[ntot_dense].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_dense[ntot_dense] = h_tile_indj[j];
	ntot_dense++;
	sparse_i_tiles = false;
      }

    }

    if (sparse_i_tiles) {
      // Sparse
    } else {
      h_ientry_dense[ni_dense] = h_ientry[i];
      h_ientry_dense[ni_dense].startj = startj_dense;
      h_ientry_dense[ni_dense].endj = ntot_dense - 1;
      ni_dense++;
    }
  }

  ni = ni_dense;
  ntot = ntot_dense;

  copy_HtoD<ientry_t>(h_ientry_dense, ientry, ni);
  copy_HtoD<int>(h_tile_indj_dense, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl_dense, tile_excl, ntot);

  allocate<ientry_t>(&ientry_sparse, ni_sparse);
  allocate<int>(&tile_indj_sparse, ntot_sparse);
  allocate< pairs_t<tilesize> >(&pairs, ntot_sparse);
  ientry_sparse_len = ni_sparse;
  tile_indj_sparse_len = ntot_sparse;
  pairs_len = ntot_sparse;

  copy_HtoD<ientry_t>(h_ientry_sparse, ientry_sparse, ni_sparse);
  copy_HtoD<int>(h_tile_indj_sparse, tile_indj_sparse, ntot_sparse);
  copy_HtoD< pairs_t<tilesize> >(h_pairs, pairs, ntot_sparse);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_dense;
  delete [] h_tile_indj_dense;
  delete [] h_tile_excl_dense;

  delete [] h_ientry_sparse;
  delete [] h_tile_indj_sparse;
  delete [] h_pairs;

}

//
// Removes empty tiles
//
template <int tilesize>
void NeighborList<tilesize>::remove_empty_tiles() {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_noempty = new ientry_t[ni];
  int *h_tile_indj_noempty = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl_noempty = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  int ni_noempty = 0;
  int ntot_noempty = 0;
  for (int i=0;i < ni;i++) {
    bool empty_i_tiles = true;
    int startj_noempty = ntot_noempty;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	if (n1bit != 32) empty_tile = false;
      }

      if (!empty_tile) {
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_noempty[ntot_noempty].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_noempty[ntot_noempty] = h_tile_indj[j];
	ntot_noempty++;
	empty_i_tiles = false;
      }
    }

    if (!empty_i_tiles) {
      h_ientry_noempty[ni_noempty] = h_ientry[i];
      h_ientry_noempty[ni_noempty].startj = startj_noempty;
      h_ientry_noempty[ni_noempty].endj = ntot_noempty - 1;
      ni_noempty++;
    }
  }

  ni = ni_noempty;
  ntot = ntot_noempty;

  copy_HtoD<ientry_t>(h_ientry_noempty, ientry, ni);
  copy_HtoD<int>(h_tile_indj_noempty, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl_noempty, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_noempty;
  delete [] h_tile_indj_noempty;
  delete [] h_tile_excl_noempty;

}

//
// Analyzes the neighbor list and prints info
//
template <int tilesize>
void NeighborList<tilesize>::analyze() {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  std::cout << "Number of i-tiles = " << ni << ", total number of tiles = " << ntot << std::endl;

  std::ofstream file_npair("npair.txt", std::ofstream::out);
  std::ofstream file_nj("nj.txt", std::ofstream::out);

  unsigned int nexcl_bit = 0;
  unsigned int nexcl_bit_self = 0;
  unsigned int nempty_tile = 0;
  unsigned int nempty_line = 0;
  for (int i=0;i < ni;i++) {
    file_nj << h_ientry[i].endj - h_ientry[i].startj + 1 << std::endl;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);

	if (n1bit > 32) {
	  std::cerr << n1bit << " " << std::hex << h_tile_excl[j].excl[k] << std::endl;
	  exit(1);
	}

	if (n1bit == 32)
	  nempty_line++;
	else
	  empty_tile = false;

	nexcl_bit += n1bit;
	npair += 32 - n1bit;

	if (h_ientry[i].indi == h_tile_indj[j]) nexcl_bit_self += n1bit;
      }
      if (empty_tile) nempty_tile++;
      file_npair << npair << std::endl;
    }
  }

  file_npair.close();
  file_nj.close();

  unsigned int ntot_pairs = ntot*tilesize*tilesize;
  std::cout << "Total number of pairs = " << ntot_pairs << std::endl;
  std::cout << "Number of excluded pairs = " << nexcl_bit << " (" << 
    ((double)nexcl_bit*100)/(double)ntot_pairs << "%)" << std::endl;
  std::cout << "Number of excluded pairs in self (i==j) tiles = " << nexcl_bit_self << " (" << 
    ((double)nexcl_bit_self*100)/(double)ntot_pairs << "%)" << std::endl;
  std::cout << "Number of empty lines = " << nempty_line << " (" <<
    ((double)nempty_line*100)/((double)(ntot*tilesize)) << "%)" << std::endl;
  std::cout << "Number of empty tiles = " << nempty_tile << " (" <<
    ((double)nempty_tile*100)/(double)ntot << "%)" << std::endl;

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

}

//
// Load neighbor list from file
//
template <int tilesize>
void NeighborList<tilesize>::load(const char *filename) {

  ientry_t *h_ientry;
  int *h_tile_indj;
  tile_excl_t<tilesize> *h_tile_excl;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> ni >> ntot;

    h_ientry = new ientry_t[ni];
    h_tile_indj = new int[ntot];
    h_tile_excl = new tile_excl_t<tilesize>[ntot];

    for (int i=0;i < ni;i++) {
      file >> std::dec >> h_ientry[i].indi >> h_ientry[i].ish >> 
	h_ientry[i].startj >> h_ientry[i].endj;
      for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
	file >> std::dec >> h_tile_indj[j];
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  file >> std::hex >> h_tile_excl[j].excl[k];
	}
      }
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.2f);
  reallocate<int>(&tile_indj, &tile_indj_len, ntot, 1.2f);
  reallocate< tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, ntot, 1.2f);

  copy_HtoD<ientry_t>(h_ientry, ientry, ni);
  copy_HtoD<int>(h_tile_indj, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;
}

//
// Explicit instances of DirectForce
//
template class NeighborList<16>;
template class NeighborList<32>;
