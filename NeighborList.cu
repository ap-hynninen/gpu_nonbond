#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"

//
// Calculates tilex index for each atom
//
__global__ void calc_tilex_ind_kernel(const int istart, const int iend,
				      const float4* __restrict__ xyzq,
				      const int ind0,
				      const int ncellx,
				      const int ncelly,
				      const int ncellz,
				      const float x0,
				      const float y0,
				      const float z0,
				      const float inv_dx,
				      const float inv_dy,
				      const float inv_dz,
				      int* __restrict__ tilex_key,
				      int* __restrict__ tilex_val) {

  const int ind = threadIdx.x + blockIdx.x*blockDim.x + istart;
  
  if (ind <= iend) {
    float4 xyzq_val = xyzq[ind];
    float x = xyzq_val.x;
    float y = xyzq_val.y;
    float z = xyzq_val.z;
    int ix = (int)((x - x0)*inv_dx);
    int iy = (int)((y - y0)*inv_dy);
    int iz = (int)((z - z0)*inv_dz);
    int key = ind0 + (ix + iy*ncellx)*ncellz + iz;

    tilex_key[ind] = key;
    tilex_val[ind] = ind;
  }

}

//
// Sort atoms into z-columns
//
// col_n[0..ncellx*ncelly-1] = number of atoms in each column
// col_ind[istart..iend]     = column index for atoms 
//
__global__ void calc_z_column_index_kernel(const int istart, const int iend,
					   const float4* __restrict__ xyzq,
					   const int ind0,
					   const int ncellx,
					   const int ncelly,
					   const float x0,
					   const float y0,
					   const float inv_dx,
					   const float inv_dy,
					   int* __restrict__ col_n,
					   int* __restrict__ col_ind) {

  const int i = threadIdx.x + blockIdx.x*blockDim.x + istart;
  
  if (i <= iend) {
    float4 xyzq_val = xyzq[i];
    float x = xyzq_val.x;
    float y = xyzq_val.y;
    int ix = (int)((x - x0)*inv_dx);
    int iy = (int)((y - y0)*inv_dy);
    int ind = ind0 + ix + iy*ncellx;
    atomicAdd(&col_n[ind], 1);
    col_ind[i] = ind;
  }
  
}

//
// Computes z column position using parallel exclusive prefix sum
// NOTE: Must have nblock = 1, we loop over buckets to avoid multiple kernel calls
//
__global__ void calc_z_column_pos_kernel(const int ncol_tot,
					 int* __restrict__ col_n,
					 int* __restrict__ col_pos) {
  // Shared memory
  // Requires: blockDim.x*sizeof(int)
  extern __shared__ int shpos[];

  if (threadIdx.x == 0) col_pos[0] = 0;

  int offset = 0;
  for (int base=0;base < ncol_tot;base += blockDim.x) {
    int i = base + threadIdx.x;
    shpos[threadIdx.x] = (i < ncol_tot) ? col_n[i] : 0;
    if (i < ncol_tot) col_n[i] = 0;
    __syncthreads();

    for (int d=1;d < blockDim.x; d *= 2) {
      int tmp = (threadIdx.x >= d) ? shpos[threadIdx.x-d] : 0;
      __syncthreads();
      shpos[threadIdx.x] += tmp;
      __syncthreads();
    }

    if (i < ncol_tot) col_pos[i+1] = shpos[threadIdx.x] + offset;

    offset += shpos[blockDim.x-1];
  }

}

struct keyval_t {
  float key;
  int val;
};

//
// Sorts atoms according to z coordinate
//
// Uses bitonic sort, see:
// http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
//
// Each thread block sorts a single z column
//
__global__ void sort_z_column_kernel(const int* __restrict__ col_pos,
				     float4* __restrict__ xyzq) {

  // Shared memory
  // Requires: blockDim.x*sizeof(keyval_t)
  extern __shared__ keyval_t sh_keyval[];

  int col_pos0 = col_pos[blockIdx.x];
  int n = col_pos[blockIdx.x+1] - col_pos0;

  // Read keys and values into shared memory
  keyval_t keyval;
  keyval.key = (threadIdx.x < n) ? xyzq[threadIdx.x + col_pos0].z : 1.0e38;
  keyval.val = (threadIdx.x < n) ? (threadIdx.x + col_pos0) : (n-1);
  sh_keyval[threadIdx.x] = keyval;
  __syncthreads();

  for (int k = 2;k <= blockDim.x;k *= 2) {
    for (int j = k/2; j > 0;j /= 2) {
      int ixj = threadIdx.x ^ j;
      if (ixj > threadIdx.x && ixj < n) {
	// asc = true for ascending order
	bool asc = ((threadIdx.x & k) == 0);
	
	// Read data
	keyval_t keyval1 = sh_keyval[threadIdx.x];
	keyval_t keyval2 = sh_keyval[ixj];
	
	float lo_key = asc ? keyval1.key : keyval2.key;
	float hi_key = asc ? keyval2.key : keyval1.key;
	
	if (lo_key > hi_key) {
	  // keys are in wrong order => exchange
	  sh_keyval[threadIdx.x] = keyval2;
	  sh_keyval[ixj]         = keyval1;
	}
	
	//if ((i&k)==0 && get(i)>get(ixj)) exchange(i,ixj);
	//if ((i&k)!=0 && get(i)<get(ixj)) exchange(i,ixj);
      }
      __syncthreads();
    }
  }

  float4 xyzq_val;
  if (threadIdx.x < n) xyzq_val = xyzq[sh_keyval[threadIdx.x].val];
  __syncthreads();
  if (threadIdx.x < n) xyzq[threadIdx.x + col_pos0] = xyzq_val;

}


//
// Re-order atoms according to pos
//
__global__ void reorder_atoms_z_column_kernel(const int ncoord,
					      const int* col_ind,
					      int* col_n,
					      const int* col_pos,
					      const float4* __restrict__ xyzq_in,
					      float4* __restrict__ xyzq_out) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < ncoord) {
    int ind = col_ind[i];
    int pos = col_pos[ind];
    int n = atomicAdd(&col_n[ind], 1);
    // new position = pos + n
    float4 xyzq_val = xyzq_in[i];
    xyzq_out[pos+n] = xyzq_val;
  }

}

//
// Re-order atoms according to tilex_val
//
__global__ void reorder_atoms_kernel(const int ncoord,
				     const int* tilex_val,
				     const float4* __restrict__ xyzq_in,
				     float4* __restrict__ xyzq_out) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (tid < ncoord) {
    int ind = tilex_val[tid];
    float4 xyzq_val = xyzq_in[ind];
    xyzq_out[tid] = xyzq_val;
  }

}

//
//
//
template <int tilesize>
void NeighborList<tilesize>::set_cell_sizes(const int *zonelist,
					    const float3 *max_xyz, const float3 *min_xyz,
					    int *ncellx, int *ncelly, int *ncellz,
					    float *celldx, float *celldy, float *celldz) {

  for (int izone=0;izone < 8;izone++) {
    int nstart;
    if (izone > 0) {
      nstart = zonelist[izone-1] + 1;
    } else {
      nstart = 1;
    }
    // ncoord_zone = number of atoms in this zone
    int ncoord_zone = zonelist[izone] - nstart + 1;
    if (ncoord_zone > 0) {
      // NOTE: we increase the cell sizes here by 0.001 to make sure no atoms drop outside cells
      float xsize = max_xyz[izone].x - min_xyz[izone].x + 0.001f;
      float ysize = max_xyz[izone].y - min_xyz[izone].y + 0.001f;
      float zsize = max_xyz[izone].z - min_xyz[izone].z + 0.001f;
      float delta = powf(xsize*ysize*zsize*tilesize/(float)ncoord_zone, 1.0f/3.0f);
      ncellx[izone] = max(1, (int)(xsize/delta));
      ncelly[izone] = max(1, (int)(ysize/delta));
      // Approximation for ncellz = "uniform distribution of atoms"
      ncellz[izone] = max(1, ncoord_zone/(ncellx[izone]*ncelly[izone]*tilesize));
      celldx[izone] = xsize/(float)(ncellx[izone]);
      celldy[izone] = ysize/(float)(ncelly[izone]);
      celldz[izone] = zsize/(float)(ncellz[izone]);
    } else {
      ncellx[izone] = 0;
      ncelly[izone] = 0;
      ncellz[izone] = 0;
      celldx[izone] = 1.0f;
      celldy[izone] = 1.0f;
      celldz[izone] = 1.0f;
    }
  }

}

//
// Sorts atoms into tiles
//
template <int tilesize>
void NeighborList<tilesize>::sort(const int *zonelist,
				  const float3 *max_xyz, const float3 *min_xyz,
				  float4 *xyzq,
				  float4 *xyzq_sorted,
				  hipStream_t stream) {

  int ncellx[8], ncelly[8], ncellz[8];
  float celldx[8], celldy[8], celldz[8];
  float inv_dx[8], inv_dy[8], inv_dz[8];

  int ncoord = zonelist[7];

  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_cell_sizes(zonelist, max_xyz, min_xyz, ncellx, ncelly, ncellz, celldx, celldy, celldz);

  int ncol_tot = 0;
  for (int i=0;i < 8;i++) ncol_tot += ncellx[i]*ncelly[i];

  reallocate<int>(&col_n, &col_n_len, ncol_tot, 1.2f);
  reallocate<int>(&col_pos, &col_pos_len, ncol_tot+1, 1.2f);
  reallocate<int>(&col_ind, &col_ind_len, ncoord, 1.2f);

  clear_gpu_array<int>(col_n, ncol_tot, stream);

  for (int izone=0;izone < 8;izone++) {
    inv_dx[izone] = 1.0f/celldx[izone];
    inv_dy[izone] = 1.0f/celldy[izone];
    inv_dz[izone] = 1.0f/celldz[izone];
  }

  //
  // Calculate number of atoms in each z-column
  //
  int ind0 = 0;
  for (int izone=0;izone < 8;izone++) {
    int istart, iend;
    if (izone > 0) {
      istart = zonelist[izone-1];
    } else {
      istart = 0;
    }
    iend = zonelist[izone] - 1;
    if (iend >= istart) {

      calc_z_column_index_kernel<<< nblock, nthread, 0, stream >>>
	(istart, iend, xyzq, ind0, ncellx[izone], ncelly[izone], 
	 min_xyz[izone].x, min_xyz[izone].y,
	 inv_dx[izone], inv_dy[izone], col_n, col_ind);
      cudaCheck(hipGetLastError());

      ind0 += ncellx[izone]*ncelly[izone];
    }
  }

  /*
  thrust::device_ptr<int> col_n_ptr(col_n);
  thrust::device_ptr<int> col_pos_ptr(col_pos);
  thrust::exclusive_scan(col_n_ptr, col_n_ptr + ncol_tot, col_pos_ptr);
  clear_gpu_array<int>(col_n, ncol_tot, stream);
  */

  /*
  {
    int *h_tmp = new int[ncol_tot];
    copy_DtoH<int>(col_n, h_tmp, ncol_tot);
    for (int i=0;i < ncol_tot;i++)
      std::cout << h_tmp[i] << " ";
    std::cout << std::endl;
    delete [] h_tmp;
  }
  */

  //
  // Calculate positions
  //
  nthread = min(((ncol_tot-1)/32+1)*32, get_max_nthread());
  //std::cout << "nthread = " << nthread << std::endl;
  int shmem_size = nthread*sizeof(int);
  calc_z_column_pos_kernel<<< 1, nthread, shmem_size, stream >>>(ncol_tot, col_n, col_pos);

  /*
  std::cout << "--------------------------------------------------------" << std::endl;
  {
    int *h_tmp = new int[ncol_tot];
    copy_DtoH<int>(col_pos, h_tmp, ncol_tot);
    for (int i=0;i < ncol_tot;i++)
      std::cout << h_tmp[i] << " ";
    std::cout << std::endl;
    delete [] h_tmp;
  }
  */

  nthread = 512;
  nblock = (ncoord-1)/nthread+1;
  reorder_atoms_z_column_kernel<<< nblock, nthread, 0, stream >>>
    (ncoord, col_ind, col_n, col_pos, xyzq, xyzq_sorted);
  cudaCheck(hipGetLastError());

  // Now sort according to z coordinate
  nthread = 11*tilesize;
  nblock = ncellx[0]*ncelly[0];
  if (nthread < get_max_nthread()) {
    shmem_size = nthread*sizeof(keyval_t);
    sort_z_column_kernel<<< nblock, nthread, shmem_size, stream >>>
      (col_pos, xyzq_sorted);
    cudaCheck(hipGetLastError());
  } else {
    std::cerr << "Neighborlist::sort, this version of sort_z_column_kernel not implemented yet"
	      << std::endl;
  }

  //  reorder_atoms_kernel<<< nblock, nthread, 0, stream >>>
  //    (ncoord, tilex_val, xyzq, xyzq_sorted);
  //cudaCheck(hipGetLastError());

}

//
// Calculates bounding box
//
template <int tilesize>
__global__ void calc_bounding_box_kernel(const int ncell,
					 const int* __restrict__ cell_start,
					 const float4* __restrict__ xyzq,
					 bb_t* __restrict__ bb) {

  const int icell = threadIdx.x + blockIdx.x*blockDim.x;

  if (icell < ncell) {
    int base = cell_start[icell];
    float4 xyzq_val = xyzq[base];
    float x0 = xyzq_val.x;
    float y0 = xyzq_val.y;
    float z0 = xyzq_val.z;
    float x1 = xyzq_val.x;
    float y1 = xyzq_val.y;
    float z1 = xyzq_val.z;
    for (int i=1;i < tilesize;i++) {
      xyzq_val = xyzq[base + i];
      x0 = min(x0, xyzq_val.x);
      y0 = min(y0, xyzq_val.y);
      z0 = min(z0, xyzq_val.z);
      x1 = max(x1, xyzq_val.x);
      y1 = max(y1, xyzq_val.y);
      z1 = max(z1, xyzq_val.z);
    }
    bb[icell].x = 0.5f*(x0 + x1);
    bb[icell].y = 0.5f*(y0 + y1);
    bb[icell].z = 0.5f*(z0 + z1);
    bb[icell].wx = 0.5f*(x1 - x0);
    bb[icell].wy = 0.5f*(y1 - y0);
    bb[icell].wz = 0.5f*(z1 - z0);
  }

}

//
// Calculates bounding boxes for tiles
//
template <int tilesize>
void NeighborList<tilesize>::calc_bounding_box(const int ncell,
					       const int *cell_start,
					       const float4 *xyzq,
					       hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncell-1)/nthread+1;

  calc_bounding_box_kernel<tilesize> <<< nblock, nthread >>>
    (ncell, cell_start, xyzq, bb);

  cudaCheck(hipGetLastError());
}

//#######################################################################

//
// Class creator
//
template <int tilesize>
NeighborList<tilesize>::NeighborList() {
  ni = 0;
  ntot = 0;

  tile_excl = NULL;
  tile_excl_len = 0;

  ientry = NULL;
  ientry_len = 0;

  tile_indj = NULL;
  tile_indj_len = 0;

  // Sparse
  ni_sparse = 0;
  ntot_sparse = 0;

  pairs_len = 0;
  pairs = NULL;
  
  ientry_sparse_len = 0;
  ientry_sparse = NULL;

  tile_indj_sparse_len = NULL;
  tile_indj_sparse = NULL;

  // Neighbor list building
  col_n_len = 0;
  col_n = NULL;

  col_pos_len = 0;
  col_pos = NULL;

  col_ind_len = 0;
  col_ind = NULL;
}

//
// Class destructor
//
template <int tilesize>
NeighborList<tilesize>::~NeighborList() {
  if (tile_excl != NULL) deallocate< tile_excl_t<tilesize> > (&tile_excl);
  if (ientry != NULL) deallocate<ientry_t>(&ientry);
  if (tile_indj != NULL) deallocate<int>(&tile_indj);
  // Sparse
  if (pairs != NULL) deallocate< pairs_t<tilesize> > (&pairs);
  if (ientry_sparse != NULL) deallocate<ientry_t>(&ientry_sparse);
  if (tile_indj_sparse != NULL) deallocate<int>(&tile_indj_sparse);
  // Neighbor list building
  if (col_n != NULL) deallocate<int>(&col_n);
  if (col_pos != NULL) deallocate<int>(&col_pos);
  if (col_ind != NULL) deallocate<int>(&col_ind);
}

//
// Sets ientry from host memory array
//
template <int tilesize>
void NeighborList<tilesize>::set_ientry(int ni, ientry_t *h_ientry, hipStream_t stream) {

  this->ni = ni;

  // Allocate & reallocate d_ientry
  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.4f);

  // Copy to device
  copy_HtoD<ientry_t>(h_ientry, ientry, ni, stream);
}

//----------------------------------------------------------------------------------------
//
// Builds neighborlist
//

struct cell_t {
  int izone;
  int icellx;
  int icelly;
  int icellz;
};

#ifdef NOTREADY

//
// The entire warp enters here
// If IvsI = true, search within I zone
//
template <bool IvsI>
__device__ void get_cell_bounds(const int izone, const int jzone, const int icell, const int ncell,
				const float x0, const float x1, const float* bx, const float rcut,
				int& jcell0, int& jcell1, float *dist) {

  int jcell_start_left, jcell_start_right;

  if (izone == jzone) {
    // Search within a single zone (I)
    if (icell < 0) {
      // This is one of the image cells on the left =>
      // set the left cell boundary (jcell0) to 1 and start looking for the right
      // boundary from 1
      jcell_start_left = 0;       // with this value, we don't look for cells on the left
      jcell_start_right = 1;      // start looking for cells at right from 1
      jcell0 = 1;                  // left boundary set to minimum value
      jcell1 = 0;                    // set to "no cells" value
      dist[1] = 0.0f;
    } else if (icell >= ncell) {
      // This is one of the image cells on the right =>
      // set the right cell boundary (icell1) to ncell and start looking for the left
      // boundary from ncell
      jcell_start_left = ncell;      // start looking for cells at left from ncell
      jcell_start_right = ncell + 1; // with this value, we don't look for cells on the right
      jcell0 = ncell + 1;            // set to "no cells" value
      jcell1 = ncell;                // right boundary set to maximum value
      dist[ncell] = 0.0f;
    } else {
      jcell_start_left = icell - 1;
      jcell_start_right = icell + 1;
      jcell0 = icell;
      jcell1 = icell;
      dist[icell] = 0.0f;
    }
  } else {
    if (bx(0) >= x1 || (bx(0) < x1 && bx(0) > x0)) {
      // j-zone is to the right of i-zone
      // => no left search, start right search from 1
      jcell_start_left = 0;
      jcell_start_right = 1;
      jcell0 = 1;
      jcell1 = 0;
    } else if (bx[ncell] <= x0 || (bx[ncell] > x0 && bx[ncell] < x1)) {
      // j-zone is to the left of i-zone
      // => no right search, start left search from ncell
      jcell_start_left = ncell;
      jcell_start_right = ncell + 1;
      jcell0 = ncell + 1;
      jcell1 = ncell;
    } else {
      // i-zone is between j-zones
      // => safe choice is to search the entire range
      jcell_start_left = ncell;
      jcell_start_right = 1;
      jcell0 = ncell;
      jcell1 = 1;
    }
  }

  // Check cells at left, stop once the distance to the cell right boundary 
  // is greater than the cutoff.
  //
  // Cell right boundary is at bx(i)
  for (int j=jcell_start_left;j >= 1;j--) {
    float d = x0 - bx[j];
    if (d > cut) break;
    dist[j] = max(0.0f, d);
    jcell0 = j;
  }

  // Check cells at right, stop once the distance to the cell left boundary
  // is greater than the cutoff.
  //
  // Cell left boundary is at bx(i-1)
  for (int j=jcell_start_right;j <= ncell;j++) {
    float d = bx[j-1] - x1;
    if (d > cut) break;
    dist[j] = max(0.0f, d);
    jcell1 = j;
  }

  // Cell bounds are jcell0:jcell1
      
}

//
// Build neighborlist for one zone at the time
// One warp takes care of one cell
//
template < int tilesize, bool IvsI >
__global__
void build_nlist_kernel(const int ncell, const int izone, const int n_jzone,
			const int *cellx, const int *celly, const int *cellz,
			const bb_t * bb,
			const float *cellbx, const float *cellby, const float *cellbz) {

  // Shared memory
  extern __shared__ char shbuf[];
  volatile int *jcellx0;
  volatile int *jcelly0;
  volatile int *jcellz0;
  volatile int *jcellx1;
  volatile int *jcelly1;
  volatile int *jcellz1;

  // Index of the i-cell
  const int icell = (threadId.x + blockIdx.x*blockDim.x)/WARPSIZE;

  if (icell >= ncell) return;

  int icellx = cellx[icell];
  int icelly = celly[icell];
  int icellz = cellz[icell];

  bb_t ibb = bb[icell];

  for (int imx=imx_lo;imx <= imx_hi;imx++) {
    float imbbx0 = ibb.x + imx*boxx;
    int n_jcellx = 0;
    for (int jjzone=0;jjzone < n_jzone;jjzone++) {
      int jzone = int_zone[izone][jjzone];
      int jcellx0_t, jcellx1_t;
      get_cell_bounds<IvsI>(izone, jzone, icellx + imx*ncellx[izone], ncellx[jzone],
			    imbbx0-ibb.wx, imbbx0+ibb.wx, cellbx[jzone], rcut,
			    jcellx0_t, jcellx1_t);
      n_jcellx += max(0, jcellx1_t-jcellx0_t+1);
      jcellx0[jzone] = jcellx0_t;
      jcellx1[jzone] = jcellx1_t;
    }

    for (int imy=imy_lo;imy <= imy_hi;imy++) {
      float imbby0 = ibb.y + imy*boxy;
      int n_jcelly = 0;
      for (int jjzone=0;jjzone < n_jzone;jjzone++) {
	int jzone = int_zone[izone][jjzone];
	int jcelly0_t, jcelly1_t;
	get_cell_bounds<IvsI>(izone, jzone, icelly + imy*ncelly[izone], ncelly[jzone],
			      imbby0-ibb.wy, imbby0+ibb.wy, cellby[jzone], rcut,
			      jcelly0_t, jcelly1_t);
	n_jcelly += max(0, jcelly1_t-jcelly0_t+1);
	jcelly0[jzone] = jcelly0_t;
	jcelly1[jzone] = jcelly1_t;
      }
    } // for (int imy=imy_lo;imy <= imy_hi;imy++)

    for (int imz=imz_lo;imz <= imz_hi;imz++) {
	float imbbz0 = ibb.z + imz*boxz;
	
	int ish = imx+1 + 3*(imy+1 + 3*(imz+1));
	
	for (int jjzone=0;jjzone < n_jzone;jjzone++) {
	  int jzone = int_zone[izone][jjzone];


	  if (jcelly1[jzone] >= jcelly0[jzone] && jcellx1[jzone] >= jcellx0[jzone]) {
	    // Loop over j-cells
	    // NOTE: we do this in order y, x, z so that the resulting tile list
	    //       is ordered
	    for (int jcelly=jcelly0[jzone]; jcelly <= jcelly1(jzone);jcelly++) {
	      float celldist1 = ydist[ydist_pos + jcelly];
	      celldist1 *= celldist1;
	      jcellx0_t = jcellx0[jzone];
	      for (int jcellx=jcellx0_t; jcellx <= jcellx1[jzone]; jcellx++) {
		float celldist2 = celldist1 + xdist[xdist_pos + jcellx];
		celldist2 *= celldist2;
		if (celldist2 > cutsq) continue;
		// Get jcellz limits (jcellz0, jcellz1)
		pos_xy = jcellx + (jcelly-1)*ncellx[jzone];
		pos_cellbz = (max_ncellz(jzone)+1)*(pos_xy - 1);
		pos_ncellz = pos_xy + startcol_zone[jzone];
		get_cell_bounds<IvsI>(izone, jzone, icellz_im,
				      ncellz[pos_ncellz], imbbz0-ibb.wz, imbbz0+ibb.wz,
				      cellbz[jzone]%array(pos_cellbz:), cut, jcellz0, jcellz1, zdist);
		for (int jcellz=jcellz0; jcellz <= jcellz1; jcellz++) {
		  if (celldist2 + zdist(jcellz)**2 > cutsq) continue;
		  // j-cell index is calculated as jcellz + start of the column cells
		  jcell = jcellz + startcell_col[pos_ncellz];

		  // Read bounding box for j-cell
		  bb_t jbb = bb[jcell];
                               
		  // Calculate distance between i- and j-cell bounding boxes
		  float bbxdist = max(0.0f, fabs(imbbx0 - jbb.x) - ibb.wx - jbb.wx);
		  float bbydist = max(0.0f, fabs(imbby0 - jbb.y) - ibb.wy - jbb.wy);
		  float bbzdist = max(0.0f, fabs(imbbz0 - jbb.z) - ibb.wz - jbb.wz);

		  if (bbxdist**2 + bbydist**2 + bbzdist**2 < cutsq) {
		  }


	}
	
    } // for (int imz=imz_lo;imz <= imz_hi;imz++)


  } // for (int imx=imx_lo;imx <= imx_hi;imx++)

}

template <int tilesize>
void NeighborList<tilesize>::build_nlist(const float boxx, const float boxy, const float boxz,
					 const float roff,
					 const int n_ijlist, const int3 *ijlist,
					 const int *cell_start,
					 const float4 *xyzq,
					 hipStream_t stream) {

  build_nlist_kernel<tilesize, true>
    <<< nblock, nthread, shmem_size, stream >>>
    ();

  build_nlist_kernel<tilesize, false>
    <<< nblock, nthread, shmem_size, stream >>>
    ();

}
#endif // NOTREADY

//----------------------------------------------------------------------------------------
//
// Builds tilex exclusion mask from ijlist[] based on distance and index
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
// Uses 32 threads to calculate the distances for a single ijlist -entry.
//
const int nwarp_build_excl_dist = 8;

template < int tilesize >
__global__ void build_excl_kernel(const unsigned int base_tid, const int n_ijlist, const int3 *ijlist,
				  const int *cell_start, const float4 *xyzq,
				  int *tile_indj,
				  tile_excl_t<tilesize> *tile_excl,
				  const float boxx, const float boxy, const float boxz,
				  const float roff2) {
  const int num_thread_per_excl = (32/(num_excl<tilesize>::val));

  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x + base_tid;
  // Global warp index
  const unsigned int wid = gtid / warpsize;
  // Local thread index (0...warpsize-1)
  const unsigned int tid = gtid % warpsize;
  // local thread index (0...tilesize-1)
  const unsigned int stid = gtid % tilesize;

  // Shared memory
  extern __shared__ char shmem[];
  float3 *sh_xyzi = (float3 *)&shmem[0];    // nwarp_build_excl_dist*tilesize
  unsigned int *sh_excl = (unsigned int *)&sh_xyzi[nwarp_build_excl_dist*tilesize];

  //  __shared__ float3 sh_xyzi[nwarp_build_excl_dist*tilesize];
  //#if (tilesize == 16)
  //  __shared__ unsigned int sh_excl[nwarp_build_excl_dist*num_excl];
  //#endif

  if (wid >= n_ijlist) return;

  // Each warp computes one ijlist entry
  int3 ijlist_val = ijlist[wid];
  int icell = ijlist_val.x - 1;
  int ish   = ijlist_val.y;
  int jcell = ijlist_val.z - 1;

  int istart = cell_start[icell] - 1;
  int iend   = cell_start[icell+1] - 2;

  int jstart = cell_start[jcell] - 1;
  int jend   = cell_start[jcell+1] - 2;

  const unsigned int load_ij = threadIdx.x % tilesize;
  const int sh_start = (threadIdx.x/warpsize)*tilesize;

  // Load atom i coordinates to shared memory
  float4 xyzq_i;

  if (tilesize == 32 || tid < 16) {
    if (istart + load_ij <= iend) {
      xyzq_i = xyzq[istart + load_ij];
    } else {
      xyzq_i.x = -100000000.0f;
      xyzq_i.y = -100000000.0f;
      xyzq_i.z = -100000000.0f;
    }
    sh_xyzi[sh_start + load_ij].x = xyzq_i.x;
    sh_xyzi[sh_start + load_ij].y = xyzq_i.y;
    sh_xyzi[sh_start + load_ij].z = xyzq_i.z;
  }

  // Load atom j coordinates
  float xj, yj, zj;
  //  const unsigned int loadj = (tid + (tid/TILESIZE)*(TILESIZE-1)) % TILESIZE;
  //  const unsigned int loadj = threadIdx.x % TILESIZE;
  if (jstart + load_ij <= jend) {
    float4 xyzq_j = xyzq[jstart + load_ij];
    xj = xyzq_j.x;
    yj = xyzq_j.y;
    zj = xyzq_j.z;
  } else {
    xj = 100000000.0f;
    yj = 100000000.0f;
    zj = 100000000.0f;
  }

  // Calculate shift
  float zsh = (ish/9 - 1)*boxz;
  ish -= (ish/9)*9;
  float ysh = (ish/3 - 1)*boxy;
  ish -= (ish/3)*3;
  float xsh = (ish - 1)*boxx;

  xj -= xsh;
  yj -= ysh;
  zj -= zsh;
  
  // Make sure shared memory has been written
  // NOTE: since we're only operating within the warp, this __syncthreads() is just to make sure
  //       all values are actually written in shared memory and not kept in registers etc.
  __syncthreads();

  int q_samecell = (icell == jcell);

  unsigned int excl = 0;
  int t;

  if (tilesize == 32) {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int i = ((threadIdx.x + t) % tilesize);
      int ii = sh_start + i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (tid <= i)) ) << t;
    }
    tile_indj[wid] = jstart;
    tile_excl[wid].excl[stid] = excl;

  } else {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int load_i = (tid + t*2 + (tid/tilesize)*(tilesize-1)) % tilesize;
      int ii = sh_start + load_i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (load_ij <= load_i)) ) << t;
    }
    // excl is a 8 bit exclusion mask.
    // The full 32 bit exclusion mask is contained in 4 threads:
    // thread 0 contains the lowest 8 bits
    // thread 1 contains the next 8 bits, etc..
    
    excl <<= (threadIdx.x % num_thread_per_excl)*(num_excl<tilesize>::val);
    
    // Combine excl using shared memory
    const unsigned int sh_excl_ind = (threadIdx.x/warpsize)*(num_excl<tilesize>::val) + 
      (threadIdx.x % warpsize)/num_thread_per_excl;
    
    sh_excl[sh_excl_ind] = 0;
    __syncthreads();
    
    atomicOr(&sh_excl[sh_excl_ind], excl);
    
    // Make sure shared memory is written
    __syncthreads();
    
    // index to tile_excl.excl[] (0...7)
    const unsigned int excl_ind = (threadIdx.x % warpsize)/num_thread_per_excl;
    
    tile_indj[wid] = jstart;
    
    if ((threadIdx.x % num_thread_per_excl) == 0) {
      tile_excl[wid].excl[excl_ind] = sh_excl[sh_excl_ind];
    }
  }

}

/*
void test_excl_dist_index(const int n_ijlist, const int3 *d_ijlist,
			  const int *d_cell_start, const float4 *d_xyzq,
			  int *d_tile_indj,
			  tile_excl_t *d_tile_excl,
			  const float boxx, const float boxy, const float boxz,
			  const float roff2) {

  int3 *h_ijlist;
  int *h_cell_start;
  float4 *h_xyzq;
  int *h_tile_indj;
  tile_excl_t *h_tile_excl;

  h_ijlist = (int3 *)malloc(n_ijlist*sizeof(int3));
  h_cell_start = (int *)malloc(mdsim.ncell*sizeof(int));
  h_xyzq = (float4 *)malloc(mdsim.ncoord*sizeof(float4));
  h_tile_indj = (int *)malloc(n_ijlist*sizeof(int));
  h_tile_excl = (tile_excl_t *)malloc(n_ijlist*sizeof(tile_excl_t));

  cudaCheck(hipMemcpy(h_ijlist, d_ijlist, sizeof(int3)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_cell_start, d_cell_start, sizeof(int)*mdsim.ncell,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_xyzq, d_xyzq, sizeof(float4)*mdsim.ncoord,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_indj, d_tile_indj, sizeof(int)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_excl, d_tile_excl, sizeof(tile_excl_t)*n_ijlist,
		       hipMemcpyDeviceToHost));

  for (int wid=0;wid < n_ijlist;wid++) {

    int3 ijlist_val = h_ijlist[wid];
    int icell = ijlist_val.x - 1;
    int ish   = ijlist_val.y;
    int jcell = ijlist_val.z - 1;

    int istart = h_cell_start[icell] - 1;
    int iend   = h_cell_start[icell+1] - 2;

    int jstart = h_cell_start[jcell] - 1;
    int jend   = h_cell_start[jcell+1] - 2;

    int q_samecell = (icell == jcell);

    // Calculate shift
    float zsh = (ish/9 - 1)*boxz;
    ish -= (ish/9)*9;
    float ysh = (ish/3 - 1)*boxy;
    ish -= (ish/3)*3;
    float xsh = (ish - 1)*boxx;
    
    int i,j,ii,jj;

    for (ii=istart,i=0;ii <= iend;ii++,i++) {
      float4 xyzq_i = h_xyzq[ii];
      float xi = xyzq_i.x;
      float yi = xyzq_i.y;
      float zi = xyzq_i.z;
      for (jj=jstart,j=0;jj <= jend;jj++,j++) {
	float4 xyzq_j = h_xyzq[jj];
	float xj = xyzq_j.x - xsh;
	float yj = xyzq_j.y - ysh;
	float zj = xyzq_j.z - zsh;
	float dx = xi - xj;
	float dy = yi - yj;
	float dz = zi - zj;
	float r2 = dx*dx + dy*dy + dz*dz;
#if (TILESIZE == 16)
	int ttid = ((i+j) % 2)*16 + j;
	int iexcl = ttid/4;
	int tmp = i + 1 + j*15;
	int shbit = ((tmp/2) % 8) + (j % 4)*8;
#else
	int ij = i + j*TILESIZE - j;
	int iexcl = j;
	int shbit = (ij % TILESIZE);
#endif
	unsigned int ibit = 1 << shbit;
	unsigned int excl = ((r2 >= roff2) | (q_samecell && (j <= i)) ) << shbit;
	unsigned int excl_gpu = h_tile_excl[wid].excl[iexcl];
	if ( ((excl_gpu & ibit) ^ excl) != 0 && fabsf(r2-roff2) > 7.0e-5) {
	  printf("Error found in test_excl_dist_index:\n");
	  printf("wid = %d i = %d j = %d iexcl = %d shbit = %d\n",wid,i,j,iexcl,shbit);
	  printf("ii = %d jj = %d %d %d %d %d\n",ii,jj,r2 >= roff2,
		 (q_samecell && (j <= i)),icell,jcell);
	  printf("%x ^ %x = %x \n",excl_gpu & ibit, excl, (excl_gpu & ibit) ^ excl);
	  printf("i:  %f %f %f\n",xi,yi,zi);
	  printf("j:  %f %f %f\n",xj,yj,zj);
	  printf("jo: %f %f %f\n",xyzq_j.x,xyzq_j.y,xyzq_j.z);
	  printf("sh: %f %f %f\n",xsh,ysh,zsh);
	  printf("dx: %1.8f %1.8f %1.8f\n",dx,dy,dz);
	  printf("r2: %f %e\n",r2,fabsf(r2-roff2));
	  exit(1);
	}
      }
    }

  }

  free(h_ijlist);
  free(h_cell_start);
  free(h_xyzq);
  free(h_tile_indj);
  free(h_tile_excl);

  printf("test_excl_dist_index OK\n");
}
*/

//
// Host wrapper for build_tilex_kernel
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
template <int tilesize>
void NeighborList<tilesize>::build_excl(const float boxx, const float boxy, const float boxz,
					const float roff,
					const int n_ijlist, const int3 *ijlist,
					const int *cell_start,
					const float4 *xyzq,
					hipStream_t stream) {

  if (n_ijlist == 0) return;

  // Allocate & re-allocate (d_tile_indj, d_tile_excl)
  reallocate<int>(&tile_indj, &tile_indj_len, n_ijlist, 1.2f);
  reallocate<tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_ijlist, 1.2f);

  float roff2 = roff*roff;

  int nthread = nwarp_build_excl_dist*warpsize;
  int nblock_tot = (n_ijlist-1)/(nthread/warpsize) + 1;
  size_t shmem_size = nwarp_build_excl_dist*tilesize*sizeof(float3); 

  if (tilesize == 16) {
    shmem_size += nwarp_build_excl_dist*(num_excl<tilesize>::val)*sizeof(unsigned int);
  }

  int3 max_nblock3 = get_max_nblock();
  unsigned int max_nblock = max_nblock3.x;
  unsigned int base_tid = 0;

  while (nblock_tot != 0) {

    int nblock = (nblock_tot > max_nblock) ? max_nblock : nblock_tot;
    nblock_tot -= nblock;

    build_excl_kernel<tilesize>
      <<< nblock, nthread, shmem_size, stream >>>
      (base_tid, n_ijlist, ijlist, cell_start,
       xyzq, tile_indj, tile_excl,
       boxx, boxy, boxz,
       roff2);

    base_tid += nblock*nthread;

    cudaCheck(hipGetLastError());
  }

  /*
  if (mdsim.q_test != 0) {
    test_excl_dist_index(mdsim.n_ijlist, mdsim.ijlist, mdsim.cell_start,
			 mdsim.xyzq.xyzq, mdsim.tile_indj, mdsim.tile_excl,
			 boxx, boxy, boxz,
			 roff2);
  }
  */

}

//----------------------------------------------------------------------------------------
//
// Combines tile_excl_top on GPU
// One thread takes care of one integer in the exclusion mask, therefore:
//
// 32x32 tile, 32 integers per tile
// 16x16 tile, 8 integers per tile
//
template <int tilesize>
__global__ void add_tile_top_kernel(const int ntile_top,
				    const int *tile_ind_top,
				    const tile_excl_t<tilesize> *tile_excl_top,
				    tile_excl_t<tilesize> *tile_excl) {
  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x;
  // Index to tile_ind_top[]
  const unsigned int i = gtid / (num_excl<tilesize>::val);
  // Index to exclusion mask
  const unsigned int ix = gtid % (num_excl<tilesize>::val);

  if (i < ntile_top) {
    int ind = tile_ind_top[i];
    tile_excl[ind].excl[ix] |= tile_excl_top[i].excl[ix];
  }

}

//
// Host wrapper for add_tile_top_kernel
//
template <int tilesize>
void NeighborList<tilesize>::add_tile_top(const int ntile_top, const int *tile_ind_top,
					  const tile_excl_t<tilesize> *tile_excl_top,
					  hipStream_t stream) {
  int nthread = 256;
  int nblock = (ntile_top*(num_excl<tilesize>::val) - 1)/nthread + 1;
  
  add_tile_top_kernel<tilesize>
    <<< nblock, nthread, 0, stream >>>
    (ntile_top, tile_ind_top, tile_excl_top, tile_excl);
  
  cudaCheck(hipGetLastError());
}

static int BitCount(unsigned int u)
 {
         unsigned int uCount;

         uCount = u
                  - ((u >> 1) & 033333333333)
                  - ((u >> 2) & 011111111111);
         return
           ((uCount + (uCount >> 3))
            & 030707070707) % 63;
 }

/*
static int BitCount_ref(unsigned int u) {
  unsigned int x = u;
  int res = 0;
  while (x != 0) {
    res += (x & 1);
    x >>= 1;
  }
  return res;
}
*/

//
// Splits neighbor list into dense and sparse parts
//
template <int tilesize>
void NeighborList<tilesize>::split_dense_sparse(int npair_cutoff) {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_dense = new ientry_t[ni];
  int *h_tile_indj_dense = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl_dense = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_sparse = new ientry_t[ni];
  int *h_tile_indj_sparse = new int[ntot];
  pairs_t<tilesize> *h_pairs = new pairs_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  int ni_dense = 0;
  int ntot_dense = 0;
  ni_sparse = 0;
  ntot_sparse = 0;
  for (int i=0;i < ni;i++) {
    bool sparse_i_tiles = true;
    int startj_dense = ntot_dense;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	npair += 32 - n1bit;
      }

      if (npair <= npair_cutoff) {
	// Sparse
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  
	}
	h_tile_indj_sparse[ntot_sparse] = h_tile_indj[j];
	ntot_sparse++;
      } else {
	// Dense
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_dense[ntot_dense].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_dense[ntot_dense] = h_tile_indj[j];
	ntot_dense++;
	sparse_i_tiles = false;
      }

    }

    if (sparse_i_tiles) {
      // Sparse
    } else {
      h_ientry_dense[ni_dense] = h_ientry[i];
      h_ientry_dense[ni_dense].startj = startj_dense;
      h_ientry_dense[ni_dense].endj = ntot_dense - 1;
      ni_dense++;
    }
  }

  ni = ni_dense;
  ntot = ntot_dense;

  copy_HtoD<ientry_t>(h_ientry_dense, ientry, ni);
  copy_HtoD<int>(h_tile_indj_dense, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl_dense, tile_excl, ntot);

  allocate<ientry_t>(&ientry_sparse, ni_sparse);
  allocate<int>(&tile_indj_sparse, ntot_sparse);
  allocate< pairs_t<tilesize> >(&pairs, ntot_sparse);
  ientry_sparse_len = ni_sparse;
  tile_indj_sparse_len = ntot_sparse;
  pairs_len = ntot_sparse;

  copy_HtoD<ientry_t>(h_ientry_sparse, ientry_sparse, ni_sparse);
  copy_HtoD<int>(h_tile_indj_sparse, tile_indj_sparse, ntot_sparse);
  copy_HtoD< pairs_t<tilesize> >(h_pairs, pairs, ntot_sparse);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_dense;
  delete [] h_tile_indj_dense;
  delete [] h_tile_excl_dense;

  delete [] h_ientry_sparse;
  delete [] h_tile_indj_sparse;
  delete [] h_pairs;

}

//
// Removes empty tiles
//
template <int tilesize>
void NeighborList<tilesize>::remove_empty_tiles() {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_noempty = new ientry_t[ni];
  int *h_tile_indj_noempty = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl_noempty = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  int ni_noempty = 0;
  int ntot_noempty = 0;
  for (int i=0;i < ni;i++) {
    bool empty_i_tiles = true;
    int startj_noempty = ntot_noempty;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	if (n1bit != 32) empty_tile = false;
      }

      if (!empty_tile) {
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_noempty[ntot_noempty].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_noempty[ntot_noempty] = h_tile_indj[j];
	ntot_noempty++;
	empty_i_tiles = false;
      }
    }

    if (!empty_i_tiles) {
      h_ientry_noempty[ni_noempty] = h_ientry[i];
      h_ientry_noempty[ni_noempty].startj = startj_noempty;
      h_ientry_noempty[ni_noempty].endj = ntot_noempty - 1;
      ni_noempty++;
    }
  }

  ni = ni_noempty;
  ntot = ntot_noempty;

  copy_HtoD<ientry_t>(h_ientry_noempty, ientry, ni);
  copy_HtoD<int>(h_tile_indj_noempty, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl_noempty, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_noempty;
  delete [] h_tile_indj_noempty;
  delete [] h_tile_excl_noempty;

}

//
// Analyzes the neighbor list and prints info
//
template <int tilesize>
void NeighborList<tilesize>::analyze() {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  std::cout << "Number of i-tiles = " << ni << ", total number of tiles = " << ntot << std::endl;

  std::ofstream file_npair("npair.txt", std::ofstream::out);
  std::ofstream file_nj("nj.txt", std::ofstream::out);

  unsigned int nexcl_bit = 0;
  unsigned int nexcl_bit_self = 0;
  unsigned int nempty_tile = 0;
  unsigned int nempty_line = 0;
  for (int i=0;i < ni;i++) {
    file_nj << h_ientry[i].endj - h_ientry[i].startj + 1 << std::endl;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);

	if (n1bit > 32) {
	  std::cerr << n1bit << " " << std::hex << h_tile_excl[j].excl[k] << std::endl;
	  exit(1);
	}

	if (n1bit == 32)
	  nempty_line++;
	else
	  empty_tile = false;

	nexcl_bit += n1bit;
	npair += 32 - n1bit;

	if (h_ientry[i].indi == h_tile_indj[j]) nexcl_bit_self += n1bit;
      }
      if (empty_tile) nempty_tile++;
      file_npair << npair << std::endl;
    }
  }

  file_npair.close();
  file_nj.close();

  unsigned int ntot_pairs = ntot*tilesize*tilesize;
  std::cout << "Total number of pairs = " << ntot_pairs << std::endl;
  std::cout << "Number of excluded pairs = " << nexcl_bit << " (" << 
    ((double)nexcl_bit*100)/(double)ntot_pairs << "%)" << std::endl;
  std::cout << "Number of excluded pairs in self (i==j) tiles = " << nexcl_bit_self << " (" << 
    ((double)nexcl_bit_self*100)/(double)ntot_pairs << "%)" << std::endl;
  std::cout << "Number of empty lines = " << nempty_line << " (" <<
    ((double)nempty_line*100)/((double)(ntot*tilesize)) << "%)" << std::endl;
  std::cout << "Number of empty tiles = " << nempty_tile << " (" <<
    ((double)nempty_tile*100)/(double)ntot << "%)" << std::endl;

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

}

//
// Load neighbor list from file
//
template <int tilesize>
void NeighborList<tilesize>::load(const char *filename) {

  ientry_t *h_ientry;
  int *h_tile_indj;
  tile_excl_t<tilesize> *h_tile_excl;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> ni >> ntot;

    h_ientry = new ientry_t[ni];
    h_tile_indj = new int[ntot];
    h_tile_excl = new tile_excl_t<tilesize>[ntot];

    for (int i=0;i < ni;i++) {
      file >> std::dec >> h_ientry[i].indi >> h_ientry[i].ish >> 
	h_ientry[i].startj >> h_ientry[i].endj;
      for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
	file >> std::dec >> h_tile_indj[j];
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  file >> std::hex >> h_tile_excl[j].excl[k];
	}
      }
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.2f);
  reallocate<int>(&tile_indj, &tile_indj_len, ntot, 1.2f);
  reallocate< tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, ntot, 1.2f);

  copy_HtoD<ientry_t>(h_ientry, ientry, ni);
  copy_HtoD<int>(h_tile_indj, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;
}

//
// Explicit instances of DirectForce
//
template class NeighborList<16>;
template class NeighborList<32>;
