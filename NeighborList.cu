#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cassert>
#include <fstream>
#include <vector>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"

// IF defined, uses strict (Factor = 1.0f) memory reallocation. Used for debuggin memory problems.
#define STRICT_MEMORY_REALLOC

static __device__ NeighborListParam_t d_nlist_param;

static int BitCount(unsigned int u)
 {
         unsigned int uCount;

         uCount = u
                  - ((u >> 1) & 033333333333)
                  - ((u >> 2) & 011111111111);
         return
           ((uCount + (uCount >> 3))
            & 030707070707) % 63;
 }

/*
static int BitCount_ref(unsigned int u) {
  unsigned int x = u;
  int res = 0;
  while (x != 0) {
    res += (x & 1);
    x >>= 1;
  }
  return res;
}
*/

//
// The entire warp enters here
// If IvsI = true, search within I zone
//
template <bool IvsI>
__device__
void get_cell_bounds_z(const int icell, const int ncell, const float minx,
		       const float x0, const float x1, const float* __restrict__ bx,
		       const float rcut, int& jcell0, int& jcell1) {

  int jcell_start_left, jcell_start_right;

  if (IvsI) {
    // Search within a single zone (I)
    if (icell < 0) {
      // This is one of the image cells on the left =>
      // set the left cell boundary (jcell0) to 1 and start looking for the right
      // boundary from 1
      jcell_start_left = -1;         // with this value, we don't look for cells on the left
      jcell_start_right = 0;         // start looking for cells at right from 0
      jcell0 = 0;                    // left boundary set to minimum value
      jcell1 = -1;                   // set to "no cells" value
    } else if (icell >= ncell) {
      // This is one of the image cells on the right =>
      // set the right cell boundary (icell1) to ncell and start looking for the left
      // boundary from ncell
      jcell_start_left = ncell-1;    // start looking for cells at left from ncell
      jcell_start_right = ncell;     // with this value, we don't look for cells on the right
      jcell0 = ncell;                // set to "no cells" value
      jcell1 = ncell-1;              // right boundary set to maximum value
    } else {
      jcell_start_left = icell - 1;
      jcell_start_right = icell + 1;
      jcell0 = icell;
      jcell1 = icell;
    }
  } else {
    // Search between two different zones
    if (bx[0] >= x1 || (bx[0] < x1 && bx[0] > x0)) {
      // j-zone is to the right of i-zone
      // => no left search, start right search from 0
      jcell_start_left = -1;
      jcell_start_right = 0;
      jcell0 = 0;
      jcell1 = -1;
    } else if (bx[ncell] <= x0 || (bx[ncell] > x0 && bx[ncell] < x1)) {
      // j-zone is to the left of i-zone
      // => no right search, start left search from ncell
      jcell_start_left = ncell-1;
      jcell_start_right = ncell;
      jcell0 = ncell;
      jcell1 = ncell-1;
    } else {
      // i-zone is between j-zones
      // => safe choice is to search the entire range
      jcell_start_left = ncell-1;
      jcell_start_right = 0;
      jcell0 = ncell-1;
      jcell1 = 0;
    }
  }

  //
  // Check cells at left, stop once the distance to the cell right boundary
  // is greater than the cutoff.
  //
  // Cell right boundary is at bx[i]
  //
  for (int j=jcell_start_left;j >= 0;j--) {
    float d = x0 - bx[j];
    if (d > rcut) break;
    jcell0 = j;
  }

  //
  // Check cells at right, stop once the distance to the cell left boundary
  // is greater than the cutoff.
  //
  // Cell left boundary is at bx[i-1]
  //
  for (int j=jcell_start_right;j < ncell;j++) {
    float bx_j = (j > 0) ? bx[j-1] : minx;
    float d = bx_j - x1;
    if (d > rcut) break;
    jcell1 = j;
  }

  // Cell bounds are jcell0:jcell1
}

//
// The entire warp enters here
// If IvsI = true, search within I zone
//
template <bool IvsI>
__device__
void get_cell_bounds_xy(const int ncell, const float minx,
			const float x0, const float x1,
			const float inv_dx, const float rcut,
			int& jcell0, int& jcell1) {

  if (IvsI) {
    // Search within a single zone (I)

    //
    // Check cells at left, stop once the distance to the cell right boundary 
    // is greater than the cutoff.
    //
    // Cell right boundary is at bx
    // portion inside i-cell is (x0-bx)
    // => what is left of rcut on the left of i-cell is rcut-(x0-bx)
    //
    //float bx = minx + icell*dx;
    //jcell0 = max(0, icell - (int)ceilf((rcut - (x0 - bx))/dx));

    //
    // Check cells at right, stop once the distance to the cell left boundary
    // is greater than the cutoff.
    //
    // Cell left boundary is at bx
    // portion inside i-cell is (bx-x1)
    // => what is left of rcut on the right of i-cell is rcut-(bx-x1)
    //
    //bx = minx + (icell+1)*dx;
    //jcell1 = min(ncell-1, icell + (int)ceilf((rcut - (bx - x1))/dx));

    // Find first left boundary that is < x0-rcut
    jcell0 = max(0, (int)floorf((x0-rcut-minx)*inv_dx));

    // Find first right boundary that is > x1+rcut
    jcell1 = min(ncell-1, (int)ceilf((x1+rcut-minx)*inv_dx) - 1);

    //
    // Take care of the boundaries:
    //
    //if (icell < 0) jcell0 = 0;
    //if (icell >= ncell) jcell1 = ncell - 1;

  } else {
    //
    // Search between zones izone and jzone
    // (x0, x1) are for izone
    // (dx, minx, ncell) are for jzone
    //

    //
    // jzone left boundaries are given by: minx + jcell*dx
    // jzone right boundaries are given by: minx + (jcell+1)*dx
    //
    // izone overlap region is: x0-rcut ... x1+rcut
    //

    // Find first left boundary that is < x0-rcut
    jcell0 = max(0, (int)floorf((x0-rcut-minx)*inv_dx));

    // Find first right boundary that is > x1+rcut
    jcell1 = min(ncell-1, (int)ceilf((x1+rcut-minx)*inv_dx) - 1);
  }

  // Cell bounds are jcell0:jcell1
      
}

//
// Finds minimum of z0 and maximum of z1 across warp using __shfl -command
//
__forceinline__ __device__ void minmax_shfl(int z0, int z1, int &z0_min, int &z1_max) {
#if __CUDA_ARCH__ >= 300
  z0_min = z0;
  z1_max = z1;
  for (int i=16;i >= 1;i/=2) {
    z0_min = min(z0_min, __shfl_xor(z0, i));
    z1_max = max(z1_max, __shfl_xor(z1, i));
  }
#endif
}

__forceinline__ __device__ int min_shfl(int val) {
#if __CUDA_ARCH__ >= 300
  for (int i=16;i >= 1;i/=2) val = min(val, __shfl_xor(val, i));
#else
  val = 0;
#endif
  return val;
}

__forceinline__ __device__ int max_shfl(int val) {
#if __CUDA_ARCH__ >= 300
  for (int i=16;i >= 1;i/=2) val = max(val, __shfl_xor(val, i));
#else
  val = 0;
#endif
  return val;
}

__forceinline__ __device__ int min_shmem(int val, const int wid, volatile int* shbuf) {
  shbuf[wid] = val;
  for (int i=16;i >= 1;i/=2) {
    int n = shbuf[i ^ wid];
    shbuf[wid] = min(shbuf[wid], n);
  }
  return shbuf[wid];
}

__forceinline__ __device__ int max_shmem(int val, const int wid, volatile int* shbuf) {
  shbuf[wid] = val;
  for (int i=16;i >= 1;i/=2) {
    int n = shbuf[i ^ wid];
    shbuf[wid] = max(shbuf[wid], n);
  }
  return shbuf[wid];
}

//
// Broadcasts value from a single lane to all lanes
//
__forceinline__ __device__ int bcast_shfl(int val, const int srclane) {
#if __CUDA_ARCH__ >= 300
  return __shfl(val, srclane);
#else
  return 0;
#endif
}

__forceinline__ __device__ int bcast_shmem(int val, const int srclane, const int wid, 
					   volatile int* shbuf) {
  if (wid == srclane) shbuf[0] = val;
  return shbuf[0];
}

#if __CUDA_ARCH__ >= 300
//
// Checks that the value of integer is the warp, used for debugging
//
__forceinline__ __device__ bool check_int(int val) {
  int val0 = bcast_shfl(val, 0);
  return __all(val == val0);
}
#endif

//
// Calculates inclusive plus scan across warp
//
__forceinline__ __device__ int incl_scan_shfl(int val, const int wid, const int scansize=warpsize) {
#if __CUDA_ARCH__ >= 300
  for (int i=1;i < scansize;i*=2) {
    int n = __shfl_up(val, i, scansize);
    if (wid >= i) val += n;
  }
#else
  val = 0;
#endif
  return val;
}

__forceinline__ __device__ int incl_scan_shmem(int val, const int wid, volatile int* shbuf,
					       const int scansize=warpsize) {
  shbuf[wid] = val;
  for (int i=1;i < scansize;i*=2) {
    int n = (wid >= i) ? shbuf[wid - i] : 0;
    shbuf[wid] += n;
  }
  return shbuf[wid];
}

//
// Calculates the sum and places the result in all threads
//
__forceinline__ __device__ int sum_shfl(int val) {
#if __CUDA_ARCH__ >= 300
  for (int i=16;i >= 1;i /= 2)
    val += __shfl_xor(val, i);
#else
  val = 0;
#endif
  return val;
}

__forceinline__ __device__ int sum_shmem(int val, const int wid, volatile int* shbuf) {
  shbuf[wid] = val;
  for (int i=16;i >= 1;i /= 2) {
    int n = shbuf[i ^ wid];
    shbuf[wid] += n;
  }
  return val;
}

//
// Calculates exclusive plus-scan across warp for binary (0 or 1) values
//
// wid = warp ID = threadIdx.x % warpsize
//
__forceinline__ __device__ int binary_excl_scan(int val, int wid) {
  return __popc( __ballot(val) & ((1 << wid) - 1) );
}

//
// Calculates reduction across warp for binary (0 or 1) values
//
__forceinline__ __device__ int binary_reduce(int val) {
  return __popc(__ballot(val));
}

//
// Calculates distance exclusion mask using a single warp
//
// exclusion bits:
// 0 = no exclusion
// 1 = exclusion
//
// wid = warp thread index (0...warpSize-1)
//
template <int tilesize>
__device__ int get_dist_excl_mask(const int wid,
				  const int istart, const int iend,
				  const int jstart, const int jend,
				  const int ish,
				  const float boxx, const float boxy, const float boxz,
				  const float rcut2,
				  const float4* __restrict__ xyzq,
				  volatile float3* __restrict__ sh_xyzi
				  ) {

  // Load atom i coordinates to shared memory
  // NOTE: volatile -keyword 
  float4 xyzq_i;

  const unsigned int load_ij = threadIdx.x % tilesize;

  if (tilesize == 32 || wid < 16) {
    if (istart + load_ij <= iend) {
      xyzq_i = xyzq[istart + load_ij];
    } else {
      xyzq_i.x = -100000000.0f;
      xyzq_i.y = -100000000.0f;
      xyzq_i.z = -100000000.0f;
    }
    sh_xyzi[load_ij].x = xyzq_i.x;
    sh_xyzi[load_ij].y = xyzq_i.y;
    sh_xyzi[load_ij].z = xyzq_i.z;
  }

  // Load atom j coordinates
  float xj, yj, zj;
  //  const unsigned int loadj = (wid + (wid/TILESIZE)*(TILESIZE-1)) % TILESIZE;
  //  const unsigned int loadj = threadIdx.x % TILESIZE;
  if (jstart + load_ij <= jend) {
    float4 xyzq_j = xyzq[jstart + load_ij];
    xj = xyzq_j.x;
    yj = xyzq_j.y;
    zj = xyzq_j.z;
  } else {
    xj = 100000000.0f;
    yj = 100000000.0f;
    zj = 100000000.0f;
  }

  int q_samecell = (istart == jstart);

  // Calculate shift
  int ish_t = ish;
  float zsh = (ish_t/9 - 1)*boxz;
  ish_t -= (ish_t/9)*9;
  float ysh = (ish_t/3 - 1)*boxy;
  ish_t -= (ish_t/3)*3;
  float xsh = (ish_t - 1)*boxx;

  xj -= xsh;
  yj -= ysh;
  zj -= zsh;
  
  unsigned int excl = 0;
  int t;
  if (tilesize == 32) {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int i = ((threadIdx.x + t) % tilesize);
      float dx = sh_xyzi[i].x - xj;
      float dy = sh_xyzi[i].y - yj;
      float dz = sh_xyzi[i].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= rcut2) | (q_samecell && (wid <= i)) ) << t;
    }

  } else {

    /*
    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int load_i = (wid + t*2 + (wid/tilesize)*(tilesize-1)) % tilesize;
      int ii = sh_start + load_i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= rcut2) | (q_samecell && (load_ij <= load_i)) ) << t;
    }
    // excl is a 8 bit exclusion mask.
    // The full 32 bit exclusion mask is contained in 4 threads:
    // thread 0 contains the lowest 8 bits
    // thread 1 contains the next 8 bits, etc..
    
    excl <<= (threadIdx.x % num_thread_per_excl)*(num_excl<tilesize>::val);
    
    // Combine excl using shared memory
    const unsigned int sh_excl_ind = (threadIdx.x/warpsize)*(num_excl<tilesize>::val) + 
      (threadIdx.x % warpsize)/num_thread_per_excl;
    
    sh_excl[sh_excl_ind] = 0;
    __syncthreads();
    
    atomicOr(&sh_excl[sh_excl_ind], excl);
    
    // Make sure shared memory is written
    __syncthreads();
    
    // index to tile_excl.excl[] (0...7)
    const unsigned int excl_ind = (threadIdx.x % warpsize)/num_thread_per_excl;
    
    tile_indj[wid] = jstart;
    
    if ((threadIdx.x % num_thread_per_excl) == 0) {
      tile_excl[wid].excl[excl_ind] = sh_excl[sh_excl_ind];
    }
    */
  }

  return excl;
}

//
// Sort atoms into z-columns
//
// col_natom[0..ncellx*ncelly-1] = number of atoms in each column
// atom_icol[istart..iend]     = column index for atoms 
//
__global__ void calc_z_column_index_kernel(const float4* __restrict__ xyzq,
					   int* __restrict__ col_natom,
					   int* __restrict__ atom_icol,
					   int3* __restrict__ col_xy_zone) {

  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  int ind0 = 0;
  for (int izone=0;izone < 8;izone++) {
    if (i < d_nlist_param.zone_patom[izone+1]) {
      float4 xyzq_val = xyzq[i];
      float x = xyzq_val.x;
      float y = xyzq_val.y;
      float3 min_xyz = d_nlist_param.min_xyz[izone];
      int ix = (int)((x - min_xyz.x)*d_nlist_param.inv_celldx[izone]);
      int iy = (int)((y - min_xyz.y)*d_nlist_param.inv_celldy[izone]);
      int ind = ind0 + ix + iy*d_nlist_param.ncellx[izone];
      atomicAdd(&col_natom[ind], 1);
      atom_icol[i] = ind;
      int3 col_xy_zone_val;
      col_xy_zone_val.x = ix;
      col_xy_zone_val.y = iy;
      col_xy_zone_val.z = izone;
      col_xy_zone[ind] = col_xy_zone_val;
      break;
    }
    ind0 += d_nlist_param.ncellx[izone]*d_nlist_param.ncelly[izone];
  }

}

//
// Computes z column position using parallel exclusive prefix sum
// Also computes the cell_patom, col_ncellz, col_cell, and ncell
//
// NOTE: Must have nblock = 1, we loop over buckets to avoid multiple kernel calls
//
template <int tilesize>
__global__ void calc_z_column_pos_kernel(const int ncol_tot,
					 const int3* __restrict__ col_xy_zone,
					 int* __restrict__ col_natom,
					 int* __restrict__ col_patom,
					 int* __restrict__ cell_patom,
					 int* __restrict__ col_ncellz,
					 int4* __restrict__ cell_xyz_zone,
					 int* __restrict__ col_cell) {
  // Shared memory
  // Requires: blockDim.x*sizeof(int2)
  extern __shared__ int2 shpos2[];

  if (threadIdx.x == 0) {
    col_patom[0] = 0;
  }

  int2 offset = make_int2(0, 0);
  for (int base=0;base < ncol_tot;base += blockDim.x) {
    int i = base + threadIdx.x;
    int2 tmpval;
    tmpval.x = (i < ncol_tot) ? col_natom[i] : 0;  // Number of atoms in this column
    tmpval.y = (i < ncol_tot) ? (tmpval.x - 1)/tilesize + 1 : 0; // Number of z-cells in this column
    if (i < ncol_tot) col_ncellz[i] = tmpval.y;    // Set col_ncellz[icol]
    shpos2[threadIdx.x] = tmpval;
    if (i < ncol_tot) col_natom[i] = 0;
    __syncthreads();

    for (int d=1;d < blockDim.x; d *= 2) {
      int2 tmp = (threadIdx.x >= d) ? shpos2[threadIdx.x-d] : make_int2(0, 0);
      __syncthreads();
      shpos2[threadIdx.x].x += tmp.x;
      shpos2[threadIdx.x].y += tmp.y;
      __syncthreads();
    }

    if (i < ncol_tot) {
      // Write col_patom in global memory
      int2 val1 = shpos2[threadIdx.x];
      val1.x += offset.x;
      val1.y += offset.y;
      col_patom[i+1] = val1.x;
      // Write cell_patom in global memory
      // OPTIMIZATION NOTE: Is this looping too slow? Should we move this into a separate kernel?
      int2 val0 = (threadIdx.x > 0) ? shpos2[threadIdx.x - 1] : make_int2(0, 0);
      val0.x += offset.x;
      val0.y += offset.y;
      int icell0 = val0.y;
      int icell1 = val1.y;
      int iatom  = val0.x;
      // Write col_cell
      col_cell[i] = icell0;
      // col_xy_zone[icol].x = x coordinate for each column
      // col_xy_zone[icol].y = y coordinate for each column
      // col_xy_zone[icol].z = zone for each column
      int4 cell_xyz_zone_val;
      int3 col_xy_zone_val = col_xy_zone[i];
      cell_xyz_zone_val.x = col_xy_zone_val.x;   // icellx
      cell_xyz_zone_val.y = col_xy_zone_val.y;   // icelly
      cell_xyz_zone_val.z = 0;                   // icellz (set in the loop below)
      cell_xyz_zone_val.w = col_xy_zone_val.z;   // izone
      for (int icell=icell0;icell < icell1;icell++,iatom+=tilesize,cell_xyz_zone_val.z++) {
	cell_patom[icell] = iatom;
	cell_xyz_zone[icell] = cell_xyz_zone_val;
      }
    }
    
    // Add the last value to the offset for the next block
    int2 lastval = shpos2[blockDim.x-1];
    offset.x += lastval.x;
    offset.y += lastval.y;

    // Sync threads so that the next iteration can start writing in shared memory
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    // Cap off cell_patom
    cell_patom[offset.y] = offset.x;
    // Write ncell into global GPU buffer
    d_nlist_param.ncell = offset.y;
    // Clear nexcl
    d_nlist_param.nexcl = 0;
  }

  // Set zone_cell = starting cell for each zone
  if (threadIdx.x < 8) {
    int icol = d_nlist_param.zone_col[threadIdx.x];
    d_nlist_param.zone_cell[threadIdx.x] = (icol < ncol_tot) ? col_cell[icol] :  d_nlist_param.ncell;
  }

}

//
// Calculates ncellz_max[izone].
//
// blockDim.x = max number of columns over all zones
// Each thread block calculates one zone (blockIdx.x = izone)
//
__global__ void calc_ncellz_max_kernel(const int* __restrict__ col_ncellz) {

  // Shared memory
  // Requires: blockDim.x*sizeof(int)
  extern __shared__ int sh_col_ncellz[];

  // ncol[izone] gives the cumulative sum of ncellx[izone]*ncelly[izone]
  int start = d_nlist_param.ncol[blockIdx.x];
  int end   = d_nlist_param.ncol[blockIdx.x+1] - 1;

  int ncellz_max = 0;

  for (;start <= end;start += blockDim.x) {
    // Load col_ncellz into shared memory
    int pos = start + threadIdx.x;
    int col_ncellz_val = 0;
    if (pos <= end) col_ncellz_val = col_ncellz[pos];
    sh_col_ncellz[threadIdx.x] = col_ncellz_val;
    __syncthreads();
      
    // Reduce
    int n = end - start;
    for (int d=1;d < n;d *= 2) {
      int t = threadIdx.x + d;
      int val = (t < n) ? sh_col_ncellz[t] : 0;
      __syncthreads();
      sh_col_ncellz[threadIdx.x] = max(sh_col_ncellz[threadIdx.x], val);
      __syncthreads();
    }
    
    // Store into register
    if (threadIdx.x == 0) ncellz_max = max(ncellz_max, sh_col_ncellz[0]);
  }

  // Write into global memory
  if (threadIdx.x == 0) d_nlist_param.ncellz_max[blockIdx.x] = ncellz_max;
}

/*
//
// Calculates celldz_min[izone], where izone = blockIdx.x = 0...7
//
__global__ void calc_celldz_min_kernel() {

  // Shared memory
  // Requires: blockDim.x*sizeof(float)
  extern __shared__ float sh_celldz_min[];

  // ncol[izone] gives the cumulative sum of ncellx[izone]*ncelly[izone]
  int start = d_nlist_param.ncell[blockIdx.x];
  int end   = d_nlist_param.ncell[blockIdx.x+1] - 1;

  float celldz_min = (float)(1.0e20);

  for (;start <= end;start += blockDim.x) {
    // Load value into shared memory
    float celldz_min_val = (float)(1.0e20);
    int pos = start + threadIdx.x;
    if (pos <= end) celldz_min_val = ;
    sh_celldz_min[threadIdx.x] = celldz_min_val;
    __synthreads();

    // Reduce
    int n = end - start;
    for (int d=1;d < n;d *= 2) {
      int t = threadIdx.x + d;
      float val = (t < n) ? sh_celldz_min[t] : (float)(1.0e20);
      __syncthreads();
      sh_celldz_min[threadIdx.x] = min(sh_celldz_min[threadIdx.x], val);
      __syncthreads();
    }

    // Store into register
    if (threadIdx.x == 0) celldz_min = min(celldz_min, sh_celldz_min[0]);
  }

  // Write into global memory
  if (threadIdx.x == 0) d_nlist_param.celldz_min[blockIdx.x] = celldz_min;

}
*/

//
// Finds the min_xyz and max_xyz for zone "izone"
//
__global__ void calc_minmax_xyz_kernel(const int ncoord, const int izone,
				       const float4* __restrict__ xyzq) {

  // Shared memory
  // Requires: 6*blockDim.x*sizeof(float)
  extern __shared__ float sh_minmax_xyz[];
  volatile float* sh_min_x = &sh_minmax_xyz[0];
  volatile float* sh_min_y = &sh_minmax_xyz[blockDim.x];
  volatile float* sh_min_z = &sh_minmax_xyz[blockDim.x*2];
  volatile float* sh_max_x = &sh_minmax_xyz[blockDim.x*3];
  volatile float* sh_max_y = &sh_minmax_xyz[blockDim.x*4];
  volatile float* sh_max_z = &sh_minmax_xyz[blockDim.x*5];

  // Load data into shared memory
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  float4 xyzq_i = xyzq[min(i,ncoord-1)];
  float x = xyzq_i.x;
  float y = xyzq_i.y;
  float z = xyzq_i.z;
  sh_min_x[threadIdx.x] = x;
  sh_min_y[threadIdx.x] = y;
  sh_min_z[threadIdx.x] = z;
  sh_max_x[threadIdx.x] = x;
  sh_max_y[threadIdx.x] = y;
  sh_max_z[threadIdx.x] = z;
  __syncthreads();

  // Reduce
  for (int d=1;d < blockDim.x;d *= 2) {
    int t = threadIdx.x + d;
    float min_x = (t < blockDim.x) ? sh_min_x[t] : (float)(1.0e20);
    float min_y = (t < blockDim.x) ? sh_min_y[t] : (float)(1.0e20);
    float min_z = (t < blockDim.x) ? sh_min_z[t] : (float)(1.0e20);
    float max_x = (t < blockDim.x) ? sh_max_x[t] : (float)(-1.0e20);
    float max_y = (t < blockDim.x) ? sh_max_y[t] : (float)(-1.0e20);
    float max_z = (t < blockDim.x) ? sh_max_z[t] : (float)(-1.0e20);
    __syncthreads();
    sh_min_x[threadIdx.x] = min(sh_min_x[threadIdx.x], min_x);
    sh_min_y[threadIdx.x] = min(sh_min_y[threadIdx.x], min_y);
    sh_min_z[threadIdx.x] = min(sh_min_z[threadIdx.x], min_z);
    sh_max_x[threadIdx.x] = max(sh_max_x[threadIdx.x], max_x);
    sh_max_y[threadIdx.x] = max(sh_max_y[threadIdx.x], max_y);
    sh_max_z[threadIdx.x] = max(sh_max_z[threadIdx.x], max_z);
    __syncthreads();
  }

  // Store into global memory
  if (threadIdx.x == 0) {
    atomicMin(&d_nlist_param.min_xyz[izone].x, sh_min_x[0]);
    atomicMin(&d_nlist_param.min_xyz[izone].y, sh_min_y[0]);
    atomicMin(&d_nlist_param.min_xyz[izone].z, sh_min_z[0]);
    atomicMax(&d_nlist_param.max_xyz[izone].x, sh_max_x[0]);
    atomicMax(&d_nlist_param.max_xyz[izone].y, sh_max_y[0]);
    atomicMax(&d_nlist_param.max_xyz[izone].z, sh_max_z[0]);
  }

}

//
// Re-order atoms according to pos. Non-deterministic version (because of atomicAdd())
//
__global__ void reorder_atoms_z_column_kernel(const int ncoord,
					      const int* atom_icol,
					      int* col_natom,
					      const int* col_patom,
					      const float4* __restrict__ xyzq_in,
					      float4* __restrict__ xyzq_out,
					      int* __restrict__ ind_sorted) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < ncoord) {
    // Column index
    int icol = atom_icol[i];
    int pos = col_patom[icol];
    int n = atomicAdd(&col_natom[icol], 1);
    // new position = pos + n
    int newpos = pos + n;
    ind_sorted[newpos] = i;
    xyzq_out[newpos] = xyzq_in[i];
  }

}

//
// Reorders loc2glo
//
__global__ void build_loc2glo_kernel(const int ncoord,
				     const int* __restrict__ ind_sorted,
				     const int* __restrict__ loc2glo_in,
				     int* __restrict__ loc2glo_out) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < ncoord) {
    int j = ind_sorted[i];
    loc2glo_out[i] = loc2glo_in[j];
  }
}


//
// Builds glo2loc using loc2glo
//
__global__ void build_glo2loc_kernel(const int ncoord,
				     const int* __restrict__ loc2glo,
				     int* __restrict__ glo2loc) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < ncoord) {
    int ig = loc2glo[i];
    glo2loc[ig] = i;
  }
}

//
// Builds atom_pcell. Single warp takes care of single cell
//
__global__ void build_atom_pcell_kernel(const int* __restrict__ cell_patom,
					int* __restrict__ atom_pcell) {
  const int icell = (threadIdx.x + blockIdx.x*blockDim.x)/warpsize;
  const int wid = threadIdx.x % warpsize;

  if (icell < d_nlist_param.ncell) {
    int istart = cell_patom[icell];
    int iend   = cell_patom[icell+1] - 1;
    if (istart + wid <= iend) atom_pcell[istart + wid] = icell;
  }

}

//
// Sorts atoms according to z coordinate
//
// Uses bitonic sort, see:
// http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
//
// Each thread block sorts a single z column.
// Each z-column can only have up to blockDim.x number of atoms
//
struct keyval_t {
  float key;
  int val;
};
__global__ void sort_z_column_kernel(const int* __restrict__ col_patom,
				     float4* __restrict__ xyzq,
				     int* __restrict__ ind_sorted) {

  // Shared memory
  // Requires: blockDim.x*sizeof(keyval_t)
  extern __shared__ keyval_t sh_keyval[];

  int col_patom0 = col_patom[blockIdx.x];
  int n = col_patom[blockIdx.x+1] - col_patom0;

  // Read keys and values into shared memory
  keyval_t keyval;
  keyval.key = (threadIdx.x < n) ? xyzq[threadIdx.x + col_patom0].z : 1.0e38;
  keyval.val = (threadIdx.x < n) ? (threadIdx.x + col_patom0) : (n-1);
  sh_keyval[threadIdx.x] = keyval;
  __syncthreads();

  for (int k = 2;k <= blockDim.x;k *= 2) {
    for (int j = k/2; j > 0;j /= 2) {
      int ixj = threadIdx.x ^ j;
      if (ixj > threadIdx.x && ixj < blockDim.x) {
	// asc = true for ascending order
	bool asc = ((threadIdx.x & k) == 0);
	
	// Read data
	keyval_t keyval1 = sh_keyval[threadIdx.x];
	keyval_t keyval2 = sh_keyval[ixj];
	
	float lo_key = asc ? keyval1.key : keyval2.key;
	float hi_key = asc ? keyval2.key : keyval1.key;
	
	if (lo_key > hi_key) {
	  // keys are in wrong order => exchange
	  sh_keyval[threadIdx.x] = keyval2;
	  sh_keyval[ixj]         = keyval1;
	}
	
	//if ((i&k)==0 && get(i)>get(ixj)) exchange(i,ixj);
	//if ((i&k)!=0 && get(i)<get(ixj)) exchange(i,ixj);
      }
      __syncthreads();
    }
  }

  // sh_keyval[threadIdx.x].val gives the mapping:
  //
  // xyzq_new[threadIdx.x + col_patom0]        = xyzq[sh_keyval[threadIdx.x].val]
  // loc2glo_new[threadIdx.x + col_patom0] = loc2glo[sh_keyval[threadIdx.x].val]
  //

  float4 xyzq_val;
  int ind_val;
  if (threadIdx.x < n) {
    int i = sh_keyval[threadIdx.x].val;    
    ind_val = ind_sorted[i];
    xyzq_val = xyzq[i];
  }
  __syncthreads();
  if (threadIdx.x < n) {
    int newpos = threadIdx.x + col_patom0;
    ind_sorted[newpos] = ind_val;
    xyzq[newpos] = xyzq_val;
  }

}

//
// Calculates bounding box (bb) and cell z-boundaries (cell_bz)
// NOTE: Each thread calculates one bounding box
//
template <int tilesize>
__global__ void calc_bb_cell_bz_kernel(const int* __restrict__ cell_patom,
				       const float4* __restrict__ xyzq,
				       bb_t* __restrict__ bb,
				       float* __restrict__ cell_bz) {

  const int icell = threadIdx.x + blockIdx.x*blockDim.x;

  if (icell < d_nlist_param.ncell) {
    int istart = cell_patom[icell];
    int iend   = cell_patom[icell+1] - 1;
    float4 xyzq_val = xyzq[istart];
    float minx = xyzq_val.x;
    float miny = xyzq_val.y;
    float minz = xyzq_val.z;
    float maxx = xyzq_val.x;
    float maxy = xyzq_val.y;
    float maxz = xyzq_val.z;

    for (int i=istart+1;i <= iend;i++) {
      xyzq_val = xyzq[i];
      minx = min(minx, xyzq_val.x);
      miny = min(miny, xyzq_val.y);
      minz = min(minz, xyzq_val.z);
      maxx = max(maxx, xyzq_val.x);
      maxy = max(maxy, xyzq_val.y);
      maxz = max(maxz, xyzq_val.z);
    }
    // Set the cell z-boundary equal to the z-coordinate of the last atom
    cell_bz[icell] = xyzq_val.z;
    bb_t bb_val;
    bb_val.x = 0.5f*(minx + maxx);
    bb_val.y = 0.5f*(miny + maxy);
    bb_val.z = 0.5f*(minz + maxz);
    bb_val.wx = 0.5f*(maxx - minx);
    bb_val.wy = 0.5f*(maxy - miny);
    bb_val.wz = 0.5f*(maxz - minz);
    bb[icell] = bb_val;
  }
  
  // Zero n_ientry and n_tile -counters in preparation for neighbor list build
  if (icell == 0) {
    d_nlist_param.n_ientry = 0;
    d_nlist_param.n_tile = 0;
    //d_nlist_param.tmp = 0;
  }

}

//#define INLINE_OFF

//
//
//
template <int tilesize>
#ifndef INLINE_OFF
__forceinline__
#endif
__device__ void flush_atomj(const int wid, const int istart,
			    volatile int* __restrict__ sh_jlist,
			    const int* __restrict__ cell_patom,
			    const int min_atomj, const int max_atomj,
			    const int n_atomj, volatile int* __restrict__ sh_atomj,
			    const int min_excl_atom, const int max_excl_atom,
			    const int n_excl_atom, const int* __restrict__ excl_atom,
			    const int jtile_start,
			    tile_excl_t<tilesize>* __restrict__ tile_excl) {
  if ((min_atomj <= max_excl_atom) && (max_atomj >= min_excl_atom)) {
    int atomj = (wid < n_atomj) ? (sh_atomj[wid] >> n_jlist_max_shift) : -1;
    for (int ibase=0;ibase < n_excl_atom;ibase+=warpsize) {
      int i = ibase + wid;
      // Load excluded atom from global memory and check if there are any possible exclusions
      int excl_atomi = (i < n_excl_atom) ? (excl_atom[i] >> 5) : -1;
      int has_excl = __ballot((excl_atomi >= min_atomj) && (excl_atomi <= max_atomj));
      // Loop through possible exclusions
      while (has_excl) {
	// Get bit position for the exclusion
	int bitpos = __ffs(has_excl) - 1;
	i = ibase + bitpos;
	excl_atomi = excl_atom[i];
	// Check excl_atomi vs. sh_atomj[0...warpsize-1]
	if ((excl_atomi >> 5) == atomj) {
	  // Thread wid found exclusion between atomj and (excl_atomi & 31)
	  // NOTE: Only a single thread per warp enters here
	  int i_jlist = (sh_atomj[wid] & n_jlist_max_mask);
	  int jtile = jtile_start + i_jlist;
	  int jcell = sh_jlist[i_jlist];
	  int jstart = cell_patom[jcell];
	  int excl_ind  = atomj - jstart;
	  int excl_shift = ( (excl_atomi & 31) - excl_ind + tilesize) % tilesize;
	  unsigned int excl_mask = 1 << excl_shift;
	  tile_excl[jtile].excl[excl_ind] |= excl_mask;
	}
	// Remove bit from has_excl
	has_excl ^= (1 << bitpos);
      }
    }
  }
}
#undef INLINE_OFF

//
// Flush jlist into global memory
//
template <int tilesize>
__device__ void flush_jlist(const int wid, const int istart, const int iend,
			    const int n_jlist, volatile int* __restrict__ sh_jlist,
			    const int ish,
			    const float rcut2, const float xi, const float yi, const float zi,
			    const float4* __restrict__ xyzq,
			    const int* __restrict__ cell_patom,
			    volatile int* __restrict__ sh_atomj,
			    const int min_excl_atom, const int max_excl_atom,
			    const int n_excl_atom, const int* __restrict__ excl_atom,
			    int* __restrict__ tile_indj,
			    tile_excl_t<tilesize>* __restrict__ tile_excl,
			    ientry_t* __restrict__ ientry
#if __CUDA_ARCH__ < 300
			    ,volatile int* __restrict__ shflmem,
			    volatile float3* __restrict__ sh_xyzj
#endif
			    ) {

  // Allocate space on the global tile_excl and tile_indj -lists
  // NOTE: we are allocating space for n_jlist entries. However, not all of these are used
  //       because some of the i-j tiles will be empty. If we don't want to keep these
  //       "ghost" tiles in the list, we need to setup another shared memory buffer for
  //       exclusion masks and then only add the tiles that are non-empty.
  int jtile_start;
  if (wid == 0) jtile_start = atomicAdd(&d_nlist_param.n_tile, n_jlist);
#if __CUDA_ARCH__ >= 300
  jtile_start = bcast_shfl(jtile_start, 0);
#else
  jtile_start = bcast_shmem(jtile_start, 0, wid, shflmem);
#endif

  int min_atomj = 1 << 30;
  int max_atomj = 0;
  int n_atomj = 0;
  int n_jlist_new = 0;
  // Loop through j-cells
  for (int i_jlist=0;i_jlist < n_jlist;i_jlist++) {
    int jcell = sh_jlist[i_jlist];
    //if (jcell < 0 || jcell >= d_nlist_param.ncell) atomicOr(&d_nlist_param.tmp, jcell);
    int jstart = cell_patom[jcell];
    int jend   = cell_patom[jcell + 1] - 1;

    //---------------------------------------------------------------------------------------
    //
    // Exclusion check with jcell
    //

    // Load j-cell atoms
    float4 xyzq_j;
    if (jstart + wid <= jend) xyzq_j = xyzq[jstart + wid];
#if __CUDA_ARCH__ >= 300
    float xj = xyzq_j.x;
    float yj = xyzq_j.y;
    float zj = xyzq_j.z;
#else
    sh_xyzj[wid].x = xyzq_j.x;
    sh_xyzj[wid].y = xyzq_j.y;
    sh_xyzj[wid].z = xyzq_j.z;
#endif

    bool first = true;
    for (int j=0;j <= jend-jstart;j++) {
#if __CUDA_ARCH__ >= 300
      float xt = __shfl(xj, j);
      float yt = __shfl(yj, j);
      float zt = __shfl(zj, j);
#else
      float xt = sh_xyzj[j].x;
      float yt = sh_xyzj[j].y;
      float zt = sh_xyzj[j].z;
#endif
      float dx = xi - xt;
      float dy = yi - yt;
      float dz = zi - zt;
      
      float r2 = dx*dx + dy*dy + dz*dz;

      if (__any((r2 < rcut2))) {

	if (first) {
	  first = false;
	  // ----------------------------
	  // Set initial exclusion masks
	  // ----------------------------
	  int jtile = jtile_start + n_jlist_new;
	  // NOTE: In case i,j cells are less than tilesize atoms, add exclusions
	  int ni = (iend-istart+1);
	  unsigned int mask = (jstart + wid <= jend) ? 0 : 0xffffffff;   // j contribution
	  int up = (ni >= wid) ? ni-wid : tilesize + ni-wid;
	  int dw = (wid >= ni) ? wid-ni : tilesize + wid-ni;
	  unsigned int imask = (1 << (tilesize-ni)) - 1;
	  mask |= (imask << up) | (imask >> dw);                // i contribution
	  // Diagonal tile, exclude i >= j
	  if (istart == jstart) {
	    mask |= (0xffffffff >> wid);
	  }
	  tile_excl[jtile].excl[wid] = mask;
	  // --------------------------
	  // Keep in sh_jlist
	  // --------------------------
	  if (wid == 0) tile_indj[jtile] = jstart;
	  // Re-store jcell so that flush_atomj can read it off
	  sh_jlist[n_jlist_new] = jcell;
	  n_jlist_new++;
	}

	// This j-atom is within rcut of one of the i-atoms => add to exclusion check list
	// Add j-atom to the exclusion check list
	int atomj = jstart + j;
	min_atomj = min(min_atomj, atomj);
	max_atomj = max(max_atomj, atomj);
	sh_atomj[n_atomj++] = (atomj << n_jlist_max_shift) | (n_jlist_new-1);
	
	// Check sh_atomj[0...warpsize-1] for exclusions with any
	// of the i atoms in excl_atom[0...n_excl_atom-1]
	if (n_atomj == warpsize) {
	  // Check for topological exclusions
	  flush_atomj<tilesize>(wid, istart, sh_jlist, cell_patom,
				min_atomj, max_atomj, n_atomj, sh_atomj,
				min_excl_atom, max_excl_atom, n_excl_atom, excl_atom,
				jtile_start, tile_excl);
	  min_atomj = 1 << 30;
	  max_atomj = 0;
	  n_atomj = 0;
	} // if (natomj == warpsize)
      } // if (__any((r2 < rcut2)))
    } // for (int j=0;j <= jend-jstart;j++)

    //---------------------------------------------------------------------------------------

  } // for (int i_jlist=0;i_jlist < n_jlist;i_jlist++)

  if (n_atomj > 0) {
    flush_atomj<tilesize>(wid, istart, sh_jlist, cell_patom,
			  min_atomj, max_atomj, n_atomj, sh_atomj,
			  min_excl_atom, max_excl_atom, n_excl_atom, excl_atom,
			  jtile_start, tile_excl);
  }

  // Add to ientry list in global memory
  if (wid == 0) {
    int ientry_ind = atomicAdd(&d_nlist_param.n_ientry, 1);
    int jtile_end = jtile_start + n_jlist_new - 1;
    ientry_t ientry_val;
    ientry_val.indi    = istart;
    ientry_val.ish     = ish;
    ientry_val.startj  = jtile_start;
    ientry_val.endj    = jtile_end;
    ientry[ientry_ind] = ientry_val;
  }

}

//
// Build neighborlist for one zone at the time
//
// NOTE: One warp takes care of one cell
//
//template < int tilesize, bool IvsI >
template < int tilesize >
__global__
void build_kernel(const int maxNumExcl,
		  const int4* __restrict__ cell_xyz_zone,
		  const int* __restrict__ col_ncellz,
		  const int* __restrict__ col_cell,
		  const float* __restrict__ cell_bz,
		  const int* __restrict__ cell_patom,
		  const int* __restrict__ loc2glo,
		  const int* __restrict__ glo2loc,
		  const int* __restrict__ atom_excl_pos,
		  const int* __restrict__ atom_excl,
		  const float4* __restrict__ xyzq,
		  const float boxx, const float boxy, const float boxz,
		  const float rcut, const float rcut2,
		  const bb_t* __restrict__ bb,
		  int* __restrict__ excl_atom_heap,
		  int* __restrict__ tile_indj,
		  tile_excl_t<tilesize>* __restrict__ tile_excl,
		  ientry_t* __restrict__ ientry) {

  // Shared memory
  extern __shared__ char shbuf[];

  // Index of the i-cell
  const int icell = (threadIdx.x + blockIdx.x*blockDim.x)/warpsize;
  // Warp index
  const int wid = threadIdx.x % warpsize;

  if (icell >= d_nlist_param.ncell) return;

  // Get (icellx, icelly, icellz, izone):
  int4 icell_xyz_zone = cell_xyz_zone[icell];
  //int icellx = icell_xyz_zone.x;
  //int icelly = icell_xyz_zone.y;
  int icellz = icell_xyz_zone.z;
  //int izone  = IvsI ? 0 : icell_xyz_zone.w;
  int izone  = icell_xyz_zone.w;
  bool IvsI = (izone == 0) ? true : false;

  int n_jzone = IvsI ? 1 : d_nlist_param.n_int_zone[izone];
  
  if (n_jzone == 0) return;

  // Load bounding box
  bb_t ibb = bb[icell];

  // ----------------------------------------------------------------
  // Calculate shared memory pointers:
  //
  // Total memory requirement:
  // (blockDim.x/warpsize)*( (!IvsI)*n_jzone*sizeof(int2) + n_jlist_max*sizeof(int) 
  //                         + tilesize*sizeof(float3))
  //
  // Required space:
  // shflmem:         blockDim.x*sizeof(int)                           (Only for __CUDA_ARCH__ < 300)  
  // sh_jcellxy_min:  (blockDim.x/warpsize)*n_jzone*sizeof(int2)       (Only for IvsI = false)
  // sh_jlist:        (blockDim.x/warpsize)*n_jlist_max*sizeof(int)
  // sh_xyzj:         (blockDim.x/warpsize)*tilesize*sizeof(float3)    (Only for __CUDA_ARCH__ < 300)
  // sh_atomj:        blockDim.x*sizeof(int)
  //
  // NOTE: Each warp has its own sh_jcellxy_min[]
  int shbuf_pos = 0;
#if __CUDA_ARCH__ < 300
  // Shuffle memory buffer
  volatile int* shflmem = (int *)&shbuf[(threadIdx.x/warpsize)*warpsize*sizeof(int)];
  shbuf_pos += blockDim.x*sizeof(int);
  // j coordinates (x, y, z) for flush_jlist
  volatile float3* sh_xyzj = (float3 *)&shbuf[shbuf_pos + 
					      (threadIdx.x/warpsize)*tilesize*sizeof(float3)];
  shbuf_pos += (blockDim.x/warpsize)*tilesize*sizeof(float3);
#endif

  // jcellx and jcelly minimum values
  volatile int2 *sh_jcellxy_min;
  //if (!IvsI) {
    sh_jcellxy_min = (int2 *)&shbuf[shbuf_pos + 
				    (threadIdx.x/warpsize)*n_jzone*sizeof(int2)];
    shbuf_pos += (blockDim.x/warpsize)*n_jzone*sizeof(int2);
    //}

  // Temporary j-cell list. Each warp has its own jlist
  volatile int *sh_jlist = (int *)&shbuf[shbuf_pos +
					 (threadIdx.x/warpsize)*n_jlist_max*sizeof(int)];
  shbuf_pos += (blockDim.x/warpsize)*n_jlist_max*sizeof(int);

  // j atoms for flush_jlist
  volatile int* sh_atomj = (int *)&shbuf[shbuf_pos + 
					 (threadIdx.x/warpsize)*warpsize*sizeof(int)];
  shbuf_pos += blockDim.x*sizeof(int);
  // ----------------------------------------------------------------

  for (int ii=0;ii < n_jlist_max;ii++) sh_jlist[ii] = -1;

  //
  // Load exclusions for atoms in icell
  //

  // Allocate space for exclusions in global memory
  // Each warp (icell) has tilesize*maxNumExcl amount of space
  int* __restrict__ excl_atom = &excl_atom_heap[icell*tilesize*maxNumExcl];

  int istart = cell_patom[icell];
  int iend   = cell_patom[icell+1] - 1;
  int iatom = istart + wid;
  int jstart = 0;
  int jend = -1;
  float4 xyzq_i;
  if (iatom <= iend) {
    int ig = loc2glo[iatom];
    jstart = atom_excl_pos[ig];
    jend   = atom_excl_pos[ig+1] - 1;
    xyzq_i = xyzq[istart + wid];
  }
  float xi = xyzq_i.x;
  float yi = xyzq_i.y;
  float zi = xyzq_i.z;
  int jlen = jend - jstart + 1;
#if __CUDA_ARCH__ >= 300
  int pos = incl_scan_shfl(jlen, wid);
#else
  int pos = incl_scan_shmem(jlen, wid, shflmem);
#endif
  // Get the total number of excluded atoms by broadcasting the last value
  // across all threads in the warp
#if __CUDA_ARCH__ >= 300
  int n_excl_atom = bcast_shfl(pos, warpsize-1);
#else
  int n_excl_atom = bcast_shmem(pos, warpsize-1, wid, shflmem);
#endif
  // Get the exclusive sum position
  pos -= jlen;
  // Loop through excluded atoms:
  // Find min and max indices
  // Store atom indices to excl_atom -buffer
  int min_excl_atom = (1 << 30);                    // (= big number)
  int max_excl_atom = 0;
  int nexcl = 0;
  for (int jatom=jstart;jatom <= jend;jatom++) {
    int atom = glo2loc[atom_excl[jatom]];
    // Atoms that are not on this node are marked in glo2loc[] by value -1
    if (atom >= 0) {
      min_excl_atom = min(min_excl_atom, atom);
      max_excl_atom = max(max_excl_atom, atom);
    }
    // Store excluded atom index (atom) and atom i index
    excl_atom[pos + nexcl++] = (atom << 5) | wid;
  }
  // Reduce min_excl_atom and max_excl_atom across the warp
#if __CUDA_ARCH__ >= 300
  min_excl_atom = min_shfl(min_excl_atom);
  max_excl_atom = max_shfl(max_excl_atom);
#else
  min_excl_atom = min_shmem(min_excl_atom, wid, shflmem);
  max_excl_atom = max_shmem(max_excl_atom, wid, shflmem);
#endif

  for (int imx=d_nlist_param.imx_lo;imx <= d_nlist_param.imx_hi;imx++) {
    float imbbx0 = ibb.x + imx*boxx;
    int n_jcellx = 0;
    int jcellx_min, jcellx_max;
    if (IvsI) {
      get_cell_bounds_xy<true>(d_nlist_param.ncellx[0], d_nlist_param.min_xyz[0].x,
			       imbbx0-ibb.wx, imbbx0+ibb.wx,
			       d_nlist_param.inv_celldx[0], rcut, jcellx_min, jcellx_max);
      n_jcellx = max(0, jcellx_max - jcellx_min + 1);
      if (n_jcellx == 0) continue;
    } else {
      if (wid < n_jzone) {
	int jzone = d_nlist_param.int_zone[izone][wid];
	int jcellx0_t, jcellx1_t;
	get_cell_bounds_xy<false>(d_nlist_param.ncellx[jzone], d_nlist_param.min_xyz[jzone].x,
				  imbbx0-ibb.wx, imbbx0+ibb.wx,
				  d_nlist_param.inv_celldx[jzone], rcut, jcellx0_t, jcellx1_t);
	n_jcellx = max(0, jcellx1_t-jcellx0_t+1);
	sh_jcellxy_min[wid].x = jcellx0_t;
      }
      if (__all(n_jcellx == 0)) continue;
    }
    
    for (int imy=d_nlist_param.imy_lo;imy <= d_nlist_param.imy_hi;imy++) {
      float imbby0 = ibb.y + imy*boxy;
      int n_jcelly = 0;
      int jcelly_min, jcelly_max;
      if (IvsI) {
	get_cell_bounds_xy<true>(d_nlist_param.ncelly[0], d_nlist_param.min_xyz[0].y,
				 imbby0-ibb.wy, imbby0+ibb.wy,
				 d_nlist_param.inv_celldy[0], rcut, jcelly_min, jcelly_max);
	n_jcelly = max(0, jcelly_max - jcelly_min + 1);
	if (n_jcelly == 0) continue;
      } else {
	if (wid < n_jzone) {
	  int jzone = d_nlist_param.int_zone[izone][wid];
	  int jcelly0_t, jcelly1_t;
	  get_cell_bounds_xy<false>(d_nlist_param.ncelly[jzone], d_nlist_param.min_xyz[jzone].y,
				    imbby0-ibb.wy, imbby0+ibb.wy,
				    d_nlist_param.inv_celldy[jzone], rcut, jcelly0_t, jcelly1_t);
	  n_jcelly = max(0, jcelly1_t-jcelly0_t+1);
	  sh_jcellxy_min[wid].y = jcelly0_t;
	}
	if (__all(n_jcelly == 0)) continue;
      }

      for (int imz=d_nlist_param.imz_lo;imz <= d_nlist_param.imz_hi;imz++) {
	float imbbz0 = ibb.z + imz*boxz;
	int ish = imx+1 + 3*(imy+1 + 3*(imz+1));

	float imxi = xi + imx*boxx;
	float imyi = yi + imy*boxy;
	float imzi = zi + imz*boxz;

	int jzone_counter;
	if (!IvsI) jzone_counter = 0;
	do {
	  int n_jlist = 0;
	  int n_jcellx_t = n_jcellx;
	  int n_jcelly_t = n_jcelly;
	  int jzone;
	  if (!IvsI) {
#if __CUDA_ARCH__ >= 300
	    n_jcellx_t = bcast_shfl(n_jcellx_t, jzone_counter);
	    n_jcelly_t = bcast_shfl(n_jcelly_t, jzone_counter);
#else
	    n_jcellx_t = bcast_shmem(n_jcellx_t, jzone_counter, wid, shflmem);
	    n_jcelly_t = bcast_shmem(n_jcelly_t, jzone_counter, wid, shflmem);
#endif
	    jcellx_min = sh_jcellxy_min[jzone_counter].x;
	    jcelly_min = sh_jcellxy_min[jzone_counter].y;
	    jzone = d_nlist_param.int_zone[izone][jzone_counter];
	  }
	  int total_xy = n_jcellx_t*n_jcelly_t;
	  if (total_xy > 0) {
	    int jcellz_min = 1<<30;
	    int jcellz_max = 0;
	    for (int ibase=0;ibase < total_xy;ibase+=warpsize) {
	      int i = ibase + wid;
	      // Find new (jcellz0_t, jcellz1_t) -range
	      int jcellz0_t = 1<<30;
	      int jcellz1_t = 0;
	      if (i < total_xy) {
		int jcelly = i/n_jcellx_t;
		int jcellx = i - jcelly*n_jcellx_t;
		jcellx += jcellx_min;
		jcelly += jcelly_min;
		int jcol = jcellx + jcelly*d_nlist_param.ncellx[IvsI ? 0 : jzone] + 
		  (IvsI ? 0 : d_nlist_param.zone_col[jzone]);
		// jcell0 = beginning of cells for column jcol
		int jcell0 = col_cell[jcol];
		if (IvsI) {
		  get_cell_bounds_z<true>(icellz + imz*col_ncellz[jcol], col_ncellz[jcol],
					  d_nlist_param.min_xyz[IvsI ? 0 : jzone].z,
					  imbbz0-ibb.wz, imbbz0+ibb.wz,
					  &cell_bz[jcell0], rcut, jcellz0_t, jcellz1_t);
		} else {
		  get_cell_bounds_z<false>(icellz + imz*col_ncellz[jcol], col_ncellz[jcol],
					   d_nlist_param.min_xyz[IvsI ? 0 : jzone].z,
					   imbbz0-ibb.wz, imbbz0+ibb.wz,
					   &cell_bz[jcell0], rcut, jcellz0_t, jcellz1_t);
		}
	      } // if (i < total_xy)
	      jcellz_min = min(jcellz_min, jcellz0_t);
	      jcellz_max = max(jcellz_max, jcellz1_t);
	    } // for (int ibase...)

	    // Here all threads have their own (jcellz_min, jcellz_max),
	    // find the minimum and maximum among all threads:
#if __CUDA_ARCH__ >= 300
	    jcellz_min = min_shfl(jcellz_min);
	    jcellz_max = max_shfl(jcellz_max);
#else
	    jcellz_min = min_shmem(jcellz_min, wid, shflmem);
	    jcellz_max = max_shmem(jcellz_max, wid, shflmem);
#endif

	    int n_jcellz_max = jcellz_max - jcellz_min + 1;
	    int total_xyz = total_xy*n_jcellz_max;

	    if (total_xyz > 0) {

	      //
	      // Final loop that goes through the cells
	      //
	      // Cells are ordered in (y, x, z). (i.e. z first, x second, y third)
	      //

	      for (int ibase=0;ibase < total_xyz;ibase+=warpsize) {
		int i = ibase + wid;
		int ok = 0;
		int jcell;
		if (i < total_xyz) {
		  // Calculate (jcellx, jcelly, jcellz)
		  int it = i;	    
		  int jcelly = it/(n_jcellx_t*n_jcellz_max);
		  it -= jcelly*(n_jcellx_t*n_jcellz_max);
		  int jcellx = it/n_jcellz_max;
		  int jcellz = it - jcellx*n_jcellz_max;
		  jcellx += jcellx_min;
		  jcelly += jcelly_min;
		  jcellz += jcellz_min;
		  // Calculate column index "jcol" and final cell index "jcell"
		  int jcol = jcellx + jcelly*d_nlist_param.ncellx[IvsI ? 0 : jzone] + 
		    (IvsI ? 0 : d_nlist_param.zone_col[jzone]);
		  jcell = col_cell[jcol] + jcellz;
		  // NOTE: jcellz can be out of bounds here, so we need to check
		  if ( ((IvsI && (icell <= jcell)) || !IvsI) && jcellz >= 0 && jcellz < col_ncellz[jcol]) {
		    // Read bounding box for j-cell
		    bb_t jbb = bb[jcell];
		    // Calculate distance between i-cell and j-cell bounding boxes
		    float dx = max(0.0f, fabsf(imbbx0 - jbb.x) - ibb.wx - jbb.wx);
		    float dy = max(0.0f, fabsf(imbby0 - jbb.y) - ibb.wy - jbb.wy);
		    float dz = max(0.0f, fabsf(imbbz0 - jbb.z) - ibb.wz - jbb.wz);
		    float r2 = dx*dx + dy*dy + dz*dz;
		    ok = (r2 < rcut2);
		  }
		} // if (i < total_xyz)
		//
		// Add j-cells into temporary list (in shared memory)
		//
		// First reduce to calculate position for each thread in warp
		int pos = binary_excl_scan(ok, wid);
		int n_jlist_add = binary_reduce(ok);

		//#define DO_NOT_FLUSH
#ifndef DO_NOT_FLUSH
		// Flush if the sh_jlist[] buffer would become full
		if ((n_jlist + n_jlist_add) > n_jlist_max) {
		  flush_jlist<tilesize>(wid, istart, iend, n_jlist, sh_jlist, ish,
					rcut2, imxi, imyi, imzi, xyzq, cell_patom,
					sh_atomj,
					min_excl_atom, max_excl_atom, n_excl_atom, excl_atom,
					tile_indj, tile_excl, ientry
#if __CUDA_ARCH__ < 300
					,shflmem, sh_xyzj
#endif
					);
		  n_jlist = 0;
		}


		// Add to list
		if (ok) sh_jlist[n_jlist + pos] = jcell;
		n_jlist += n_jlist_add;

#endif
	      } // for (int ibase...)

#ifndef DO_NOT_FLUSH
	      if (n_jlist > 0) {
		flush_jlist<tilesize>(wid, istart, iend, n_jlist, sh_jlist, ish,
				      rcut2, imxi, imyi, imzi, xyzq, cell_patom,
				      sh_atomj,
				      min_excl_atom, max_excl_atom, n_excl_atom, excl_atom,
				      tile_indj, tile_excl, ientry
#if __CUDA_ARCH__ < 300
				      ,shflmem, sh_xyzj
#endif
				      );
	      }
#endif
#undef DO_NOT_FLUSH
	    } // if (total_xyz > 0)
	  } // if (total_xy > 0)

	  if (!IvsI) jzone_counter++;
	} while (!IvsI && (jzone_counter < n_jzone));

      } // for (int imz=imz_lo;imz <= imz_hi;imz++)
    } // for (int imy=imy_lo;imy <= imy_hi;imy++)
  } // for (int imx=imx_lo;imx <= imx_hi;imx++)

}

//----------------------------------------------------------------------------------------
//
// Builds tilex exclusion mask from ijlist[] based on distance and index
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
// Uses 32 threads to calculate the distances for a single ijlist -entry.
//
const int nwarp_build_excl_dist = 8;

template < int tilesize >
__global__ void build_excl_kernel(const unsigned int base_tid, const int n_ijlist,
				  const int3 *ijlist,
				  const int *cell_patom, const float4 *xyzq,
				  int *tile_indj,
				  tile_excl_t<tilesize> *tile_excl,
				  const float boxx, const float boxy, const float boxz,
				  const float rcut2) {
  const int num_thread_per_excl = (32/(num_excl<tilesize>::val));

  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x + base_tid;
  // Global warp index
  const unsigned int wid = gtid / warpsize;
  // Local thread index (0...warpsize-1)
  const unsigned int tid = gtid % warpsize;
  // local thread index (0...tilesize-1)
  const unsigned int stid = gtid % tilesize;

  // Shared memory
  extern __shared__ char shmem[];
  volatile float3 *sh_xyzi = (float3 *)&shmem[0];    // nwarp_build_excl_dist*tilesize
  unsigned int *sh_excl = (unsigned int *)&sh_xyzi[nwarp_build_excl_dist*tilesize];

  //  __shared__ float3 sh_xyzi[nwarp_build_excl_dist*tilesize];
  //#if (tilesize == 16)
  //  __shared__ unsigned int sh_excl[nwarp_build_excl_dist*num_excl];
  //#endif

  if (wid >= n_ijlist) return;

  // Each warp computes one ijlist entry
  int3 ijlist_val = ijlist[wid];
  int icell = ijlist_val.x - 1;
  int ish   = ijlist_val.y;
  int jcell = ijlist_val.z - 1;

  int istart = cell_patom[icell] - 1;
  int iend   = cell_patom[icell+1] - 2;

  int jstart = cell_patom[jcell] - 1;
  int jend   = cell_patom[jcell+1] - 2;

  const unsigned int load_ij = threadIdx.x % tilesize;
  const int sh_start = (threadIdx.x/warpsize)*tilesize;

  // Load atom i coordinates to shared memory
  // NOTE: volatile qualifier in "sh_xyzi" guarantees that values are actually read/written from
  //       shared memory. Therefore, no __syncthreads() is needed.
  float4 xyzq_i;

  if (tilesize == 32 || tid < 16) {
    if (istart + load_ij <= iend) {
      xyzq_i = xyzq[istart + load_ij];
    } else {
      xyzq_i.x = -100000000.0f;
      xyzq_i.y = -100000000.0f;
      xyzq_i.z = -100000000.0f;
    }
    sh_xyzi[sh_start + load_ij].x = xyzq_i.x;
    sh_xyzi[sh_start + load_ij].y = xyzq_i.y;
    sh_xyzi[sh_start + load_ij].z = xyzq_i.z;
  }

  // Load atom j coordinates
  float xj, yj, zj;
  //  const unsigned int loadj = (tid + (tid/TILESIZE)*(TILESIZE-1)) % TILESIZE;
  //  const unsigned int loadj = threadIdx.x % TILESIZE;
  if (jstart + load_ij <= jend) {
    float4 xyzq_j = xyzq[jstart + load_ij];
    xj = xyzq_j.x;
    yj = xyzq_j.y;
    zj = xyzq_j.z;
  } else {
    xj = 100000000.0f;
    yj = 100000000.0f;
    zj = 100000000.0f;
  }

  // Calculate shift
  float zsh = (ish/9 - 1)*boxz;
  ish -= (ish/9)*9;
  float ysh = (ish/3 - 1)*boxy;
  ish -= (ish/3)*3;
  float xsh = (ish - 1)*boxx;

  xj -= xsh;
  yj -= ysh;
  zj -= zsh;
  
  // Make sure shared memory has been written
  // NOTE: since we're only operating within the warp, this __syncthreads() is just to make sure
  //       all values are actually written in shared memory and not kept in registers etc.
  //__syncthreads();

  int q_samecell = (icell == jcell);

  unsigned int excl = 0;
  int t;

  if (tilesize == 32) {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int i = ((threadIdx.x + t) % tilesize);
      int ii = sh_start + i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= rcut2) | (q_samecell && (tid <= i)) ) << t;
    }
    tile_indj[wid] = jstart;
    tile_excl[wid].excl[stid] = excl;

  } else {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int load_i = (tid + t*2 + (tid/tilesize)*(tilesize-1)) % tilesize;
      int ii = sh_start + load_i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= rcut2) | (q_samecell && (load_ij <= load_i)) ) << t;
    }
    // excl is a 8 bit exclusion mask.
    // The full 32 bit exclusion mask is contained in 4 threads:
    // thread 0 contains the lowest 8 bits
    // thread 1 contains the next 8 bits, etc..
    
    excl <<= (threadIdx.x % num_thread_per_excl)*(num_excl<tilesize>::val);
    
    // Combine excl using shared memory
    const unsigned int sh_excl_ind = (threadIdx.x/warpsize)*(num_excl<tilesize>::val) + 
      (threadIdx.x % warpsize)/num_thread_per_excl;
    
    sh_excl[sh_excl_ind] = 0;
    __syncthreads();
    
    atomicOr(&sh_excl[sh_excl_ind], excl);
    
    // Make sure shared memory is written
    __syncthreads();
    
    // index to tile_excl.excl[] (0...7)
    const unsigned int excl_ind = (threadIdx.x % warpsize)/num_thread_per_excl;
    
    tile_indj[wid] = jstart;
    
    if ((threadIdx.x % num_thread_per_excl) == 0) {
      tile_excl[wid].excl[excl_ind] = sh_excl[sh_excl_ind];
    }
  }

}

//----------------------------------------------------------------------------------------
//
// Combines tile_excl_top on GPU
// One thread takes care of one integer in the exclusion mask, therefore:
//
// 32x32 tile, 32 integers per tile
// 16x16 tile, 8 integers per tile
//
template <int tilesize>
__global__ void add_tile_top_kernel(const int ntile_top,
				    const int *tile_ind_top,
				    const tile_excl_t<tilesize> *tile_excl_top,
				    tile_excl_t<tilesize> *tile_excl) {
  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x;
  // Index to tile_ind_top[]
  const unsigned int i = gtid / (num_excl<tilesize>::val);
  // Index to exclusion mask
  const unsigned int ix = gtid % (num_excl<tilesize>::val);

  if (i < ntile_top) {
    int ind = tile_ind_top[i];
    tile_excl[ind].excl[ix] |= tile_excl_top[i].excl[ix];
  }

}

//########################################################################################
//########################################################################################
//########################################################################################

//
// Class creator
//
template <int tilesize>
NeighborList<tilesize>::NeighborList(const int ncoord_glo, const CudaTopExcl& topExcl,
				     const int nx, const int ny, const int nz) : topExcl(topExcl) {
  this->ncoord_glo = ncoord_glo;
  this->init(nx, ny, nz);
}

template <int tilesize>
NeighborList<tilesize>::NeighborList(const int ncoord_glo, const CudaTopExcl& topExcl,
				     const char *filename,
				     const int nx, const int ny, const int nz) : topExcl(topExcl) {
  this->ncoord_glo = ncoord_glo;
  this->init(nx, ny, nz);
  load(filename);
}

//
// Class destructor
//
template <int tilesize>
NeighborList<tilesize>::~NeighborList() {
  if (tile_excl != NULL) deallocate< tile_excl_t<tilesize> > (&tile_excl);
  if (ientry != NULL) deallocate<ientry_t>(&ientry);
  if (tile_indj != NULL) deallocate<int>(&tile_indj);
  // Sparse
  if (pairs != NULL) deallocate< pairs_t<tilesize> > (&pairs);
  if (ientry_sparse != NULL) deallocate<ientry_t>(&ientry_sparse);
  if (tile_indj_sparse != NULL) deallocate<int>(&tile_indj_sparse);
  // Neighbor list building
  if (col_natom != NULL) deallocate<int>(&col_natom);
  if (col_patom != NULL) deallocate<int>(&col_patom);
  if (atom_icol != NULL) deallocate<int>(&atom_icol);
  if (ind_sorted != NULL) deallocate<int>(&ind_sorted);
  if (cell_patom != NULL) deallocate<int>(&cell_patom);
  if (atom_pcell != NULL) deallocate<int>(&atom_pcell);
  if (col_ncellz != NULL) deallocate<int>(&col_ncellz);
  if (col_xy_zone != NULL) deallocate<int3>(&col_xy_zone);
  if (col_cell != NULL) deallocate<int>(&col_cell);
  if (cell_xyz_zone != NULL) deallocate<int4>(&cell_xyz_zone);
  if (cell_bz != NULL) deallocate<float>(&cell_bz);
  if (excl_atom_heap != NULL) deallocate<int>(&excl_atom_heap);
  if (cell_excl_pos != NULL) deallocate<int>(&cell_excl_pos);
  if (cell_excl != NULL) deallocate<int>(&cell_excl);
  if (bb != NULL) deallocate<bb_t>(&bb);
  deallocate_host<NeighborListParam_t>(&h_nlist_param);
}

template <int tilesize>
void NeighborList<tilesize>::init(const int nx, const int ny, const int nz) {
  n_ientry = 0;
  n_tile = 0;

  tile_excl = NULL;
  tile_excl_len = 0;

  ientry = NULL;
  ientry_len = 0;

  tile_indj = NULL;
  tile_indj_len = 0;

  // Sparse
  n_ientry_sparse = 0;
  n_tile_sparse = 0;

  pairs_len = 0;
  pairs = NULL;
  
  ientry_sparse_len = 0;
  ientry_sparse = NULL;

  tile_indj_sparse_len = 0;
  tile_indj_sparse = NULL;

  // Neighbor list building
  col_natom_len = 0;
  col_natom = NULL;

  col_patom_len = 0;
  col_patom = NULL;

  ind_sorted_len = 0;
  ind_sorted = NULL;

  atom_icol_len = 0;
  atom_icol = NULL;

  col_ncellz_len = 0;
  col_ncellz = NULL;

  col_xy_zone_len = 0;
  col_xy_zone = NULL;

  col_cell_len = 0;
  col_cell = NULL;

  cell_patom_len = 0;
  cell_patom = NULL;

  atom_pcell_len = 0;
  atom_pcell = NULL;

  cell_xyz_zone_len = 0;
  cell_xyz_zone = NULL;

  cell_bz_len = 0;
  cell_bz = NULL;

  excl_atom_heap_len = 0;
  excl_atom_heap = NULL;

  cell_excl_pos_len = 0;
  cell_excl_pos = NULL;

  cell_excl_len = 0;
  cell_excl = NULL;

  bb_len = 0;
  bb = NULL;

  allocate_host<NeighborListParam_t>(&h_nlist_param, 1);

  h_nlist_param->imx_lo = 0;
  h_nlist_param->imx_hi = 0;
  h_nlist_param->imy_lo = 0;
  h_nlist_param->imy_hi = 0;
  h_nlist_param->imz_lo = 0;
  h_nlist_param->imz_hi = 0;
  if (nx == 1) {
    h_nlist_param->imx_lo = -1;
    h_nlist_param->imx_hi = 1;
  }
  if (ny == 1) {
    h_nlist_param->imy_lo = -1;
    h_nlist_param->imy_hi = 1;
  }
  if (nz == 1) {
    h_nlist_param->imz_lo = -1;
    h_nlist_param->imz_hi = 1;
  }

  test = false;
}

//
// Setup n_int_zone[0:7] and int_zone[0:7][0:7]
// zone ordering is: I,FZ,FY,EX,FX,EZ,EY,C = 0,...7
//
template <int tilesize>
void NeighborList<tilesize>::set_int_zone(const int *zone_natom, int *n_int_zone,
					  int int_zone[][8]) {
  const int I=0,FZ=1,FY=2,EX=3/*,FX=4,EZ=5,EY=6,C=7*/;
  // Setup interaction order that maximizes communication-computation overlap
  const int zones[8][5] = { {I, -1, -1, -1, -1},  // I-I
			    {I, -1, -1, -1, -1},  // I-FZ
			    {I, FZ, -1, -1, -1},  // I-FY, FZ-FY
			    {I, -1, -1, -1, -1},  // I-EX
			    {I, FZ, FY, EX, -1},  // I-FX, FZ-FX, FY-FX, EX-FX
			    {I, FZ, -1, -1, -1},  // I-EZ, FZ-EZ
			    {I, FY, -1, -1, -1},  // I-EY, FY-EY
			    {I, -1, -1, -1, -1}}; // I-C

  n_int_zone_max = 0;
  for (int izone=0;izone < 8;izone++) {
    n_int_zone[izone] = 0;
    if (zone_natom[izone] > 0) {
      int j = 0;
      while (zones[izone][j] > -1) {
	if (zone_natom[zones[izone][j]] > 0) {
	  int_zone[izone][n_int_zone[izone]] = zones[izone][j];
	  n_int_zone[izone]++;
	}
	j++;
      }
    }
    n_int_zone_max = max(n_int_zone_max, n_int_zone[izone]);
  }

}

//
// Setup xy-cell sizes
//
template <int tilesize>
void NeighborList<tilesize>::set_cell_sizes(const int *zone_natom,
					    const float3 *max_xyz, const float3 *min_xyz,
					    int *ncellx, int *ncelly, int *ncellz_max,
					    float *celldx, float *celldy, float *celldz_min) {

  for (int izone=0;izone < 8;izone++) {
    if (zone_natom[izone] > 0) {
      // NOTE: we increase the cell sizes here by 0.001 to make sure no atoms drop outside cells
      float xsize = max_xyz[izone].x - min_xyz[izone].x + 0.001f;
      float ysize = max_xyz[izone].y - min_xyz[izone].y + 0.001f;
      float zsize = max_xyz[izone].z - min_xyz[izone].z + 0.001f;
      float delta = powf(xsize*ysize*zsize*tilesize/(float)zone_natom[izone], 1.0f/3.0f);
      ncellx[izone] = max(1, (int)(xsize/delta));
      ncelly[izone] = max(1, (int)(ysize/delta));
      // Approximation for ncellz = 2 x "uniform distribution of atoms"
      ncellz_max[izone] = max(1, 2*zone_natom[izone]/(ncellx[izone]*ncelly[izone]*tilesize));
      celldx[izone] = xsize/(float)(ncellx[izone]);
      celldy[izone] = ysize/(float)(ncelly[izone]);
      celldz_min[izone] = zsize/(float)(ncellz_max[izone]);
      if (test) {
	std::cerr << izone << ": " << min_xyz[izone].z << " ... " << max_xyz[izone].z << std::endl;
      }
    } else {
      ncellx[izone] = 0;
      ncelly[izone] = 0;
      ncellz_max[izone] = 0;
      celldx[izone] = 1.0f;
      celldy[izone] = 1.0f;
      celldz_min[izone] = 1.0f;
    }
  }

  /*
  std::cout << "celldx = " << celldx[0] << " ncellx[0] = " << ncellx[0] 
	    << " xsize = " << max_xyz[0].x - min_xyz[0].x + 0.001f << std::endl;

  std::cout << "celldy = " << celldy[0] << " ncelly[0] = " << ncelly[0] 
	    << " ysize = " << max_xyz[0].y - min_xyz[0].y + 0.001f << std::endl;
  */

}

//
// Tests for z columns
//
template <int tilesize>
bool NeighborList<tilesize>::test_z_columns(const int* zone_patom,
					    const int* ncellx, const int* ncelly,
					    const int ncol_tot,
					    const float3* min_xyz,
					    const float* inv_celldx, const float* inv_celldy,
					    const float4* xyzq, const float4* xyzq_sorted,
					    const int* col_patom) {

  int ncoord = zone_patom[8];
  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);
  float4 *h_xyzq_sorted = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq_sorted, h_xyzq_sorted, ncoord);

  int *h_col_patom = new int[ncol_tot+1];
  copy_DtoH_sync<int>(col_patom, h_col_patom, ncol_tot+1);
  int *h_ind_sorted = new int[ncoord];
  copy_DtoH_sync<int>(ind_sorted, h_ind_sorted, ncoord);

  bool ok = true;

  int izone, i, j;
  float x, y, xj, yj;
  int ix, iy, ind, lo_ind, hi_ind;
  int ind0 = 0;
  for (izone=0;izone < 8;izone++) {
    int istart = zone_patom[izone];
    int iend   = zone_patom[izone+1] - 1;
    if (iend >= istart) {
      float x0 = min_xyz[izone].x;
      float y0 = min_xyz[izone].y;
      for (i=istart;i <= iend;i++) {
	x = h_xyzq_sorted[i].x;
	y = h_xyzq_sorted[i].y;
	ix = (int)((x - x0)*inv_celldx[izone]);
	iy = (int)((y - y0)*inv_celldy[izone]);
	ind = ind0 + ix + iy*ncellx[izone];
	lo_ind = h_col_patom[ind];
	hi_ind = h_col_patom[ind+1] - 1;
	if (i < lo_ind || i > hi_ind) {
	  std::cout << "test_z_columns FAILED at i=" << i << " izone = " << izone << std::endl;
	  std::cout << "ind, lo_ind, hi_ind = " << ind << " " << lo_ind << " " << hi_ind << std::endl;
	  std::cout << "x,y = " << x << " " << y << " x0,y0 = " << x0 << " " << y0 << std::endl;
	  std::cout << "inv_celldx/y = " << inv_celldx[izone] << " " << inv_celldy[izone] << std::endl;
	  std::cout << "ix,iy =" << ix << " " << iy << " ind0 = " << ind0
		    << " ncellx = " << ncellx[izone] << " ncelly = " << ncelly[izone] << std::endl;
	  exit(1);
	}
      }
      for (i=istart;i <= iend;i++) {
	j = h_ind_sorted[i];
	x = h_xyzq_sorted[i].x;
	y = h_xyzq_sorted[i].y;
	xj = h_xyzq[j].x;
	yj = h_xyzq[j].y;
	if (x != xj || y != yj) {
	  std::cout << "test_z_columns FAILED at i=" << i << std::endl;
	  std::cout << "x,y   =" << x << " " << y << std::endl;
	  std::cout << "xj,yj =" << xj << " " << yj << std::endl;
	  exit(1);
	}
      }
      ind0 += ncellx[izone]*ncelly[izone];
    }
  }

  if (ok) std::cout << "test_z_columns OK" << std::endl;

  delete [] h_xyzq;
  delete [] h_xyzq_sorted;
  delete [] h_col_patom;
  delete [] h_ind_sorted;

  return ok;
}

//
// Tests sort
//
template <int tilesize>
bool NeighborList<tilesize>::test_sort(const int* zone_patom,
				       const int* ncellx, const int* ncelly,
				       const int ncol_tot, const int ncell_max,
				       const float3* min_xyz,
				       const float* inv_celldx, const float* inv_celldy,
				       const float4* xyzq, const float4* xyzq_sorted,
				       const int* col_patom, const int* cell_patom) {

  cudaCheck(hipDeviceSynchronize());

  int ncoord = zone_patom[8];
  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);
  float4 *h_xyzq_sorted = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq_sorted, h_xyzq_sorted, ncoord);
  int *h_col_patom = new int[ncol_tot+1];
  copy_DtoH_sync<int>(col_patom, h_col_patom, ncol_tot+1);
  int *h_ind_sorted = new int[ncoord];
  copy_DtoH_sync<int>(ind_sorted, h_ind_sorted, ncoord);
  int *h_cell_patom = new int[ncell_max];
  copy_DtoH_sync<int>(cell_patom, h_cell_patom, ncell_max);

  bool ok = true;
  
  int izone, i, j, k, prev_ind;
  float x, y, z, prev_z;
  float xj, yj, zj;
  int ix, iy, ind, lo_ind, hi_ind;

  k = 0;
  for (i=1;i < ncol_tot+1;i++) {
    for (j=h_col_patom[i-1];j < h_col_patom[i];j+=32) {
      if (j != h_cell_patom[k]) {
	std::cout << "test_sort FAILED at i=" << i << std::endl;
	std::cout << "j,k=" << j << " " << k << "cell_patom[k]=" << h_cell_patom[k] << std::endl;
	exit(1);
      }
      k++;
    }
  }
  int ind0 = 0;
  for (izone=0;izone < 8;izone++) {
    int istart = zone_patom[izone];
    int iend   = zone_patom[izone+1] - 1;
    if (iend >= istart) {
      float x0 = min_xyz[izone].x;
      float y0 = min_xyz[izone].y;
      prev_z = min_xyz[izone].z;
      prev_ind = ind0;
      for (i=istart;i <= iend;i++) {
	x = h_xyzq_sorted[i].x;
	y = h_xyzq_sorted[i].y;
	z = h_xyzq_sorted[i].z;
	  
	ix = (int)((x - x0)*inv_celldx[izone]);
	iy = (int)((y - y0)*inv_celldy[izone]);
	ind = ind0 + ix + iy*ncellx[izone];

	if (prev_ind != ind) {
	  prev_z = min_xyz[izone].z;
	}

	lo_ind = h_col_patom[ind];
	hi_ind = h_col_patom[ind+1] - 1;
	if (i < lo_ind || i > hi_ind) {
	  std::cout << "test_sort FAILED at i=" << i << std::endl;
	  std::cout << "ind, lo_ind, hi_ind = " << ind << " " << lo_ind << " " << hi_ind << std::endl;
	  exit(1);
	}
	if (z < prev_z) {
	  std::cout << "test_sort FAILED at i=" << i << std::endl;
	  std::cout << "prev_z, z = " << prev_z << " " << z << std::endl;
	  exit(1);
	}
	prev_z = z;
	prev_ind = ind;
      }
      
      for (i=istart;i <= iend;i++) {
	j = h_ind_sorted[i];
	x = h_xyzq_sorted[i].x;
	y = h_xyzq_sorted[i].y;
	z = h_xyzq_sorted[i].z;
	xj = h_xyzq[j].x;
	yj = h_xyzq[j].y;
	zj = h_xyzq[j].z;
	if (x != xj || y != yj || z != zj) {
	  std::cout << "test_sort FAILED at i=" << i << std::endl;
	  std::cout << "x,y,z   =" << x << " " << y << " " << z << std::endl;
	  std::cout << "xj,yj,zj=" << xj << " " << yj << " " << zj << std::endl;
	  exit(1);
	}
      }
      ind0 += ncellx[izone]*ncelly[izone];
    }
  }

  if (ok) std::cout << "test_sort OK" << std::endl;

  delete [] h_xyzq;
  delete [] h_xyzq_sorted;
  delete [] h_col_patom;
  delete [] h_cell_patom;
  delete [] h_ind_sorted;

  return ok;
}

//
// Copies h_nlist_param (CPU) -> d_nlist_param (GPU)
//
template <int tilesize>
void NeighborList<tilesize>::set_nlist_param(hipStream_t stream) {
  cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_nlist_param), h_nlist_param, sizeof(NeighborListParam_t),
  				    0, hipMemcpyHostToDevice, stream));
}

//
// Copies d_nlist_param (GPU) -> h_nlist_param (CPU)
//
template <int tilesize>
void NeighborList<tilesize>::get_nlist_param() {
  cudaCheck(hipMemcpyFromSymbol(h_nlist_param, HIP_SYMBOL(d_nlist_param), sizeof(NeighborListParam_t),
				 0, hipMemcpyDeviceToHost));
}

//
// Resets n_tile and n_ientry variables for build() -call
//
template <int tilesize>
void NeighborList<tilesize>::reset() {
  get_nlist_param();
  cudaCheck(hipDeviceSynchronize());
  h_nlist_param->n_tile = 0;
  h_nlist_param->n_ientry = 0;
  set_nlist_param(0);
  cudaCheck(hipDeviceSynchronize());
}

//
// Returns an estimate for the number of tiles
//
template <int tilesize>
void NeighborList<tilesize>::get_tile_ientry_est(int *n_int_zone, int int_zone[][8],
						 int *ncellx, int *ncelly, int *ncellz_max,
						 float *celldx, float *celldy, float *celldz_min,
						 float rcut, int &n_tile_est, int &n_ientry_est) {
  n_tile_est = 0;
  // Loop over all zone-zone interactions
  for (int izone=0;izone < 8;izone++) {
    for (int j=0;j < n_int_zone[izone];j++) {
      int jzone = int_zone[izone][j];
      if (izone != jzone) {
	// Calculate the amount of volume overlap on zone j
	double dx_j, dy_j, dz_j;
	calc_volume_overlap(h_nlist_param->min_xyz[izone].x,
			    h_nlist_param->min_xyz[izone].y,
			    h_nlist_param->min_xyz[izone].z,
			    h_nlist_param->max_xyz[izone].x,
			    h_nlist_param->max_xyz[izone].y,
			    h_nlist_param->max_xyz[izone].z, rcut,
			    h_nlist_param->min_xyz[jzone].x,
			    h_nlist_param->min_xyz[jzone].y,
			    h_nlist_param->min_xyz[jzone].z,
			    h_nlist_param->max_xyz[jzone].x,
			    h_nlist_param->max_xyz[jzone].y,
			    h_nlist_param->max_xyz[jzone].z, dx_j, dy_j, dz_j);
	// Calculate the amount of volume overlap on zone i
	double dx_i, dy_i, dz_i;
	calc_volume_overlap(h_nlist_param->min_xyz[jzone].x,
			    h_nlist_param->min_xyz[jzone].y,
			    h_nlist_param->min_xyz[jzone].z,
			    h_nlist_param->max_xyz[jzone].x,
			    h_nlist_param->max_xyz[jzone].y,
			    h_nlist_param->max_xyz[jzone].z, rcut,
			    h_nlist_param->min_xyz[izone].x,
			    h_nlist_param->min_xyz[izone].y,
			    h_nlist_param->min_xyz[izone].z,
			    h_nlist_param->max_xyz[izone].x,
			    h_nlist_param->max_xyz[izone].y,
			    h_nlist_param->max_xyz[izone].z, dx_i, dy_i, dz_i);
	// Number of cells in each direction that are needed to fill the overlap volume
	int ncellx_j = (int)ceil(dx_j/celldx[jzone]);
	int ncelly_j = (int)ceil(dy_j/celldy[jzone]);
	int ncellz_j = (int)ceil(dz_j/celldz_min[jzone]);
	int ncell_j = ncellx_j*ncelly_j*ncellz_j;
	int ncellx_i = (int)ceil(dx_i/celldx[izone]);
	int ncelly_i = (int)ceil(dy_i/celldy[izone]);
	int ncellz_i = (int)ceil(dz_i/celldz_min[izone]);
	int ncell_i = ncellx_i*ncelly_i*ncellz_i;
	n_tile_est += ncell_j*ncell_i;
      } else {
	int ncell_i = ncellx[izone]*ncelly[izone]*ncellz_max[izone];
	// Estimate the number of neighbors in each direction for the positive direction and multiply
	// by the number of cells
	int n_neigh_ij = ((int)ceilf(rcut/celldx[izone])+1)*((int)ceilf(rcut/celldy[izone])+1)
	  *((int)ceilf(rcut/celldz_min[izone])+1)*ncell_i;
	n_tile_est += n_neigh_ij;
      }
    }
  }

  // Assume every i-j tile is in a separate ientry (worst case)
  n_ientry_est = n_tile_est;
}

//
// Calculates overlap between volumes
//
template <int tilesize>
double NeighborList<tilesize>::calc_volume_overlap(double Ax0, double Ay0, double Az0, 
						   double Ax1, double Ay1, double Az1, double rcut,
						   double Bx0, double By0, double Bz0, 
						   double Bx1, double By1, double Bz1,
						   double& dx, double& dy, double& dz) {
  double x0 = Ax0-rcut;
  double y0 = Ay0-rcut;
  double z0 = Az0-rcut;
  double x1 = Ax1+rcut;
  double y1 = Ay1+rcut;
  double z1 = Az1+rcut;

  dx = min(x1, Bx1) - max(x0, Bx0);
  dy = min(y1, By1) - max(y0, By0);
  dz = min(z1, Bz1) - max(z0, Bz0);
  dx = (dx > 0.0) ? dx : 0.0;
  dy = (dy > 0.0) ? dy : 0.0;
  dz = (dz > 0.0) ? dz : 0.0;

  return dx*dy*dz;
}


//
// Sorts atoms, when minimum and maximum coordinate values are known
//
template <int tilesize>
void NeighborList<tilesize>::sort(const int *zone_patom,
				  const float3 *max_xyz, const float3 *min_xyz,
				  float4 *xyzq,
				  float4 *xyzq_sorted,
				  int *loc2glo,
				  hipStream_t stream) {
  int ncoord = zone_patom[8];
  assert(ncoord <= max_ncoord);
  int ncol_tot;

  if (ncoord > ncoord_glo) {
    std::cerr << "NeighborList::sort(1), Invalid value for ncoord" << std::endl;
    exit(1);
  }

  // -------------------------- Setup -----------------------------
  sort_setup(zone_patom, max_xyz, min_xyz, ncol_tot, stream);
  // --------------------------------------------------------------

  // ------------------ Allocate/Reallocate memory ----------------
  sort_alloc_realloc(ncol_tot, ncoord);
  // --------------------------------------------------------------

  // ---------------------- Do actual sorting ---------------------
  sort_core(ncol_tot, ncoord, xyzq, xyzq_sorted, stream);
  // --------------------------------------------------------------

  // ------------------ Build indices etc. after sort -------------
  sort_build_indices(ncoord, xyzq_sorted, loc2glo, stream);
  // --------------------------------------------------------------

  // Test sort
  if (test) {
    test_sort(h_nlist_param->zone_patom, h_nlist_param->ncellx, h_nlist_param->ncelly,
	      ncol_tot, ncell_max, min_xyz, h_nlist_param->inv_celldx, h_nlist_param->inv_celldy,
	      xyzq, xyzq_sorted, col_patom, cell_patom);
  }

}

//
// Sorts atoms
//
template <int tilesize>
void NeighborList<tilesize>::sort(const int *zone_patom,
				  float4 *xyzq,
				  float4 *xyzq_sorted,
				  int *loc2glo,
				  hipStream_t stream) {
  const int ncoord = zone_patom[8];
  assert(ncoord <= max_ncoord);
  int ncol_tot;

  if (ncoord > ncoord_glo) {
    std::cerr << "NeighborList::sort(1), Invalid value for ncoord" << std::endl;
    exit(1);
  }

  for (int izone=0;izone < 8;izone++) {
    h_nlist_param->min_xyz[izone].x = (float)1.0e20;
    h_nlist_param->min_xyz[izone].y = (float)1.0e20;
    h_nlist_param->min_xyz[izone].z = (float)1.0e20;
    h_nlist_param->max_xyz[izone].x = (float)(-1.0e20);
    h_nlist_param->max_xyz[izone].y = (float)(-1.0e20);
    h_nlist_param->max_xyz[izone].z = (float)(-1.0e20);
  }

  set_nlist_param(stream);

  for (int izone=0;izone < 8;izone++) {
    int nstart = zone_patom[izone];
    int ncoord_zone = zone_patom[izone+1] - nstart;
    if (ncoord_zone > 0) {
      int nthread = 512;
      int nblock = (ncoord_zone-1)/nthread+1;
      int shmem_size = 6*nthread*sizeof(float);
      calc_minmax_xyz_kernel<<< nblock, nthread, shmem_size, stream >>>
	(ncoord_zone, izone, &xyzq[nstart]);
    }
  }
  cudaCheck(hipStreamSynchronize(stream));

  get_nlist_param();

  /*
  std::cout << "min_xyz = " << h_nlist_param->min_xyz[0].x << " "
	    << h_nlist_param->min_xyz[0].y << " "
	    << h_nlist_param->min_xyz[0].z << " " << std::endl;

  std::cout << "max_xyz = " << h_nlist_param->max_xyz[0].x << " "
	    << h_nlist_param->max_xyz[0].y << " "
	    << h_nlist_param->max_xyz[0].z << " " << std::endl;
  */

  // -------------------------- Setup -----------------------------
  sort_setup(zone_patom, h_nlist_param->max_xyz, h_nlist_param->min_xyz, ncol_tot, stream);
  // --------------------------------------------------------------

  // ------------------ Allocate/Reallocate memory ----------------
  sort_alloc_realloc(ncol_tot, ncoord);
  // --------------------------------------------------------------

  // ---------------------- Do actual sorting ---------------------
  sort_core(ncol_tot, ncoord, xyzq, xyzq_sorted, stream);
  // --------------------------------------------------------------

  // ------------------ Build indices etc. after sort -------------
  sort_build_indices(ncoord, xyzq_sorted, loc2glo, stream);
  // --------------------------------------------------------------

  // Test sort
  if (test) {
    test_sort(h_nlist_param->zone_patom, h_nlist_param->ncellx, h_nlist_param->ncelly,
	      ncol_tot, ncell_max, h_nlist_param->min_xyz,
	      h_nlist_param->inv_celldx, h_nlist_param->inv_celldy,
	      xyzq, xyzq_sorted, col_patom, cell_patom);
  }

}

//
// Setups sort parameters: nlist_param, ncol_tot, ncell_max
//
// NOTE: ncell_max is an approximate upper bound for the number of cells,
//       it is possible to blow this bound, so we should check for it
template < int tilesize >
void NeighborList<tilesize>::sort_setup(const int *zone_patom,
					const float3 *max_xyz, const float3 *min_xyz,
					int &ncol_tot, hipStream_t stream) {

  int zone_natom[8];
  //
  // Calculate zone_natom
  //
  // zone_natom[izone] = number of atoms in zone "izone"
  for (int izone=0;izone < 8;izone++) {
    zone_natom[izone] = zone_patom[izone+1] - zone_patom[izone];
  }

  set_int_zone(zone_natom, h_nlist_param->n_int_zone, h_nlist_param->int_zone);
  set_cell_sizes(zone_natom, max_xyz, min_xyz,
		 h_nlist_param->ncellx, h_nlist_param->ncelly, h_nlist_param->ncellz_max,
		 h_nlist_param->celldx, h_nlist_param->celldy, h_nlist_param->celldz_min);

  // Setup nlist_param and copy it to GPU
  int ncol = 0;
  int max_ncellxy = 0;
  h_nlist_param->zone_col[0] = 0;
  for (int izone=0;izone < 8;izone++) {
    h_nlist_param->zone_patom[izone] = zone_patom[izone];
    h_nlist_param->ncol[izone] = ncol;
    max_ncellxy = max(max_ncellxy, h_nlist_param->ncellx[izone]*h_nlist_param->ncelly[izone]);
    ncol += h_nlist_param->ncellx[izone]*h_nlist_param->ncelly[izone];
    h_nlist_param->inv_celldx[izone] = 1.0f/h_nlist_param->celldx[izone];
    h_nlist_param->inv_celldy[izone] = 1.0f/h_nlist_param->celldy[izone];
    h_nlist_param->min_xyz[izone].x = min_xyz[izone].x;
    h_nlist_param->min_xyz[izone].y = min_xyz[izone].y;
    h_nlist_param->min_xyz[izone].z = min_xyz[izone].z;
    if (izone > 0) {
      h_nlist_param->zone_col[izone] = h_nlist_param->zone_col[izone-1] +
	h_nlist_param->ncellx[izone-1]*h_nlist_param->ncelly[izone-1];
    }
  }
  h_nlist_param->ncol[8] = ncol;
  h_nlist_param->zone_patom[8] = zone_patom[8];

  // Copy h_nlist_param => d_nlist_param
  set_nlist_param(stream);

  ncol_tot = 0;
  ncell_max = 0;
  for (int izone=0;izone < 8;izone++) {
    int ncellxy = h_nlist_param->ncellx[izone]*h_nlist_param->ncelly[izone];
    ncol_tot += ncellxy;
    ncell_max += ncellxy*h_nlist_param->ncellz_max[izone];
  }

  // Wait till set_nlist_param finishes 
  cudaCheck(hipStreamSynchronize(stream));
}

//
// Allocates / Re-allocates memory for sort
//
template <int tilesize>
void NeighborList<tilesize>::sort_alloc_realloc(const int ncol_tot, const int ncoord) {

#ifdef STRICT_MEMORY_REALLOC
  float fac = 1.0f;
#else
  float fac = 1.2f;
#endif

  reallocate<int>(&atom_icol, &atom_icol_len, ncoord, fac);
  reallocate<int>(&atom_pcell, &atom_pcell_len, ncoord, fac);

  reallocate<int>(&cell_patom, &cell_patom_len, ncell_max+1, fac);
  reallocate<int4>(&cell_xyz_zone, &cell_xyz_zone_len, ncell_max, fac);
  reallocate<float>(&cell_bz, &cell_bz_len, ncell_max, fac);
  reallocate<bb_t>(&bb, &bb_len, ncell_max, fac);

  reallocate<int>(&col_natom, &col_natom_len, ncol_tot, fac);
  reallocate<int>(&col_patom, &col_patom_len, ncol_tot+1, fac);
  reallocate<int>(&col_ncellz, &col_ncellz_len, ncol_tot, fac);
  reallocate<int3>(&col_xy_zone, &col_xy_zone_len, ncol_tot, fac);
  reallocate<int>(&col_cell, &col_cell_len, ncol_tot, fac);

  reallocate<int>(&ind_sorted, &ind_sorted_len, ncoord, fac);
}

//
// Builds indices etc. after sort. xyzq is the sorted array
//
template <int tilesize>
void NeighborList<tilesize>::sort_build_indices(const int ncoord, float4 *xyzq, int *loc2glo,
						hipStream_t stream) {
  int nthread, nblock, shmem_size;

  //
  // Build loc2glo (really we are reordering it with ind_sorted)
  //
  // Make a copy of loc2glo to glo2loc
  // NOTE: This is temporary, glo2loc will be used for a different purpose later
  copy_DtoD<int>(loc2glo, topExcl.get_glo2loc(), ncoord, stream);
  nthread = 512;
  nblock = (ncoord - 1)/nthread + 1;
  build_loc2glo_kernel<<< nblock, nthread, 0, stream >>>(ncoord, ind_sorted,
							 topExcl.get_glo2loc(), loc2glo);
  cudaCheck(hipGetLastError());

  // Build glo2loc
  // NOTE: We mark atoms that do not exist with -1
  set_gpu_array<int>(topExcl.get_glo2loc(), topExcl.get_ncoord(), -1, stream);
  nthread = 512;
  nblock = (ncoord - 1)/nthread + 1;
  build_glo2loc_kernel<<< nblock, nthread, 0, stream >>>(ncoord, loc2glo, topExcl.get_glo2loc());
  cudaCheck(hipGetLastError());

  // Build atom_pcell
  nthread = 1024;
  nblock = (ncell_max - 1)/(nthread/warpsize) + 1;
  build_atom_pcell_kernel<<< nblock, nthread, 0, stream >>>(cell_patom, atom_pcell);
  cudaCheck(hipGetLastError());

  // Build bounding box (bb) and cell boundaries (cell_bz)
  nthread = 512;
  nblock = (ncell_max-1)/nthread + 1;
  shmem_size = 0;
  calc_bb_cell_bz_kernel<tilesize> <<< nblock, nthread, shmem_size, stream >>>
    (cell_patom, xyzq, bb, cell_bz);
  cudaCheck(hipGetLastError());

  /*
  //
  // Calculate ncellz_max[izone]
  // NOTE: This is only needed in order to get a better estimate for n_tile_est
  //
  nthread = min(((max_ncellxy-1)/warpsize+1)*warpsize, get_max_nthread());
  nblock = 8;
  shmem_size = nthread*sizeof(int);
  calc_ncellz_max_kernel<<< nblock, nthread, shmem_size, stream >>>(col_ncellz);
  */

}

//
// Sorts atoms, core subroutine.
//
template <int tilesize>
void NeighborList<tilesize>::sort_core(const int ncol_tot, const int ncoord,
				       float4 *xyzq,
				       float4 *xyzq_sorted,
				       hipStream_t stream) {

  int nthread, nblock, shmem_size;

  // Clear col_natom
  clear_gpu_array<int>(col_natom, ncol_tot, stream);

  // Make a copy of loc2glo to glo2loc
  // NOTE: This is temporary, glo2loc will be used for a different purpose later
  //copy_DtoD<int>(loc2glo, glo2loc, ncoord, stream);

  //
  // Calculate number of atoms in each z-column (col_natom)
  // and the column index for each atom (atom_icol)
  //
  nthread = 512;
  nblock = (ncoord-1)/nthread+1;
  calc_z_column_index_kernel<<< nblock, nthread, 0, stream >>>
    (xyzq, col_natom, atom_icol, col_xy_zone);
  cudaCheck(hipGetLastError());

  //
  // Calculate positions in z columns
  // NOTE: Clears col_natom and sets (col_patom, cell_patom, col_ncellz, d_nlist_param.ncell)
  //
  nthread = min(((ncol_tot-1)/tilesize+1)*tilesize, get_max_nthread());
  shmem_size = nthread*sizeof(int2);
  calc_z_column_pos_kernel<tilesize> <<< 1, nthread, shmem_size, stream >>>
    (ncol_tot, col_xy_zone, col_natom, col_patom, cell_patom, col_ncellz, cell_xyz_zone,
     col_cell);

  //
  // Reorder atoms into z-columns
  // NOTE: also sets up startcell_zone[izone]
  //
  nthread = 512;
  nblock = (ncoord-1)/nthread+1;
  reorder_atoms_z_column_kernel<<< nblock, nthread, 0, stream >>>
    (ncoord, atom_icol, col_natom, col_patom, xyzq, xyzq_sorted, ind_sorted);
  cudaCheck(hipGetLastError());

  // Test z columns
  if (test) {
    cudaCheck(hipDeviceSynchronize());
    test_z_columns(h_nlist_param->zone_patom, h_nlist_param->ncellx, h_nlist_param->ncelly,
		   ncol_tot, h_nlist_param->min_xyz, h_nlist_param->inv_celldx, h_nlist_param->inv_celldy,
		   xyzq, xyzq_sorted, col_patom);
  }

  // Now sort according to z coordinate
  nthread = 0;
  nblock = 0;
  for (int izone=0;izone < 8;izone++) {
    nblock += h_nlist_param->ncellx[izone]*h_nlist_param->ncelly[izone];
    nthread = max(nthread, h_nlist_param->ncellz_max[izone]*tilesize);
  }
  if (nthread < get_max_nthread()) {
    shmem_size = nthread*sizeof(keyval_t);
    sort_z_column_kernel<<< nblock, nthread, shmem_size, stream >>>
      (col_patom, xyzq_sorted, ind_sorted);
    cudaCheck(hipGetLastError());
  } else {
    std::cerr << "Neighborlist::sort_core, this version of sort_z_column_kernel not implemented yet"
	      << std::endl;
    exit(1);
  }

}

//
// Sets ientry from host memory array
//
template <int tilesize>
void NeighborList<tilesize>::set_ientry(int n_ientry, ientry_t *h_ientry) {

  this->n_ientry = n_ientry;

  // Allocate & reallocate d_ientry
#ifdef STRICT_MEMORY_REALLOC
  reallocate<ientry_t>(&ientry, &ientry_len, n_ientry, 1.0f);
#else
  reallocate<ientry_t>(&ientry, &ientry_len, n_ientry, 1.4f);
#endif

  // Copy to device
  copy_HtoD_sync<ientry_t>(h_ientry, ientry, n_ientry);
}

//
// Builds neighborlist
//
template <int tilesize>
void NeighborList<tilesize>::build(const float boxx, const float boxy, const float boxz,
				   const float rcut,
				   const float4 *xyzq, const int *loc2glo,
				   hipStream_t stream) {
  int nthread, nblock, shmem_size;

  get_nlist_param();
  
  int n_tile_est, n_ientry_est;
  get_tile_ientry_est(h_nlist_param->n_int_zone, h_nlist_param->int_zone,
		      h_nlist_param->ncellx, h_nlist_param->ncelly, h_nlist_param->ncellz_max,
		      h_nlist_param->celldx, h_nlist_param->celldy, h_nlist_param->celldz_min,
		      rcut, n_tile_est, n_ientry_est);
  //std::cout << "n_ientry_est = " << n_ientry_est << " n_tile_est = " << n_tile_est << std::endl;

  if (test) {
    std::cout << "ncell = " << h_nlist_param->ncell << " ncell_max = " << ncell_max
	      << " n_tile_est = " << n_tile_est << std::endl;
    for (int izone=0;izone < 8;izone++) {
      std::cout << izone << ": " << h_nlist_param->ncellx[izone]
		<< " " << h_nlist_param->ncelly[izone]
		<< " " << h_nlist_param->ncellz_max[izone]
		<< std::endl;
    }
  }

  reallocate<ientry_t>(&ientry, &ientry_len, n_ientry_est, 1.0f);
  reallocate<tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_tile_est, 1.0f);
  reallocate<int>(&tile_indj, &tile_indj_len, n_tile_est, 1.0f);

#ifdef STRICT_MEMORY_REALLOC
  reallocate<int>(&excl_atom_heap, &excl_atom_heap_len, ncell_max*tilesize*topExcl.getMaxNumExcl(), 1.0f);
#else
  reallocate<int>(&excl_atom_heap, &excl_atom_heap_len, ncell_max*tilesize*topExcl.getMaxNumExcl(), 1.2f);
#endif

  //clear_gpu_array< tile_excl_t<tilesize> >(tile_excl, tile_excl_len, stream);

  // Shared memory requirements:
  // (blockDim.x/warpsize)*( (!IvsI)*n_jzone*sizeof(int2) + n_jlist_max*sizeof(int) 
  //                         + tilesize*sizeof(float3))

  // I vs. I
  nthread = 512;
  //nblock = (ncell_max-1)/(nthread/warpsize) + 1;
  nblock = (h_nlist_param->ncell-1)/(nthread/warpsize) + 1;
  shmem_size = (nthread/warpsize)*n_jlist_max*sizeof(int);     // sh_jlist[]
  shmem_size += nthread*sizeof(int);                           // sh_atomj[]
  if (get_cuda_arch() < 300) {
    shmem_size += nthread*sizeof(int);                         // shflmem[]
    shmem_size += (nthread/warpsize)*tilesize*sizeof(float3);  // sh_xyzj[]
  }
  // For !IvsI, shmem_size += (nthread/warpsize)*n_int_zone_max*sizeof(int2)
  shmem_size += (nthread/warpsize)*n_int_zone_max*sizeof(int2);// sh_jcellxy_min[]

  //std::cout << "NeighborList::build, shmem_size = " << shmem_size << std::endl;
  build_kernel<tilesize>
    <<< nblock, nthread, shmem_size, stream >>>
    (topExcl.getMaxNumExcl(), cell_xyz_zone, col_ncellz, col_cell, cell_bz, cell_patom,
     loc2glo, topExcl.get_glo2loc(), topExcl.getAtomExclPos(), topExcl.getAtomExcl(),
     xyzq, boxx, boxy, boxz, rcut, rcut*rcut, bb, excl_atom_heap,
     tile_indj, tile_excl, ientry);
  cudaCheck(hipGetLastError());

  /*
  // Rest
  nthread = 512;
  nblock = (ncell_max-1)/(nthread/warpsize) + 1;
  shmem_size = (nthread/warpsize)*( n_jlist_max*sizeof(int) + tilesize*sizeof(float3)) + 
    nthread*sizeof(int);
  if (get_cuda_arch() < 300) shmem_size += nthread*sizeof(int);
  shmem_size += (nthread/warpsize)*n_int_zone_max*sizeof(int2)
  //std::cout << "NeighborList::build, shmem_size = " << shmem_size << std::endl;
  build_kernel<tilesize, false>
    <<< nblock, nthread, shmem_size, stream >>>
    (max_nexcl, cell_xyz_zone, col_ncellz, col_cell, cell_bz, cell_patom, loc2glo, glo2loc,
     atom_excl_pos, atom_excl, xyzq, boxx, boxy, boxz, rcut, rcut*rcut, bb, excl_atom_heap,
     tile_indj, tile_excl, ientry);
  cudaCheck(hipGetLastError());
  */

  cudaCheck(hipDeviceSynchronize());
  get_nlist_param();

  n_ientry = h_nlist_param->n_ientry;
  n_tile = h_nlist_param->n_tile;

  if (n_tile > n_tile_est) {
    std::cout << "NeighborList::build, Limit blown: n_tile > n_tile_est"<< std::endl;
    exit(1);
  }

  if (n_ientry > n_ientry_est) {
    std::cout << "NeighborList::build, Limit blown: n_ientry > n_ientry_est"<< std::endl;
    exit(1);
  }

  if (test) test_build(boxx, boxy, boxz, rcut, xyzq, loc2glo);
}

struct tileinfo_t {
  int excl;
  double dx, dy, dz;
  double r2;
};

template <int tilesize>
bool compare(tileinfo_t* tile1, tileinfo_t* tile2, std::vector<int2>& ijvec) {
  ijvec.clear();
  bool ok = true;
  for (int jt=0;jt < tilesize;jt++) {
    for (int it=0;it < tilesize;it++) {
      if (tile1[it + jt*tilesize].excl != tile2[it + jt*tilesize].excl) {
	int2 ijval;
	ijval.x = it;
	ijval.y = jt;
	ijvec.push_back(ijval);
	ok = false;
      }
    }
  }
  return ok;
}

template <int tilesize>
void set_excl(tileinfo_t* tile1) {
  for (int jt=0;jt < tilesize;jt++) {
    for (int it=0;it < tilesize;it++) {
      tile1[it + jt*tilesize].excl = 1;
    }
  }
}

template<int tilesize>
void print_excl(tileinfo_t* tile1) {
  for (int jt=0;jt < tilesize;jt++) {
    for (int it=0;it < tilesize;it++) {
      fprintf(stderr,"%d ",tile1[it + jt*tilesize].excl);
    }
    fprintf(stderr,"\n");
  }
}

std::ostream& operator<< (std::ostream &o, const bb_t& b) {
  o << "x,y,z= " << b.x << " " << b.y << " "<< b.z
    << " wx,wy,wz= " << b.wx << " " << b.wy << " "<< b.wz;
  return o;
}

//
// Test neighbor list building with a simple N^2 algorithm
//
template <int tilesize>
void NeighborList<tilesize>::test_build(const double boxx, const double boxy, const double boxz,
					const double rcut, const float4 *xyzq, const int* loc2glo) {

  cudaCheck(hipDeviceSynchronize());
  get_nlist_param();

  int n_ientry = h_nlist_param->n_ientry;
  int n_tile = h_nlist_param->n_tile;
  int ncell = h_nlist_param->ncell;

  int *h_atom_excl_pos = new int[topExcl.getAtomExclPosLen()];
  int *h_atom_excl = new int[topExcl.getAtomExclLen()];
  copy_DtoH_sync<int>(topExcl.getAtomExclPos(), h_atom_excl_pos, topExcl.getAtomExclPosLen());
  copy_DtoH_sync<int>(topExcl.getAtomExcl(), h_atom_excl, topExcl.getAtomExclLen());

  int ncoord = h_nlist_param->zone_patom[8];

  int *h_loc2glo = new int[ncoord];
  copy_DtoH_sync<int>(loc2glo, h_loc2glo, ncoord);

  float4* h_xyzq = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);

  bb_t *h_bb = new bb_t[ncell];
  copy_DtoH_sync<bb_t>(bb, h_bb, ncell);

  float* h_cell_bz = new float[ncell];
  copy_DtoH_sync<float>(cell_bz, h_cell_bz, ncell);

  double rcut2 = rcut*rcut;

  //float boxxf = (float)boxx;
  //float boxyf = (float)boxy;
  //float boxzf = (float)boxz;

  double hboxx = 0.5*boxx;
  double hboxy = 0.5*boxy;
  double hboxz = 0.5*boxz;
  //float hboxxf = (float)hboxx;
  //float hboxyf = (float)hboxy;
  //float hboxzf = (float)hboxz;

  // Calculate number of pairs
  int npair_cpu = calc_cpu_pairlist<double>(h_nlist_param->zone_patom, h_xyzq, h_loc2glo,
					    h_atom_excl_pos, h_atom_excl,
					    boxx, boxy, boxz, rcut);

  std::cout << "npair_cpu=" << npair_cpu << std::endl;

  ientry_t* h_ientry = new ientry_t[n_ientry];
  tile_excl_t<tilesize>* h_tile_excl = new tile_excl_t<tilesize>[n_tile];
  int* h_tile_indj = new int[n_tile];
  int* h_cell_patom = new int[ncell+1];

  copy_DtoH_sync<ientry_t>(ientry, h_ientry, n_ientry);
  copy_DtoH_sync<tile_excl_t<tilesize> >(tile_excl, h_tile_excl, n_tile);
  copy_DtoH_sync<int>(tile_indj, h_tile_indj, n_tile);
  copy_DtoH_sync<int>(cell_patom, h_cell_patom, ncell+1);

  // Calculate number of pairs on the GPU list
  int npair_gpu = calc_gpu_pairlist<double>(n_ientry, h_ientry, h_tile_indj, h_tile_excl,
					    h_xyzq, boxx, boxy, boxz, rcut);

  std::cout << "npair_gpu=" << npair_gpu << std::endl;

  tileinfo_t *tileinfo = new tileinfo_t[tilesize*tilesize];
  tileinfo_t *tileinfo2 = new tileinfo_t[tilesize*tilesize];
  std::vector<int2> ijvec;

  //
  // Go through all cell pairs and check that the gpu caught all of them
  //
  int npair_gpu2 = 0;
  int ncell_pair = 0;
  bool okloop = true;
  for (int izone=0;izone < 8;izone++) {
    for (int jzone=0;jzone < 8;jzone++) {
      if (izone == 1 && jzone != 5) continue;
      if (izone == 2 && jzone != 1 && jzone != 6) continue;
      if (izone == 4 && jzone != 1 && jzone != 2 && jzone != 3) continue;

      int icell_start = h_nlist_param->zone_cell[izone];
      int icell_end   = (izone+1 < 8) ? h_nlist_param->zone_cell[izone+1] : h_nlist_param->ncell;

      for (int icell=icell_start;icell < icell_end;icell++) {
	int jcell_start = h_nlist_param->zone_cell[jzone];
	int jcell_end   = (jzone+1 < 8) ? h_nlist_param->zone_cell[jzone+1] : h_nlist_param->ncell;
	if (izone == 0 && jzone == 0) {
	  jcell_start = icell;
	}
	for (int jcell=jcell_start;jcell < jcell_end;jcell++) {
	  int istart = h_cell_patom[icell];
	  int iend   = h_cell_patom[icell+1]-1;
	  int jstart = h_cell_patom[jcell];
	  int jend   = h_cell_patom[jcell+1]-1;
	  int npair_tile1 = 0;
	  bool pair = false;
	  double min_diff = 1.0e10;
	  set_excl<tilesize>(tileinfo);
	  for (int i=istart;i <= iend;i++) {
	    double xi = h_xyzq[i].x;
	    double yi = h_xyzq[i].y;
	    double zi = h_xyzq[i].z;
	    int ig = h_loc2glo[i];
	    int excl_start = h_atom_excl_pos[ig];
	    int excl_end   = h_atom_excl_pos[ig+1]-1;
	    for (int j=jstart;j <= jend;j++) {
	      tileinfo_t tileinfo_val;
	      tileinfo_val.excl = 1;
	      if (icell != jcell || i < j) {
		double xj = h_xyzq[j].x;
		double yj = h_xyzq[j].y;
		double zj = h_xyzq[j].z;
		double dx = xi - xj;
		double dy = yi - yj;
		double dz = zi - zj;
		double shx = 0.0;
		double shy = 0.0;
		double shz = 0.0;
		if (dx > hboxx) {
		  shx = -boxx;
		} else if (dx < -hboxx) {
		  shx = boxx;
		}
		if (dy > hboxy) {
		  shy = -boxy;
		} else if (dy < -hboxy) {
		  shy = boxy;
		}
		if (dz > hboxz) {
		  shz = -boxz;
		} else if (dz < -hboxz) {
		  shz = boxz;
		}
		double xis = xi + shx;
		double yis = yi + shy;
		double zis = zi + shz;
		dx = xis - xj;
		dy = yis - yj;
		dz = zis - zj;
		double r2 = dx*dx + dy*dy + dz*dz;
		min_diff = min(min_diff, fabs(r2-rcut2));

		int jg = h_loc2glo[j];
		bool excl_flag = false;
		for (int excl=excl_start;excl <= excl_end;excl++) {
		  if (h_atom_excl[excl] == jg) {
		    excl_flag = true;
		    break;
		  }
		}
		if (excl_flag == false) {
		  tileinfo_val.excl = 0;
		} else {
		  tileinfo_val.excl = 1;
		}
		if (r2 < rcut2 && !excl_flag) {
		  npair_gpu2++;
		  npair_tile1++;
		  pair = true;
		}
		tileinfo_val.dx = dx;
		tileinfo_val.dy = dy;
		tileinfo_val.dz = dz;
		tileinfo_val.r2 = r2;
	      }
	      int it = i-istart;
	      int jt = j-jstart;
	      tileinfo[it + jt*tilesize] = tileinfo_val;
	    }
	  } // for (int i=istart;i <= iend;i++)

	  if (pair) {
	    // Pair of cells with atoms starting at istart and jstart
	    bool found_this_pair = false;
	    int ind, jtile;
	    for (ind=0;ind < n_ientry;ind++) {
	      if (h_ientry[ind].indi != istart &&
		  h_ientry[ind].indi != jstart) continue;
	      int startj = h_ientry[ind].startj;
	      int endj   = h_ientry[ind].endj;
	      for (jtile=startj;jtile <= endj;jtile++) {
		if ((h_ientry[ind].indi == istart && h_tile_indj[jtile] == jstart) ||
		    (h_ientry[ind].indi == jstart && h_tile_indj[jtile] == istart)) {
		  found_this_pair = true;
		  break;
		}
	      }
	      if (found_this_pair) break;
	    }

	    if (found_this_pair) {
	      // Check the tile we found (ind, jtile)
	      int istart0, jstart0;
	      istart0 = h_ientry[ind].indi;
	      jstart0 = h_tile_indj[jtile];
	
	      int ish     = h_ientry[ind].ish;
	      int ish_tmp = ish;
	      double shz = (ish_tmp/9 - 1)*boxz;
	      ish_tmp -= (ish_tmp/9)*9;
	      double shy = (ish_tmp/3 - 1)*boxy;
	      ish_tmp -= (ish_tmp/3)*3;
	      double shx = (ish_tmp - 1)*boxx;

	      int npair_tile2 = 0;
	      for (int i=istart0;i < istart0+tilesize;i++) {
		double xi = (double)h_xyzq[i].x + shx;
		double yi = (double)h_xyzq[i].y + shy;
		double zi = (double)h_xyzq[i].z + shz;
		for (int j=jstart0;j < jstart0+tilesize;j++) {
		  int bitpos = ((i-istart0) - (j-jstart0) + tilesize) % tilesize;
		  unsigned int excl = h_tile_excl[jtile].excl[j-jstart0] >> bitpos;
		  double xj = h_xyzq[j].x;
		  double yj = h_xyzq[j].y;
		  double zj = h_xyzq[j].z;
		  double dx = xi - xj;
		  double dy = yi - yj;
		  double dz = zi - zj;
		  double r2 = dx*dx + dy*dy + dz*dz;

		  int it, jt;
		  if (istart0 == istart) {
		    it = i-istart0;
		    jt = j-jstart0;
		  } else {
		    jt = i-istart0;
		    it = j-jstart0;
		  }

		  tileinfo_t tileinfo_val;
		  tileinfo_val.excl = (excl & 1);
		  tileinfo_val.dx = dx;
		  tileinfo_val.dy = dy;
		  tileinfo_val.dz = dz;
		  tileinfo_val.r2 = r2;
		  tileinfo2[it + jt*tilesize] = tileinfo_val;

		  if (r2 < rcut2 && !(excl & 1)) {
		    npair_tile2++;
		  }
		}
	      }

	      //if (abs(npair_tile1 - npair_tile2) > 0) {
	      if (!compare<tilesize>(tileinfo, tileinfo2, ijvec)) {

		bool ok = true;
		for (int k=0;k < ijvec.size();k++) {
		  int it = ijvec.at(k).x;
		  int jt = ijvec.at(k).y;
		  tileinfo_t tileinfo_val;
		  tileinfo_val = tileinfo[it + jt*tilesize];
		  if (tileinfo_val.r2 >= rcut2) {
		    ok = false;
		    break;
		  }
		}
		if (!ok) continue;

		//std::cerr << "tile pair ERROR: icell = " << icell << " jcell = " << jcell 
		//	  << " npair_tile1 = " << npair_tile1 << " npair_tile2 = " << npair_tile2
		//	  << std::endl;
		//std::cerr << " istart0 = " << istart0 << " jstart0 = " << jstart0 
		//	  << " izone = " << izone << " jzone = " << jzone << std::endl;
		//std::cerr << " istart,iend  = " << istart << " " << iend 
		//	  << " jstart,jend  = " << jstart << " " << jend
		//	  << " min_diff=" << min_diff << std::endl;

		//fprintf(stderr,"tileinfo:\n");
		//print_excl<tilesize>(tileinfo);
		//fprintf(stderr,"tileinfo2:\n");
		//print_excl<tilesize>(tileinfo2);

		for (int k=0;k < ijvec.size();k++) {
		  int it = ijvec.at(k).x;
		  int jt = ijvec.at(k).y;

		  tileinfo_t tileinfo_val;
		  tileinfo_val = tileinfo[it + jt*tilesize];
		  tileinfo_t tileinfo2_val;
		  tileinfo2_val = tileinfo2[it + jt*tilesize];

		  if (tileinfo_val.r2 < rcut2) {
		    fprintf(stderr,"----------------------------------------------\n");
		    fprintf(stderr,"it,jt=%d %d dx,dy,dz=%lf %lf %lf r2=%lf | %d %d\n",it,jt,
			    tileinfo_val.dx,tileinfo_val.dy,tileinfo_val.dz,tileinfo_val.r2,
			    tileinfo_val.excl, tileinfo2_val.excl);
		    int ig = h_loc2glo[it];
		    int excl_start = h_atom_excl_pos[ig];
		    int excl_end   = h_atom_excl_pos[ig+1]-1;
		    int jg = h_loc2glo[jt];
		    //bool excl_flag = false;
		    for (int excl=excl_start;excl <= excl_end;excl++) {
		      if (h_atom_excl[excl] == jg) {
			fprintf(stderr,"======================= EXCLUSION FOUND! ==================\n");
			break;
		      }
		    }
		  }
		}
		//exit(1);
	      }
	  
	    } else {
	      std::cerr << "tile pair with istart = " << istart << " jstart = " << jstart
			<< " NOT FOUND" << std::endl;
	      std::cerr << "min_diff = " << min_diff << " npair_tile1 = " << npair_tile1
			<< " ind = " << ind << std::endl;
	      std::cerr << h_bb[icell] << " | " << icell << std::endl;
	      std::cerr << h_bb[jcell] << " | " << jcell << std::endl;
	      //exit(1);
	      okloop = false;
	    }
	  }

	  if (pair) ncell_pair++;
	} // for (int jcell...)
      } // for (int icell...)
    }
  }

  delete [] tileinfo;
  delete [] tileinfo2;

  delete [] h_atom_excl_pos;
  delete [] h_atom_excl;

  delete [] h_loc2glo;

  delete [] h_xyzq;
  delete [] h_ientry;
  delete [] h_tile_excl;
  delete [] h_tile_indj;
  delete [] h_cell_patom;

  delete [] h_bb;
  delete [] h_cell_bz;

  if (npair_cpu != npair_gpu || !okloop) {
    std::cout << "##################################################" << std::endl;
    std::cout << "test_build FAILED" << std::endl;
    std::cout << "n_ientry = " << n_ientry << " n_tile = " << n_tile << std::endl;
    std::cout << "npair_cpu = " << npair_cpu << " npair_gpu = " << npair_gpu 
	      << " npair_gpu2 = " << npair_gpu2 << std::endl;
    std::cout << "##################################################" << std::endl;
  } else {
    std::cout << "test_build OK" << std::endl;
  }

  if (!okloop) exit(1);

}

//
// Calculates GPU pair list
//
template <int tilesize> template <typename T>
int NeighborList<tilesize>::calc_gpu_pairlist(const int n_ientry, const ientry_t* ientry,
					      const int* tile_indj,
					      const tile_excl_t<tilesize>* tile_excl, const float4* xyzq,
					      const double boxx, const double boxy, const double boxz,
					      const double rcut) {
  T rcut2 = rcut*rcut;
  T boxxT = boxx;
  T boxyT = boxy;
  T boxzT = boxz;

  int npair = 0;
  for (int ind=0;ind < n_ientry;ind++) {
    int istart = ientry[ind].indi;
    int ish    = ientry[ind].ish;
    int startj = ientry[ind].startj;
    int endj   = ientry[ind].endj;

    int ish_tmp = ish;
    T shz = (ish_tmp/9 - 1)*boxzT;
    ish_tmp -= (ish_tmp/9)*9;
    T shy = (ish_tmp/3 - 1)*boxyT;
    ish_tmp -= (ish_tmp/3)*3;
    T shx = (ish_tmp - 1)*boxxT;

    for (int jtile=startj;jtile <= endj;jtile++) {
      for (int i=istart;i < istart+tilesize;i++) {
	T xi = (T)xyzq[i].x + shx;
	T yi = (T)xyzq[i].y + shy;
	T zi = (T)xyzq[i].z + shz;
	int jstart = tile_indj[jtile];
	for (int j=jstart;j < jstart+tilesize;j++) {
	  int bitpos = ((i-istart) - (j-jstart) + tilesize) % tilesize;
	  int excl = tile_excl[jtile].excl[j-jstart] >> bitpos;
	  T xj = xyzq[j].x;
	  T yj = xyzq[j].y;
	  T zj = xyzq[j].z;
	  T dx = xi - xj;
	  T dy = yi - yj;
	  T dz = zi - zj;
	  T r2 = dx*dx + dy*dy + dz*dz;
	  if (r2 < rcut2 && !(excl & 1)) npair++;
	}
      }
    }
  }

  return npair;
}

//
// Calculates CPU pair list
//
template <int tilesize> template <typename T>
int NeighborList<tilesize>::calc_cpu_pairlist(const int* zone_patom, const float4* xyzq,
					      const int* loc2glo, const int* atom_excl_pos,
					      const int* atom_excl, const double boxx,
					      const double boxy, const double boxz, const double rcut) {
  T rcut2 = rcut*rcut;
  T boxxT = boxx;
  T boxyT = boxy;
  T boxzT = boxz;
  T hboxx = 0.5*boxx;
  T hboxy = 0.5*boxy;
  T hboxz = 0.5*boxz;

  int npair = 0;
  for (int izone=0;izone < 8;izone++) {
    for (int jzone=0;jzone < 8;jzone++) {
      if (izone == 1 && jzone != 5) continue;
      if (izone == 2 && jzone != 1 && jzone != 6) continue;
      if (izone == 4 && jzone != 1 && jzone != 2 && jzone != 3) continue;

      int istart = zone_patom[izone];
      int iend   = zone_patom[izone+1] - 1;
      int jstart = zone_patom[jzone];
      int jend   = zone_patom[jzone+1] - 1;

      for (int i=istart;i <= iend;i++) {
	T xi = xyzq[i].x;
	T yi = xyzq[i].y;
	T zi = xyzq[i].z;
	int ig = loc2glo[i];
	int excl_start = atom_excl_pos[ig];
	int excl_end   = atom_excl_pos[ig+1]-1;
	if (izone == 0 && jzone == 0) jstart = i + 1;
	for (int j=jstart;j <= jend;j++) {
	  T xj = xyzq[j].x;
	  T yj = xyzq[j].y;
	  T zj = xyzq[j].z;
	  T dx = xi - xj;
	  T dy = yi - yj;
	  T dz = zi - zj;
	  if (dx > hboxx) {
	    dx = (xi-boxxT) - xj;
	  } else if (dx < -hboxx) {
	    dx = (xi+boxxT) - xj;
	  }
	  if (dy > hboxy) {
	    dy = (yi-boxyT) - yj;
	  } else if (dy < -hboxy) {
	    dy = (yi+boxyT) - yj;
	  }
	  if (dz > hboxz) {
	    dz = (zi-boxzT) - zj;
	  } else if (dz < -hboxz) {
	    dz = (zi+boxzT) - zj;
	  }
	  T r2 = dx*dx + dy*dy + dz*dz;

	  if (r2 < rcut2) {
	    int jg = loc2glo[j];
	    bool excl_flag = false;
	    for (int excl=excl_start;excl <= excl_end;excl++) {
	      if (atom_excl[excl] == jg) {
	      	excl_flag = true;
		break;
	      }
	    }
	    if (excl_flag == false) npair++;
	  }

	}
	//
      }
    }
  }

  return npair;
}

/*
void test_excl_dist_index(const int n_ijlist, const int3 *d_ijlist,
			  const int *d_cell_patom, const float4 *d_xyzq,
			  int *d_tile_indj,
			  tile_excl_t *d_tile_excl,
			  const float boxx, const float boxy, const float boxz,
			  const float rcut2) {

  int3 *h_ijlist;
  int *h_cell_patom;
  float4 *h_xyzq;
  int *h_tile_indj;
  tile_excl_t *h_tile_excl;

  h_ijlist = (int3 *)malloc(n_ijlist*sizeof(int3));
  h_cell_patom = (int *)malloc(mdsim.ncell*sizeof(int));
  h_xyzq = (float4 *)malloc(mdsim.ncoord*sizeof(float4));
  h_tile_indj = (int *)malloc(n_ijlist*sizeof(int));
  h_tile_excl = (tile_excl_t *)malloc(n_ijlist*sizeof(tile_excl_t));

  cudaCheck(hipMemcpy(h_ijlist, d_ijlist, sizeof(int3)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_cell_patom, d_cell_patom, sizeof(int)*mdsim.ncell,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_xyzq, d_xyzq, sizeof(float4)*mdsim.ncoord,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_indj, d_tile_indj, sizeof(int)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_excl, d_tile_excl, sizeof(tile_excl_t)*n_ijlist,
		       hipMemcpyDeviceToHost));

  for (int wid=0;wid < n_ijlist;wid++) {

    int3 ijlist_val = h_ijlist[wid];
    int icell = ijlist_val.x - 1;
    int ish   = ijlist_val.y;
    int jcell = ijlist_val.z - 1;

    int istart = h_cell_patom[icell] - 1;
    int iend   = h_cell_patom[icell+1] - 2;

    int jstart = h_cell_patom[jcell] - 1;
    int jend   = h_cell_patom[jcell+1] - 2;

    int q_samecell = (icell == jcell);

    // Calculate shift
    float zsh = (ish/9 - 1)*boxz;
    ish -= (ish/9)*9;
    float ysh = (ish/3 - 1)*boxy;
    ish -= (ish/3)*3;
    float xsh = (ish - 1)*boxx;
    
    int i,j,ii,jj;

    for (ii=istart,i=0;ii <= iend;ii++,i++) {
      float4 xyzq_i = h_xyzq[ii];
      float xi = xyzq_i.x;
      float yi = xyzq_i.y;
      float zi = xyzq_i.z;
      for (jj=jstart,j=0;jj <= jend;jj++,j++) {
	float4 xyzq_j = h_xyzq[jj];
	float xj = xyzq_j.x - xsh;
	float yj = xyzq_j.y - ysh;
	float zj = xyzq_j.z - zsh;
	float dx = xi - xj;
	float dy = yi - yj;
	float dz = zi - zj;
	float r2 = dx*dx + dy*dy + dz*dz;
#if (TILESIZE == 16)
	int ttid = ((i+j) % 2)*16 + j;
	int iexcl = ttid/4;
	int tmp = i + 1 + j*15;
	int shbit = ((tmp/2) % 8) + (j % 4)*8;
#else
	int ij = i + j*TILESIZE - j;
	int iexcl = j;
	int shbit = (ij % TILESIZE);
#endif
	unsigned int ibit = 1 << shbit;
	unsigned int excl = ((r2 >= rcut2) | (q_samecell && (j <= i)) ) << shbit;
	unsigned int excl_gpu = h_tile_excl[wid].excl[iexcl];
	if ( ((excl_gpu & ibit) ^ excl) != 0 && fabsf(r2-rcut2) > 7.0e-5) {
	  printf("Error found in test_excl_dist_index:\n");
	  printf("wid = %d i = %d j = %d iexcl = %d shbit = %d\n",wid,i,j,iexcl,shbit);
	  printf("ii = %d jj = %d %d %d %d %d\n",ii,jj,r2 >= rcut2,
		 (q_samecell && (j <= i)),icell,jcell);
	  printf("%x ^ %x = %x \n",excl_gpu & ibit, excl, (excl_gpu & ibit) ^ excl);
	  printf("i:  %f %f %f\n",xi,yi,zi);
	  printf("j:  %f %f %f\n",xj,yj,zj);
	  printf("jo: %f %f %f\n",xyzq_j.x,xyzq_j.y,xyzq_j.z);
	  printf("sh: %f %f %f\n",xsh,ysh,zsh);
	  printf("dx: %1.8f %1.8f %1.8f\n",dx,dy,dz);
	  printf("r2: %f %e\n",r2,fabsf(r2-rcut2));
	  exit(1);
	}
      }
    }

  }

  free(h_ijlist);
  free(h_cell_patom);
  free(h_xyzq);
  free(h_tile_indj);
  free(h_tile_excl);

  printf("test_excl_dist_index OK\n");
}
*/

//
// Host wrapper for build_tilex_kernel
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
template <int tilesize>
void NeighborList<tilesize>::build_excl(const float boxx, const float boxy, const float boxz,
					const float rcut,
					const int n_ijlist, const int3 *ijlist,
					const int *cell_patom,
					const float4 *xyzq,
					hipStream_t stream) {

  if (n_ijlist == 0) return;

  // Allocate & re-allocate (d_tile_indj, d_tile_excl)
#ifdef STRICT_MEMORY_REALLOC
  reallocate<int>(&tile_indj, &tile_indj_len, n_ijlist, 1.0f);
  reallocate<tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_ijlist, 1.0f);
#else
  reallocate<int>(&tile_indj, &tile_indj_len, n_ijlist, 1.2f);
  reallocate<tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_ijlist, 1.2f);
#endif

  float rcut2 = rcut*rcut;

  int nthread = nwarp_build_excl_dist*warpsize;
  int nblock_tot = (n_ijlist-1)/(nthread/warpsize) + 1;
  size_t shmem_size = nwarp_build_excl_dist*tilesize*sizeof(float3); 

  if (tilesize == 16) {
    shmem_size += nwarp_build_excl_dist*(num_excl<tilesize>::val)*sizeof(unsigned int);
  }

  int3 max_nblock3 = get_max_nblock();
  unsigned int max_nblock = max_nblock3.x;
  unsigned int base_tid = 0;

  while (nblock_tot != 0) {

    int nblock = (nblock_tot > max_nblock) ? max_nblock : nblock_tot;
    nblock_tot -= nblock;

    build_excl_kernel<tilesize>
      <<< nblock, nthread, shmem_size, stream >>>
      (base_tid, n_ijlist, ijlist, cell_patom,
       xyzq, tile_indj, tile_excl,
       boxx, boxy, boxz,
       rcut2);

    base_tid += nblock*nthread;

    cudaCheck(hipGetLastError());
  }

  /*
  if (mdsim.q_test != 0) {
    test_excl_dist_index(mdsim.n_ijlist, mdsim.ijlist, mdsim.cell_patom,
			 mdsim.xyzq.xyzq, mdsim.tile_indj, mdsim.tile_excl,
			 boxx, boxy, boxz,
			 rcut2);
  }
  */

}

//
// Host wrapper for add_tile_top_kernel
//
template <int tilesize>
void NeighborList<tilesize>::add_tile_top(const int ntile_top, const int *tile_ind_top,
					  const tile_excl_t<tilesize> *tile_excl_top,
					  hipStream_t stream) {
  int nthread = 256;
  int nblock = (ntile_top*(num_excl<tilesize>::val) - 1)/nthread + 1;
  
  add_tile_top_kernel<tilesize>
    <<< nblock, nthread, 0, stream >>>
    (ntile_top, tile_ind_top, tile_excl_top, tile_excl);
  
  cudaCheck(hipGetLastError());
}

//
// Splits neighbor list into dense and sparse parts
//
template <int tilesize>
void NeighborList<tilesize>::split_dense_sparse(int npair_cutoff) {

  ientry_t *h_ientry = new ientry_t[n_ientry];
  int *h_tile_indj = new int[n_tile];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[n_tile];

  ientry_t *h_ientry_dense = new ientry_t[n_ientry];
  int *h_tile_indj_dense = new int[n_tile];
  tile_excl_t<tilesize> *h_tile_excl_dense = new tile_excl_t<tilesize>[n_tile];

  ientry_t *h_ientry_sparse = new ientry_t[n_ientry];
  int *h_tile_indj_sparse = new int[n_tile];
  pairs_t<tilesize> *h_pairs = new pairs_t<tilesize>[n_tile];

  copy_DtoH_sync<ientry_t>(ientry, h_ientry, n_ientry);
  copy_DtoH_sync<int>(tile_indj, h_tile_indj, n_tile);
  copy_DtoH_sync< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, n_tile);

  int n_ientry_dense = 0;
  int n_tile_dense = 0;
  n_ientry_sparse = 0;
  n_tile_sparse = 0;
  for (int i=0;i < n_ientry;i++) {
    bool sparse_i_tiles = true;
    int startj_dense = n_tile_dense;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	npair += 32 - n1bit;
      }

      if (npair <= npair_cutoff) {
	// Sparse
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  
	}
	h_tile_indj_sparse[n_tile_sparse] = h_tile_indj[j];
	n_tile_sparse++;
      } else {
	// Dense
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_dense[n_tile_dense].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_dense[n_tile_dense] = h_tile_indj[j];
	n_tile_dense++;
	sparse_i_tiles = false;
      }

    }

    if (sparse_i_tiles) {
      // Sparse
    } else {
      h_ientry_dense[n_ientry_dense] = h_ientry[i];
      h_ientry_dense[n_ientry_dense].startj = startj_dense;
      h_ientry_dense[n_ientry_dense].endj = n_tile_dense - 1;
      n_ientry_dense++;
    }
  }

  n_ientry = n_ientry_dense;
  n_tile = n_tile_dense;

  copy_HtoD_sync<ientry_t>(h_ientry_dense, ientry, n_ientry);
  copy_HtoD_sync<int>(h_tile_indj_dense, tile_indj, n_tile);
  copy_HtoD_sync< tile_excl_t<tilesize> >(h_tile_excl_dense, tile_excl, n_tile);

  allocate<ientry_t>(&ientry_sparse, n_ientry_sparse);
  allocate<int>(&tile_indj_sparse, n_tile_sparse);
  allocate< pairs_t<tilesize> >(&pairs, n_tile_sparse);
  ientry_sparse_len = n_ientry_sparse;
  tile_indj_sparse_len = n_tile_sparse;
  pairs_len = n_tile_sparse;

  copy_HtoD_sync<ientry_t>(h_ientry_sparse, ientry_sparse, n_ientry_sparse);
  copy_HtoD_sync<int>(h_tile_indj_sparse, tile_indj_sparse, n_tile_sparse);
  copy_HtoD_sync< pairs_t<tilesize> >(h_pairs, pairs, n_tile_sparse);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_dense;
  delete [] h_tile_indj_dense;
  delete [] h_tile_excl_dense;

  delete [] h_ientry_sparse;
  delete [] h_tile_indj_sparse;
  delete [] h_pairs;

}

//
// Removes empty tiles
//
template <int tilesize>
void NeighborList<tilesize>::remove_empty_tiles() {

  ientry_t *h_ientry = new ientry_t[n_ientry];
  int *h_tile_indj = new int[n_tile];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[n_tile];

  ientry_t *h_ientry_noempty = new ientry_t[n_ientry];
  int *h_tile_indj_noempty = new int[n_tile];
  tile_excl_t<tilesize> *h_tile_excl_noempty = new tile_excl_t<tilesize>[n_tile];

  copy_DtoH_sync<ientry_t>(ientry, h_ientry, n_ientry);
  copy_DtoH_sync<int>(tile_indj, h_tile_indj, n_tile);
  copy_DtoH_sync< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, n_tile);

  int n_ientry_noempty = 0;
  int n_tile_noempty = 0;
  for (int i=0;i < n_ientry;i++) {
    bool empty_i_tiles = true;
    int startj_noempty = n_tile_noempty;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	if (n1bit != 32) empty_tile = false;
      }

      if (!empty_tile) {
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_noempty[n_tile_noempty].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_noempty[n_tile_noempty] = h_tile_indj[j];
	n_tile_noempty++;
	empty_i_tiles = false;
      }
    }

    if (!empty_i_tiles) {
      h_ientry_noempty[n_ientry_noempty] = h_ientry[i];
      h_ientry_noempty[n_ientry_noempty].startj = startj_noempty;
      h_ientry_noempty[n_ientry_noempty].endj = n_tile_noempty - 1;
      n_ientry_noempty++;
    }
  }

  n_ientry = n_ientry_noempty;
  n_tile = n_tile_noempty;

  copy_HtoD_sync<ientry_t>(h_ientry_noempty, ientry, n_ientry);
  copy_HtoD_sync<int>(h_tile_indj_noempty, tile_indj, n_tile);
  copy_HtoD_sync< tile_excl_t<tilesize> >(h_tile_excl_noempty, tile_excl, n_tile);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_noempty;
  delete [] h_tile_indj_noempty;
  delete [] h_tile_excl_noempty;

}

//
// Analyzes the neighbor list and prints info
//
template <int tilesize>
void NeighborList<tilesize>::analyze() {

  ientry_t *h_ientry = new ientry_t[n_ientry];
  int *h_tile_indj = new int[n_tile];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[n_tile];

  copy_DtoH_sync<ientry_t>(ientry, h_ientry, n_ientry);
  copy_DtoH_sync<int>(tile_indj, h_tile_indj, n_tile);
  copy_DtoH_sync< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, n_tile);

  std::cout << "Number of i-tiles = " << n_ientry << ", total number of tiles = " 
	    << n_tile << std::endl;

  std::ofstream file_npair("npair.txt", std::ofstream::out);
  std::ofstream file_nj("nj.txt", std::ofstream::out);

  unsigned int nexcl_bit = 0;
  unsigned int nexcl_bit_self = 0;
  unsigned int nempty_tile = 0;
  unsigned int nempty_line = 0;
  unsigned int npair_tot = 0;
  for (int i=0;i < n_ientry;i++) {
    file_nj << h_ientry[i].endj - h_ientry[i].startj + 1 << std::endl;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);

	if (n1bit > 32) {
	  std::cerr << n1bit << " " << std::hex << h_tile_excl[j].excl[k] << std::endl;
	  exit(1);
	}

	if (n1bit == 32)
	  nempty_line++;
	else
	  empty_tile = false;

	nexcl_bit += n1bit;
	npair += 32 - n1bit;

	if (h_ientry[i].indi == h_tile_indj[j]) nexcl_bit_self += n1bit;
      }
      if (empty_tile) nempty_tile++;
      file_npair << npair << std::endl;
      npair_tot += npair;
    }
  }

  file_npair.close();
  file_nj.close();

  unsigned int n_tile_pairs = n_tile*tilesize*tilesize;
  std::cout << "Total number of pairs = " << npair_tot 
	    << " (" << (double)npair_tot*100.0/(double)n_tile_pairs << "% full)" << std::endl;
  std::cout << "Total number of pairs in tiles = " << n_tile_pairs << std::endl;
  std::cout << "Number of excluded pairs = " << nexcl_bit << " (" << 
    ((double)nexcl_bit*100)/(double)n_tile_pairs << "%)" << std::endl;
  std::cout << "Number of excluded pairs in self (i==j) tiles = " << nexcl_bit_self << " (" << 
    ((double)nexcl_bit_self*100)/(double)n_tile_pairs << "%)" << std::endl;
  std::cout << "Number of empty lines = " << nempty_line << " (" <<
    ((double)nempty_line*100)/((double)(n_tile*tilesize)) << "%)" << std::endl;
  std::cout << "Number of empty tiles = " << nempty_tile << " (" <<
    ((double)nempty_tile*100)/(double)n_tile << "%)" << std::endl;

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

}

//
// Load neighbor list from file
//
template <int tilesize>
void NeighborList<tilesize>::load(const char *filename) {

  ientry_t *h_ientry;
  int *h_tile_indj;
  tile_excl_t<tilesize> *h_tile_excl;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> n_ientry >> n_tile;

    h_ientry = new ientry_t[n_ientry];
    h_tile_indj = new int[n_tile];
    h_tile_excl = new tile_excl_t<tilesize>[n_tile];

    for (int i=0;i < n_ientry;i++) {
      file >> std::dec >> h_ientry[i].indi >> h_ientry[i].ish >> 
	h_ientry[i].startj >> h_ientry[i].endj;
      for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
	file >> std::dec >> h_tile_indj[j];
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  file >> std::hex >> h_tile_excl[j].excl[k];
	}
      }
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

#ifdef STRICT_MEMORY_REALLOC
  reallocate<ientry_t>(&ientry, &ientry_len, n_ientry, 1.0f);
  reallocate<int>(&tile_indj, &tile_indj_len, n_tile, 1.0f);
  reallocate< tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_tile, 1.0f);
#else
  reallocate<ientry_t>(&ientry, &ientry_len, n_ientry, 1.2f);
  reallocate<int>(&tile_indj, &tile_indj_len, n_tile, 1.2f);
  reallocate< tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_tile, 1.2f);
#endif

  copy_HtoD_sync<ientry_t>(h_ientry, ientry, n_ientry);
  copy_HtoD_sync<int>(h_tile_indj, tile_indj, n_tile);
  copy_HtoD_sync< tile_excl_t<tilesize> >(h_tile_excl, tile_excl, n_tile);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;
}

//
// Explicit instances of NeighborList
//
//template class NeighborList<16>;
template class NeighborList<32>;

template int NeighborList<32>::calc_gpu_pairlist<double>(const int n_ientry, const ientry_t* ientry,
							 const int* tile_indj,
							 const tile_excl_t<32>* tile_excl,
							 const float4* xyzq,
							 const double boxx, const double boxy,
							 const double boxz,
							 const double rcut);

template int NeighborList<32>::calc_cpu_pairlist<double>(const int* zone_patom, const float4* xyzq,
							 const int* loc2glo, const int* atom_excl_pos,
							 const int* atom_excl, const double boxx,
							 const double boxy, const double boxz,
							 const double rcut);

template int NeighborList<32>::calc_gpu_pairlist<float>(const int n_ientry, const ientry_t* ientry,
							const int* tile_indj,
							const tile_excl_t<32>* tile_excl,
							const float4* xyzq,
							const double boxx, const double boxy,
							const double boxz,
							const double rcut);

template int NeighborList<32>::calc_cpu_pairlist<float>(const int* zone_patom, const float4* xyzq,
							const int* loc2glo, const int* atom_excl_pos,
							const int* atom_excl, const double boxx,
							const double boxy, const double boxz,
							const double rcut);
