#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"

//
// Class creator
//
template <int tilesize>
NeighborList<tilesize>::NeighborList() {
  ni = 0;

  tile_excl = NULL;
  tile_excl_len = 0;

  ientry = NULL;
  ientry_len = 0;

  tile_indj = NULL;
  tile_indj_len = 0;
}

//
// Class destructor
//
template <int tilesize>
NeighborList<tilesize>::~NeighborList() {
  if (tile_excl != NULL) deallocate< tile_excl_t<tilesize> > (&tile_excl);
  if (ientry != NULL) deallocate<ientry_t>(&ientry);
  if (tile_indj != NULL) deallocate<int>(&tile_indj);
}

static unsigned int count_1bits(unsigned int x)
{
  x = x - ((x >> 1) & 0x55555555);
  x = (x & 0x33333333) + ((x >> 2) & 0x33333333);
  x = x + (x >> 8);
  x = x + (x >> 16);
  return x & 0x0000003F;
}

//
// Analyzes the neighbor list and prints info
//
template <int tilesize>
void NeighborList<tilesize>::analyze() {

  ientry_t *h_ientry;
  int *h_tile_indj;
  tile_excl_t<tilesize> *h_tile_excl;

  h_ientry = new ientry_t[ni];
  h_tile_indj = new int[ntot];
  h_tile_excl = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  std::cout << "ni = " << ni << " ntot = " << ntot << std::endl;

  unsigned int nexcl_bit = 0;
  for (int i=0;i < ni;i++) {
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      for (int k=0;k < num_excl;k++) {
	nexcl_bit += count_1bits(h_tile_excl[j].excl[k]);
      }
    }
  }

  unsigned int ntot_pairs = ntot*tilesize*tilesize;
  std::cout << "Total number of pairs = " << ntot_pairs << std::endl;
  std::cout << "Number of excluded pairs = " << nexcl_bit << " (" << 
    ((double)nexcl_bit*100)/(double)ntot_pairs << "%)" << std::endl;

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

}

//
// Load neighbor list from file
//
template <int tilesize>
void NeighborList<tilesize>::load(const char *filename) {

  ientry_t *h_ientry;
  int *h_tile_indj;
  tile_excl_t<tilesize> *h_tile_excl;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> ni >> ntot;

    h_ientry = new ientry_t[ni];
    h_tile_indj = new int[ntot];
    h_tile_excl = new tile_excl_t<tilesize>[ntot];

    for (int i=0;i < ni;i++) {
      file >> std::dec >> h_ientry[i].indi >> h_ientry[i].ish >> 
	h_ientry[i].startj >> h_ientry[i].endj;
      for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
	file >> std::dec >> h_tile_indj[j];
	for (int k=0;k < num_excl;k++) {
	  file >> std::hex >> h_tile_excl[j].excl[k];
	}
      }
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.2f);
  reallocate<int>(&tile_indj, &tile_indj_len, ntot, 1.2f);
  reallocate< tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, ntot, 1.2f);

  copy_HtoD<ientry_t>(h_ientry, ientry, ni);
  copy_HtoD<int>(h_tile_indj, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;
}

//
// Explicit instances of DirectForce
//
template class NeighborList<16>;
template class NeighborList<32>;
