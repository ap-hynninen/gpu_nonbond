#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <thrust/sort.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "NeighborList.h"

//
// Calculates tilex index for each atom
//
__global__ void calc_tilex_ind_kernel(const int istart, const int iend,
				      const float4* __restrict__ xyzq,
				      const int ind0,
				      const int ncellx,
				      const int ncelly,
				      const int ncellz,
				      const float x0,
				      const float y0,
				      const float z0,
				      const float inv_dx,
				      const float inv_dy,
				      const float inv_dz,
				      int* __restrict__ tilex_key,
				      int* __restrict__ tilex_val) {

  const int ind = threadIdx.x + blockIdx.x*blockDim.x + istart;
  
  if (ind <= iend) {
    float4 xyzq_val = xyzq[ind];
    float x = xyzq_val.x;
    float y = xyzq_val.y;
    float z = xyzq_val.z;
    int ix = (int)((x - x0)*inv_dx);
    int iy = (int)((y - y0)*inv_dy);
    int iz = (int)((z - z0)*inv_dz);
    int key = ind0 + (ix + iy*ncellx)*ncellz + iz;

    tilex_key[ind] = key;
    tilex_val[ind] = ind;
  }

}

//
// Re-order atoms according to tilex_val
//
__global__ void reorder_atoms_kernel(const int ncoord,
				     const int* tilex_val,
				     const float4* __restrict__ xyzq_in,
				     float4* __restrict__ xyzq_out) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (tid < ncoord) {
    int ind = tilex_val[tid];
    float4 xyzq_val = xyzq_in[ind];
    xyzq_out[tid] = xyzq_val;
  }

}

//
//
//
template <int tilesize>
void NeighborList<tilesize>::set_cell_sizes(const int *zonelist_atom,
					    const float3 *max_xyz, const float3 *min_xyz,
					    int *ncellx, int *ncelly,
					    float *celldx, float *celldy) {

  for (int izone=0;izone < 8;izone++) {
    int nstart;
    if (izone > 0) {
      nstart = zonelist_atom[izone-1] + 1;
    } else {
      nstart = 1;
    }
    // ncoord_zone = number of atoms in this zone
    int ncoord_zone = zonelist_atom[izone] - nstart + 1;
    if (ncoord_zone > 0) {
      // NOTE: we increase the cell sizes here by 0.001 to make sure no atoms drop outside cells
      float xsize = max_xyz[izone].x - min_xyz[izone].x + 0.001f;
      float ysize = max_xyz[izone].y - min_xyz[izone].y + 0.001f;
      float zsize = max_xyz[izone].z - min_xyz[izone].z + 0.001f;
      float delta = powf(xsize*ysize*zsize*tilesize/(float)ncoord_zone, 1.0f/3.0f);
      ncellx[izone] = max(1, (int)(xsize/delta));
      ncelly[izone] = max(1, (int)(ysize/delta));
      celldx[izone] = xsize/(float)(ncellx[izone]);
      celldy[izone] = ysize/(float)(ncelly[izone]);
      // Increase ncellx and ncelly by one to account for bonded atoms outside the box
    } else {
      ncellx[izone] = 0;
      ncelly[izone] = 0;
      celldx[izone] = 1.0f;
      celldy[izone] = 1.0f;
    }
  }

}

//
// Sorts atoms into tiles
//
template <int tilesize>
void NeighborList<tilesize>::sort_tilex(const int *zonelist_atom,
					const int ncoord,
					const float3 *max_xyz, const float3 *min_xyz,
					const float4 *xyzq,
					float4 *xyzq_sorted,
					hipStream_t stream) {

  int ncellx[8], ncelly[8], ncellz[8];
  float celldx[8], celldy[8], celldz[8];
  float inv_dx[8], inv_dy[8], inv_dz[8];

  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_cell_sizes(zonelist_atom, max_xyz, min_xyz, ncellx, ncelly, celldx, celldy);

  for (int i=0;i < 8;i++) {
    inv_dx[i] = 1.0f/celldx[i];
    inv_dy[i] = 1.0f/celldy[i];
  }

  for (int i=0;i < 8;i++) {
    int istart, iend;
    if (i > 0) {
      istart = zonelist_atom[i-1];
    } else {
      istart = 0;
    }
    iend = zonelist_atom[i] - 1;
    calc_tilex_ind_kernel<<< nblock, nthread >>>
      (istart, iend, xyzq, 0, ncellx[i], ncelly[i], ncellz[i],
       min_xyz[i].x, min_xyz[i].y, min_xyz[i].z,
       inv_dx[i], inv_dy[i], inv_dz[i], tilex_key, tilex_val);

    cudaCheck(hipGetLastError());
  }

  thrust::sort_by_key(tilex_key, tilex_key + ncoord, tilex_val);

  reorder_atoms_kernel<<< nblock, nthread >>>
    (ncoord, tilex_val, xyzq, xyzq_sorted);

  cudaCheck(hipGetLastError());

}

//
// Calculates bounding box
//
template <int tilesize>
__global__ void calc_bounding_box_kernel(const int ncell,
					 const int* __restrict__ cell_start,
					 const float4* __restrict__ xyzq,
					 bb_t* __restrict__ bb) {

  const int icell = threadIdx.x + blockIdx.x*blockDim.x;

  if (icell < ncell) {
    int base = cell_start[icell];
    float4 xyzq_val = xyzq[base];
    float x0 = xyzq_val.x;
    float y0 = xyzq_val.y;
    float z0 = xyzq_val.z;
    float x1 = xyzq_val.x;
    float y1 = xyzq_val.y;
    float z1 = xyzq_val.z;
    for (int i=1;i < tilesize;i++) {
      xyzq_val = xyzq[base + i];
      x0 = min(x0, xyzq_val.x);
      y0 = min(y0, xyzq_val.y);
      z0 = min(z0, xyzq_val.z);
      x1 = max(x1, xyzq_val.x);
      y1 = max(y1, xyzq_val.y);
      z1 = max(z1, xyzq_val.z);
    }
    bb[icell].x = 0.5f*(x0 + x1);
    bb[icell].y = 0.5f*(y0 + y1);
    bb[icell].z = 0.5f*(z0 + z1);
    bb[icell].wx = 0.5f*(x1 - x0);
    bb[icell].wy = 0.5f*(y1 - y0);
    bb[icell].wz = 0.5f*(z1 - z0);
  }

}

//
// Calculates bounding boxes for tiles
//
template <int tilesize>
void NeighborList<tilesize>::calc_bounding_box(const int ncell,
					       const int *cell_start,
					       const float4 *xyzq,
					       hipStream_t stream) {
  int nthread = 512;
  int nblock = (ncell-1)/nthread+1;

  calc_bounding_box_kernel<tilesize> <<< nblock, nthread >>>
    (ncell, cell_start, xyzq, bb);

  cudaCheck(hipGetLastError());
}

//#######################################################################

//
// Class creator
//
template <int tilesize>
NeighborList<tilesize>::NeighborList() {
  ni = 0;
  ntot = 0;

  tile_excl = NULL;
  tile_excl_len = 0;

  ientry = NULL;
  ientry_len = 0;

  tile_indj = NULL;
  tile_indj_len = 0;

  // Sparse
  ni_sparse = 0;
  ntot_sparse = 0;

  pairs_len = 0;
  pairs = NULL;
  
  ientry_sparse_len = 0;
  ientry_sparse = NULL;

  tile_indj_sparse_len = NULL;
  tile_indj_sparse = NULL;
}

//
// Class destructor
//
template <int tilesize>
NeighborList<tilesize>::~NeighborList() {
  if (tile_excl != NULL) deallocate< tile_excl_t<tilesize> > (&tile_excl);
  if (ientry != NULL) deallocate<ientry_t>(&ientry);
  if (tile_indj != NULL) deallocate<int>(&tile_indj);
  // Sparse
  if (pairs != NULL) deallocate< pairs_t<tilesize> > (&pairs);
  if (ientry_sparse != NULL) deallocate<ientry_t>(&ientry_sparse);
  if (tile_indj_sparse != NULL) deallocate<int>(&tile_indj_sparse);
}

//
// Sets ientry from host memory array
//
template <int tilesize>
void NeighborList<tilesize>::set_ientry(int ni, ientry_t *h_ientry, hipStream_t stream) {

  this->ni = ni;

  // Allocate & reallocate d_ientry
  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.4f);

  // Copy to device
  copy_HtoD<ientry_t>(h_ientry, ientry, ni, stream);
}

//----------------------------------------------------------------------------------------
//
// Builds neighborlist
//

struct cell_t {
  int izone;
  int icellx;
  int icelly;
  int icellz;
};

#ifdef NOTREADY

//
// The entire warp enters here
// If IvsI = true, search within I zone
//
template <bool IvsI>
__device__ void get_cell_bounds(const int izone, const int jzone, const int icell, const int ncell,
				const float x0, const float x1, const float* bx, const float rcut,
				int& jcell0, int& jcell1, float *dist) {

  int jcell_start_left, jcell_start_right;

  if (izone == jzone) {
    // Search within a single zone (I)
    if (icell < 0) {
      // This is one of the image cells on the left =>
      // set the left cell boundary (jcell0) to 1 and start looking for the right
      // boundary from 1
      jcell_start_left = 0;       // with this value, we don't look for cells on the left
      jcell_start_right = 1;      // start looking for cells at right from 1
      jcell0 = 1;                  // left boundary set to minimum value
      jcell1 = 0;                    // set to "no cells" value
      dist[1] = 0.0f;
    } else if (icell >= ncell) {
      // This is one of the image cells on the right =>
      // set the right cell boundary (icell1) to ncell and start looking for the left
      // boundary from ncell
      jcell_start_left = ncell;      // start looking for cells at left from ncell
      jcell_start_right = ncell + 1; // with this value, we don't look for cells on the right
      jcell0 = ncell + 1;            // set to "no cells" value
      jcell1 = ncell;                // right boundary set to maximum value
      dist[ncell] = 0.0f;
    } else {
      jcell_start_left = icell - 1;
      jcell_start_right = icell + 1;
      jcell0 = icell;
      jcell1 = icell;
      dist[icell] = 0.0f;
    }
  } else {
    if (bx(0) >= x1 || (bx(0) < x1 && bx(0) > x0)) {
      // j-zone is to the right of i-zone
      // => no left search, start right search from 1
      jcell_start_left = 0;
      jcell_start_right = 1;
      jcell0 = 1;
      jcell1 = 0;
    } else if (bx[ncell] <= x0 || (bx[ncell] > x0 && bx[ncell] < x1)) {
      // j-zone is to the left of i-zone
      // => no right search, start left search from ncell
      jcell_start_left = ncell;
      jcell_start_right = ncell + 1;
      jcell0 = ncell + 1;
      jcell1 = ncell;
    } else {
      // i-zone is between j-zones
      // => safe choice is to search the entire range
      jcell_start_left = ncell;
      jcell_start_right = 1;
      jcell0 = ncell;
      jcell1 = 1;
    }
  }

  // Check cells at left, stop once the distance to the cell right boundary 
  // is greater than the cutoff.
  //
  // Cell right boundary is at bx(i)
  for (int j=jcell_start_left;j >= 1;j--) {
    float d = x0 - bx[j];
    if (d > cut) break;
    dist[j] = max(0.0f, d);
    jcell0 = j;
  }

  // Check cells at right, stop once the distance to the cell left boundary
  // is greater than the cutoff.
  //
  // Cell left boundary is at bx(i-1)
  for (int j=jcell_start_right;j <= ncell;j++) {
    float d = bx[j-1] - x1;
    if (d > cut) break;
    dist[j] = max(0.0f, d);
    jcell1 = j;
  }

  // Cell bounds are jcell0:jcell1
      
}

//
// Build neighborlist for one zone at the time
// One warp takes care of one cell
//
template < int tilesize, bool IvsI >
__global__
void build_nlist_kernel(const int ncell, const int izone, const int n_jzone,
			const int *cellx, const int *celly, const int *cellz,
			const bb_t * bb,
			const float *cellbx, const float *cellby, const float *cellbz) {

  // Shared memory
  extern __shared__ char shbuf[];
  volatile int *jcellx0;
  volatile int *jcelly0;
  volatile int *jcellz0;
  volatile int *jcellx1;
  volatile int *jcelly1;
  volatile int *jcellz1;

  // Index of the i-cell
  const int icell = (threadId.x + blockIdx.x*blockDim.x)/WARPSIZE;

  if (icell >= ncell) return;

  int icellx = cellx[icell];
  int icelly = celly[icell];
  int icellz = cellz[icell];

  bb_t ibb = bb[icell];

  for (int imx=imx_lo;imx <= imx_hi;imx++) {
    float imbbx0 = ibb.x + imx*boxx;
    int n_jcellx = 0;
    for (int jjzone=0;jjzone < n_jzone;jjzone++) {
      int jzone = int_zone[izone][jjzone];
      int jcellx0_t, jcellx1_t;
      get_cell_bounds<IvsI>(izone, jzone, icellx + imx*ncellx[izone], ncellx[jzone],
			    imbbx0-ibb.wx, imbbx0+ibb.wx, cellbx[jzone], rcut,
			    jcellx0_t, jcellx1_t);
      n_jcellx += max(0, jcellx1_t-jcellx0_t+1);
      jcellx0[jzone] = jcellx0_t;
      jcellx1[jzone] = jcellx1_t;
    }

    for (int imy=imy_lo;imy <= imy_hi;imy++) {
      float imbby0 = ibb.y + imy*boxy;
      int n_jcelly = 0;
      for (int jjzone=0;jjzone < n_jzone;jjzone++) {
	int jzone = int_zone[izone][jjzone];
	int jcelly0_t, jcelly1_t;
	get_cell_bounds<IvsI>(izone, jzone, icelly + imy*ncelly[izone], ncelly[jzone],
			      imbby0-ibb.wy, imbby0+ibb.wy, cellby[jzone], rcut,
			      jcelly0_t, jcelly1_t);
	n_jcelly += max(0, jcelly1_t-jcelly0_t+1);
	jcelly0[jzone] = jcelly0_t;
	jcelly1[jzone] = jcelly1_t;
      }
    } // for (int imy=imy_lo;imy <= imy_hi;imy++)

    for (int imz=imz_lo;imz <= imz_hi;imz++) {
	float imbbz0 = ibb.z + imz*boxz;
	
	int ish = imx+1 + 3*(imy+1 + 3*(imz+1));
	
	for (int jjzone=0;jjzone < n_jzone;jjzone++) {
	  int jzone = int_zone[izone][jjzone];


	  if (jcelly1[jzone] >= jcelly0[jzone] && jcellx1[jzone] >= jcellx0[jzone]) {
	    // Loop over j-cells
	    // NOTE: we do this in order y, x, z so that the resulting tile list
	    //       is ordered
	    for (int jcelly=jcelly0[jzone]; jcelly <= jcelly1(jzone);jcelly++) {
	      float celldist1 = ydist[ydist_pos + jcelly];
	      celldist1 *= celldist1;
	      jcellx0_t = jcellx0[jzone];
	      for (int jcellx=jcellx0_t; jcellx <= jcellx1[jzone]; jcellx++) {
		float celldist2 = celldist1 + xdist[xdist_pos + jcellx];
		celldist2 *= celldist2;
		if (celldist2 > cutsq) continue;
		// Get jcellz limits (jcellz0, jcellz1)
		pos_xy = jcellx + (jcelly-1)*ncellx[jzone];
		pos_cellbz = (max_ncellz(jzone)+1)*(pos_xy - 1);
		pos_ncellz = pos_xy + startcol_zone[jzone];
		get_cell_bounds<IvsI>(izone, jzone, icellz_im,
				      ncellz[pos_ncellz], imbbz0-ibb.wz, imbbz0+ibb.wz,
				      cellbz[jzone]%array(pos_cellbz:), cut, jcellz0, jcellz1, zdist);
		for (int jcellz=jcellz0; jcellz <= jcellz1; jcellz++) {
		  if (celldist2 + zdist(jcellz)**2 > cutsq) continue;
		  // j-cell index is calculated as jcellz + start of the column cells
		  jcell = jcellz + startcell_col[pos_ncellz];

		  // Read bounding box for j-cell
		  bb_t jbb = bb[jcell];
                               
		  // Calculate distance between i- and j-cell bounding boxes
		  float bbxdist = max(0.0f, fabs(imbbx0 - jbb.x) - ibb.wx - jbb.wx);
		  float bbydist = max(0.0f, fabs(imbby0 - jbb.y) - ibb.wy - jbb.wy);
		  float bbzdist = max(0.0f, fabs(imbbz0 - jbb.z) - ibb.wz - jbb.wz);

		  if (bbxdist**2 + bbydist**2 + bbzdist**2 < cutsq) {
		  }


	}
	
    } // for (int imz=imz_lo;imz <= imz_hi;imz++)


  } // for (int imx=imx_lo;imx <= imx_hi;imx++)

}

template <int tilesize>
void NeighborList<tilesize>::build_nlist(const float boxx, const float boxy, const float boxz,
					 const float roff,
					 const int n_ijlist, const int3 *ijlist,
					 const int *cell_start,
					 const float4 *xyzq,
					 hipStream_t stream) {

  build_nlist_kernel<tilesize, true>
    <<< nblock, nthread, shmem_size, stream >>>
    ();

  build_nlist_kernel<tilesize, false>
    <<< nblock, nthread, shmem_size, stream >>>
    ();

}
#endif // NOTREADY

//----------------------------------------------------------------------------------------
//
// Builds tilex exclusion mask from ijlist[] based on distance and index
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
// Uses 32 threads to calculate the distances for a single ijlist -entry.
//
const int nwarp_build_excl_dist = 8;

template < int tilesize >
__global__ void build_excl_kernel(const unsigned int base_tid, const int n_ijlist, const int3 *ijlist,
				  const int *cell_start, const float4 *xyzq,
				  int *tile_indj,
				  tile_excl_t<tilesize> *tile_excl,
				  const float boxx, const float boxy, const float boxz,
				  const float roff2) {
  const int num_thread_per_excl = (32/(num_excl<tilesize>::val));

  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x + base_tid;
  // Global warp index
  const unsigned int wid = gtid / warpsize;
  // Local thread index (0...warpsize-1)
  const unsigned int tid = gtid % warpsize;
  // local thread index (0...tilesize-1)
  const unsigned int stid = gtid % tilesize;

  // Shared memory
  extern __shared__ char shmem[];
  float3 *sh_xyzi = (float3 *)&shmem[0];    // nwarp_build_excl_dist*tilesize
  unsigned int *sh_excl = (unsigned int *)&sh_xyzi[nwarp_build_excl_dist*tilesize];

  //  __shared__ float3 sh_xyzi[nwarp_build_excl_dist*tilesize];
  //#if (tilesize == 16)
  //  __shared__ unsigned int sh_excl[nwarp_build_excl_dist*num_excl];
  //#endif

  if (wid >= n_ijlist) return;

  // Each warp computes one ijlist entry
  int3 ijlist_val = ijlist[wid];
  int icell = ijlist_val.x - 1;
  int ish   = ijlist_val.y;
  int jcell = ijlist_val.z - 1;

  int istart = cell_start[icell] - 1;
  int iend   = cell_start[icell+1] - 2;

  int jstart = cell_start[jcell] - 1;
  int jend   = cell_start[jcell+1] - 2;

  const unsigned int load_ij = threadIdx.x % tilesize;
  const int sh_start = (threadIdx.x/warpsize)*tilesize;

  // Load atom i coordinates to shared memory
  float4 xyzq_i;

  if (tilesize == 32 || tid < 16) {
    if (istart + load_ij <= iend) {
      xyzq_i = xyzq[istart + load_ij];
    } else {
      xyzq_i.x = -100000000.0f;
      xyzq_i.y = -100000000.0f;
      xyzq_i.z = -100000000.0f;
    }
    sh_xyzi[sh_start + load_ij].x = xyzq_i.x;
    sh_xyzi[sh_start + load_ij].y = xyzq_i.y;
    sh_xyzi[sh_start + load_ij].z = xyzq_i.z;
  }

  // Load atom j coordinates
  float xj, yj, zj;
  //  const unsigned int loadj = (tid + (tid/TILESIZE)*(TILESIZE-1)) % TILESIZE;
  //  const unsigned int loadj = threadIdx.x % TILESIZE;
  if (jstart + load_ij <= jend) {
    float4 xyzq_j = xyzq[jstart + load_ij];
    xj = xyzq_j.x;
    yj = xyzq_j.y;
    zj = xyzq_j.z;
  } else {
    xj = 100000000.0f;
    yj = 100000000.0f;
    zj = 100000000.0f;
  }

  // Calculate shift
  float zsh = (ish/9 - 1)*boxz;
  ish -= (ish/9)*9;
  float ysh = (ish/3 - 1)*boxy;
  ish -= (ish/3)*3;
  float xsh = (ish - 1)*boxx;

  xj -= xsh;
  yj -= ysh;
  zj -= zsh;
  
  // Make sure shared memory has been written
  // NOTE: since we're only operating within the warp, this __syncthreads() is just to make sure
  //       all values are actually written in shared memory and not kept in registers etc.
  __syncthreads();

  int q_samecell = (icell == jcell);

  unsigned int excl = 0;
  int t;

  if (tilesize == 32) {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int i = ((threadIdx.x + t) % tilesize);
      int ii = sh_start + i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (tid <= i)) ) << t;
    }
    tile_indj[wid] = jstart;
    tile_excl[wid].excl[stid] = excl;

  } else {

    for (t=0;t < (num_excl<tilesize>::val);t++) {
      int load_i = (tid + t*2 + (tid/tilesize)*(tilesize-1)) % tilesize;
      int ii = sh_start + load_i;
      float dx = sh_xyzi[ii].x - xj;
      float dy = sh_xyzi[ii].y - yj;
      float dz = sh_xyzi[ii].z - zj;
      float r2 = dx*dx + dy*dy + dz*dz;
      excl |= ((r2 >= roff2) | (q_samecell && (load_ij <= load_i)) ) << t;
    }
    // excl is a 8 bit exclusion mask.
    // The full 32 bit exclusion mask is contained in 4 threads:
    // thread 0 contains the lowest 8 bits
    // thread 1 contains the next 8 bits, etc..
    
    excl <<= (threadIdx.x % num_thread_per_excl)*(num_excl<tilesize>::val);
    
    // Combine excl using shared memory
    const unsigned int sh_excl_ind = (threadIdx.x/warpsize)*(num_excl<tilesize>::val) + 
      (threadIdx.x % warpsize)/num_thread_per_excl;
    
    sh_excl[sh_excl_ind] = 0;
    __syncthreads();
    
    atomicOr(&sh_excl[sh_excl_ind], excl);
    
    // Make sure shared memory is written
    __syncthreads();
    
    // index to tile_excl.excl[] (0...7)
    const unsigned int excl_ind = (threadIdx.x % warpsize)/num_thread_per_excl;
    
    tile_indj[wid] = jstart;
    
    if ((threadIdx.x % num_thread_per_excl) == 0) {
      tile_excl[wid].excl[excl_ind] = sh_excl[sh_excl_ind];
    }
  }

}

/*
void test_excl_dist_index(const int n_ijlist, const int3 *d_ijlist,
			  const int *d_cell_start, const float4 *d_xyzq,
			  int *d_tile_indj,
			  tile_excl_t *d_tile_excl,
			  const float boxx, const float boxy, const float boxz,
			  const float roff2) {

  int3 *h_ijlist;
  int *h_cell_start;
  float4 *h_xyzq;
  int *h_tile_indj;
  tile_excl_t *h_tile_excl;

  h_ijlist = (int3 *)malloc(n_ijlist*sizeof(int3));
  h_cell_start = (int *)malloc(mdsim.ncell*sizeof(int));
  h_xyzq = (float4 *)malloc(mdsim.ncoord*sizeof(float4));
  h_tile_indj = (int *)malloc(n_ijlist*sizeof(int));
  h_tile_excl = (tile_excl_t *)malloc(n_ijlist*sizeof(tile_excl_t));

  cudaCheck(hipMemcpy(h_ijlist, d_ijlist, sizeof(int3)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_cell_start, d_cell_start, sizeof(int)*mdsim.ncell,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_xyzq, d_xyzq, sizeof(float4)*mdsim.ncoord,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_indj, d_tile_indj, sizeof(int)*n_ijlist,
		       hipMemcpyDeviceToHost));

  cudaCheck(hipMemcpy(h_tile_excl, d_tile_excl, sizeof(tile_excl_t)*n_ijlist,
		       hipMemcpyDeviceToHost));

  for (int wid=0;wid < n_ijlist;wid++) {

    int3 ijlist_val = h_ijlist[wid];
    int icell = ijlist_val.x - 1;
    int ish   = ijlist_val.y;
    int jcell = ijlist_val.z - 1;

    int istart = h_cell_start[icell] - 1;
    int iend   = h_cell_start[icell+1] - 2;

    int jstart = h_cell_start[jcell] - 1;
    int jend   = h_cell_start[jcell+1] - 2;

    int q_samecell = (icell == jcell);

    // Calculate shift
    float zsh = (ish/9 - 1)*boxz;
    ish -= (ish/9)*9;
    float ysh = (ish/3 - 1)*boxy;
    ish -= (ish/3)*3;
    float xsh = (ish - 1)*boxx;
    
    int i,j,ii,jj;

    for (ii=istart,i=0;ii <= iend;ii++,i++) {
      float4 xyzq_i = h_xyzq[ii];
      float xi = xyzq_i.x;
      float yi = xyzq_i.y;
      float zi = xyzq_i.z;
      for (jj=jstart,j=0;jj <= jend;jj++,j++) {
	float4 xyzq_j = h_xyzq[jj];
	float xj = xyzq_j.x - xsh;
	float yj = xyzq_j.y - ysh;
	float zj = xyzq_j.z - zsh;
	float dx = xi - xj;
	float dy = yi - yj;
	float dz = zi - zj;
	float r2 = dx*dx + dy*dy + dz*dz;
#if (TILESIZE == 16)
	int ttid = ((i+j) % 2)*16 + j;
	int iexcl = ttid/4;
	int tmp = i + 1 + j*15;
	int shbit = ((tmp/2) % 8) + (j % 4)*8;
#else
	int ij = i + j*TILESIZE - j;
	int iexcl = j;
	int shbit = (ij % TILESIZE);
#endif
	unsigned int ibit = 1 << shbit;
	unsigned int excl = ((r2 >= roff2) | (q_samecell && (j <= i)) ) << shbit;
	unsigned int excl_gpu = h_tile_excl[wid].excl[iexcl];
	if ( ((excl_gpu & ibit) ^ excl) != 0 && fabsf(r2-roff2) > 7.0e-5) {
	  printf("Error found in test_excl_dist_index:\n");
	  printf("wid = %d i = %d j = %d iexcl = %d shbit = %d\n",wid,i,j,iexcl,shbit);
	  printf("ii = %d jj = %d %d %d %d %d\n",ii,jj,r2 >= roff2,
		 (q_samecell && (j <= i)),icell,jcell);
	  printf("%x ^ %x = %x \n",excl_gpu & ibit, excl, (excl_gpu & ibit) ^ excl);
	  printf("i:  %f %f %f\n",xi,yi,zi);
	  printf("j:  %f %f %f\n",xj,yj,zj);
	  printf("jo: %f %f %f\n",xyzq_j.x,xyzq_j.y,xyzq_j.z);
	  printf("sh: %f %f %f\n",xsh,ysh,zsh);
	  printf("dx: %1.8f %1.8f %1.8f\n",dx,dy,dz);
	  printf("r2: %f %e\n",r2,fabsf(r2-roff2));
	  exit(1);
	}
      }
    }

  }

  free(h_ijlist);
  free(h_cell_start);
  free(h_xyzq);
  free(h_tile_indj);
  free(h_tile_excl);

  printf("test_excl_dist_index OK\n");
}
*/

//
// Host wrapper for build_tilex_kernel
// Builds exclusion mask based on atom-atom distance and index (i >= j excluded)
//
template <int tilesize>
void NeighborList<tilesize>::build_excl(const float boxx, const float boxy, const float boxz,
					const float roff,
					const int n_ijlist, const int3 *ijlist,
					const int *cell_start,
					const float4 *xyzq,
					hipStream_t stream) {

  if (n_ijlist == 0) return;

  // Allocate & re-allocate (d_tile_indj, d_tile_excl)
  reallocate<int>(&tile_indj, &tile_indj_len, n_ijlist, 1.2f);
  reallocate<tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, n_ijlist, 1.2f);

  float roff2 = roff*roff;

  int nthread = nwarp_build_excl_dist*warpsize;
  int nblock_tot = (n_ijlist-1)/(nthread/warpsize) + 1;
  size_t shmem_size = nwarp_build_excl_dist*tilesize*sizeof(float3); 

  if (tilesize == 16) {
    shmem_size += nwarp_build_excl_dist*(num_excl<tilesize>::val)*sizeof(unsigned int);
  }

  int3 max_nblock3 = get_max_nblock();
  unsigned int max_nblock = max_nblock3.x;
  unsigned int base_tid = 0;

  while (nblock_tot != 0) {

    int nblock = (nblock_tot > max_nblock) ? max_nblock : nblock_tot;
    nblock_tot -= nblock;

    build_excl_kernel<tilesize>
      <<< nblock, nthread, shmem_size, stream >>>
      (base_tid, n_ijlist, ijlist, cell_start,
       xyzq, tile_indj, tile_excl,
       boxx, boxy, boxz,
       roff2);

    base_tid += nblock*nthread;

    cudaCheck(hipGetLastError());
  }

  /*
  if (mdsim.q_test != 0) {
    test_excl_dist_index(mdsim.n_ijlist, mdsim.ijlist, mdsim.cell_start,
			 mdsim.xyzq.xyzq, mdsim.tile_indj, mdsim.tile_excl,
			 boxx, boxy, boxz,
			 roff2);
  }
  */

}

//----------------------------------------------------------------------------------------
//
// Combines tile_excl_top on GPU
// One thread takes care of one integer in the exclusion mask, therefore:
//
// 32x32 tile, 32 integers per tile
// 16x16 tile, 8 integers per tile
//
template <int tilesize>
__global__ void add_tile_top_kernel(const int ntile_top,
				    const int *tile_ind_top,
				    const tile_excl_t<tilesize> *tile_excl_top,
				    tile_excl_t<tilesize> *tile_excl) {
  // Global thread index
  const unsigned int gtid = threadIdx.x + blockDim.x*blockIdx.x;
  // Index to tile_ind_top[]
  const unsigned int i = gtid / (num_excl<tilesize>::val);
  // Index to exclusion mask
  const unsigned int ix = gtid % (num_excl<tilesize>::val);

  if (i < ntile_top) {
    int ind = tile_ind_top[i];
    tile_excl[ind].excl[ix] |= tile_excl_top[i].excl[ix];
  }

}

//
// Host wrapper for add_tile_top_kernel
//
template <int tilesize>
void NeighborList<tilesize>::add_tile_top(const int ntile_top, const int *tile_ind_top,
					  const tile_excl_t<tilesize> *tile_excl_top,
					  hipStream_t stream) {
  int nthread = 256;
  int nblock = (ntile_top*(num_excl<tilesize>::val) - 1)/nthread + 1;
  
  add_tile_top_kernel<tilesize>
    <<< nblock, nthread, 0, stream >>>
    (ntile_top, tile_ind_top, tile_excl_top, tile_excl);
  
  cudaCheck(hipGetLastError());
}

static int BitCount(unsigned int u)
 {
         unsigned int uCount;

         uCount = u
                  - ((u >> 1) & 033333333333)
                  - ((u >> 2) & 011111111111);
         return
           ((uCount + (uCount >> 3))
            & 030707070707) % 63;
 }

/*
static int BitCount_ref(unsigned int u) {
  unsigned int x = u;
  int res = 0;
  while (x != 0) {
    res += (x & 1);
    x >>= 1;
  }
  return res;
}
*/

//
// Splits neighbor list into dense and sparse parts
//
template <int tilesize>
void NeighborList<tilesize>::split_dense_sparse(int npair_cutoff) {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_dense = new ientry_t[ni];
  int *h_tile_indj_dense = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl_dense = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_sparse = new ientry_t[ni];
  int *h_tile_indj_sparse = new int[ntot];
  pairs_t<tilesize> *h_pairs = new pairs_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  int ni_dense = 0;
  int ntot_dense = 0;
  ni_sparse = 0;
  ntot_sparse = 0;
  for (int i=0;i < ni;i++) {
    bool sparse_i_tiles = true;
    int startj_dense = ntot_dense;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	npair += 32 - n1bit;
      }

      if (npair <= npair_cutoff) {
	// Sparse
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  
	}
	h_tile_indj_sparse[ntot_sparse] = h_tile_indj[j];
	ntot_sparse++;
      } else {
	// Dense
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_dense[ntot_dense].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_dense[ntot_dense] = h_tile_indj[j];
	ntot_dense++;
	sparse_i_tiles = false;
      }

    }

    if (sparse_i_tiles) {
      // Sparse
    } else {
      h_ientry_dense[ni_dense] = h_ientry[i];
      h_ientry_dense[ni_dense].startj = startj_dense;
      h_ientry_dense[ni_dense].endj = ntot_dense - 1;
      ni_dense++;
    }
  }

  ni = ni_dense;
  ntot = ntot_dense;

  copy_HtoD<ientry_t>(h_ientry_dense, ientry, ni);
  copy_HtoD<int>(h_tile_indj_dense, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl_dense, tile_excl, ntot);

  allocate<ientry_t>(&ientry_sparse, ni_sparse);
  allocate<int>(&tile_indj_sparse, ntot_sparse);
  allocate< pairs_t<tilesize> >(&pairs, ntot_sparse);
  ientry_sparse_len = ni_sparse;
  tile_indj_sparse_len = ntot_sparse;
  pairs_len = ntot_sparse;

  copy_HtoD<ientry_t>(h_ientry_sparse, ientry_sparse, ni_sparse);
  copy_HtoD<int>(h_tile_indj_sparse, tile_indj_sparse, ntot_sparse);
  copy_HtoD< pairs_t<tilesize> >(h_pairs, pairs, ntot_sparse);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_dense;
  delete [] h_tile_indj_dense;
  delete [] h_tile_excl_dense;

  delete [] h_ientry_sparse;
  delete [] h_tile_indj_sparse;
  delete [] h_pairs;

}

//
// Removes empty tiles
//
template <int tilesize>
void NeighborList<tilesize>::remove_empty_tiles() {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  ientry_t *h_ientry_noempty = new ientry_t[ni];
  int *h_tile_indj_noempty = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl_noempty = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  int ni_noempty = 0;
  int ntot_noempty = 0;
  for (int i=0;i < ni;i++) {
    bool empty_i_tiles = true;
    int startj_noempty = ntot_noempty;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);
	if (n1bit != 32) empty_tile = false;
      }

      if (!empty_tile) {
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  h_tile_excl_noempty[ntot_noempty].excl[k] = h_tile_excl[j].excl[k];
	}
	h_tile_indj_noempty[ntot_noempty] = h_tile_indj[j];
	ntot_noempty++;
	empty_i_tiles = false;
      }
    }

    if (!empty_i_tiles) {
      h_ientry_noempty[ni_noempty] = h_ientry[i];
      h_ientry_noempty[ni_noempty].startj = startj_noempty;
      h_ientry_noempty[ni_noempty].endj = ntot_noempty - 1;
      ni_noempty++;
    }
  }

  ni = ni_noempty;
  ntot = ntot_noempty;

  copy_HtoD<ientry_t>(h_ientry_noempty, ientry, ni);
  copy_HtoD<int>(h_tile_indj_noempty, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl_noempty, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

  delete [] h_ientry_noempty;
  delete [] h_tile_indj_noempty;
  delete [] h_tile_excl_noempty;

}

//
// Analyzes the neighbor list and prints info
//
template <int tilesize>
void NeighborList<tilesize>::analyze() {

  ientry_t *h_ientry = new ientry_t[ni];
  int *h_tile_indj = new int[ntot];
  tile_excl_t<tilesize> *h_tile_excl = new tile_excl_t<tilesize>[ntot];

  copy_DtoH<ientry_t>(ientry, h_ientry, ni);
  copy_DtoH<int>(tile_indj, h_tile_indj, ntot);
  copy_DtoH< tile_excl_t<tilesize> >(tile_excl, h_tile_excl, ntot);

  std::cout << "Number of i-tiles = " << ni << ", total number of tiles = " << ntot << std::endl;

  std::ofstream file_npair("npair.txt", std::ofstream::out);
  std::ofstream file_nj("nj.txt", std::ofstream::out);

  unsigned int nexcl_bit = 0;
  unsigned int nexcl_bit_self = 0;
  unsigned int nempty_tile = 0;
  unsigned int nempty_line = 0;
  for (int i=0;i < ni;i++) {
    file_nj << h_ientry[i].endj - h_ientry[i].startj + 1 << std::endl;
    for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
      int npair = 0;
      bool empty_tile = true;
      for (int k=0;k < (num_excl<tilesize>::val);k++) {
	unsigned int n1bit = BitCount(h_tile_excl[j].excl[k]);

	if (n1bit > 32) {
	  std::cerr << n1bit << " " << std::hex << h_tile_excl[j].excl[k] << std::endl;
	  exit(1);
	}

	if (n1bit == 32)
	  nempty_line++;
	else
	  empty_tile = false;

	nexcl_bit += n1bit;
	npair += 32 - n1bit;

	if (h_ientry[i].indi == h_tile_indj[j]) nexcl_bit_self += n1bit;
      }
      if (empty_tile) nempty_tile++;
      file_npair << npair << std::endl;
    }
  }

  file_npair.close();
  file_nj.close();

  unsigned int ntot_pairs = ntot*tilesize*tilesize;
  std::cout << "Total number of pairs = " << ntot_pairs << std::endl;
  std::cout << "Number of excluded pairs = " << nexcl_bit << " (" << 
    ((double)nexcl_bit*100)/(double)ntot_pairs << "%)" << std::endl;
  std::cout << "Number of excluded pairs in self (i==j) tiles = " << nexcl_bit_self << " (" << 
    ((double)nexcl_bit_self*100)/(double)ntot_pairs << "%)" << std::endl;
  std::cout << "Number of empty lines = " << nempty_line << " (" <<
    ((double)nempty_line*100)/((double)(ntot*tilesize)) << "%)" << std::endl;
  std::cout << "Number of empty tiles = " << nempty_tile << " (" <<
    ((double)nempty_tile*100)/(double)ntot << "%)" << std::endl;

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;

}

//
// Load neighbor list from file
//
template <int tilesize>
void NeighborList<tilesize>::load(const char *filename) {

  ientry_t *h_ientry;
  int *h_tile_indj;
  tile_excl_t<tilesize> *h_tile_excl;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    file >> ni >> ntot;

    h_ientry = new ientry_t[ni];
    h_tile_indj = new int[ntot];
    h_tile_excl = new tile_excl_t<tilesize>[ntot];

    for (int i=0;i < ni;i++) {
      file >> std::dec >> h_ientry[i].indi >> h_ientry[i].ish >> 
	h_ientry[i].startj >> h_ientry[i].endj;
      for (int j=h_ientry[i].startj;j <= h_ientry[i].endj;j++) {
	file >> std::dec >> h_tile_indj[j];
	for (int k=0;k < (num_excl<tilesize>::val);k++) {
	  file >> std::hex >> h_tile_excl[j].excl[k];
	}
      }
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  reallocate<ientry_t>(&ientry, &ientry_len, ni, 1.2f);
  reallocate<int>(&tile_indj, &tile_indj_len, ntot, 1.2f);
  reallocate< tile_excl_t<tilesize> >(&tile_excl, &tile_excl_len, ntot, 1.2f);

  copy_HtoD<ientry_t>(h_ientry, ientry, ni);
  copy_HtoD<int>(h_tile_indj, tile_indj, ntot);
  copy_HtoD< tile_excl_t<tilesize> >(h_tile_excl, tile_excl, ntot);

  delete [] h_ientry;
  delete [] h_tile_indj;
  delete [] h_tile_excl;
}

//
// Explicit instances of DirectForce
//
template class NeighborList<16>;
template class NeighborList<32>;
